#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<vector_functions.h>
#include<stdio.h>
#include<hiprand/hiprand_kernel.h>
#include "VectorFunctions.hpp"

__global__ void DeviceRandInit(hiprandState *rngStates, uint *d_seeds, unsigned long long num){
    size_t idx = threadIdx.x + blockIdx.x*blockDim.x;
    if (idx < num){
        hiprandState rS = rngStates[idx];
        hiprand_init(d_seeds[idx], 0, 0 , &rS);
        rngStates[idx] = rS; 
    }
}



#define check_float3(a){ \
        if (!good_float3(a)){ \
        printf("uh oh hotdog\n"); \
        printf("%d, %s, %d, %d \n", __LINE__, __FILE__, blockIdx.x, threadIdx.x);   \
        asm("trap;"); \
    } \
}

__device__ float3 GetAngleForce(const float3 iPos, const float3 kPos,
                                      const float theta_o, const float k){
    float i_dot_k = dot(iPos, kPos);
    float ri_2 = mag2(iPos);
    float rk_2 = mag2(kPos);
    //if (ri_2*rk_2 - i_dot_k*i_dot_k < 0) asm("trap;");
    float c1 = -1/( sqrtf( ri_2*rk_2 - i_dot_k*i_dot_k + 1e-3));
    
    float c2 = i_dot_k/ri_2;
    
    float theta = acos(i_dot_k/(sqrtf(ri_2)*sqrtf(rk_2) + 1e-3));
    
    float3 F_i = -k * c1 * (theta - theta_o)*(kPos - c2*iPos);

    // float imag = mag(iPos);
    // float kmag = mag(kPos);
    
    // float cos_theta = dot(iPos, kPos)/(imag*kmag);
    // float cos_thetao = cos(theta_o); 

    // float3 F_i =  -2*k/(imag*kmag) * (cos_theta - cos_thetao) * kPos; 
    
    if (!good_float3(F_i)){
        printf("c1: %f, c2: %f, theta: %f, %d %d\n", c1, c2, theta, blockIdx.x, threadIdx.x);
        printf("i.k %f ri2 %f rk2 %f, %d %d\n", i_dot_k, ri_2, rk_2, blockIdx.x, threadIdx.x);
        asm("trap;");
    }

    // if (!good_float3(F_i)){
    //     printf("Angle force calculation failed for node %d in cell %d\n", blockIdx.x, threadIdx.x);
    //     asm("trap;");
    // }
    
    return F_i; 
}

// Watch me whip, whatch me...
__device__ void NeighNeighs (const int nodeInd, const int ni, int& nii, int& nij, int& nik,
                             const angles3 d_theta0[], float& theta1_o, float& theta2_o){
    if (nii == nodeInd){
        theta1_o = d_theta0[ni].aij;
        theta2_o = d_theta0[ni].aik; 
        nii = nij;
        nij = nik;
    } else if (nij == nodeInd){
        theta1_o = d_theta0[ni].aij;
        theta2_o = d_theta0[ni].ajk; 
        nij = nik;
    } else {
        theta1_o = d_theta0[ni].aik;
        theta2_o = d_theta0[ni].ajk; 
    }
}

// This fucking function will break if we decide to make cell geometry more interesting
__device__ float3 CalculateAngleForce(int nodeInd, int d_C180_nn[],
                                      float d_X[], float d_Y[], float d_Z[],
                                      const angles3 d_theta0[], float k, int cellInd){
    // First get the first angle contribution
    int ni = d_C180_nn[0*192 + nodeInd];
    int nj = d_C180_nn[1*192 + nodeInd];
    int nk = d_C180_nn[2*192 + nodeInd];
    
    float3 nodePos, nodeForce;

    nodeForce = make_float3(0, 0, 0);
    nodePos = make_float3(d_X[cellInd*192 + nodeInd], 
                          d_Y[cellInd*192 + nodeInd], 
                          d_Z[cellInd*192 + nodeInd]);

    float3 niPos, njPos, nkPos;
    niPos = make_float3(d_X[cellInd*192 + ni],
                        d_Y[cellInd*192 + ni],
                        d_Z[cellInd*192 + ni]);
    
    njPos = make_float3(d_X[cellInd*192 + nj],
                        d_Y[cellInd*192 + nj],
                        d_Z[cellInd*192 + nj]);

    nkPos = make_float3(d_X[cellInd*192 + nk],
                        d_Y[cellInd*192 + nk],
                        d_Z[cellInd*192 + nk]);
    
    angles3 nodeAngles = d_theta0[nodeInd]; 
    
    nodeForce = nodeForce - 
        (GetAngleForce(niPos-nodePos, njPos-nodePos, nodeAngles.aij, k) + 
         GetAngleForce(njPos-nodePos, niPos-nodePos, nodeAngles.aij, k));
    
    nodeForce = nodeForce - 
        (GetAngleForce(njPos-nodePos, nkPos-nodePos, nodeAngles.ajk, k) + 
         GetAngleForce(nkPos-nodePos, njPos-nodePos, nodeAngles.ajk, k));

    nodeForce = nodeForce -
        (GetAngleForce(niPos-nodePos, nkPos-nodePos, nodeAngles.aik, k) +
         GetAngleForce(nkPos-nodePos, niPos-nodePos, nodeAngles.aik, k));

    
    // Now second angle contributions
    // Each neighbor will have two other neighbors + our node

    int nii = d_C180_nn[ni + 0*192];
    int nij = d_C180_nn[ni + 1*192]; 
    int nik = d_C180_nn[ni + 2*192];

    int nji = d_C180_nn[nj + 0*192];
    int njj = d_C180_nn[nj + 1*192]; 
    int njk = d_C180_nn[nj + 2*192];

    int nki = d_C180_nn[nk + 0*192];
    int nkj = d_C180_nn[nk + 1*192]; 
    int nkk = d_C180_nn[nk + 2*192];
    
    float theta1_o, theta2_o; 
    float3 tiPos, tjPos;
    tiPos = make_float3(0, 0, 0);
    tjPos = make_float3(0, 0, 0);
    
    NeighNeighs(nodeInd, ni, nii, nij, nik, d_theta0, theta1_o, theta2_o);
    tiPos = make_float3(d_X[cellInd*192 + nii],
                        d_Y[cellInd*192 + nii],
                        d_Z[cellInd*192 + nii]);

    tjPos = make_float3(d_X[cellInd*192 + nij],
                        d_Y[cellInd*192 + nij],
                        d_Z[cellInd*192 + nij]);
    
    nodeForce = nodeForce + GetAngleForce(nodePos - niPos, tiPos - niPos, theta1_o, k); 
    nodeForce = nodeForce + GetAngleForce(nodePos - niPos, tjPos - niPos, theta2_o, k);
    
    NeighNeighs(nodeInd, nj, nji, njj, njk, d_theta0, theta1_o, theta2_o);
    tiPos = make_float3(d_X[cellInd*192 + nji],
                        d_Y[cellInd*192 + nji],
                        d_Z[cellInd*192 + nji]);

    tjPos = make_float3(d_X[cellInd*192 + njj],
                        d_Y[cellInd*192 + njj],
                        d_Z[cellInd*192 + njj]);

    nodeForce = nodeForce + GetAngleForce(nodePos - njPos, tiPos - njPos, theta1_o, k); 
    nodeForce = nodeForce + GetAngleForce(nodePos - njPos, tjPos - njPos, theta2_o, k);

    NeighNeighs(nodeInd, nk, nki, nkj, nkk, d_theta0, theta1_o, theta2_o);
    tiPos = make_float3(d_X[cellInd*192 + nki],
                        d_Y[cellInd*192 + nki],
                        d_Z[cellInd*192 + nki]);
                                 
    tjPos = make_float3(d_X[cellInd*192 + nkj],
                        d_Y[cellInd*192 + nkj],
                        d_Z[cellInd*192 + nkj]);

    nodeForce = nodeForce + GetAngleForce(nodePos - nkPos, tiPos - nkPos, theta1_o, k); 
    nodeForce = nodeForce + GetAngleForce(nodePos - nkPos, tjPos - nkPos, theta2_o, k);

    return nodeForce;
}
        
__global__ void CalculateConForce( int No_of_C180s, int d_C180_nn[], int d_C180_sign[],
                           float d_X[],  float d_Y[],  float d_Z[],
                           float *d_CMx, float *d_CMy, float *d_CMz,
                           float* d_R0, float* d_pressList, float* d_stiffness , float bondingYoungsMod, 
                           float internal_damping, const float *d_time,
                           float attraction_strength, float attraction_range,
                           float repulsion_strength, float repulsion_range,
                           float viscotic_damping, float mass,
                           int Xdiv, int Ydiv, int Zdiv, bool usePBCs,float3 boxMax,
                           int *d_NoofNNlist, int *d_NNlist, float DL, float gamma_visc,
                           float wall1, float wall2,
                           float threshDist, bool useWalls, 
                           float* d_velListX, float* d_velListY, float* d_velListZ,
                           bool useRigidSimulationBox, float boxLength, float* d_boxMin, float Youngs_mod, 
                                bool constrainAngles, const angles3 d_theta0[], R3Nptrs d_forceList, float r_CM_o, R3Nptrs d_contactForces, const float* volList, const float div_vol)
{
    // __shared__ hiprandState rngState;
    // if (threadIdx.x == 0){
    //     rngState = rngStates[threadIdx.x + blockDim.x*blockIdx.x];
    // }
#ifdef FORCE_DEBUG
        __shared__ float FX_sum;
        __shared__ float FY_sum;
        __shared__ float FZ_sum;
        
        if (threadIdx.x == 0){
            FX_sum = 0;
            FY_sum = 0;
            FZ_sum = 0;
        }

        __syncthreads();

#endif
        
    int rank, atom, nn_rank, nn_atom;
    int N1, N2, N3;
    int NooflocalNN;
    int localNNs[10];
    float deltaX, deltaY, deltaZ;
    float A1, A2, A3;
    float B1, B2, B3;
    float TX, TY, TZ;
    float NORM, R;
    float NX, NY, NZ;
    //float setPress;

    float3 disForce = make_float3(0, 0, 0);

    rank = blockIdx.x;
    atom = threadIdx.x;
    float Pressure = d_pressList[rank]; 
    int cellOffset = rank*192;
    int atomInd = cellOffset + atom;
    float stiffness;
    if ( rank < No_of_C180s && atom < 180 )
    {
        if (isnan(d_X[rank*192+atom]) ||
            isnan(d_Y[rank*192+atom]) || 
            isnan(d_Z[rank*192+atom])){
            printf("OH SHIT: we have a nan\n");
            printf("Particle index: %d, Cell: %d\n", atom, rank);
            printf("Crash now :(\n"); 
            asm("trap;"); 
        }

        float R0=0;


        stiffness = d_stiffness[rank];
        // printf("stiffness: %f\n", stiffness);
        // asm("trap;"); 
        
        N1 = d_C180_nn[  0+atom];
        N2 = d_C180_nn[192+atom];
        N3 = d_C180_nn[384+atom];

        A1 = d_X[rank*192+N2]-d_X[rank*192+N1];
        A2 = d_Y[rank*192+N2]-d_Y[rank*192+N1];
        A3 = d_Z[rank*192+N2]-d_Z[rank*192+N1];

        B1 = d_X[rank*192+N3]-d_X[rank*192+N1];
        B2 = d_Y[rank*192+N3]-d_Y[rank*192+N1];
        B3 = d_Z[rank*192+N3]-d_Z[rank*192+N1];

        TX = A2*B3-A3*B2;
        TY = A3*B1-A1*B3;
        TZ = A1*B2-A2*B1;

        NORM = sqrt(TX*TX+TY*TY+TZ*TZ);

        NX = d_C180_sign[atom]*TX/NORM;
        NY = d_C180_sign[atom]*TY/NORM;
        NZ = d_C180_sign[atom]*TZ/NORM;

        float X = d_X[rank*192+atom];
        float Y = d_Y[rank*192+atom];
        float Z = d_Z[rank*192+atom];

        float FX = 0.f;
        float FY = 0.f;
        float FZ = 0.f;

        int nnAtomInd;
        
        
        float velX = d_velListX[atomInd];
        float velY = d_velListY[atomInd];
        float velZ = d_velListZ[atomInd];


        //  Spring Force calculation within cell
        //  go through three nearest neighbors

        for ( int i = 0; i < 3 ; ++i ) // Better to open this loop
        {
            N1 = d_C180_nn[i*192+atom];

            deltaX = d_X[rank*192+N1]-d_X[rank*192+atom];
            deltaY = d_Y[rank*192+N1]-d_Y[rank*192+atom];
            deltaZ = d_Z[rank*192+N1]-d_Z[rank*192+atom];

            R  = sqrt(deltaX*deltaX+deltaY*deltaY+deltaZ*deltaZ);

            R0 = d_R0[i*192 + atom];

            //spring forces
            FX += +stiffness*(R-R0)/R0*deltaX/R;
            FY += +stiffness*(R-R0)/R0*deltaY/R;
            FZ += +stiffness*(R-R0)/R0*deltaZ/R;
        }

        // new growth force

        float3 r_CM = make_float3(d_X[atomInd] - d_CMx[rank], 
                                  d_Y[atomInd] - d_CMy[rank], 
                                  d_Z[atomInd] - d_CMz[rank]);
        float3 gForce  = make_float3(0.f, 0.f, 0.f);

        gForce = 3*Pressure*calcUnitVec(r_CM);

        //gForce = -10*(volList[rank] - div_vol)*calcUnitVec(r_CM);
        //gForce = -10*(mag(r_CM) - r_CM_o)*calcUnitVec(r_CM);
        
        FX += gForce.x; 
        FY += gForce.y; 
        FZ += gForce.z; 

        if (constrainAngles){
            float3 t = CalculateAngleForce(atom, d_C180_nn,
                                           d_X, d_Y, d_Z,
                                           d_theta0, 1000 /*Youngs_mod*/, rank);
            FX += t.x; FY += t.y; FZ += t.z;
        }
        
        
        
#ifdef FORCE_DEBUG

        atomicAdd(&FX_sum, FX);
        __syncthreads();
        atomicAdd(&FY_sum, FY);
        __syncthreads();
        atomicAdd(&FZ_sum, FZ);
        __syncthreads();
        if (threadIdx.x == 0){
            printf("Spring, pressure, internal\n");
            printf("Fx = %f, Fy = %f, Fz = %f\n", FX_sum, FY_sum, FZ_sum);
        }

#endif

        // interfullerene attraction and repulsion
        
        NooflocalNN = 0;
        
        int posX = 0;    
        int posY = 0;
        int posZ = 0;
        
        if(usePBCs){
            posX = (int) ((X - floor( X / boxMax.x) * boxMax.x )/DL);
            posX = posX - floor((float)posX/(float)Xdiv) * Xdiv;
        } else {   
            posX = (int)(X/DL);
            if ( posX < 0 ) posX = 0;
            if ( posX > Xdiv ) posX = Xdiv;
        }

		if(usePBCs){
            posY = (int) ((Y - floor( Y / boxMax.y) * boxMax.y )/DL);
            posY = posY - floor((float)posY/(float)Ydiv) * Ydiv;
        } else {   
            posY = (int)(Y/DL);
            if ( posY < 0 ) posY = 0;
            if ( posY > Ydiv ) posY = Ydiv;
        }

        if(usePBCs){
            posZ = (int) ((Z - floor( Z / boxMax.z) * boxMax.z )/DL);
            posZ = posZ - floor((float)posZ/(float)Zdiv) * Zdiv;
        } else {   
            posZ = (int)(Z/DL);
            if ( posZ < 0 ) posZ = 0;
            if ( posZ > Zdiv ) posZ = Zdiv;
        }


        int index = posZ*Xdiv*Ydiv + posY*Xdiv + posX;



        float3 contactForce = make_float3(0.f, 0.f, 0.f);
        
        for ( int nn_rank1 = 1 ; nn_rank1 <= d_NoofNNlist[index] ; ++nn_rank1 )
        {
            nn_rank = d_NNlist[32*index+nn_rank1-1];
            
            if ( nn_rank == rank )
                continue;
                
            deltaX  = X - d_CMx[nn_rank];
            if(usePBCs) deltaX = deltaX - nearbyint( deltaX / boxMax.x) * boxMax.x;
            // deltaX += (d_bounding_xyz[nn_rank*6+0]-X>0.0f)*(d_bounding_xyz[nn_rank*6+0]-X);
    
            deltaY  = Y - d_CMy[nn_rank];
            if(usePBCs) deltaY = deltaY - nearbyint( deltaY / boxMax.y) * boxMax.y; 
            // deltaY += (d_bounding_xyz[nn_rank*6+2]-Y>0.0f)*(d_bounding_xyz[nn_rank*6+2]-Y);
                
            deltaZ  = Z - d_CMz[nn_rank];
            if(usePBCs) deltaZ = deltaZ - nearbyint( deltaZ / boxMax.z) * boxMax.z;
            // deltaZ += (d_bounding_xyz[nn_rank*6+4]-Z>0.0f)*(d_bounding_xyz[nn_rank*6+4]-Z);
    

               
            if ( deltaX*deltaX + deltaY*deltaY + deltaZ*deltaZ - r_CM_o > attraction_range*attraction_range )
                continue;

            ++NooflocalNN;

            //printf("NooflocalNN %d\n", NooflocalNN);

            if ( NooflocalNN > MAX_NN ){
                printf("Recoverable error: NooflocalNN = %d, should be < 8\n",NooflocalNN);
                continue;
            }

            localNNs[NooflocalNN-1] = nn_rank;
        }

        for ( int i = 0; i < NooflocalNN; ++i )
        {
            nn_rank =localNNs[i];
            nnAtomInd = nn_rank*192;

            for ( nn_atom = 0; nn_atom < 180 ; ++nn_atom )
            {
                nnAtomInd += nn_atom;

                deltaX = d_X[rank*192+atom]-d_X[nn_rank*192+nn_atom];
                if(usePBCs) deltaX = deltaX - nearbyint( deltaX / boxMax.x) * boxMax.x;
            
                deltaY = d_Y[rank*192+atom]-d_Y[nn_rank*192+nn_atom];
                if(usePBCs) deltaY = deltaY - nearbyint( deltaY / boxMax.y) * boxMax.y;
            
                deltaZ = d_Z[rank*192+atom]-d_Z[nn_rank*192+nn_atom];
                if(usePBCs) deltaZ = deltaZ - nearbyint( deltaZ / boxMax.z) * boxMax.z;
            
                R = deltaX*deltaX+deltaY*deltaY+deltaZ*deltaZ;

                if ( R >= attraction_range*attraction_range )
                    continue;

                R = sqrt(R);

                if ( R < attraction_range )
                {
                    contactForce.x += -attraction_strength*Youngs_mod*(attraction_range-R)/R*deltaX;
                    contactForce.y += -attraction_strength*Youngs_mod*(attraction_range-R)/R*deltaY;
                    contactForce.z += -attraction_strength*Youngs_mod*(attraction_range-R)/R*deltaZ;
                }
                if ( R <= repulsion_range )
                {
                    //if (R < (repulsion_range-0.01)) R = repulsion_range-0.01; 
                    contactForce.x += +repulsion_strength*Youngs_mod*(repulsion_range-R)/R*deltaX;
                    contactForce.y += +repulsion_strength*Youngs_mod*(repulsion_range-R)/R*deltaY;
                    contactForce.z += +repulsion_strength*Youngs_mod*(repulsion_range-R)/R*deltaZ;
                }

            }

        }

        d_contactForces.x[atomInd] = contactForce.x;
        d_contactForces.y[atomInd] = contactForce.y;
        d_contactForces.z[atomInd] = contactForce.z;

        FX += contactForce.x;
        FY += contactForce.y;
        FZ += contactForce.z; 

#ifdef FORCE_DEBUG

        if (threadIdx.x == 0){
            FX_sum = 0;
            FY_sum = 0;
            FZ_sum = 0;
        }
        __syncthreads();

        atomicAdd(&FX_sum, FX);
        __syncthreads();
        atomicAdd(&FY_sum, FY);
        __syncthreads();
        atomicAdd(&FZ_sum, FZ);
        __syncthreads();
        if (threadIdx.x == 0){
            printf("neighbours\n");
            printf("Fx = %f, Fy = %f, Fz = %f\n", FX_sum, FY_sum, FZ_sum);
        }

#endif
        // add forces from simulation box if needed:
        if (useRigidSimulationBox){
            float gap1, gap2; 

            gap1 = d_X[atomInd] /* - 0 */;
            gap2 = boxMax.x - d_X[atomInd]; 

            if (gap1 < threshDist){
                FX += -100*Youngs_mod*(gap1 - threshDist);
            }

            if (gap2 < threshDist){
                FX += 100*Youngs_mod*(gap2 - threshDist);
            }
            
            gap1 = d_Y[atomInd];
            gap2 = boxMax.y - d_Y[atomInd];

            if (gap1 < threshDist){
                FY += -100*Youngs_mod*(gap1 - threshDist);
            }

            if (gap2 < threshDist){
                FY += 100*Youngs_mod*(gap2 - threshDist);
            }

            gap1 = d_Z[atomInd];
            gap2 = boxMax.z - d_Z[atomInd];

            if (gap1 < threshDist){
                FZ += -100*Youngs_mod*(gap1 - threshDist);
            }

            if (gap2 < threshDist){
                FZ += 100*Youngs_mod*(gap2 - threshDist);
            }

        }

        d_forceList.x[atomInd] = FX;
        d_forceList.y[atomInd] = FY;
        d_forceList.z[atomInd] = FZ;

        d_contactForces.x[atomInd] = FX;
        d_contactForces.y[atomInd] = FY;
        d_contactForces.z[atomInd] = FZ;
    }
}


__global__ void CalculateDisForce( int No_of_C180s, int d_C180_nn[], int d_C180_sign[],
                                   float d_X[],  float d_Y[],  float d_Z[],
                                   float *d_CMx, float *d_CMy, float *d_CMz,float r_CM_o,
                                   float gamma_int,
                                   float attraction_range,
                                   float gamma_ext,
                                   int Xdiv, int Ydiv, int Zdiv, bool usePBCs, float3 boxMax,
                                   int *d_NoofNNlist, int *d_NNlist, float DL, float gamma_o,
                                   float* d_velListX, float* d_velListY, float* d_velListZ,
                                   R3Nptrs d_fDisList){
    size_t cellInd = blockIdx.x;
    size_t nodeInd = threadIdx.x;

    if (cellInd < No_of_C180s && nodeInd < 180){
        size_t globalNodeInd = cellInd*192 + nodeInd;
        size_t N = 0;
        float3 force = make_float3(0, 0, 0);
        float3 nodeVelocity = make_float3(d_velListX[globalNodeInd],
                                          d_velListY[globalNodeInd],
                                          d_velListZ[globalNodeInd]);

        float3 neighVelocity = make_float3(0, 0, 0);
        
        // dampen bonding
        for (int i = 0; i < 3; ++i){
            N = d_C180_nn[i*192+nodeInd];
            neighVelocity = make_float3(d_velListX[cellInd*192+N],
                                        d_velListY[cellInd*192+N],
                                        d_velListZ[cellInd*192+N]);
                
            force = force - gamma_int*(nodeVelocity - neighVelocity);
        }

        // inter-cellular friction
        // This break if cell geometry every changes
        int N1 = d_C180_nn[  0+nodeInd];
        int N2 = d_C180_nn[192+nodeInd];
        int N3 = d_C180_nn[384+nodeInd];

        float3 A = make_float3(d_X[cellInd*192+N2]-d_X[cellInd*192+N1],
                               d_Y[cellInd*192+N2]-d_Y[cellInd*192+N1],
                               d_Z[cellInd*192+N2]-d_Z[cellInd*192+N1]);

        float3 B = make_float3(d_X[cellInd*192+N3]-d_X[cellInd*192+N1],
                               d_Y[cellInd*192+N3]-d_Y[cellInd*192+N1],
                               d_Z[cellInd*192+N3]-d_Z[cellInd*192+N1]);

        float3 normal = calcUnitVec(cross(A, B));

        normal = d_C180_sign[nodeInd]*normal;

        
        float X = d_X[globalNodeInd];
        float Y = d_Y[globalNodeInd];
        float Z = d_Z[globalNodeInd];
        

        float deltaX = 0;
        float deltaY = 0;
        float deltaZ = 0;
        float R = 0;

        int nn_rank = 0;
        int nnAtomInd = 0;
        
        int NooflocalNN = 0;
        int localNNs[10];

        
        int posX = 0;    
        int posY = 0;
        int posZ = 0;

        if(usePBCs){
            posX = (int) ((X - floor( X / boxMax.x) * boxMax.x )/DL);
            posX = posX - floor((float)posX/(float)Xdiv) * Xdiv;
        } else {   
            posX = (int)(X/DL);
            if ( posX < 0 ) posX = 0;
            if ( posX > Xdiv ) posX = Xdiv;
        }

		if(usePBCs){
            posY = (int) ((Y - floor( Y / boxMax.y) * boxMax.y )/DL);
            posY = posY - floor((float)posY/(float)Ydiv) * Ydiv;
        } else {   
            posY = (int)(Y/DL);
            if ( posY < 0 ) posY = 0;
            if ( posY > Ydiv ) posY = Ydiv;
        }

        if(usePBCs){
            posZ = (int) ((Z - floor( Z / boxMax.z) * boxMax.z )/DL);
            posZ = posZ - floor((float)posZ/(float)Zdiv) * Zdiv;
        } else {   
            posZ = (int)(Z/DL);
            if ( posZ < 0 ) posZ = 0;
            if ( posZ > Zdiv ) posZ = Zdiv;
        }


        int index = posZ*Xdiv*Ydiv + posY*Xdiv + posX;
        
        for ( int nn_rank1 = 1 ; nn_rank1 <= d_NoofNNlist[index] ; ++nn_rank1 )
        {
            nn_rank = d_NNlist[32*index+nn_rank1-1]; // MAGIC NUMBER!!
            if ( nn_rank == cellInd ) continue;

            deltaX  = X - d_CMx[nn_rank];
            if(usePBCs) deltaX = deltaX - nearbyint( deltaX / boxMax.x) * boxMax.x;
            // deltaX += (d_bounding_xyz[nn_rank*6+0]-X>0.0f)*(d_bounding_xyz[nn_rank*6+0]-X);

            deltaY  = Y - d_CMy[nn_rank];
            if(usePBCs) deltaY = deltaY - nearbyint( deltaY / boxMax.y) * boxMax.y; 
            // deltaY += (d_bounding_xyz[nn_rank*6+2]-Y>0.0f)*(d_bounding_xyz[nn_rank*6+2]-Y);
            
            deltaZ  = Z - d_CMz[nn_rank];
            if(usePBCs) deltaZ = deltaZ - nearbyint( deltaZ / boxMax.z) * boxMax.z;
            // deltaZ += (d_bounding_xyz[nn_rank*6+4]-Z>0.0f)*(d_bounding_xyz[nn_rank*6+4]-Z);

            if ( deltaX*deltaX + deltaY*deltaY + deltaZ*deltaZ - r_CM_o > attraction_range*attraction_range )
                continue;

            ++NooflocalNN;

            if ( NooflocalNN > MAX_NN ){
                printf("Recoverable error: NooflocalNN = %d, should be < 8\n",NooflocalNN);
                continue;
            }
            localNNs[NooflocalNN-1] = nn_rank;
        }

        for ( int i = 0; i < NooflocalNN; ++i )
        {
            nn_rank =localNNs[i];

            for ( int nn_atom = 0; nn_atom < 180 ; ++nn_atom )
            {
                deltaX = X - d_X[nn_rank*192+nn_atom];
                if(usePBCs) deltaX = deltaX - nearbyint( deltaX / boxMax.x) * boxMax.x;
                   
                deltaY = Y - d_Y[nn_rank*192+nn_atom];
                if(usePBCs) deltaY = deltaY - nearbyint( deltaY / boxMax.y) * boxMax.y;
                  
                deltaZ = Z - d_Z[nn_rank*192+nn_atom];
                if(usePBCs) deltaZ = deltaZ - nearbyint( deltaZ / boxMax.z) * boxMax.z;                
                

                R = deltaX*deltaX + deltaY*deltaY + deltaZ*deltaZ;

                if ( R > attraction_range*attraction_range )
                    continue;

                neighVelocity = make_float3(d_velListX[nn_rank*192+nn_atom],
                                            d_velListY[nn_rank*192+nn_atom],
                                            d_velListZ[nn_rank*192+nn_atom]);

                float3 v_ij = nodeVelocity - neighVelocity;

                // Tangential component of relative velocity
                float3 vTau = v_ij - dot(v_ij, normal)*normal;
                force = force - gamma_ext*vTau;
            }

        }

        // viscous drag
        force = force - gamma_o*nodeVelocity;
        
        // write force to global memory
        d_fDisList.x[globalNodeInd] = force.x; 
        d_fDisList.y[globalNodeInd] = force.y; 
        d_fDisList.z[globalNodeInd] = force.z; 
    }
}


__global__ void CalculateRanForce(int No_of_C180s, hiprandState *d_rngStates, float rand_scale_factor,
                                  R3Nptrs d_fRanList){
    size_t nodeIdx = blockIdx.x*blockDim.x + threadIdx.x;
    if (nodeIdx < No_of_C180s*192){
        hiprandState rngState = d_rngStates[nodeIdx];
        d_fRanList.x[nodeIdx] = rand_scale_factor*hiprand_normal(&rngState); 
        d_fRanList.y[nodeIdx] = rand_scale_factor*hiprand_normal(&rngState); 
        d_fRanList.z[nodeIdx] = rand_scale_factor*hiprand_normal(&rngState);
        d_rngStates[nodeIdx] = rngState;
    }
}


__global__ void Integrate(float *d_XP, float *d_YP, float *d_ZP,
                          float *d_X, float *d_Y, float *d_Z, 
                          float *d_XM, float *d_YM, float *d_ZM,
                          float *d_velListX, float *d_velListY, float *d_velListZ, 
                          float *d_time, float* MassArray,
                          R3Nptrs d_fConList, R3Nptrs d_fDisList, R3Nptrs d_fRanList,
                          int numCells, bool add_rands,
                          hiprandState *rngStates, float rand_scale_factor){
    const int cellInd = blockIdx.x;
    const int node = threadIdx.x;


    __shared__ float m;
        
    if (threadIdx.x == 0){
    	m = MassArray[cellInd];
    }
    __syncthreads();
    
    
    if (cellInd < numCells && node < 180){
        int nodeInd = cellInd*192 + node;
        const float dt = d_time[0];
        const float root_dt = sqrtf(dt);
        
        d_velListX[nodeInd] = d_velListX[nodeInd] + 0.5*(dt*d_fConList.x[nodeInd] + dt*d_fDisList.x[nodeInd] + \
                                                         root_dt*d_fRanList.x[nodeInd])/m;
        
        d_velListY[nodeInd] = d_velListY[nodeInd] + 0.5*(dt*d_fConList.y[nodeInd] + dt*d_fDisList.y[nodeInd] + \
                                                         root_dt*d_fRanList.y[nodeInd])/m;
        
        d_velListZ[nodeInd] = d_velListZ[nodeInd] + 0.5*(dt*d_fConList.z[nodeInd] + dt*d_fDisList.z[nodeInd] + \
                                                         root_dt*d_fRanList.z[nodeInd])/m;

        d_XP[nodeInd] = d_X[nodeInd] + d_velListX[nodeInd]*dt; 
        d_YP[nodeInd] = d_Y[nodeInd] + d_velListY[nodeInd]*dt; 
        d_ZP[nodeInd] = d_Z[nodeInd] + d_velListZ[nodeInd]*dt; 

    }
}

__global__ void VelocityUpdateA(float* d_VX, float* d_VY, float* d_VZ,
                                R3Nptrs fConList, R3Nptrs fRanList,
                                float dt, long int num_nodes, float* MassArray){
   
    long int nodeInd = blockIdx.x*blockDim.x + threadIdx.x;

    __shared__ float m;
        
   if (threadIdx.x == 0){
   	m = MassArray[blockIdx.x];
   }
  
   __syncthreads();


    if (nodeInd < num_nodes){
        float root_dt = sqrtf(dt);
        d_VX[nodeInd] = d_VX[nodeInd] + 0.5*(dt*fConList.x[nodeInd] + root_dt*fRanList.x[nodeInd])/m;
        d_VY[nodeInd] = d_VY[nodeInd] + 0.5*(dt*fConList.y[nodeInd] + root_dt*fRanList.y[nodeInd])/m;
        d_VZ[nodeInd] = d_VZ[nodeInd] + 0.5*(dt*fConList.z[nodeInd] + root_dt*fRanList.z[nodeInd])/m;
    }
}


__global__ void VelocityUpdateB(float* d_VX, float* d_VY, float* d_VZ,
                                R3Nptrs fDisList, float dt, long int num_nodes, float* MassArray){
    
    long int nodeInd = blockIdx.x*blockDim.x + threadIdx.x;

    __shared__ float m;
        
        if (threadIdx.x == 0){
            m = MassArray[blockIdx.x];
        }
        __syncthreads();

    if (nodeInd < num_nodes){
        d_VX[nodeInd] = d_VX[nodeInd] + 0.5*dt*(fDisList.x[nodeInd])/m;
        d_VY[nodeInd] = d_VY[nodeInd] + 0.5*dt*(fDisList.y[nodeInd])/m;
        d_VZ[nodeInd] = d_VZ[nodeInd] + 0.5*dt*(fDisList.z[nodeInd])/m;
    }
}


__global__ void ForwardTime(float *d_XP, float *d_YP, float *d_ZP,
                            float *d_X, float *d_Y, float *d_Z,
                            float *d_XM, float *d_YM, float *d_ZM, 
                            int numCells){
    
    const int nodeInd = blockIdx.x*blockDim.x + threadIdx.x;
    if (nodeInd < 192*numCells){
        // if (d_XP[nodeInd] != d_XM[nodeInd] ||
        //     d_YP[nodeInd] != d_YM[nodeInd] || 
        //     d_ZP[nodeInd] != d_ZM[nodeInd] ){
        //     printf("%.20f != %.20f or\n%.20f != %.20f or\n%.20f != %.20f\nnodeInd=%d\n",
        //            d_XP[nodeInd], d_XM[nodeInd],
        //            d_YP[nodeInd], d_YM[nodeInd],
        //            d_ZP[nodeInd], d_ZM[nodeInd], nodeInd);
        //     asm("trap;");
        // }

        d_XM[nodeInd] = d_X[nodeInd]; 
        d_YM[nodeInd] = d_Y[nodeInd]; 
        d_ZM[nodeInd] = d_Z[nodeInd];
    
        d_X[nodeInd] = d_XP[nodeInd];
        d_Y[nodeInd] = d_YP[nodeInd];
        d_Z[nodeInd] = d_ZP[nodeInd];
    }
}


__global__ void CorrectCoMMotion(float* d_X, float* d_Y, float* d_Z,
                                 float sysCMx, float sysCMy, float sysCMz, long int numParts){
    
    long int partInd = blockIdx.x*blockDim.x + threadIdx.x;

    if (partInd < numParts){
        d_X[partInd] -= sysCMx;
        d_Y[partInd] -= sysCMy;
        d_Z[partInd] -= sysCMz;
    }
}


__global__ void SumForces(R3Nptrs fConList, R3Nptrs fDisList, R3Nptrs fRanList,
                          R3Nptrs fList, 
                          long int numNodes){
    size_t idx = blockIdx.x*blockDim.x + threadIdx.x;
    if (idx < numNodes){
        fList.x[idx] = fConList.x[idx] + fDisList.x[idx] + fRanList.x[idx]; 
        fList.y[idx] = fConList.y[idx] + fDisList.y[idx] + fRanList.y[idx]; 
        fList.z[idx] = fConList.z[idx] + fDisList.z[idx] + fRanList.z[idx];
    }
}



__global__ void CoorUpdatePBC (float *d_X, float *d_Y, float *d_Z, 
                               float *d_XM, float *d_YM, float *d_ZM,
                               float *d_CMx, float *d_CMy, float *d_CMz,
                               float3 boxMax, float divVol, int numCells){

    
    int cellInd = blockIdx.x;
    int node = threadIdx.x;
    int nodeInd = cellInd*192 + node;

    
    if (cellInd < numCells && node < 180){

        
        if(d_CMx[cellInd] > boxMax.x + divVol ){ 

            d_X[nodeInd] = d_X[nodeInd] - boxMax.x;
            d_XM[nodeInd] = d_XM[nodeInd] - boxMax.x;
     

        } 

        if(d_CMx[cellInd] < -divVol){

            d_X[nodeInd] = d_X[nodeInd] + boxMax.x;
            d_XM[nodeInd] = d_XM[nodeInd] + boxMax.x;

        }
    
    
        if(d_CMy[cellInd] > boxMax.y + divVol) {

            d_Y[nodeInd] = d_Y[nodeInd] - boxMax.y;
            d_YM[nodeInd] = d_YM[nodeInd] - boxMax.y;

        }
    
        if(d_CMy[cellInd] < -divVol){

            d_Y[nodeInd] = d_Y[nodeInd] + boxMax.y;
            d_YM[nodeInd] = d_YM[nodeInd] + boxMax.y;

        }


    
        if(d_CMz[cellInd] > boxMax.z + divVol){

            d_Z[nodeInd] = d_Z[nodeInd] - boxMax.z;
            d_ZM[nodeInd] = d_ZM[nodeInd] - boxMax.z;

        }
    
    
        if(d_CMz[cellInd] < -divVol){

            d_Z[nodeInd] = d_Z[nodeInd] + boxMax.z;
            d_ZM[nodeInd] = d_ZM[nodeInd] + boxMax.z;

        }
    
    }

}
