#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "postscript.h"
#include "VectorFunctions.hpp"

//#define PRINT_TOO_SHORT_ERROR

__global__ void bounding_boxes( int No_of_C180s,
               float *d_XP, float *d_YP, float *d_ZP,
//               float *d_X,  float *d_Y,  float *d_Z,
//               float *d_XM, float *d_YM, float *d_ZM,
               float *d_bounding_xyz, float *CMx,
			   float *CMy, float *CMz)
{
  __shared__ float  minx[32];
  __shared__ float  maxx[32];
  __shared__ float  miny[32];
  __shared__ float  maxy[32];
  __shared__ float  minz[32];
  __shared__ float  maxz[32];

  int rank = blockIdx.x;
  int tid  = threadIdx.x;
  int atom = tid;

  if ( rank < No_of_C180s )
    {
	  minx[tid] = d_XP[rank*192+atom];
	  maxx[tid] = d_XP[rank*192+atom];
	  miny[tid] = d_YP[rank*192+atom];
	  maxy[tid] = d_YP[rank*192+atom];
	  minz[tid] = d_ZP[rank*192+atom];
	  maxz[tid] = d_ZP[rank*192+atom];

	  // // move present value to past value
	  // d_XM[rank*192+atom] =  d_X[rank*192+atom];
	  // d_YM[rank*192+atom] =  d_Y[rank*192+atom];
	  // d_ZM[rank*192+atom] =  d_Z[rank*192+atom];

	  // // move future value to present value
	  // d_X[rank*192+atom] = d_XP[rank*192+atom];
	  // d_Y[rank*192+atom] = d_YP[rank*192+atom];
	  // d_Z[rank*192+atom] = d_ZP[rank*192+atom];


	  while ( atom + 32 < 180 )
        {
		  atom += 32;
		  if ( minx[tid] > d_XP[rank*192+atom] )
		       minx[tid] = d_XP[rank*192+atom];
		  if ( maxx[tid] < d_XP[rank*192+atom] )
		       maxx[tid] = d_XP[rank*192+atom];
		  if ( miny[tid] > d_YP[rank*192+atom] )
		       miny[tid] = d_YP[rank*192+atom];
		  if ( maxy[tid] < d_YP[rank*192+atom] )
		       maxy[tid] = d_YP[rank*192+atom];
		  if ( minz[tid] > d_ZP[rank*192+atom] )
		       minz[tid] = d_ZP[rank*192+atom];
		  if ( maxz[tid] < d_ZP[rank*192+atom] )
		       maxz[tid] = d_ZP[rank*192+atom];

		  // // move present value to past value
		  // d_XM[rank*192+atom] =  d_X[rank*192+atom];
		  // d_YM[rank*192+atom] =  d_Y[rank*192+atom];
		  // d_ZM[rank*192+atom] =  d_Z[rank*192+atom];

		  // // move future value to present value
		  // d_X[rank*192+atom]  = d_XP[rank*192+atom];
		  // d_Y[rank*192+atom]  = d_YP[rank*192+atom];
		  // d_Z[rank*192+atom]  = d_ZP[rank*192+atom];

        }

	  if ( tid < 16 )
        {
		  if ( minx[tid] > minx[tid+16] ) minx[tid] = minx[tid+16];
		  if ( maxx[tid] < maxx[tid+16] ) maxx[tid] = maxx[tid+16];

		  if ( miny[tid] > miny[tid+16] ) miny[tid] = miny[tid+16];
		  if ( maxy[tid] < maxy[tid+16] ) maxy[tid] = maxy[tid+16];

		  if ( minz[tid] > minz[tid+16] ) minz[tid] = minz[tid+16];
		  if ( maxz[tid] < maxz[tid+16] ) maxz[tid] = maxz[tid+16];
        }

	  if ( tid < 8 )
        {
		  if ( minx[tid] > minx[tid+8] ) minx[tid] = minx[tid+8];
		  if ( maxx[tid] < maxx[tid+8] ) maxx[tid] = maxx[tid+8];
		  if ( miny[tid] > miny[tid+8] ) miny[tid] = miny[tid+8];
		  if ( maxy[tid] < maxy[tid+8] ) maxy[tid] = maxy[tid+8];
		  if ( minz[tid] > minz[tid+8] ) minz[tid] = minz[tid+8];
		  if ( maxz[tid] < maxz[tid+8] ) maxz[tid] = maxz[tid+8];
        }

	  if ( tid < 4 )
        {
		  if ( minx[tid] > minx[tid+4] ) minx[tid] = minx[tid+4];
		  if ( maxx[tid] < maxx[tid+4] ) maxx[tid] = maxx[tid+4];
		  if ( miny[tid] > miny[tid+4] ) miny[tid] = miny[tid+4];
		  if ( maxy[tid] < maxy[tid+4] ) maxy[tid] = maxy[tid+4];
		  if ( minz[tid] > minz[tid+4] ) minz[tid] = minz[tid+4];
		  if ( maxz[tid] < maxz[tid+4] ) maxz[tid] = maxz[tid+4];
        }

	  if ( tid < 2 )
        {
		  if ( minx[tid] > minx[tid+2] ) minx[tid] = minx[tid+2];
		  if ( maxx[tid] < maxx[tid+2] ) maxx[tid] = maxx[tid+2];
		  if ( miny[tid] > miny[tid+2] ) miny[tid] = miny[tid+2];
		  if ( maxy[tid] < maxy[tid+2] ) maxy[tid] = maxy[tid+2];
		  if ( minz[tid] > minz[tid+2] ) minz[tid] = minz[tid+2];
		  if ( maxz[tid] < maxz[tid+2] ) maxz[tid] = maxz[tid+2];
        }

	  if ( tid == 0  )
        {
		  if ( minx[0] > minx[1] ) minx[0] = minx[1];
		  d_bounding_xyz[rank*6+0] = minx[0];

		  if ( maxx[0] < maxx[1] ) maxx[0] = maxx[1];
		  d_bounding_xyz[rank*6+1] = maxx[0];

		  if ( miny[0] > miny[1] ) miny[0] = miny[1];
		  d_bounding_xyz[rank*6+2] = miny[0];

		  if ( maxy[0] < maxy[1] ) maxy[0] = maxy[1];
		  d_bounding_xyz[rank*6+3] = maxy[0];

		  if ( minz[0] > minz[1] ) minz[0] = minz[1];
		  d_bounding_xyz[rank*6+4] = minz[0];

		  if ( maxz[0] < maxz[1] ) maxz[0] = maxz[1];
		  d_bounding_xyz[rank*6+5] = maxz[0];
        }

    }

}



__global__ void minmaxpre( int No_of_C180s, float *d_bounding_xyz,
                    float *Minx, float *Maxx,
					float *Miny, float *Maxy,
					float *Minz, float *Maxz)
{

  __shared__ float  minx[1024];
  __shared__ float  maxx[1024];
  __shared__ float  miny[1024];
  __shared__ float  maxy[1024];
  __shared__ float  minz[1024];
  __shared__ float  maxz[1024];

  int fullerene = blockIdx.x*blockDim.x+threadIdx.x;
  int tid       = threadIdx.x;

  minx[tid] = +1.0E8f;
  maxx[tid] = -1.0E8f;
  miny[tid] = +1.0E8f;
  maxy[tid] = -1.0E8f;
  minz[tid] = +1.0E8f;
  maxz[tid] = -1.0E8f;

  if ( fullerene < No_of_C180s )
    {
	  minx[tid] = d_bounding_xyz[6*fullerene+0];
	  maxx[tid] = d_bounding_xyz[6*fullerene+1];
	  miny[tid] = d_bounding_xyz[6*fullerene+2];
	  maxy[tid] = d_bounding_xyz[6*fullerene+3];
	  minz[tid] = d_bounding_xyz[6*fullerene+4];
	  maxz[tid] = d_bounding_xyz[6*fullerene+5];
    }

  __syncthreads();

  for ( int s = blockDim.x/2; s > 0; s>>=1)
	{
	  if ( tid < s )
		{
		  minx[tid] = fminf(minx[tid],minx[tid+s]);
		  maxx[tid] = fmaxf(maxx[tid],maxx[tid+s]);
		  miny[tid] = fminf(miny[tid],miny[tid+s]);
		  maxy[tid] = fmaxf(maxy[tid],maxy[tid+s]);
		  minz[tid] = fminf(minz[tid],minz[tid+s]);
		  maxz[tid] = fmaxf(maxz[tid],maxz[tid+s]);
		}
	  __syncthreads();
	}

  if ( tid == 0 )
	{
	  Minx[blockIdx.x]  = minx[0];
	  Maxx[blockIdx.x]  = maxx[0];
	  Miny[blockIdx.x]  = miny[0];
	  Maxy[blockIdx.x]  = maxy[0];
	  Minz[blockIdx.x]  = minz[0];
	  Maxz[blockIdx.x]  = maxz[0];
	}

}


__global__ void makeNNlist(int No_of_C180s, float *CMx, float *CMy,float *CMz,
                           int Xdiv, int Ydiv, int Zdiv, float3 BoxMin,
                           int *d_NoofNNlist, int *d_NNlist, float DL)
{


	int fullerene = blockIdx.x*blockDim.x+threadIdx.x;
//  printf("(%d, %d, %d) %d %d\n", blockIdx.x, blockDim.x, threadIdx.x, fullerene, No_of_C180s);


	if ( fullerene < No_of_C180s )
	{
	  
		int posx = 0;
		int posy = 0;
		int posz = 0;		


	 	posx = (int)((CMx[fullerene] - BoxMin.x)/DL);
	  	if ( posx < 0 ) posx = 0;
	  	if ( posx > Xdiv - 1 ) posx = Xdiv - 1;
	  	

	  	posy = (int)((CMy[fullerene]-BoxMin.y)/DL);
	  	if ( posy < 0 ) posy = 0;
	  	if ( posy > Ydiv - 1 ) posy = Ydiv - 1;

	   	posz = (int)((CMz[fullerene]-BoxMin.z)/DL);
	  	if ( posz < 0 ) posz = 0;
	  	if ( posz > Zdiv - 1 ) posz = Zdiv - 1;
	  	
	 
		int j1 = 0;
	  	int j2 = 0;
	  	int j3 = 0;
	 
	  	for (  int i = -1; i < 2 ; ++i ){
				
			j1 = posx + i;
			if(j1 < 0 || j1 > Xdiv-1) continue;
			

			for (  int j = -1; j < 2; ++j ){
					
				j2 = posy + j;
				if(j2 < 0 || j2 > Ydiv-1) continue;
				
	
				for (  int k = -1 ; k < 2; ++k ){
			
					j3 = posz + k;
					if(j3 < 0 || j3 > Zdiv-1) continue;
		

			  		int index = atomicAdd( &d_NoofNNlist[j3*Xdiv*Ydiv+j2*Xdiv+j1] , 1); //returns old
#ifdef PRINT_TOO_SHORT_ERROR
			  		if ( index > 64 )
					{
                         			printf("Fullerene %d, NN-list too short, atleast %d\n", fullerene, index);
                                  			// for ( int k = 0; k < 32; ++k )
                                  			//     printf("%d ",d_NNlist[ 32*(j2*Xdiv+j1) + k]); 
                                 			// printf("\n");
						 continue;
					}
#endif
			  		d_NNlist[ 64*(j3*Xdiv*Ydiv+j2*Xdiv+j1)+index] = fullerene;
					
				}
	
			}
		}	
		
	}

}




__global__ void makeNNlistPBC(int No_of_C180s, float *CMx, float *CMy,float *CMz,
                           float attrac, int Xdiv, int Ydiv, int Zdiv, float3 boxMax,
                           int *d_NoofNNlist, int *d_NNlist, float3 DLp, bool useRigidBoxZ, bool useRigidBoxY)
{


	int fullerene = blockIdx.x*blockDim.x+threadIdx.x;
//  printf("(%d, %d, %d) %d %d\n", blockIdx.x, blockDim.x, threadIdx.x, fullerene, No_of_C180s);


	if ( fullerene < No_of_C180s )
	{
	  
		int posx = 0;
		int posy = 0;
		int posz = 0;	
		float boxX = boxMax.x;
		float boxY = boxMax.y;
		float boxZ = boxMax.z;	
		float DX = DLp.x;
		float DY = DLp.y;
		float DZ = DLp.z;	

	  	
	  	posx = (int)(( CMx[fullerene] - floor( CMx[fullerene] / boxX) * boxX )/DX); 	
 	  	posy = (int)(( CMy[fullerene] - floor( CMy[fullerene] / boxY) * boxY )/DY); 	
	  	posz = (int)(( CMz[fullerene] - floor( CMz[fullerene] / boxZ) * boxZ )/DZ); 		
	 
		int j1 = 0;
	  	int j2 = 0;
	  	int j3 = 0;
	 
	  	for (  int i = -1; i < 2 ; ++i ){
				
			j1 = posx + i;
			j1 = j1 - floor((float)j1/(float)Xdiv) * Xdiv;	 

			for (  int j = -1; j < 2; ++j ){ 


				j2 = posy + j;
					
				if(useRigidBoxY){
					
					if(j2 < 0 || j2 > Ydiv-1) continue;
					
				}else{	
					
					j2 = j2 - floor((float)j2/(float)Ydiv) * Ydiv;	 
					
				}
	
				for (  int k = -1 ; k < 2; ++k ){
			
			
					j3 = posz + k;
					
					if(useRigidBoxZ){
					
						if(j3 < 0 || j3 > Zdiv-1) continue;
					
					}else{	
					
						j3 = j3 - floor((float)j3/(float)Zdiv) * Zdiv;	 
					
					}

			  		int index = atomicAdd( &d_NoofNNlist[j3*Xdiv*Ydiv+j2*Xdiv+j1] , 1); //returns old
#ifdef PRINT_TOO_SHORT_ERROR
			  		if ( index > 64 )
					{
                         			printf("Fullerene %d, NN-list too short, atleast %d\n", fullerene, index);
                                  		// for ( int k = 0; k < 32; ++k )
                                  		// printf("%d ",d_NNlist[ 32*(j2*Xdiv+j1) + k]);
                                  		// printf("\n");
						continue;
					}
#endif
			  		d_NNlist[ 64*(j3*Xdiv*Ydiv+j2*Xdiv+j1)+index] = fullerene;
					
				}
	
			}
		}	

		
		
	}
		
}



__global__ void makeNNlistLEbc(int No_of_C180s, float *CMx, float *CMy,float *CMz,
                           float attrac, int Xdiv, int Ydiv, int Zdiv, float3 boxMax,
                           int *d_NoofNNlist, int *d_NNlist, float3 DLp, float Pshift,bool useRigidBoxZ)
{


	int fullerene = blockIdx.x*blockDim.x+threadIdx.x;
//  printf("(%d, %d, %d) %d %d\n", blockIdx.x, blockDim.x, threadIdx.x, fullerene, No_of_C180s);


	if ( fullerene < No_of_C180s )
	{
	  
		int posx = 0;
		int posy = 0;
		int posz = 0;		

	  	
	  	posx = (int)(( CMx[fullerene] - floor( CMx[fullerene] / boxMax.x) * boxMax.x )/DLp.x); 	
 	  	posy = (int)(( CMy[fullerene] - floor( CMy[fullerene] / boxMax.y) * boxMax.y )/DLp.y); 	
	  	posz = (int)(( CMz[fullerene] - floor( CMz[fullerene] / boxMax.z) * boxMax.z )/DLp.z); 		
	 
		int j1 = 0;
	  	int j2 = 0;
	  	int j3 = 0;
	 
	  	for (  int i = -1; i < 2 ; ++i ){
				
			j1 = posx + i;
			if (j1 >= Xdiv || j1 <= -1) continue; 
				 

			for (  int j = -1; j < 2; ++j ){
					
				j2 = posy + j;
				j2 = j2 - floor((float)j2/(float)Ydiv) * Ydiv;	 
	
				for (  int k = -1 ; k < 2; ++k ){
			
			
					j3 = posz + k;
					
					if(useRigidBoxZ){
					
						if(j3 < 0 || j3 > Zdiv-1) continue;
					
					}else{	
					
						j3 = j3 - floor((float)j3/(float)Zdiv) * Zdiv;	 
					
					}

			  		int index = atomicAdd( &d_NoofNNlist[j3*Xdiv*Ydiv+j2*Xdiv+j1] , 1); //returns old
#ifdef PRINT_TOO_SHORT_ERROR
			  		if ( index > 64 )
					{
                         			printf("Fullerene %d, NN-list too short, atleast %d\n", fullerene, index);
                                  		// for ( int k = 0; k < 32; ++k )
                                  		// printf("%d ",d_NNlist[ 32*(j2*Xdiv+j1) + k]);
                                  		// printf("\n");
						continue;
					}
#endif
			  		d_NNlist[ 64*(j3*Xdiv*Ydiv+j2*Xdiv+j1)+index] = fullerene;
					
				}
	
			}
		}
		
				
		if(posx == Xdiv-1 ){
			
			posy = (int)(( (CMy[fullerene] - Pshift) - floor((CMy[fullerene] - Pshift) / boxMax.y) * boxMax.y )/DLp.y);	
			
			j1 = 0;
			j3 = posz;
			if(useRigidBoxZ){
				if ( posz < 0 ) posz = 0;
	  			if ( posz > Zdiv - 1 ) posz = Zdiv - 1;	
			}else{						
				j3 = j3 - floor((float)j3/(float)Zdiv) * Zdiv;	 
			}
				 
			for (  int i = -1; i < 2 ; ++i ){
				
				j2 = posy + i;
				j2 = j2 - floor((float)j2/(float)Ydiv) * Ydiv;
		
				int index = atomicAdd( &d_NoofNNlist[j3*Xdiv*Ydiv+j2*Xdiv+j1] , 1);
				d_NNlist[ 64*(j3*Xdiv*Ydiv+j2*Xdiv+j1)+index] = fullerene;
		
			}
		}
			
			
			
			
		if (posx == 0){
			
			posy = (int)(( (CMy[fullerene] + Pshift) - floor( (CMy[fullerene] + Pshift) / boxMax.y) * boxMax.y )/DLp.y);	
			
			j1 = Xdiv - 1;
			j3 = posz;
			if(useRigidBoxZ){
				if ( posz < 0 ) posz = 0;
	  			if ( posz > Zdiv - 1 ) posz = Zdiv - 1;	
			}else{						
				j3 = j3 - floor((float)j3/(float)Zdiv) * Zdiv;	 
			}
				
			for (  int i = -1; i < 2 ; ++i ){
				
				j2 = posy + i;
				j2 = j2 - floor((float)j2/(float)Ydiv) * Ydiv;
		
				int index = atomicAdd( &d_NoofNNlist[j3*Xdiv*Ydiv+j2*Xdiv+j1] , 1);
				d_NNlist[ 64*(j3*Xdiv*Ydiv+j2*Xdiv+j1)+index] = fullerene;
		
			}	
			
				

		}
			

		
		
	}
		
}


// Pinning
__global__ void makeNNlistPin(int impurityNum, float *CMx, float *CMy,float *CMz,
                           int Xdiv, int Ydiv, int Zdiv, float3 BoxMin,
                           int *d_NoofNNlistPin, int *d_NNlistPin, float DL)
{


	int fullerene = blockIdx.x*blockDim.x+threadIdx.x;


	if ( fullerene < impurityNum )
	{
	  
		int posx = 0;
		int posy = 0;
		int posz = 0;		


	 	posx = (int)((CMx[fullerene] - BoxMin.x)/DL);
	  	if ( posx < 0 ) posx = 0;
	  	if ( posx > Xdiv - 1 ) posx = Xdiv - 1;
	  	

	  	posy = (int)((CMy[fullerene]-BoxMin.y)/DL);
	  	if ( posy < 0 ) posy = 0;
	  	if ( posy > Ydiv - 1 ) posy = Ydiv - 1;

	   	posz = (int)((CMz[fullerene]-BoxMin.z)/DL);
	  	if ( posz < 0 ) posz = 0;
	  	if ( posz > Zdiv - 1 ) posz = Zdiv - 1;
	  	
	 
		int j1 = 0;
	  	int j2 = 0;
	  	int j3 = 0;
	 
	  	for (  int i = -1; i < 2 ; ++i ){
				
			j1 = posx + i;
			if(j1 < 0 || j1 > Xdiv-1) continue;
			

			for (  int j = -1; j < 2; ++j ){
					
				j2 = posy + j;
				if(j2 < 0 || j2 > Ydiv-1) continue;
				
	
				for (  int k = -1 ; k < 2; ++k ){
			
					j3 = posz + k;
					if(j3 < 0 || j3 > Zdiv-1) continue;
		

			  		int index = atomicAdd( &d_NoofNNlistPin[j3*Xdiv*Ydiv+j2*Xdiv+j1] , 1); //returns old
#ifdef PRINT_TOO_SHORT_ERROR
			  		if ( index > 32 )
					{
                         			printf("Fullerene %d, NN-listPin too short, atleast %d\n", fullerene, index);
                                  			// for ( int k = 0; k < 32; ++k )
                                  			//     printf("%d ",d_NNlist[ 32*(j2*Xdiv+j1) + k]); 
                                 			// printf("\n");
						 continue;
					}
#endif
			  		d_NNlistPin[ 32*(j3*Xdiv*Ydiv+j2*Xdiv+j1)+index] = fullerene;
					
				}
	
			}
		}	
		
	}

}




__global__ void makeNNlistPBCPin(int impurityNum, float *CMx, float *CMy,float *CMz,
                           float attrac, int Xdiv, int Ydiv, int Zdiv, float3 boxMax,
                           int *d_NoofNNlistPin, int *d_NNlistPin, float3 DLp, bool useRigidBoxZ, bool useRigidBoxY)
{


	int fullerene = blockIdx.x*blockDim.x+threadIdx.x;
//  printf("(%d, %d, %d) %d %d\n", blockIdx.x, blockDim.x, threadIdx.x, fullerene, No_of_C180s);


	if ( fullerene < impurityNum )
	{
	  
		int posx = 0;
		int posy = 0;
		int posz = 0;	
		float boxX = boxMax.x;
		float boxY = boxMax.y;
		float boxZ = boxMax.z;	
		float DX = DLp.x;
		float DY = DLp.y;
		float DZ = DLp.z;	

	  	
	  	posx = (int)(( CMx[fullerene] - floor( CMx[fullerene] / boxX) * boxX )/DX); 	
 	  	posy = (int)(( CMy[fullerene] - floor( CMy[fullerene] / boxY) * boxY )/DY); 	
	  	posz = (int)(( CMz[fullerene] - floor( CMz[fullerene] / boxZ) * boxZ )/DZ); 		
	 
		int j1 = 0;
	  	int j2 = 0;
	  	int j3 = 0;
	 
	  	for (  int i = -1; i < 2 ; ++i ){
				
			j1 = posx + i;
			j1 = j1 - floor((float)j1/(float)Xdiv) * Xdiv;	 

			for (  int j = -1; j < 2; ++j ){ 


				j2 = posy + j;
					
				if(useRigidBoxY){
					
					if(j2 < 0 || j2 > Ydiv-1) continue;
					
				}else{	
					
					j2 = j2 - floor((float)j2/(float)Ydiv) * Ydiv;	 
					
				}
	
				for (  int k = -1 ; k < 2; ++k ){
			
			
					j3 = posz + k;
					
					if(useRigidBoxZ){
					
						if(j3 < 0 || j3 > Zdiv-1) continue;
					
					}else{	
					
						j3 = j3 - floor((float)j3/(float)Zdiv) * Zdiv;	 
					
					}

			  		int index = atomicAdd( &d_NoofNNlistPin[j3*Xdiv*Ydiv+j2*Xdiv+j1] , 1); //returns old
#ifdef PRINT_TOO_SHORT_ERROR
			  		if ( index > 32 )
					{
                         			printf("Fullerene %d, NN-listPin too short, atleast %d\n", fullerene, index);
                                  		// for ( int k = 0; k < 32; ++k )
                                  		// printf("%d ",d_NNlist[ 32*(j2*Xdiv+j1) + k]);
                                  		// printf("\n");
						continue;
					}
#endif
			  		d_NNlistPin[ 32*(j3*Xdiv*Ydiv+j2*Xdiv+j1)+index] = fullerene;
					
				}
	
			}
		}	

		
		
	}
		

}



__global__ void makeNNlistLEbcPin(int impurityNum, float *CMx, float *CMy,float *CMz,
                           float attrac, int Xdiv, int Ydiv, int Zdiv, float3 boxMax,
                           int *d_NoofNNlistPin, int *d_NNlistPin, float3 DLp, float Pshift,bool useRigidBoxZ)
{


	int fullerene = blockIdx.x*blockDim.x+threadIdx.x;


	if ( fullerene < impurityNum )
	{
	  
		int posx = 0;
		int posy = 0;
		int posz = 0;		

	  	
	  	posx = (int)(( CMx[fullerene] - floor( CMx[fullerene] / boxMax.x) * boxMax.x )/DLp.x); 	
 	  	posy = (int)(( CMy[fullerene] - floor( CMy[fullerene] / boxMax.y) * boxMax.y )/DLp.y); 	
	  	posz = (int)(( CMz[fullerene] - floor( CMz[fullerene] / boxMax.z) * boxMax.z )/DLp.z); 		
	 
		int j1 = 0;
	  	int j2 = 0;
	  	int j3 = 0;
	 
	  	for (  int i = -1; i < 2 ; ++i ){
				
			j1 = posx + i;
			if (j1 >= Xdiv || j1 <= -1) continue; 
				 

			for (  int j = -1; j < 2; ++j ){
					
				j2 = posy + j;
				j2 = j2 - floor((float)j2/(float)Ydiv) * Ydiv;	 
	
				for (  int k = -1 ; k < 2; ++k ){
			
			
					j3 = posz + k;
					
					if(useRigidBoxZ){
					
						if(j3 < 0 || j3 > Zdiv-1) continue;
					
					}else{	
					
						j3 = j3 - floor((float)j3/(float)Zdiv) * Zdiv;	 
					
					}

			  		int index = atomicAdd( &d_NoofNNlistPin[j3*Xdiv*Ydiv+j2*Xdiv+j1] , 1); //returns old
#ifdef PRINT_TOO_SHORT_ERROR
			  		if ( index > 32 )
					{
                         			printf("Fullerene %d, NN-listPin too short, atleast %d\n", fullerene, index);
                                  		// for ( int k = 0; k < 32; ++k )
                                  		// printf("%d ",d_NNlist[ 32*(j2*Xdiv+j1) + k]);
                                  		// printf("\n");
						continue;
					}
#endif
			  		d_NNlistPin[ 32*(j3*Xdiv*Ydiv+j2*Xdiv+j1)+index] = fullerene;
					
				}
	
			}
		}
		
				
		if(posx == Xdiv-1 ){
			
			posy = (int)(( (CMy[fullerene] - Pshift) - floor((CMy[fullerene] - Pshift) / boxMax.y) * boxMax.y )/DLp.y);	
			
			j1 = 0;
			j3 = posz;
			if(useRigidBoxZ){
				if ( posz < 0 ) posz = 0;
	  			if ( posz > Zdiv - 1 ) posz = Zdiv - 1;	
			}else{						
				j3 = j3 - floor((float)j3/(float)Zdiv) * Zdiv;	 
			}
				 
			for (  int i = -1; i < 2 ; ++i ){
				
				j2 = posy + i;
				j2 = j2 - floor((float)j2/(float)Ydiv) * Ydiv;
		
				int index = atomicAdd( &d_NoofNNlistPin[j3*Xdiv*Ydiv+j2*Xdiv+j1] , 1);
				d_NNlistPin[ 32*(j3*Xdiv*Ydiv+j2*Xdiv+j1)+index] = fullerene;
		
			}
		}
			
			
			
			
		if (posx == 0){
			
			posy = (int)(( (CMy[fullerene] + Pshift) - floor( (CMy[fullerene] + Pshift) / boxMax.y) * boxMax.y )/DLp.y);	
			
			j1 = Xdiv - 1;
			j3 = posz;
			if(useRigidBoxZ){
				if ( posz < 0 ) posz = 0;
	  			if ( posz > Zdiv - 1 ) posz = Zdiv - 1;	
			}else{						
				j3 = j3 - floor((float)j3/(float)Zdiv) * Zdiv;	 
			}
				
			for (  int i = -1; i < 2 ; ++i ){
				
				j2 = posy + i;
				j2 = j2 - floor((float)j2/(float)Ydiv) * Ydiv;
		
				int index = atomicAdd( &d_NoofNNlistPin[j3*Xdiv*Ydiv+j2*Xdiv+j1] , 1);
				d_NNlistPin[ 32*(j3*Xdiv*Ydiv+j2*Xdiv+j1)+index] = fullerene;
		
			}	
			
				

		}
			

		
		
	}
		

}






__global__ void minmaxpost( int No_of_C180s,
							float *Minx, float *Maxx, float *Miny, float *Maxy,  float *Minz, float *Maxz)
{

  __shared__ float  minx[1024];
  __shared__ float  maxx[1024];
  __shared__ float  miny[1024];
  __shared__ float  maxy[1024];
  __shared__ float  minz[1024];
  __shared__ float  maxz[1024];

  int fullerene = blockIdx.x*blockDim.x+threadIdx.x;
  int tid       = threadIdx.x;

  minx[tid] = +1.0E8f;
  maxx[tid] = -1.0E8f;
  miny[tid] = +1.0E8f;
  maxy[tid] = -1.0E8f;
  minz[tid] = +1.0E8f;
  maxz[tid] = -1.0E8f;

  if ( fullerene < No_of_C180s )
    {
	  minx[tid] = Minx[fullerene];
	  maxx[tid] = Maxx[fullerene];
	  miny[tid] = Miny[fullerene];
	  maxy[tid] = Maxy[fullerene];
	  minz[tid] = Minz[fullerene];
	  maxz[tid] = Maxz[fullerene];
    }

  __syncthreads();

  for ( int s = blockDim.x/2; s > 0; s>>=1)
	{
	  if ( tid < s )
		{
		  minx[tid] = fminf(minx[tid],minx[tid+s]);
		  maxx[tid] = fmaxf(maxx[tid],maxx[tid+s]);
		  miny[tid] = fminf(miny[tid],miny[tid+s]);
		  maxy[tid] = fmaxf(maxy[tid],maxy[tid+s]);
		  minz[tid] = fminf(minz[tid],minz[tid+s]);
		  maxz[tid] = fmaxf(maxz[tid],maxz[tid+s]);
		}
	  __syncthreads();
	}

  if ( tid == 0 )
	{
	  Minx[blockIdx.x+0]  = minx[0];
	  Minx[blockIdx.x+1]  = maxx[0];
	  Minx[blockIdx.x+2]  = miny[0];
	  Minx[blockIdx.x+3]  = maxy[0];
	  Minx[blockIdx.x+4]  = minz[0];
	  Minx[blockIdx.x+5]  = maxz[0];
	}

}
