#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "postscript.h"
#include "VectorFunctions.hpp"

//#define PRINT_TOO_SHORT_ERROR

__global__ void bounding_boxes( int No_of_C180s,
               float *d_XP, float *d_YP, float *d_ZP,
//               float *d_X,  float *d_Y,  float *d_Z,
//               float *d_XM, float *d_YM, float *d_ZM,
               float *d_bounding_xyz, float *CMx,
			   float *CMy, float *CMz)
{
  __shared__ float  minx[32];
  __shared__ float  maxx[32];
  __shared__ float  miny[32];
  __shared__ float  maxy[32];
  __shared__ float  minz[32];
  __shared__ float  maxz[32];

  int rank = blockIdx.x;
  int tid  = threadIdx.x;
  int atom = tid;

  if ( rank < No_of_C180s )
    {
	  minx[tid] = d_XP[rank*192+atom];
	  maxx[tid] = d_XP[rank*192+atom];
	  miny[tid] = d_YP[rank*192+atom];
	  maxy[tid] = d_YP[rank*192+atom];
	  minz[tid] = d_ZP[rank*192+atom];
	  maxz[tid] = d_ZP[rank*192+atom];

	  // // move present value to past value
	  // d_XM[rank*192+atom] =  d_X[rank*192+atom];
	  // d_YM[rank*192+atom] =  d_Y[rank*192+atom];
	  // d_ZM[rank*192+atom] =  d_Z[rank*192+atom];

	  // // move future value to present value
	  // d_X[rank*192+atom] = d_XP[rank*192+atom];
	  // d_Y[rank*192+atom] = d_YP[rank*192+atom];
	  // d_Z[rank*192+atom] = d_ZP[rank*192+atom];


	  while ( atom + 32 < 180 )
        {
		  atom += 32;
		  if ( minx[tid] > d_XP[rank*192+atom] )
		       minx[tid] = d_XP[rank*192+atom];
		  if ( maxx[tid] < d_XP[rank*192+atom] )
		       maxx[tid] = d_XP[rank*192+atom];
		  if ( miny[tid] > d_YP[rank*192+atom] )
		       miny[tid] = d_YP[rank*192+atom];
		  if ( maxy[tid] < d_YP[rank*192+atom] )
		       maxy[tid] = d_YP[rank*192+atom];
		  if ( minz[tid] > d_ZP[rank*192+atom] )
		       minz[tid] = d_ZP[rank*192+atom];
		  if ( maxz[tid] < d_ZP[rank*192+atom] )
		       maxz[tid] = d_ZP[rank*192+atom];

		  // // move present value to past value
		  // d_XM[rank*192+atom] =  d_X[rank*192+atom];
		  // d_YM[rank*192+atom] =  d_Y[rank*192+atom];
		  // d_ZM[rank*192+atom] =  d_Z[rank*192+atom];

		  // // move future value to present value
		  // d_X[rank*192+atom]  = d_XP[rank*192+atom];
		  // d_Y[rank*192+atom]  = d_YP[rank*192+atom];
		  // d_Z[rank*192+atom]  = d_ZP[rank*192+atom];

        }

	  if ( tid < 16 )
        {
		  if ( minx[tid] > minx[tid+16] ) minx[tid] = minx[tid+16];
		  if ( maxx[tid] < maxx[tid+16] ) maxx[tid] = maxx[tid+16];

		  if ( miny[tid] > miny[tid+16] ) miny[tid] = miny[tid+16];
		  if ( maxy[tid] < maxy[tid+16] ) maxy[tid] = maxy[tid+16];

		  if ( minz[tid] > minz[tid+16] ) minz[tid] = minz[tid+16];
		  if ( maxz[tid] < maxz[tid+16] ) maxz[tid] = maxz[tid+16];
        }

	  if ( tid < 8 )
        {
		  if ( minx[tid] > minx[tid+8] ) minx[tid] = minx[tid+8];
		  if ( maxx[tid] < maxx[tid+8] ) maxx[tid] = maxx[tid+8];
		  if ( miny[tid] > miny[tid+8] ) miny[tid] = miny[tid+8];
		  if ( maxy[tid] < maxy[tid+8] ) maxy[tid] = maxy[tid+8];
		  if ( minz[tid] > minz[tid+8] ) minz[tid] = minz[tid+8];
		  if ( maxz[tid] < maxz[tid+8] ) maxz[tid] = maxz[tid+8];
        }

	  if ( tid < 4 )
        {
		  if ( minx[tid] > minx[tid+4] ) minx[tid] = minx[tid+4];
		  if ( maxx[tid] < maxx[tid+4] ) maxx[tid] = maxx[tid+4];
		  if ( miny[tid] > miny[tid+4] ) miny[tid] = miny[tid+4];
		  if ( maxy[tid] < maxy[tid+4] ) maxy[tid] = maxy[tid+4];
		  if ( minz[tid] > minz[tid+4] ) minz[tid] = minz[tid+4];
		  if ( maxz[tid] < maxz[tid+4] ) maxz[tid] = maxz[tid+4];
        }

	  if ( tid < 2 )
        {
		  if ( minx[tid] > minx[tid+2] ) minx[tid] = minx[tid+2];
		  if ( maxx[tid] < maxx[tid+2] ) maxx[tid] = maxx[tid+2];
		  if ( miny[tid] > miny[tid+2] ) miny[tid] = miny[tid+2];
		  if ( maxy[tid] < maxy[tid+2] ) maxy[tid] = maxy[tid+2];
		  if ( minz[tid] > minz[tid+2] ) minz[tid] = minz[tid+2];
		  if ( maxz[tid] < maxz[tid+2] ) maxz[tid] = maxz[tid+2];
        }

	  if ( tid == 0  )
        {
		  if ( minx[0] > minx[1] ) minx[0] = minx[1];
		  d_bounding_xyz[rank*6+0] = minx[0];

		  if ( maxx[0] < maxx[1] ) maxx[0] = maxx[1];
		  d_bounding_xyz[rank*6+1] = maxx[0];

		  if ( miny[0] > miny[1] ) miny[0] = miny[1];
		  d_bounding_xyz[rank*6+2] = miny[0];

		  if ( maxy[0] < maxy[1] ) maxy[0] = maxy[1];
		  d_bounding_xyz[rank*6+3] = maxy[0];

		  if ( minz[0] > minz[1] ) minz[0] = minz[1];
		  d_bounding_xyz[rank*6+4] = minz[0];

		  if ( maxz[0] < maxz[1] ) maxz[0] = maxz[1];
		  d_bounding_xyz[rank*6+5] = maxz[0];
        }

    }

}



__global__ void minmaxpre( int No_of_C180s, float *d_bounding_xyz,
                    float *Minx, float *Maxx,
					float *Miny, float *Maxy,
					float *Minz, float *Maxz)
{

  __shared__ float  minx[1024];
  __shared__ float  maxx[1024];
  __shared__ float  miny[1024];
  __shared__ float  maxy[1024];
  __shared__ float  minz[1024];
  __shared__ float  maxz[1024];

  int fullerene = blockIdx.x*blockDim.x+threadIdx.x;
  int tid       = threadIdx.x;

  minx[tid] = +1.0E8f;
  maxx[tid] = -1.0E8f;
  miny[tid] = +1.0E8f;
  maxy[tid] = -1.0E8f;
  minz[tid] = +1.0E8f;
  maxz[tid] = -1.0E8f;

  if ( fullerene < No_of_C180s )
    {
	  minx[tid] = d_bounding_xyz[6*fullerene+0];
	  maxx[tid] = d_bounding_xyz[6*fullerene+1];
	  miny[tid] = d_bounding_xyz[6*fullerene+2];
	  maxy[tid] = d_bounding_xyz[6*fullerene+3];
	  minz[tid] = d_bounding_xyz[6*fullerene+4];
	  maxz[tid] = d_bounding_xyz[6*fullerene+5];
    }

  __syncthreads();

  for ( int s = blockDim.x/2; s > 0; s>>=1)
	{
	  if ( tid < s )
		{
		  minx[tid] = fminf(minx[tid],minx[tid+s]);
		  maxx[tid] = fmaxf(maxx[tid],maxx[tid+s]);
		  miny[tid] = fminf(miny[tid],miny[tid+s]);
		  maxy[tid] = fmaxf(maxy[tid],maxy[tid+s]);
		  minz[tid] = fminf(minz[tid],minz[tid+s]);
		  maxz[tid] = fmaxf(maxz[tid],maxz[tid+s]);
		}
	  __syncthreads();
	}

  if ( tid == 0 )
	{
	  Minx[blockIdx.x]  = minx[0];
	  Maxx[blockIdx.x]  = maxx[0];
	  Miny[blockIdx.x]  = miny[0];
	  Maxy[blockIdx.x]  = maxy[0];
	  Minz[blockIdx.x]  = minz[0];
	  Maxz[blockIdx.x]  = maxz[0];
	}

}



__global__ void minmaxpost( int No_of_C180s,
							float *Minx, float *Maxx, float *Miny, float *Maxy,  float *Minz, float *Maxz)
{

  __shared__ float  minx[1024];
  __shared__ float  maxx[1024];
  __shared__ float  miny[1024];
  __shared__ float  maxy[1024];
  __shared__ float  minz[1024];
  __shared__ float  maxz[1024];

  int fullerene = blockIdx.x*blockDim.x+threadIdx.x;
  int tid       = threadIdx.x;

  minx[tid] = +1.0E8f;
  maxx[tid] = -1.0E8f;
  miny[tid] = +1.0E8f;
  maxy[tid] = -1.0E8f;
  minz[tid] = +1.0E8f;
  maxz[tid] = -1.0E8f;

  if ( fullerene < No_of_C180s )
    {
	  minx[tid] = Minx[fullerene];
	  maxx[tid] = Maxx[fullerene];
	  miny[tid] = Miny[fullerene];
	  maxy[tid] = Maxy[fullerene];
	  minz[tid] = Minz[fullerene];
	  maxz[tid] = Maxz[fullerene];
    }

  __syncthreads();

  for ( int s = blockDim.x/2; s > 0; s>>=1)
	{
	  if ( tid < s )
		{
		  minx[tid] = fminf(minx[tid],minx[tid+s]);
		  maxx[tid] = fmaxf(maxx[tid],maxx[tid+s]);
		  miny[tid] = fminf(miny[tid],miny[tid+s]);
		  maxy[tid] = fmaxf(maxy[tid],maxy[tid+s]);
		  minz[tid] = fminf(minz[tid],minz[tid+s]);
		  maxz[tid] = fmaxf(maxz[tid],maxz[tid+s]);
		}
	  __syncthreads();
	}

  if ( tid == 0 )
	{
	  Minx[blockIdx.x+0]  = minx[0];
	  Minx[blockIdx.x+1]  = maxx[0];
	  Minx[blockIdx.x+2]  = miny[0];
	  Minx[blockIdx.x+3]  = maxy[0];
	  Minx[blockIdx.x+4]  = minz[0];
	  Minx[blockIdx.x+5]  = maxz[0];
	}

}



__global__ void makeNNlist(int No_of_C180s, float *CMx, float *CMy,float *CMz, float *CMxNNlist, float *CMyNNlist, float *CMzNNlist,
                           int Xdiv, int Ydiv, int Zdiv, float3 BoxMin,
                           int *d_NoofNNlist, int *d_NNlist, float DL)
{


	int fullerene = blockIdx.x*blockDim.x+threadIdx.x;
//  printf("(%d, %d, %d) %d %d\n", blockIdx.x, blockDim.x, threadIdx.x, fullerene, No_of_C180s);


	if ( fullerene < No_of_C180s )
	{
	  
		int posx = 0;
		int posy = 0;
		int posz = 0;		
		

	 	posx = (int)((CMx[fullerene] - BoxMin.x)/DL);
	  	if ( posx < 0 ) posx = 0;
	  	if ( posx > Xdiv - 1 ) posx = Xdiv - 1;
	  	

	  	posy = (int)((CMy[fullerene]-BoxMin.y)/DL);
	  	if ( posy < 0 ) posy = 0;
	  	if ( posy > Ydiv - 1 ) posy = Ydiv - 1;

	   	posz = (int)((CMz[fullerene]-BoxMin.z)/DL);
	  	if ( posz < 0 ) posz = 0;
	  	if ( posz > Zdiv - 1 ) posz = Zdiv - 1;
	  	
	 
		int j1 = 0;
	  	int j2 = 0;
	  	int j3 = 0;
	 
	  	for (  int i = -1; i < 2 ; ++i ){
				
			j1 = posx + i;
			if(j1 < 0 || j1 > Xdiv-1) continue;
			

			for (  int j = -1; j < 2; ++j ){
					
				j2 = posy + j;
				if(j2 < 0 || j2 > Ydiv-1) continue;
				
	
				for (  int k = -1 ; k < 2; ++k ){
			
					j3 = posz + k;
					if(j3 < 0 || j3 > Zdiv-1) continue;
		

			  		int index = atomicAdd( &d_NoofNNlist[j3*Xdiv*Ydiv+j2*Xdiv+j1] , 1); //returns old
#ifdef PRINT_TOO_SHORT_ERROR
			  		if ( index > 64 )
					{
                         			printf("Fullerene %d, NN-list too short, atleast %d\n", fullerene, index);
                                  			// for ( int k = 0; k < 32; ++k )
                                  			//     printf("%d ",d_NNlist[ 32*(j2*Xdiv+j1) + k]); 
                                 			// printf("\n");
						 continue;
					}
#endif
			  		d_NNlist[ 64*(j3*Xdiv*Ydiv+j2*Xdiv+j1)+index] = fullerene;
					
				}
	
			}
		}	
		
	
		
		CMxNNlist[fullerene] = CMx[fullerene];
		CMyNNlist[fullerene] = CMy[fullerene];
		CMzNNlist[fullerene] = CMz[fullerene];
	
	
	}

}

__global__ void makeNNlistMultiGpu( int No_of_C180s, float *d_CMx, float *d_CMy,float *d_CMz, float *CMxNNlist, float *CMyNNlist, float *CMzNNlist,
                           int Xdiv, int Ydiv, int Zdiv, float3 Subdivision_min, float3 Subdivision_max, float3 BoxMin, float3 boxMax,
                           int *d_NoofNNlist, int *d_NNlist, float DL, int* d_counter_gc_e, int* d_counter_gc_w,
                           int* d_counter_gc_n, int* d_counter_gc_s, int* d_counter_gc_u, int* d_counter_gc_d,
                           int* d_Ghost_Cells_ind_EAST, int* d_Ghost_Cells_ind_WEST, int* d_Ghost_Cells_ind_NORTH, int* d_Ghost_Cells_ind_SOUTH,
                           int* d_Ghost_Cells_ind_UP, int* d_Ghost_Cells_ind_DOWN )
{


	int fullerene = blockIdx.x*blockDim.x+threadIdx.x;
//  printf("(%d, %d, %d) %d %d\n", blockIdx.x, blockDim.x, threadIdx.x, fullerene, No_of_C180s);


	if ( fullerene < No_of_C180s )
	{	
		
		
		float Cx = d_CMx[fullerene];
		float Cy = d_CMy[fullerene];
		float Cz = d_CMz[fullerene];
		
	 	int posx = (int)((Cx - Subdivision_min.x)/DL);
	  	if ( posx < 0 ) posx = 0;
	  	if ( posx > Xdiv - 1 ) posx = Xdiv - 1;
	  	

	  	int posy = (int)((Cy - Subdivision_min.y)/DL);
	  	if ( posy < 0 ) posy = 0;
	  	if ( posy > Ydiv - 1 ) posy = Ydiv - 1;

	   	int posz = (int)((Cz - Subdivision_min.z)/DL);
	  	if ( posz < 0 ) posz = 0;
	  	if ( posz > Zdiv - 1 ) posz = Zdiv - 1;
	  	 
	 
	 
	 		
	 	//if (Subdivision_max.x < boxMax.x) {	
	 		
	 	if( Cx >=  Subdivision_max.x - 2.0){
	 			
	 		int index = atomicAdd(d_counter_gc_e,1);
	 		d_Ghost_Cells_ind_EAST[index] = fullerene;	 			
	 	}
	 	
	 	//}
	 	
	 	//if ( BoxMin.x < Subdivision_min.x){
	 	
	 	if( Cx <=  Subdivision_min.x + 2.0 ){
	 			
	 		int index = atomicAdd(d_counter_gc_w,1);
	 		d_Ghost_Cells_ind_WEST[index] = fullerene;
	 			//if (rank == 1) printf("Xpos is:	%f\n", Cx);
	 	}
	 	
	 	//}
	 	
	 	//if (Subdivision_max.y < boxMax.y) {	
	 		
	 	if( Cy >=  Subdivision_max.y - 2.0 ){
	 			
	 		int index = atomicAdd(d_counter_gc_n,1);
	 		d_Ghost_Cells_ind_NORTH[index] = fullerene;	 			
	 	}
	 	//}
	 	
	 	
	 	//if ( BoxMin.y < Subdivision_min.y){
	 		
	 	if( Cy <=  Subdivision_min.y + 2.0 ){
	 			
	 		int index = atomicAdd(d_counter_gc_s,1);
	 		d_Ghost_Cells_ind_SOUTH[index] = fullerene;
	 	}
	 	//}
	 	
	 	//if (Subdivision_max.z < boxMax.z) {	
	 		
	 	if( Cz >=  Subdivision_max.z - 2.0 ){
	 			
	 		int index = atomicAdd(d_counter_gc_u,1);
	 		d_Ghost_Cells_ind_UP[index] = fullerene;
	 				 			
	 	}
	 	//}
	 	
	 	//if ( BoxMin.z < Subdivision_min.z){
	 	if( Cz <= Subdivision_min.z + 2.0 ){
	 			
	 		int index = atomicAdd(d_counter_gc_d,1);
	 		d_Ghost_Cells_ind_DOWN[index] = fullerene;
	 	}	
	 	//}
	 	
	 	
	 
		int j1 = 0;
	  	int j2 = 0;
	  	int j3 = 0;
	 
	  	for (  int i = -1; i < 2 ; ++i ){
				
			j1 = posx + i;
			if(j1 < 0 || j1 > Xdiv-1) continue;
			

			for (  int j = -1; j < 2; ++j ){
					
				j2 = posy + j;
				if(j2 < 0 || j2 > Ydiv-1) continue;
				
	
				for (  int k = -1 ; k < 2; ++k ){
			
					j3 = posz + k;
					if(j3 < 0 || j3 > Zdiv-1) continue;
		

			  		int index = atomicAdd( &d_NoofNNlist[j3*Xdiv*Ydiv+j2*Xdiv+j1] , 1); //returns old
#ifdef PRINT_TOO_SHORT_ERROR
			  		if ( index > 64 )
					{
                         			printf("Fullerene %d, NN-list too short, atleast %d\n", fullerene, index);
                                  			// for ( int k = 0; k < 32; ++k )
                                  			//     printf("%d ",d_NNlist[ 32*(j2*Xdiv+j1) + k]); 
                                 			// printf("\n");
						 continue;
					}
#endif
			  		d_NNlist[ 64*(j3*Xdiv*Ydiv+j2*Xdiv+j1)+index] = fullerene;
					
				}
	
			}
		}	
			
		
		CMxNNlist[fullerene] = d_CMx[fullerene];
		CMyNNlist[fullerene] = d_CMy[fullerene];
		CMzNNlist[fullerene] = d_CMz[fullerene];
	
	
	}

}




__global__ void makeNNlistPBC(int No_of_C180s, float *CMx, float *CMy,float *CMz, float *CMxNNlist, float *CMyNNlist, float *CMzNNlist,
                           float attrac, int Xdiv, int Ydiv, int Zdiv, float3 boxMax,
                           int *d_NoofNNlist, int *d_NNlist, float3 DLp, bool useRigidBoxZ, bool useRigidBoxY)
{


	int fullerene = blockIdx.x*blockDim.x+threadIdx.x;
//  printf("(%d, %d, %d) %d %d\n", blockIdx.x, blockDim.x, threadIdx.x, fullerene, No_of_C180s);


	if ( fullerene < No_of_C180s )
	{
	  
		int posx = 0;
		int posy = 0;
		int posz = 0;	
		float boxX = boxMax.x;
		float boxY = boxMax.y;
		float boxZ = boxMax.z;	
		float DX = DLp.x;
		float DY = DLp.y;
		float DZ = DLp.z;	

	  	
	  	posx = (int)(( CMx[fullerene] - floor( CMx[fullerene] / boxX) * boxX )/DX); 	
 	  	posy = (int)(( CMy[fullerene] - floor( CMy[fullerene] / boxY) * boxY )/DY); 	
	  	posz = (int)(( CMz[fullerene] - floor( CMz[fullerene] / boxZ) * boxZ )/DZ); 		
	 
		int j1 = 0;
	  	int j2 = 0;
	  	int j3 = 0;
	 
	  	for (  int i = -1; i < 2 ; ++i ){
				
			j1 = posx + i;
			j1 = j1 - floor((float)j1/(float)Xdiv) * Xdiv;	 

			for (  int j = -1; j < 2; ++j ){ 


				j2 = posy + j;
					
				if(useRigidBoxY){
					
					if(j2 < 0 || j2 > Ydiv-1) continue;
					
				}else{	
					
					j2 = j2 - floor((float)j2/(float)Ydiv) * Ydiv;	 
					
				}
	
				for (  int k = -1 ; k < 2; ++k ){
			
			
					j3 = posz + k;
					
					if(useRigidBoxZ){
					
						if(j3 < 0 || j3 > Zdiv-1) continue;
					
					}else{	
					
						j3 = j3 - floor((float)j3/(float)Zdiv) * Zdiv;	 
					
					}

			  		int index = atomicAdd( &d_NoofNNlist[j3*Xdiv*Ydiv+j2*Xdiv+j1] , 1); //returns old
#ifdef PRINT_TOO_SHORT_ERROR
			  		if ( index > 64 )
					{
                         			printf("Fullerene %d, NN-list too short, atleast %d\n", fullerene, index);
                                  		// for ( int k = 0; k < 32; ++k )
                                  		// printf("%d ",d_NNlist[ 32*(j2*Xdiv+j1) + k]);
                                  		// printf("\n");
						continue;
					}
#endif
			  		d_NNlist[ 64*(j3*Xdiv*Ydiv+j2*Xdiv+j1)+index] = fullerene;
					
				}
	
			}
		}	

		
		CMxNNlist[fullerene] = CMx[fullerene];
		CMyNNlist[fullerene] = CMy[fullerene];
		CMzNNlist[fullerene] = CMz[fullerene];
		
		
	}
		
}



__global__ void makeNNlistLEbc(int No_of_C180s, float *CMx, float *CMy,float *CMz, float *CMxNNlist, float *CMyNNlist, float *CMzNNlist,
                           float attrac, int Xdiv, int Ydiv, int Zdiv, float3 boxMax,
                           int *d_NoofNNlist, int *d_NNlist, float3 DLp, float Pshift,bool useRigidBoxZ)
{


	int fullerene = blockIdx.x*blockDim.x+threadIdx.x;
//  printf("(%d, %d, %d) %d %d\n", blockIdx.x, blockDim.x, threadIdx.x, fullerene, No_of_C180s);


	if ( fullerene < No_of_C180s )
	{
	  
		int posx = 0;
		int posy = 0;
		int posz = 0;		

	  	
	  	posx = (int)(( CMx[fullerene] - floor( CMx[fullerene] / boxMax.x) * boxMax.x )/DLp.x); 	
 	  	posy = (int)(( CMy[fullerene] - floor( CMy[fullerene] / boxMax.y) * boxMax.y )/DLp.y); 	
	  	posz = (int)(( CMz[fullerene] - floor( CMz[fullerene] / boxMax.z) * boxMax.z )/DLp.z); 		
	 
		int j1 = 0;
	  	int j2 = 0;
	  	int j3 = 0;
	 
	  	for (  int i = -1; i < 2 ; ++i ){
				
			j1 = posx + i;
			if (j1 >= Xdiv || j1 <= -1) continue; 
				 

			for (  int j = -1; j < 2; ++j ){
					
				j2 = posy + j;
				j2 = j2 - floor((float)j2/(float)Ydiv) * Ydiv;	 
	
				for (  int k = -1 ; k < 2; ++k ){
			
			
					j3 = posz + k;
					
					if(useRigidBoxZ){
					
						if(j3 < 0 || j3 > Zdiv-1) continue;
					
					}else{	
					
						j3 = j3 - floor((float)j3/(float)Zdiv) * Zdiv;	 
					
					}

			  		int index = atomicAdd( &d_NoofNNlist[j3*Xdiv*Ydiv+j2*Xdiv+j1] , 1); //returns old
#ifdef PRINT_TOO_SHORT_ERROR
			  		if ( index > 64 )
					{
                         			printf("Fullerene %d, NN-list too short, atleast %d\n", fullerene, index);
                                  		// for ( int k = 0; k < 32; ++k )
                                  		// printf("%d ",d_NNlist[ 32*(j2*Xdiv+j1) + k]);
                                  		// printf("\n");
						continue;
					}
#endif
			  		d_NNlist[ 64*(j3*Xdiv*Ydiv+j2*Xdiv+j1)+index] = fullerene;
					
				}
	
			}
		}
		
				
		if(posx == Xdiv-1 ){
			
			posy = (int)(( (CMy[fullerene] - Pshift) - floor((CMy[fullerene] - Pshift) / boxMax.y) * boxMax.y )/DLp.y);	
			
			j1 = 0;
			j3 = posz;
			if(useRigidBoxZ){
				if ( posz < 0 ) posz = 0;
	  			if ( posz > Zdiv - 1 ) posz = Zdiv - 1;	
			}else{						
				j3 = j3 - floor((float)j3/(float)Zdiv) * Zdiv;	 
			}
				 
			for (  int i = -1; i < 2 ; ++i ){
				
				j2 = posy + i;
				j2 = j2 - floor((float)j2/(float)Ydiv) * Ydiv;
		
				int index = atomicAdd( &d_NoofNNlist[j3*Xdiv*Ydiv+j2*Xdiv+j1] , 1);
				d_NNlist[ 64*(j3*Xdiv*Ydiv+j2*Xdiv+j1)+index] = fullerene;
		
			}
		}
			
			
			
			
		if (posx == 0){
			
			posy = (int)(( (CMy[fullerene] + Pshift) - floor( (CMy[fullerene] + Pshift) / boxMax.y) * boxMax.y )/DLp.y);	
			
			j1 = Xdiv - 1;
			j3 = posz;
			if(useRigidBoxZ){
				if ( posz < 0 ) posz = 0;
	  			if ( posz > Zdiv - 1 ) posz = Zdiv - 1;	
			}else{						
				j3 = j3 - floor((float)j3/(float)Zdiv) * Zdiv;	 
			}
				
			for (  int i = -1; i < 2 ; ++i ){
				
				j2 = posy + i;
				j2 = j2 - floor((float)j2/(float)Ydiv) * Ydiv;
		
				int index = atomicAdd( &d_NoofNNlist[j3*Xdiv*Ydiv+j2*Xdiv+j1] , 1);
				d_NNlist[ 64*(j3*Xdiv*Ydiv+j2*Xdiv+j1)+index] = fullerene;
		
			}	
			
				

		}
			
		
		CMxNNlist[fullerene] = CMx[fullerene];
		CMyNNlist[fullerene] = CMy[fullerene];
		CMzNNlist[fullerene] = CMz[fullerene];
		
		
	}
		
}


// Pinning
__global__ void makeNNlistPin(int impurityNum, float *CMx, float *CMy,float *CMz,
                           int Xdiv, int Ydiv, int Zdiv, float3 BoxMin,
                           int *d_NoofNNlistPin, int *d_NNlistPin, float DL)
{


	int fullerene = blockIdx.x*blockDim.x+threadIdx.x;


	if ( fullerene < impurityNum )
	{
	  
		int posx = 0;
		int posy = 0;
		int posz = 0;		


	 	posx = (int)((CMx[fullerene] - BoxMin.x)/DL);
	  	if ( posx < 0 ) posx = 0;
	  	if ( posx > Xdiv - 1 ) posx = Xdiv - 1;
	  	

	  	posy = (int)((CMy[fullerene]-BoxMin.y)/DL);
	  	if ( posy < 0 ) posy = 0;
	  	if ( posy > Ydiv - 1 ) posy = Ydiv - 1;

	   	posz = (int)((CMz[fullerene]-BoxMin.z)/DL);
	  	if ( posz < 0 ) posz = 0;
	  	if ( posz > Zdiv - 1 ) posz = Zdiv - 1;
	  	
	 
		int j1 = 0;
	  	int j2 = 0;
	  	int j3 = 0;
	 
	  	for (  int i = -1; i < 2 ; ++i ){
				
			j1 = posx + i;
			if(j1 < 0 || j1 > Xdiv-1) continue;
			

			for (  int j = -1; j < 2; ++j ){
					
				j2 = posy + j;
				if(j2 < 0 || j2 > Ydiv-1) continue;
				
	
				for (  int k = -1 ; k < 2; ++k ){
			
					j3 = posz + k;
					if(j3 < 0 || j3 > Zdiv-1) continue;
		

			  		int index = atomicAdd( &d_NoofNNlistPin[j3*Xdiv*Ydiv+j2*Xdiv+j1] , 1); //returns old
#ifdef PRINT_TOO_SHORT_ERROR
			  		if ( index > 32 )
					{
                         			printf("Fullerene %d, NN-listPin too short, atleast %d\n", fullerene, index);
                                  			// for ( int k = 0; k < 32; ++k )
                                  			//     printf("%d ",d_NNlist[ 32*(j2*Xdiv+j1) + k]); 
                                 			// printf("\n");
						 continue;
					}
#endif
			  		d_NNlistPin[ 32*(j3*Xdiv*Ydiv+j2*Xdiv+j1)+index] = fullerene;
					
				}
	
			}
		}	
		
	}

}




__global__ void makeNNlistPBCPin(int impurityNum, float *CMx, float *CMy,float *CMz,
                           float attrac, int Xdiv, int Ydiv, int Zdiv, float3 boxMax,
                           int *d_NoofNNlistPin, int *d_NNlistPin, float3 DLp, bool useRigidBoxZ, bool useRigidBoxY)
{


	int fullerene = blockIdx.x*blockDim.x+threadIdx.x;
//  printf("(%d, %d, %d) %d %d\n", blockIdx.x, blockDim.x, threadIdx.x, fullerene, No_of_C180s);


	if ( fullerene < impurityNum )
	{
	  
		int posx = 0;
		int posy = 0;
		int posz = 0;	
		float boxX = boxMax.x;
		float boxY = boxMax.y;
		float boxZ = boxMax.z;	
		float DX = DLp.x;
		float DY = DLp.y;
		float DZ = DLp.z;	

	  	
	  	posx = (int)(( CMx[fullerene] - floor( CMx[fullerene] / boxX) * boxX )/DX); 	
 	  	posy = (int)(( CMy[fullerene] - floor( CMy[fullerene] / boxY) * boxY )/DY); 	
	  	posz = (int)(( CMz[fullerene] - floor( CMz[fullerene] / boxZ) * boxZ )/DZ); 		
	 
		int j1 = 0;
	  	int j2 = 0;
	  	int j3 = 0;
	 
	  	for (  int i = -1; i < 2 ; ++i ){
				
			j1 = posx + i;
			j1 = j1 - floor((float)j1/(float)Xdiv) * Xdiv;	 

			for (  int j = -1; j < 2; ++j ){ 


				j2 = posy + j;
					
				if(useRigidBoxY){
					
					if(j2 < 0 || j2 > Ydiv-1) continue;
					
				}else{	
					
					j2 = j2 - floor((float)j2/(float)Ydiv) * Ydiv;	 
					
				}
	
				for (  int k = -1 ; k < 2; ++k ){
			
			
					j3 = posz + k;
					
					if(useRigidBoxZ){
					
						if(j3 < 0 || j3 > Zdiv-1) continue;
					
					}else{	
					
						j3 = j3 - floor((float)j3/(float)Zdiv) * Zdiv;	 
					
					}

			  		int index = atomicAdd( &d_NoofNNlistPin[j3*Xdiv*Ydiv+j2*Xdiv+j1] , 1); //returns old
#ifdef PRINT_TOO_SHORT_ERROR
			  		if ( index > 32 )
					{
                         			printf("Fullerene %d, NN-listPin too short, atleast %d\n", fullerene, index);
                                  		// for ( int k = 0; k < 32; ++k )
                                  		// printf("%d ",d_NNlist[ 32*(j2*Xdiv+j1) + k]);
                                  		// printf("\n");
						continue;
					}
#endif
			  		d_NNlistPin[ 32*(j3*Xdiv*Ydiv+j2*Xdiv+j1)+index] = fullerene;
					
				}
	
			}
		}	

		
		
	}
		

}



__global__ void makeNNlistLEbcPin(int impurityNum, float *CMx, float *CMy,float *CMz,
                           float attrac, int Xdiv, int Ydiv, int Zdiv, float3 boxMax,
                           int *d_NoofNNlistPin, int *d_NNlistPin, float3 DLp, float Pshift,bool useRigidBoxZ)
{


	int fullerene = blockIdx.x*blockDim.x+threadIdx.x;


	if ( fullerene < impurityNum )
	{
	  
		int posx = 0;
		int posy = 0;
		int posz = 0;		

	  	
	  	posx = (int)(( CMx[fullerene] - floor( CMx[fullerene] / boxMax.x) * boxMax.x )/DLp.x); 	
 	  	posy = (int)(( CMy[fullerene] - floor( CMy[fullerene] / boxMax.y) * boxMax.y )/DLp.y); 	
	  	posz = (int)(( CMz[fullerene] - floor( CMz[fullerene] / boxMax.z) * boxMax.z )/DLp.z); 		
	 
		int j1 = 0;
	  	int j2 = 0;
	  	int j3 = 0;
	 
	  	for (  int i = -1; i < 2 ; ++i ){
				
			j1 = posx + i;
			if (j1 >= Xdiv || j1 <= -1) continue; 
				 

			for (  int j = -1; j < 2; ++j ){
					
				j2 = posy + j;
				j2 = j2 - floor((float)j2/(float)Ydiv) * Ydiv;	 
	
				for (  int k = -1 ; k < 2; ++k ){
			
			
					j3 = posz + k;
					
					if(useRigidBoxZ){
					
						if(j3 < 0 || j3 > Zdiv-1) continue;
					
					}else{	
					
						j3 = j3 - floor((float)j3/(float)Zdiv) * Zdiv;	 
					
					}

			  		int index = atomicAdd( &d_NoofNNlistPin[j3*Xdiv*Ydiv+j2*Xdiv+j1] , 1); //returns old
#ifdef PRINT_TOO_SHORT_ERROR
			  		if ( index > 32 )
					{
                         			printf("Fullerene %d, NN-listPin too short, atleast %d\n", fullerene, index);
                                  		// for ( int k = 0; k < 32; ++k )
                                  		// printf("%d ",d_NNlist[ 32*(j2*Xdiv+j1) + k]);
                                  		// printf("\n");
						continue;
					}
#endif
			  		
			  		d_NNlistPin[ 32*(j3*Xdiv*Ydiv+j2*Xdiv+j1)+index] = fullerene;
					
				}
	
			}
		}
		
				
		if(posx == Xdiv-1 ){
			
			posy = (int)(( (CMy[fullerene] - Pshift) - floor((CMy[fullerene] - Pshift) / boxMax.y) * boxMax.y )/DLp.y);	
			
			j1 = 0;
			j3 = posz;
			if(useRigidBoxZ){
				if ( posz < 0 ) posz = 0;
	  			if ( posz > Zdiv - 1 ) posz = Zdiv - 1;	
			}else{						
				j3 = j3 - floor((float)j3/(float)Zdiv) * Zdiv;	 
			}
				 
			for (  int i = -1; i < 2 ; ++i ){
				
				j2 = posy + i;
				j2 = j2 - floor((float)j2/(float)Ydiv) * Ydiv;
		
				int index = atomicAdd( &d_NoofNNlistPin[j3*Xdiv*Ydiv+j2*Xdiv+j1] , 1);
				d_NNlistPin[ 32*(j3*Xdiv*Ydiv+j2*Xdiv+j1)+index] = fullerene;
		
			}
		}
			
			
			
			
		if (posx == 0){
			
			posy = (int)(( (CMy[fullerene] + Pshift) - floor( (CMy[fullerene] + Pshift) / boxMax.y) * boxMax.y )/DLp.y);	
			
			j1 = Xdiv - 1;
			j3 = posz;
			if(useRigidBoxZ){
				if ( posz < 0 ) posz = 0;
	  			if ( posz > Zdiv - 1 ) posz = Zdiv - 1;	
			}else{						
				j3 = j3 - floor((float)j3/(float)Zdiv) * Zdiv;	 
			}
				
			for (  int i = -1; i < 2 ; ++i ){
				
				j2 = posy + i;
				j2 = j2 - floor((float)j2/(float)Ydiv) * Ydiv;
		
				int index = atomicAdd( &d_NoofNNlistPin[j3*Xdiv*Ydiv+j2*Xdiv+j1] , 1);
				d_NNlistPin[ 32*(j3*Xdiv*Ydiv+j2*Xdiv+j1)+index] = fullerene;
		
			}	
			
				

		}
			

		
		
	}
		

}


__global__ void DangerousParticlesFinder(int No_of_C180s, float *CMx, float *CMy,float *CMz,
					  float *CMxNNlist, float *CMyNNlist, float *CMzNNlist,
					  float BufferDistance, int *num_cell_dang, int* cell_dang_inds, char* cell_dang,
					  float3 boxMax)
{


	int fullerene = blockIdx.x*blockDim.x+threadIdx.x;


	if ( fullerene < No_of_C180s )
	{
		
		if( cell_dang[fullerene] == 0){
			
			float deltaX, deltaY, deltaZ;
			float R;
		
		
			deltaX = CMxNNlist[fullerene] - CMx[fullerene]; 
			deltaY = CMyNNlist[fullerene] - CMy[fullerene];
			deltaZ = CMzNNlist[fullerene] - CMz[fullerene];
		
			R  = deltaX*deltaX+deltaY*deltaY+deltaZ*deltaZ;
		
			if (R >= BufferDistance){
			
				cell_dang[fullerene] = 1;
				int index = atomicAdd(&num_cell_dang[0],1);   
				cell_dang_inds[index] = fullerene;   
		
			}
		}


	}

}


__global__ void DangerousParticlesFinderPBC(int No_of_C180s, float *CMx, float *CMy,float *CMz,
					  float *CMxNNlist, float *CMyNNlist, float *CMzNNlist,
					  float BufferDistance, int *num_cell_dang, int* cell_dang_inds, char* cell_dang,
					  float3 boxMax, bool useRigidBoxZ, bool useRigidBoxY)
{


	int fullerene = blockIdx.x*blockDim.x+threadIdx.x;


	if ( fullerene < No_of_C180s )
	{
		
		if( cell_dang[fullerene] == 0){
			
			float deltaX, deltaY, deltaZ;
			float R;
		
		
			deltaX = CMxNNlist[fullerene] - CMx[fullerene];
			deltaX = deltaX - nearbyint( deltaX / boxMax.x) * boxMax.x;
			 
			deltaY = CMyNNlist[fullerene] - CMy[fullerene];
			if (!useRigidBoxY)deltaY = deltaY - nearbyint( deltaY / boxMax.y) * boxMax.y;
			
			deltaZ = CMzNNlist[fullerene] - CMz[fullerene];
			if (!useRigidBoxZ) deltaZ = deltaZ - nearbyint( deltaZ / boxMax.z) * boxMax.z;
			
		
			R  = deltaX*deltaX+deltaY*deltaY+deltaZ*deltaZ;
		
			
			if (R >= BufferDistance){
			
				cell_dang[fullerene] = 1;
				int index = atomicAdd(&num_cell_dang[0],1);   
				cell_dang_inds[index] = fullerene; 
				//printf("cell %d\n",fullerene);  
		
			}
		}


	}

}


__global__ void DangerousParticlesFinderLEbc(int No_of_C180s, float *CMx, float *CMy,float *CMz,
					  float *CMxNNlist, float *CMyNNlist, float *CMzNNlist,
					  float BufferDistance, int *num_cell_dang, int* cell_dang_inds, char* cell_dang,
					  float3 boxMax, bool useRigidBoxZ, bool useRigidBoxY )
{


	int fullerene = blockIdx.x*blockDim.x+threadIdx.x;


	if ( fullerene < No_of_C180s )
	{
		
		if( cell_dang[fullerene] == 0){
			
			float deltaX, deltaY, deltaZ;
			float R;
		
		
			if ( abs(CMx[fullerene] - CMxNNlist[fullerene]) > boxMax.x/2 ){
				
				cell_dang[fullerene] = 1;
				int index = atomicAdd(&num_cell_dang[0],1);   
				cell_dang_inds[index] = fullerene; 
				
			
			} else{
		
			deltaX = CMxNNlist[fullerene] - CMx[fullerene];
			
			deltaY = CMyNNlist[fullerene] - CMy[fullerene];
			deltaY = deltaY - nearbyint( deltaY / boxMax.y) * boxMax.y;
			
			deltaZ = CMzNNlist[fullerene] - CMz[fullerene];
			if (!useRigidBoxZ) deltaZ = deltaZ - nearbyint( deltaZ / boxMax.z) * boxMax.z;
			 
						    			
            		R  = deltaX*deltaX+deltaY*deltaY+deltaZ*deltaZ;
        	
            		if (R >= BufferDistance){
					
					
				cell_dang[fullerene] = 1;
				int index = atomicAdd(&num_cell_dang[0],1);   
				cell_dang_inds[index] = fullerene;   
		
			}
            		
            		
            		}
			
			
		}


	}

}



__global__ void UpdateNNlistDivision(int No_of_C180s, int non_divided_cells, float *CMx, float *CMy,float *CMz, float *CMxNNlist, float *CMyNNlist, float *CMzNNlist,
                           		int Xdiv, int Ydiv, int Zdiv, float3 BoxMin,
                           		int *d_NoofNNlist, int *d_NNlist, float DL)
{


    	int fullerene  = non_divided_cells + blockIdx.x*blockDim.x+threadIdx.x;

	if ( fullerene < No_of_C180s )
	{  
		
		int posx = 0;
		int posy = 0;
		int posz = 0;		
		

		posx = (int)((CMx[fullerene] - BoxMin.x)/DL);
		if ( posx < 0 ) posx = 0;
		if ( posx > Xdiv - 1 ) posx = Xdiv - 1;
	  	

	 	posy = (int)((CMy[fullerene]-BoxMin.y)/DL);
	 	if ( posy < 0 ) posy = 0;
	 	if ( posy > Ydiv - 1 ) posy = Ydiv - 1;

	 	posz = (int)((CMz[fullerene]-BoxMin.z)/DL);
	 	if ( posz < 0 ) posz = 0;
	 	if ( posz > Zdiv - 1 ) posz = Zdiv - 1;
	  	
	 
	 	int j1 = 0;
	 	int j2 = 0;
	 	int j3 = 0;
	 
	 	for (  int i = -1; i < 2 ; ++i ){
				
			j1 = posx + i;
			if(j1 < 0 || j1 > Xdiv-1) continue;
			

			for (  int j = -1; j < 2; ++j ){
					
				j2 = posy + j;
				if(j2 < 0 || j2 > Ydiv-1) continue;
				
	
				for (  int k = -1 ; k < 2; ++k ){
			
					j3 = posz + k;
					if(j3 < 0 || j3 > Zdiv-1) continue;
		

			  		int index = atomicAdd( &d_NoofNNlist[j3*Xdiv*Ydiv+j2*Xdiv+j1] , 1); //returns old
#ifdef PRINT_TOO_SHORT_ERROR
			  		if ( index > 64 )
					{
					        printf("Fullerene %d, NN-list too short, atleast %d\n", fullerene, index);
               	                   	// for ( int k = 0; k < 32; ++k )
               	                   	//     printf("%d ",d_NNlist[ 32*(j2*Xdiv+j1) + k]); 
               	                  	// printf("\n");
						 continue;
					}
#endif
				  	d_NNlist[ 64*(j3*Xdiv*Ydiv+j2*Xdiv+j1)+index] = fullerene;
					
				}
	
			}
		}	
		
	
		
		CMxNNlist[fullerene] = CMx[fullerene];
		CMyNNlist[fullerene] = CMy[fullerene];
		CMzNNlist[fullerene] = CMz[fullerene];
	
	}

}

__global__ void UpdateNNlistDivisionPBC(int No_of_C180s, int non_divided_cells, float *CMx, float *CMy,float *CMz, float *CMxNNlist, float *CMyNNlist, float *CMzNNlist,
                           		 int Xdiv, int Ydiv, int Zdiv, float3 boxMax,
                           		int *d_NoofNNlist, int *d_NNlist, float3 DLp, bool useRigidBoxZ, bool useRigidBoxY)
{


    	int fullerene  = non_divided_cells + blockIdx.x*blockDim.x+threadIdx.x;
    	
	if ( fullerene < No_of_C180s )
	{ 
		int posx = 0;
		int posy = 0;
		int posz = 0;		

	  	
		posx = (int)(( CMx[fullerene] - floor( CMx[fullerene] / boxMax.x) * boxMax.x )/DLp.x); 	
 		posy = (int)(( CMy[fullerene] - floor( CMy[fullerene] / boxMax.y) * boxMax.y )/DLp.y); 	
		posz = (int)(( CMz[fullerene] - floor( CMz[fullerene] / boxMax.z) * boxMax.z )/DLp.z); 		
	 
		int j1 = 0;
		int j2 = 0;
		int j3 = 0;
	 
		for (  int i = -1; i < 2 ; ++i ){
				
			j1 = posx + i;
			j1 = j1 - floor((float)j1/(float)Xdiv) * Xdiv;	 

			for (  int j = -1; j < 2; ++j ){ 


				j2 = posy + j;
						
				if(useRigidBoxY){
						
					if(j2 < 0 || j2 > Ydiv-1) continue;
					
				}else{	
					
					j2 = j2 - floor((float)j2/(float)Ydiv) * Ydiv;	 
					
				}
	
				for (  int k = -1 ; k < 2; ++k ){
			
			
					j3 = posz + k;
						
					if(useRigidBoxZ){
					
						if(j3 < 0 || j3 > Zdiv-1) continue;
					
					}else{	
					
						j3 = j3 - floor((float)j3/(float)Zdiv) * Zdiv;	 
					
					}

				  	int index = atomicAdd( &d_NoofNNlist[j3*Xdiv*Ydiv+j2*Xdiv+j1] , 1); //returns old
#ifdef PRINT_TOO_SHORT_ERROR
				  	if ( index > 64 )
					{
               	          		printf("Fullerene %d, NN-list too short, atleast %d\n", fullerene, index);
               	                   	// for ( int k = 0; k < 32; ++k )
               	                   	// printf("%d ",d_NNlist[ 32*(j2*Xdiv+j1) + k]);
               	                   	// printf("\n");
						continue;
					}
#endif
				  	d_NNlist[ 64*(j3*Xdiv*Ydiv+j2*Xdiv+j1)+index] = fullerene;
					
				}
	
			}
		}	

		
		CMxNNlist[fullerene] = CMx[fullerene];
		CMyNNlist[fullerene] = CMy[fullerene];
		CMzNNlist[fullerene] = CMz[fullerene];
	
	}
		
}

__global__ void UpdateNNlistDivisionLEbc(int No_of_C180s, int non_divided_cells, float *CMx, float *CMy,float *CMz, float *CMxNNlist, float *CMyNNlist, float *CMzNNlist,
                           		   int Xdiv, int Ydiv, int Zdiv, float3 boxMax,
                           		   int *d_NoofNNlist, int *d_NNlist, float3 DLp, float Pshift,bool useRigidBoxZ)
{


    	int fullerene  = non_divided_cells + blockIdx.x*blockDim.x+threadIdx.x;
	
	if ( fullerene < No_of_C180s )
	{ 	  
		int posx = 0;
		int posy = 0;
		int posz = 0;		

	  	
	  	posx = (int)(( CMx[fullerene] - floor( CMx[fullerene] / boxMax.x) * boxMax.x )/DLp.x); 	
 	  	posy = (int)(( CMy[fullerene] - floor( CMy[fullerene] / boxMax.y) * boxMax.y )/DLp.y); 	
	  	posz = (int)(( CMz[fullerene] - floor( CMz[fullerene] / boxMax.z) * boxMax.z )/DLp.z); 		
	 
		int j1 = 0;
	  	int j2 = 0;
	  	int j3 = 0;
	 
	  	for (  int i = -1; i < 2 ; ++i ){
				
			j1 = posx + i;
			if (j1 >= Xdiv || j1 <= -1) continue; 
				 

			for (  int j = -1; j < 2; ++j ){
					
				j2 = posy + j;
				j2 = j2 - floor((float)j2/(float)Ydiv) * Ydiv;	 
	
				for (  int k = -1 ; k < 2; ++k ){
			
			
					j3 = posz + k;
					
					if(useRigidBoxZ){
					
						if(j3 < 0 || j3 > Zdiv-1) continue;
					
					}else{	
					
						j3 = j3 - floor((float)j3/(float)Zdiv) * Zdiv;	 
					
					}

			  		int index = atomicAdd( &d_NoofNNlist[j3*Xdiv*Ydiv+j2*Xdiv+j1] , 1); //returns old
#ifdef PRINT_TOO_SHORT_ERROR
			  		if ( index > 64 )
					{
                         			printf("Fullerene %d, NN-list too short, atleast %d\n", fullerene, index);
                                  		// for ( int k = 0; k < 32; ++k )
                                  		// printf("%d ",d_NNlist[ 32*(j2*Xdiv+j1) + k]);
                                  		// printf("\n");
						continue;
					}
#endif
			  		d_NNlist[ 64*(j3*Xdiv*Ydiv+j2*Xdiv+j1)+index] = fullerene;
					
				}
	
			}
		}
		
				
		if(posx == Xdiv-1 ){
			
			posy = (int)(( (CMy[fullerene] - Pshift) - floor((CMy[fullerene] - Pshift) / boxMax.y) * boxMax.y )/DLp.y);	
			
			j1 = 0;
			j3 = posz;
			if(useRigidBoxZ){
				if ( posz < 0 ) posz = 0;
	  			if ( posz > Zdiv - 1 ) posz = Zdiv - 1;	
			}else{						
				j3 = j3 - floor((float)j3/(float)Zdiv) * Zdiv;	 
			}
				 
			for (  int i = -1; i < 2 ; ++i ){
				
				j2 = posy + i;
				j2 = j2 - floor((float)j2/(float)Ydiv) * Ydiv;
		
				int index = atomicAdd( &d_NoofNNlist[j3*Xdiv*Ydiv+j2*Xdiv+j1] , 1);
				d_NNlist[ 64*(j3*Xdiv*Ydiv+j2*Xdiv+j1)+index] = fullerene;
		
			}
		}
			
						
		if (posx == 0){
			
			posy = (int)(( (CMy[fullerene] + Pshift) - floor( (CMy[fullerene] + Pshift) / boxMax.y) * boxMax.y )/DLp.y);	
			
			j1 = Xdiv - 1;
			j3 = posz;
			if(useRigidBoxZ){
				if ( posz < 0 ) posz = 0;
	  			if ( posz > Zdiv - 1 ) posz = Zdiv - 1;	
			}else{						
				j3 = j3 - floor((float)j3/(float)Zdiv) * Zdiv;	 
			}
				
			for (  int i = -1; i < 2 ; ++i ){
				
				j2 = posy + i;
				j2 = j2 - floor((float)j2/(float)Ydiv) * Ydiv;
		
				int index = atomicAdd( &d_NoofNNlist[j3*Xdiv*Ydiv+j2*Xdiv+j1] , 1);
				d_NNlist[ 64*(j3*Xdiv*Ydiv+j2*Xdiv+j1)+index] = fullerene;
		
			}	
			
				

		}
			
		
		CMxNNlist[fullerene] = CMx[fullerene];
		CMyNNlist[fullerene] = CMy[fullerene];
		CMzNNlist[fullerene] = CMz[fullerene];
	}	
}



__global__ void Ghost_Cells_Pack(int No_of_Ghost_cells_buffer, int* d_Ghost_Cells_ind,
				float *d_X,  float *d_Y,  float *d_Z,
                               float* d_velListX, float* d_velListY, float* d_velListZ,
                               float* d_CMx, float* d_CMy, float* d_CMz,
				float *d_X_gc_buffer,  float *d_Y_gc_buffer,  float *d_Z_gc_buffer,
                              float* d_velListX_gc_buffer, float* d_velListY_gc_buffer, float* d_velListZ_gc_buffer,
                              float* d_CMx_gc_buffer, float* d_CMy_gc_buffer, float* d_CMz_gc_buffer){

	
	
	int ghost_cell = d_Ghost_Cells_ind[blockIdx.x];
	
	int tid = threadIdx.x;	
	int cell = blockIdx.x;	
	
	if( cell < No_of_Ghost_cells_buffer ) {
		

		d_X_gc_buffer[cell*192 + tid] = d_X[192*ghost_cell + tid];
		d_Y_gc_buffer[cell*192 + tid] = d_Y[192*ghost_cell + tid];
		d_Z_gc_buffer[cell*192 + tid] = d_Z[192*ghost_cell + tid];
	
	
		d_velListX_gc_buffer[cell*192 + tid] = d_velListX[192*ghost_cell + tid];
		d_velListY_gc_buffer[cell*192 + tid] = d_velListY[192*ghost_cell + tid];
		d_velListZ_gc_buffer[cell*192 + tid] = d_velListZ[192*ghost_cell + tid]; 
		
		
		if(tid == 0){
			
			d_CMx_gc_buffer[cell] = d_CMx[ghost_cell];
			d_CMy_gc_buffer[cell] = d_CMy[ghost_cell];
			d_CMz_gc_buffer[cell] = d_CMz[ghost_cell];
		
		
		}

	}
	
	
	
}  

__global__ void UpdateNNlistWithGhostCells(int No_of_C180s, int All_Cells, float *d_CMx, float *d_CMy,float *d_CMz,
                           int Xdiv, int Ydiv, int Zdiv, float3 Subdivision_min,
                           int *d_NoofNNlist, int *d_NNlist, float DL){
                           
	
	int atom = blockIdx.x*blockDim.x+threadIdx.x;
	

	
	if ( atom < All_Cells )
	{
	
		int fullerene = atom + No_of_C180s;	
		//printf("fullerene:	%d\n",fullerene);
		  
		int posx = 0;
		int posy = 0;
		int posz = 0;		
		

	 	posx = (int)((d_CMx[fullerene] - Subdivision_min.x)/DL);
	  	if ( posx < 0 ) posx = 0;
	  	if ( posx > Xdiv - 1 ) posx = Xdiv - 1;
	  	

	  	posy = (int)((d_CMy[fullerene] - Subdivision_min.y)/DL);
	  	if ( posy < 0 ) posy = 0;
	  	if ( posy > Ydiv - 1 ) posy = Ydiv - 1;

	   	posz = (int)((d_CMz[fullerene] - Subdivision_min.z)/DL);
	  	if ( posz < 0 ) posz = 0;
	  	if ( posz > Zdiv - 1 ) posz = Zdiv - 1;

		
		int j1 = 0;
		int j2 = 0;
		int j3 = 0;
		
		
		//printf(" my min x is %f, and posx:	%d, posy:	%d, posz:	%d\n",Subdivision_min.x, posx,posy,posz);
		
		for (  int i = -1; i < 2 ; ++i ){
				
			j1 = posx + i;
			if (j1 >= Xdiv || j1 <= -1) continue; 

			for (  int j = -1; j < 2; ++j ){
					
				j2 = posy + j;
				if(j2 < 0 || j2 > Ydiv-1) continue;
				
	
				for (  int k = -1 ; k < 2; ++k ){
			
					j3 = posz + k;
					if(j3 < 0 || j3 > Zdiv-1) continue;
		

					int index = atomicAdd( &d_NoofNNlist[j3*Xdiv*Ydiv+j2*Xdiv+j1] , 1); //returns old
			  		
#ifdef PRINT_TOO_SHORT_ERROR
					if ( index > 64 )
					{
                				printf("Fullerene %d, NN-list too short, atleast %d\n", fullerene, index);
                      				 // for ( int k = 0; k < 32; ++k )
                      				 //     printf("%d ",d_NNlist[ 32*(j2*Xdiv+j1) + k]); 
                      				 // printf("\n");
						continue;
					}
#endif
					d_NNlist[ 64*(j3*Xdiv*Ydiv+j2*Xdiv+j1)+index] = fullerene;
				}
			}
		}
	}
                           
}



__global__ void migrated_cells_finder(int No_of_C180s, float *d_CM,
                         		float Sub_max, float Sub_min, float BMin, float BMax,
                         		int* d_counter_mc_r, int* d_counter_mc_l,
                         		int* d_migrated_cells_ind_R, int* d_migrated_cells_ind_L,
                         		char* d_cell_mig){
 
	
	int fullerene = blockIdx.x*blockDim.x+threadIdx.x;


	if ( fullerene < No_of_C180s )
	{
	      
      		float C = d_CM[fullerene];
      		
      			
      		if ( BMin < Sub_min) {	
      			
      			if ( C < Sub_min - 0.1 ) {
	
	  			int index = atomicAdd(d_counter_mc_l,1);
	 			d_migrated_cells_ind_L[index] = fullerene;
	 			d_cell_mig[fullerene] = 1;
	  		
	  		}
	  	
	  	}		  	
	  	if (Sub_max < BMax){
	  		
	  		if ( C > Sub_max + 0.1) {
	  		
	  			int index = atomicAdd(d_counter_mc_r,1);
	 			d_migrated_cells_ind_R[index] = fullerene;
	 			d_cell_mig[fullerene] = 1;
	  		
	  		}
	  	}
		          		 
	}
                       		 
}



__global__ void ghost_cells_finder_Auxiliary(int No_of_C180s, int All_Cells, float *d_CM , 
						float Sub_max, float Sub_min,
						int* d_counter_gc_r, int* d_counter_gc_l,
                         			int* d_Ghost_Cells_ind_R, int* d_Ghost_Cells_ind_L)
{
 
	int atom = blockIdx.x*blockDim.x+threadIdx.x;
	
	if ( atom < All_Cells )
	{	
		int fullerene = atom + No_of_C180s;
	 	
	 	float pos = d_CM[fullerene];
	 	
	 	if( pos <=  Sub_min + 2.0 ){
	 			
	 		int index = atomicAdd(d_counter_gc_l,1);
	 		d_Ghost_Cells_ind_L[index] = fullerene;
	 	
	 	} else if( pos >=  Sub_max - 2.0 ){
	 			
	 		int index = atomicAdd(d_counter_gc_r,1);
	 		d_Ghost_Cells_ind_R[index] = fullerene;	 			
	 	}
	 	
	
	}
 
                         		 
}


__global__ void migrated_Cells_Remove_Pack(int No_of_C180s, int No_of_migration_cells_buffer, int* d_counter,
					 	int* d_migrated_cells_ind, char* d_cell_mig,
   						float *d_X,  float *d_Y,  float *d_Z,
                               		float* d_velListX, float* d_velListY, float* d_velListZ,
                               		float* d_CMx, float* d_CMy, float* d_CMz,
                               		float* d_ScaleFactor,float* d_Youngs_mod, float* d_Growth_rate, float* d_DivisionVolume,
                               		float* d_gamma_env, float* d_viscotic_damp, float* d_pressList, int* d_CellINdex, 
                               		float* d_Apo_rate, float* d_squeeze_rate,
						float *d_X_mc_buffer,  float *d_Y_mc_buffer,  float *d_Z_mc_buffer,
                               		float* d_velListX_mc_buffer, float* d_velListY_mc_buffer, float* d_velListZ_mc_buffer,
                               		float* d_CMx_mc_buffer, float* d_CMy_mc_buffer, float* d_CMz_mc_buffer,
                               		float* d_ScaleFactor_mc_buffer,float* d_Youngs_mod_mc_buffer, float* d_Growth_rate_mc_buffer, float* d_DivisionVolume_mc_buffer,
                               		float* d_gamma_env_mc_buffer, float* d_viscotic_damp_mc_buffer, float* d_pressList_mc_buffer, int* d_CellINdex_mc_buffer, 
                               		float* d_Apo_rate_mc_buffer, float* d_squeeze_rate_mc_buffer){

	
	
	int migrated_cell = d_migrated_cells_ind[blockIdx.x];
	
	int tid = threadIdx.x;	
	int cell = blockIdx.x;	
	
	if( cell < No_of_migration_cells_buffer ) {

		if( migrated_cell < No_of_C180s - No_of_migration_cells_buffer ){
		
			
			__shared__ int moving_Cell;
		
			if (tid == 0){

				int index = atomicAdd(d_counter,1);
				moving_Cell = No_of_C180s - index - 1;			
		
				while ( d_cell_mig[moving_Cell] == 1 ){
				
					index = atomicAdd(d_counter,1);
					moving_Cell = No_of_C180s - index - 1;
				}
	
			}
	
			
			__syncthreads();
			
			
			
			d_X_mc_buffer[cell*192 + tid] = d_X[192*migrated_cell + tid];
			d_X[migrated_cell*192 + tid] = d_X[192*moving_Cell + tid];
			
			d_Y_mc_buffer[cell*192 + tid] = d_Y[192*migrated_cell + tid];
			d_Y[migrated_cell*192 + tid] = d_Y[192*moving_Cell + tid];
			
			
			d_Z_mc_buffer[cell*192 + tid] = d_Z[192*migrated_cell + tid];
			d_Z[migrated_cell*192 + tid] = d_Z[192*moving_Cell + tid];
			

			d_velListX_mc_buffer[cell*192 + tid] = d_velListX[192*migrated_cell + tid];
			d_velListX[migrated_cell*192 + tid] = d_velListX[192*moving_Cell + tid];
			
			d_velListY_mc_buffer[cell*192 + tid] = d_velListY[192*migrated_cell + tid];
			d_velListY[migrated_cell*192 + tid] = d_velListY[192*moving_Cell + tid];
			
			
			d_velListZ_mc_buffer[cell*192 + tid] = d_velListZ[192*migrated_cell + tid]; 
			d_velListZ[migrated_cell*192 + tid] = d_velListZ[192*moving_Cell + tid];
			
			
			
			if(tid == 0){
			
				d_CMx_mc_buffer[cell] = d_CMx[migrated_cell];
				d_CMx[migrated_cell] = d_CMx[moving_Cell];
				
				d_CMy_mc_buffer[cell] = d_CMy[migrated_cell];
				d_CMy[migrated_cell] = d_CMy[moving_Cell];
				
				d_CMz_mc_buffer[cell] = d_CMz[migrated_cell];
				d_CMz[migrated_cell] = d_CMz[moving_Cell];
				
				d_pressList_mc_buffer[cell] = d_pressList[migrated_cell];
				d_pressList[migrated_cell] = d_pressList[moving_Cell];
				
				d_Growth_rate_mc_buffer[cell] = d_Growth_rate[migrated_cell];
				d_Growth_rate[migrated_cell] = d_Growth_rate[moving_Cell];
				
				d_Youngs_mod_mc_buffer[cell]  = d_Youngs_mod[migrated_cell];
				d_Youngs_mod[migrated_cell]  = d_Youngs_mod[moving_Cell];
				
				d_ScaleFactor_mc_buffer[cell] = d_ScaleFactor[migrated_cell];
				d_ScaleFactor[migrated_cell] = d_ScaleFactor[moving_Cell];
				
				d_DivisionVolume_mc_buffer[cell] = d_DivisionVolume[migrated_cell];
				d_DivisionVolume[migrated_cell] = d_DivisionVolume[moving_Cell];
				
				d_gamma_env_mc_buffer[cell] = d_gamma_env[migrated_cell];
				d_gamma_env[migrated_cell] = d_gamma_env[moving_Cell];
				
				d_viscotic_damp_mc_buffer[cell] = d_viscotic_damp[migrated_cell];
				d_viscotic_damp[migrated_cell] = d_viscotic_damp[moving_Cell];
				
				d_CellINdex_mc_buffer[cell] = d_CellINdex[migrated_cell];
				d_CellINdex[migrated_cell] = d_CellINdex[moving_Cell];
				
				d_Apo_rate_mc_buffer[cell] = d_Apo_rate[migrated_cell];
				d_Apo_rate[migrated_cell] = d_Apo_rate[moving_Cell];
				
				d_squeeze_rate_mc_buffer[cell] = d_squeeze_rate[migrated_cell];
				d_squeeze_rate[migrated_cell] = d_squeeze_rate[moving_Cell];
		
		
			}		
		
		
		
		
		
		} else {

			
			d_X_mc_buffer[cell*192 + tid] = d_X[192*migrated_cell + tid];
			d_Y_mc_buffer[cell*192 + tid] = d_Y[192*migrated_cell + tid];
			d_Z_mc_buffer[cell*192 + tid] = d_Z[192*migrated_cell + tid];
	
	
			d_velListX_mc_buffer[cell*192 + tid] = d_velListX[192*migrated_cell + tid];
			d_velListY_mc_buffer[cell*192 + tid] = d_velListY[192*migrated_cell + tid];
			d_velListZ_mc_buffer[cell*192 + tid] = d_velListZ[192*migrated_cell + tid]; 
		
		
			if(tid == 0){
			
				d_CMx_mc_buffer[cell] = d_CMx[migrated_cell];
				d_CMy_mc_buffer[cell] = d_CMy[migrated_cell];
				d_CMz_mc_buffer[cell] = d_CMz[migrated_cell];
				d_pressList_mc_buffer[cell] = d_pressList[migrated_cell];
				d_Growth_rate_mc_buffer[cell] = d_Growth_rate[migrated_cell];
				d_Youngs_mod_mc_buffer[cell]  = d_Youngs_mod[migrated_cell];
				d_ScaleFactor_mc_buffer[cell] = d_ScaleFactor[migrated_cell];
				d_DivisionVolume_mc_buffer[cell] = d_DivisionVolume[migrated_cell];
				d_gamma_env_mc_buffer[cell] = d_gamma_env[migrated_cell];
				d_viscotic_damp_mc_buffer[cell] = d_viscotic_damp[migrated_cell];
				d_CellINdex_mc_buffer[cell] = d_CellINdex[migrated_cell];
				d_Apo_rate_mc_buffer[cell] = d_Apo_rate[migrated_cell];
				d_squeeze_rate_mc_buffer[cell] = d_squeeze_rate[migrated_cell];
		
		
			}			


	
		}
	
	
	}
	
	
	
}  
