#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "postscript.h"


__global__ void CenterOfMass( int No_of_C180s, 
               float *d_XP, float *d_YP, float *d_ZP, 
               float *CMx, float *CMy,float *CMz)
{
__shared__ float  sumx[256];
__shared__ float  sumy[256];
__shared__ float  sumz[256];

int rank = blockIdx.x;
int tid  = threadIdx.x;

sumx[tid] = 0.0;
sumy[tid] = 0.0;
sumz[tid] = 0.0;

if ( tid < 180 )
    {
    sumx[tid] = d_XP[rank*192+tid];
    sumy[tid] = d_YP[rank*192+tid];
    sumz[tid] = d_ZP[rank*192+tid];
    }

__syncthreads();

for ( int s = blockDim.x/2; s > 0; s>>=1)
   {
   if ( tid < s )
      {
      sumx[tid] += sumx[tid+s];
      sumy[tid] += sumy[tid+s];
      sumz[tid] += sumz[tid+s];
      }
   __syncthreads();
   }

if ( tid == 0 ) 
   {
   CMx[rank] = sumx[0]/180.0f;
   CMy[rank] = sumy[0]/180.0f;
   CMz[rank] = sumz[0]/180.0f;
   }

}




__global__ void VelocityCenterOfMass( int No_of_C180s, 
               float *d_velListX, float *d_velListY, float *d_velListZ, 
               float *d_VCMx, float *d_VCMy,float *d_VCMz)
{
__shared__ float  sumx[256];
__shared__ float  sumy[256];
__shared__ float  sumz[256];

int rank = blockIdx.x;
int tid  = threadIdx.x;

sumx[tid] = 0.0;
sumy[tid] = 0.0;
sumz[tid] = 0.0;

if ( tid < 180 )
    {
    sumx[tid] = d_velListX[rank*192+tid];
    sumy[tid] = d_velListY[rank*192+tid];
    sumz[tid] = d_velListZ[rank*192+tid];
    }

__syncthreads();

for ( int s = blockDim.x/2; s > 0; s>>=1)
   {
   if ( tid < s )
      {
      sumx[tid] += sumx[tid+s];
      sumy[tid] += sumy[tid+s];
      sumz[tid] += sumz[tid+s];
      }
   __syncthreads();
   }

if ( tid == 0 ) 
   {
   d_VCMx[rank] = sumx[0]/180.0f;
   d_VCMy[rank] = sumy[0]/180.0f;
   d_VCMz[rank] = sumz[0]/180.0f;
   }

}





