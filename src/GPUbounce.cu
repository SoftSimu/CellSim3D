#include "hip/hip_runtime.h"
//#define FORCE_DEBUG
//#define PRINT_VOLUMES
//#define TURNOFF_RAN
//#define DEBUG_RAND
//#define OUTPUT_ADP_ERROR
//#degine RO_DEBUG
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <locale.h>
#include <algorithm>
#include <iostream>
#include <fstream>
#include <streambuf>
#include <cstring>
#include <string>

#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <vector_functions.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/fill.h>
#include <thrust/sort.h>
#include <thrust/extrema.h>
//#include "hip/hip_runtime_api.h"
#include "postscript.h"
#include "marsaglia.h"
//#include "IntegrationKernels.h"
#include "RandomVector.h"
#include "VectorFunctions.hpp"
#include "AdaptiveTimeKernels.cuh"

#include "json/json.h"


void CudaFailure();

#define CudaErrorCheck() { \
        hipError_t e = hipPeekAtLastError();        \
        if (e!=hipSuccess){                                            \
            printf("Cuda failure in %s, line %d, code %d %s\n", __FILE__, __LINE__, e, hipGetErrorString(e)); \
            exit(0); \
        }\
    }

float asym[1];
float mass;                                           //  M
float repulsion_range,    attraction_range;        //  LL1, LL2
float repulsion_strength, attraction_strength;     //  ST1, ST2

// variables to allow for different stiffnesses
float stiffness1;
float Youngs_mod; 
float* d_Youngs_mod;
float* youngsModArray;
float* d_Growth_rate;
float* Growth_rate; 
int* d_indSoftCells;
float softYoungsMod;
bool checkSphericity; 


int phase_count = INT_MAX;
bool write_cont_force=false;
bool write_vel_file = false;
char forces_file[256];

float shear_rate;
float Pshift;
float Vshift;
float viscotic_damping, internal_damping;          //  C, DMP
float gamma_visc;
float zOffset; // Offset from Z = 0 for starting positions.
int ranZOffset;
int   Time_steps;
float divVol;
float ApoVol;
float delta_t;
float dt_max;
float dt_tol;
bool doAdaptive_dt;
float c1 = 0; float c2 = 0; 
int Restart;
int Laststep = 0;
int Lastframe = 0;
int   trajWriteInt; // trajectory write interval
int   countOnlyInternal; // 0 - Count all new cells
                         // 1 - Count only the cells born within 0.6Rmax from
                         //     the center of mass of the system
float radFrac; // The factor to count cells within a raduys (<Rmax)

int   overWriteMitInd; // 0 No, 1 yes

int newCellCountInt; // Interval at which to count the divided cells
int equiStepCount;
const char* ptrajFileName;
char trajFileName[256];
bool binaryOutput; 

// equilibrium length of springs between fullerene atoms
float R0  = 0.13517879937327418f; // why is this hard-coded?
float* d_R0;
float* h_R0;


float L1  = 3.0f;       // the initial fullerenes are placed in
// an X x Y grid of sizne L1 x L1


// the three nearest neighbours of C180 atoms
int   C180_nn[3*192];
int   C180_sign[180];
// device: the three nearest neighbours of C180 atoms
int   *d_C180_nn;
int   *d_C180_sign;

int   CCI[2][271];       // list of nearest neighbor carbon pairs in the fullerne
// number of pairs = 270

int   C180_56[92*7];     // 12 lists of atoms forming pentagons 1 2 3 4 5 1 1 and
// 80 lists of atoms forming hexagons  1 2 3 4 5 6 1
int   *d_C180_56;

float *d_volume;
float *volume;
float *d_area; 
float *area; 
char* cell_div;
char* d_cell_div;
int num_cell_div;
int* cell_div_inds;

char* cell_Apo;
char* d_cell_Apo;
int num_cell_Apo;
int* cell_Apo_inds;


char mitIndFileName[256]; 

float *d_pressList;
float *pressList;
int* d_resetIndices;
int* resetIndices; 


float* d_velListX; 
float* d_velListY; 
float* d_velListZ;

float* velListX; 
float* velListY; 
float* velListZ; 

// Params related to population modelling
int doPopModel;
char* didCellDie;
float totalFood;
float* d_totalFood;
int haylimit;
int cellLifeTime;
float cellFoodCons; // baseline food consumption
float cellFoodConsDiv; // Extra good consumption when cell divides
float cellFoodRel; // Food released when cell dies (should < total consumed food)
float maxPressure;
float minPressure;
float rMax;
float squeeze_rate;
float maxPop; 

// Params related to having walls in the simulation
int useWalls;
char perpAxis[2];
float threshDist;
float dAxis;
float wallLen;
float wallWidth;
float wall1, wall2;
float wallWStart, wallWEnd;
float wallLStart, wallLEnd;

float boxLength, boxMin[3];
float3 boxMax;
float3 BoxMin;
bool flatbox; 
bool LineCenter; 
bool useRigidSimulationBox;
bool usePBCs; 
bool useLEbc;
bool useRigidBoxZ; 
bool useRigidBoxY; 
float* d_boxMin;
bool rand_pos;
bool rand_vel;
bool impurity;
int impurityNum;
bool line;
bool plane;
float L  = 2.5f;  

// randomness parameters

bool add_rands;
int rand_seed;
int rand_dist;
float rand_scale_factor;
hiprandState *d_rngStates;
unsigned int *d_seeds; 

int No_of_threads; // ie number of staring cells
int Side_length;
int ex, ey;


float  *X,  *Y,  *Z;     // host: atom positions

float *d_XP, *d_YP, *d_ZP;     // device: time propagated atom positions
float  *d_X,  *d_Y,  *d_Z;     // device: present atom positions
float *d_XM, *d_YM, *d_ZM;     // device: previous atom positions
float *d_XMM, *d_YMM, *d_ZMM; 

R3Nptrs d_fConList;
R3Nptrs d_fDisList;
R3Nptrs d_fRanList; 

R3Nptrs d_contactForces;
R3Nptrs h_contactForces;

R3Nptrs d_ExtForces;
R3Nptrs h_ExtForces;


float* d_Fx;
float* d_Fy;
float* d_Fz;

// float* theta0;
// float* d_theta0;

bool constrainAngles;

// host: minimal bounding box for fullerene
//float *bounding_xyz;
//float *d_bounding_xyz;   // device:  bounding_xyz

// global minimum and maximum of x and y, preprocessfirst
// global minimum and maximum of x and y, postprocesssecond
//float *d_Minx, *d_Maxx, *d_Miny, *d_Maxy, *d_Minz, *d_Maxz;
float *Minx, *Maxx, *Miny, *Maxy, *Minz, *Maxz;

float DL;
float3 DLp;
int Xdiv, Ydiv, Zdiv;

//int *d_NoofNNlist;
int *d_NNlist;
int *NoofNNlist;
int *NNlist;

int  No_of_Cell1;
int  No_of_Cell2;
int CellInApo1;
int CellInApo2;

float *d_CMx, *d_CMy, *d_CMz;
float *CMx, *CMy, *CMz;
float sysCMx = 1.0, sysCMy = 1.0, sysCMz = 1.0;
float sysCMx_old = 0.0, sysCMy_old = 0.0, sysCMz_old = 0.0;



float *d_VCMx, *d_VCMy, *d_VCMz;
float *VCMx, *VCMy, *VCMz;
float sysVCMx = 0.0, sysVCMy = 0.0, sysVCMz = 0.0;


//float Pressure;          // pressure
//float Temperature;       // equation of state relates Pressure and Temperature

int  No_of_C180s;        // the global number of C180 fullerenes
int  No_of_C180s_in;     // the number of C180s near the center of mass of the system
int MaxNoofC180s; 

float *ran2;             // host: ran2[]
float *d_ran2;           // device: ran2[], used in celldivision

int *NDIV;               // # of divisions

// Parameters related to division
bool useDivPlaneBasis;
float divPlaneBasis[3]; 

long int GPUMemory;
long int CPUMemory;


int frameCount = 1;

bool correct_com = false;
bool correct_Vcom = false;
bool asymDivision;
int Orig_No_of_C180s;
 

bool apoptosis;
float Apo_rate;
int popToStartApo;
bool WithoutApo;

int* CellINdex;
int* d_CellINdex;
int NumApoCell;

bool colloidal_dynamics;
bool dispersity;
float* ScaleFactor;
float* d_ScaleFactor;
float* DivisionVolume;
float* d_DivisionVolume;
float* gamma_env;
float* d_gamma_env;
float* viscotic_damp;
float* d_viscotic_damp;


bool useDifferentCell;
float SizeFactor;
float Stiffness2;
float gRate;
float divisionV;
float gEnv;
float gVis;
float Apo_rate2;
float squeeze_rate2;
int numberOfCells;
float fractionOfCells;
float closenessToCenter;
bool chooseRandomCellIndices;
bool daughtSame;
bool duringGrowth;
bool recalc_r0; 

int main(int argc, char *argv[])
{
  int i;
  int globalrank;
  int step = 0;
  int noofblocks, threadsperblock, prevnoofblocks;
  int newcells;
  int reductionblocks;
  
  No_of_Cell1 =0;
  No_of_Cell2 =0;
  CellInApo1 = 0;
  CellInApo2 = 0;
  NumApoCell = 0;
  
  FILE *outfile;
  FILE *trajfile; // pointer to xyz file
  hipError_t myError;

  int* dividingCells; //Cells that are about to divide
  int* totalCells; // No. of cells at every Dividing_steps


  int* num_new_cells_per_step;
  int countOffset = 0;

  //int min_no_of_cells = 10;

  printf("CellDiv version 0.9\n");

  if ( argc !=4 )
  {
      printf("Usage: CellDiv no_of_threads inpFile.json gpuid\n");
      return(0);
  }

  No_of_threads = atoi(argv[1]);

  char inpFile[256];
  strcpy(inpFile, argv[2]);

  if ( read_json_params(inpFile)          != 0 ) return(-1);

  printf("%d\n", MaxNoofC180s); 
  
  Vshift = shear_rate*boxMax.x;
  Pshift = 0;


  	
  if ( line ) {
  
  	int LineCell = 0;
  	LineCell = (int) ((boxMax.x - BoxMin.x)/L);
  	printf(" Max Number of initial cells in line is:   %d\n", LineCell); 
  	if ( No_of_threads > LineCell ) {
  	printf(" Number of initial cells in line is greater than Max \n ");
  	return(-1);
  	}	
  }	

  Side_length   = (int)( sqrt( (double)No_of_threads )+0.5);
  if ( No_of_threads > MaxNoofC180s // Side_length*Side_length != No_of_threads
      )
  {
      printf("Usage: Celldiv no_of_threads\n");
      printf("       no_of_threads should be a square, n^2, < %d\n", MaxNoofC180s);
      return(0);
  }

  if (!impurity) impurityNum = 0;	
  No_of_C180s      = No_of_threads + impurityNum;
  Orig_No_of_C180s = No_of_C180s;
  GPUMemory = 0L;
  CPUMemory = 0L;



  X = (float *)calloc(192*MaxNoofC180s,sizeof(float));
  Y = (float *)calloc(192*MaxNoofC180s,sizeof(float));
  Z = (float *)calloc(192*MaxNoofC180s,sizeof(float));
  //bounding_xyz = (float *)calloc(MaxNoofC180s*6, sizeof(float));
  velListX = (float *)calloc(192*MaxNoofC180s, sizeof(float)); 
  velListY = (float *)calloc(192*MaxNoofC180s, sizeof(float)); 
  velListZ = (float *)calloc(192*MaxNoofC180s, sizeof(float));
  youngsModArray = (float *)calloc(MaxNoofC180s, sizeof(float));
  Growth_rate = (float *)calloc(MaxNoofC180s, sizeof(float));
  pressList = (float *)calloc(MaxNoofC180s, sizeof(float));  
  CellINdex = (int *)calloc(MaxNoofC180s, sizeof(int));
  ScaleFactor = (float *)calloc(MaxNoofC180s, sizeof(float)); 
  DivisionVolume = (float *)calloc(MaxNoofC180s, sizeof(float));
  gamma_env = (float *)calloc(MaxNoofC180s, sizeof(float));
  viscotic_damp = (float *)calloc(MaxNoofC180s, sizeof(float));

  CPUMemory += 6L*192L*MaxNoofC180s*sizeof(float);
  CPUMemory += MaxNoofC180s*10L*sizeof(float);



  h_R0 = (float *)calloc(192*3, sizeof(float));
  for (int i =  0; i < MaxNoofC180s; ++i) ScaleFactor[i] = 1.0;

  //if ( read_global_params()               != 0 ) return(-1);
  if (Restart == 1 ) if( ReadRestartFile() != 0 ) return(-1); 
  if (generate_random(Orig_No_of_C180s)  != 0 ) return(-1);
  if (DispersityFunc(Orig_No_of_C180s) != 0 ) return(-1);
  if (Restart == 0 ) if ( initialize_C180s(Orig_No_of_C180s) != 0 ) return(-1);
  if ( read_fullerene_nn()                != 0 ) return(-1);
  NDIV = (int *)calloc(MaxNoofC180s,sizeof(int));
  CPUMemory += 2*MaxNoofC180s*sizeof(int);
  for ( i = 0; i < No_of_C180s; ++i ){
   NDIV[i] = 1;
  }
  for ( i = No_of_C180s; i < MaxNoofC180s; ++i ) {
   NDIV[i] = 0;
  }

  // empty the psfil from previous results
  outfile = fopen("psfil","w");
  if ( outfile == NULL ) {printf("Unable to open file psfil\n");return(-1);}
  fclose(outfile);

  /* PM
     Allocate memory for the dividingCells array that will be used to
     calculate the mitotic index.
  */

  dividingCells = (int *)calloc((Time_steps/newCellCountInt), sizeof(int));
  totalCells = (int *)calloc((Time_steps/newCellCountInt), sizeof(int));
  num_new_cells_per_step = (int *)calloc(Time_steps, sizeof(int));
  


  CPUMemory += (2L*(long)(Time_steps/newCellCountInt) + 1L + (long)Time_steps) * sizeof(int);



  CPUMemory += (long)MaxNoofC180s * sizeof(char);

  hipDeviceProp_t deviceProp = getDevice();
  if (hipSuccess != hipSetDevice(atoi(argv[3]))){
      CudaErrorCheck();
      printf("Could not set to divice %d\n", atoi(argv[3]));
      return -1;
  }

  if ( hipSuccess != hipMalloc( (void **)&d_C180_nn, 3*192*sizeof(int))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_C180_sign, 180*sizeof(int))) return(-1);
  GPUMemory +=  3*192*sizeof(int) + 180*sizeof(int);
  //  hipError_t myError = hipGetLastError();
  //     if ( hipSuccess != myError )
  //         { printf( "1: Error %d: %s!\n",myError,hipGetErrorString(myError) );return(-1);}

  if ( hipSuccess != hipMalloc( (void **)&d_XP , 192*MaxNoofC180s*sizeof(float))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_YP , 192*MaxNoofC180s*sizeof(float))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_ZP , 192*MaxNoofC180s*sizeof(float))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_X  , 192*MaxNoofC180s*sizeof(float))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_Y  , 192*MaxNoofC180s*sizeof(float))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_Z  , 192*MaxNoofC180s*sizeof(float))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_XM , 192*MaxNoofC180s*sizeof(float))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_YM , 192*MaxNoofC180s*sizeof(float))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_ZM , 192*MaxNoofC180s*sizeof(float))) return(-1);
  //if ( hipSuccess != hipMalloc( (void **)&d_bounding_xyz , MaxNoofC180s*6*sizeof(float))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_CMx ,          MaxNoofC180s*sizeof(float))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_CMy ,          MaxNoofC180s*sizeof(float))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_CMz ,          MaxNoofC180s*sizeof(float))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_VCMx ,          MaxNoofC180s*sizeof(float))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_VCMy ,          MaxNoofC180s*sizeof(float))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_VCMz ,          MaxNoofC180s*sizeof(float))) return(-1);
//  if ( hipSuccess != hipMalloc( (void **)&d_volume ,       MaxNoofC180s*sizeof(float))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_area ,       MaxNoofC180s*sizeof(float))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_cell_div ,     MaxNoofC180s*sizeof(char))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_cell_Apo ,     MaxNoofC180s*sizeof(char))) return(-1);
  //if ( hipSuccess != hipMalloc( (void **)&d_Minx ,         1024*sizeof(float))) return(-1);
  //if ( hipSuccess != hipMalloc( (void **)&d_Maxx ,         1024*sizeof(float))) return(-1);
  //if ( hipSuccess != hipMalloc( (void **)&d_Miny ,         1024*sizeof(float))) return(-1);
  //if ( hipSuccess != hipMalloc( (void **)&d_Maxy ,         1024*sizeof(float))) return(-1);
  //if ( hipSuccess != hipMalloc( (void **)&d_Minz ,         1024*sizeof(float))) return(-1);
  //if ( hipSuccess != hipMalloc( (void **)&d_Maxz ,         1024*sizeof(float))) return(-1);
  //  if ( hipSuccess != hipMalloc( (void **)&d_NoofNNlist ,   1024*1024*sizeof(int))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_NNlist ,    MAX_NN*MaxNoofC180s*sizeof(int))) return(-1); 
  if ( hipSuccess != hipMalloc( (void **)&d_C180_56,       92*7*sizeof(int))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_ran2 , 10000*sizeof(float))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_pressList, MaxNoofC180s*sizeof(float))) return(-1);
  // if ( hipSuccess != hipMalloc( (void **)&d_velListX, 192*MaxNoofC180s*sizeof(float))) return(-1);
  // if ( hipSuccess != hipMalloc( (void **)&d_velListY, 192*MaxNoofC180s*sizeof(float))) return(-1);
  // if ( hipSuccess != hipMalloc( (void **)&d_velListZ, 192*MaxNoofC180s*sizeof(float))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_resetIndices, MaxNoofC180s*sizeof(int))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_Fx, 192*MaxNoofC180s*sizeof(float))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_Fy, 192*MaxNoofC180s*sizeof(float))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_Fz, 192*MaxNoofC180s*sizeof(float))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_Youngs_mod, MaxNoofC180s*sizeof(float))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_Growth_rate, MaxNoofC180s*sizeof(float))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_CellINdex, MaxNoofC180s*sizeof(int))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_ScaleFactor, MaxNoofC180s*sizeof(float))) return(-1); 
  if ( hipSuccess != hipMalloc( (void **)&d_DivisionVolume, MaxNoofC180s*sizeof(float))) return(-1); 
  if ( hipSuccess != hipMalloc( (void **)&d_gamma_env, MaxNoofC180s*sizeof(float))) return(-1); 
  if ( hipSuccess != hipMalloc( (void **)&d_viscotic_damp, MaxNoofC180s*sizeof(float))) return(-1); 
  if ( hipSuccess != hipMalloc( (void **)&d_boxMin, 3*sizeof(float))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_R0, 192*3*sizeof(float))) return(-1);
  if ( hipSuccess != hipMalloc((void **)&d_velListX, 192*MaxNoofC180s*sizeof(float))) return -1; 
  if ( hipSuccess != hipMalloc((void **)&d_velListY, 192*MaxNoofC180s*sizeof(float))) return -1; 
  if ( hipSuccess != hipMalloc((void **)&d_velListZ, 192*MaxNoofC180s*sizeof(float))) return -1;
  if ( hipSuccess != hipMalloc((void **)&d_contactForces.x, 192*MaxNoofC180s*sizeof(float))) return -1;
  if ( hipSuccess != hipMalloc((void **)&d_contactForces.y, 192*MaxNoofC180s*sizeof(float))) return -1;
  if ( hipSuccess != hipMalloc((void **)&d_contactForces.z, 192*MaxNoofC180s*sizeof(float))) return -1;
  if ( hipSuccess != hipMalloc((void **)&d_ExtForces.x, 192*MaxNoofC180s*sizeof(float))) return -1;
  if ( hipSuccess != hipMalloc((void **)&d_ExtForces.y, 192*MaxNoofC180s*sizeof(float))) return -1;
  if ( hipSuccess != hipMalloc((void **)&d_ExtForces.z, 192*MaxNoofC180s*sizeof(float))) return -1;
  if ( hipSuccess != hipMalloc((void **)&d_fConList.x, 192*MaxNoofC180s*sizeof(float))) return -1;
  if ( hipSuccess != hipMalloc((void **)&d_fConList.y, 192*MaxNoofC180s*sizeof(float))) return -1;
  if ( hipSuccess != hipMalloc((void **)&d_fConList.z, 192*MaxNoofC180s*sizeof(float))) return -1;
  if ( hipSuccess != hipMalloc((void **)&d_fDisList.x, 192*MaxNoofC180s*sizeof(float))) return -1;
  if ( hipSuccess != hipMalloc((void **)&d_fDisList.y, 192*MaxNoofC180s*sizeof(float))) return -1;
  if ( hipSuccess != hipMalloc((void **)&d_fDisList.z, 192*MaxNoofC180s*sizeof(float))) return -1;
  if ( hipSuccess != hipMalloc((void **)&d_fRanList.x, 192*MaxNoofC180s*sizeof(float))) return -1;
  if ( hipSuccess != hipMalloc((void **)&d_fRanList.y, 192*MaxNoofC180s*sizeof(float))) return -1;
  if ( hipSuccess != hipMalloc((void **)&d_fRanList.z, 192*MaxNoofC180s*sizeof(float))) return -1;
  

  hipMemset(d_C180_nn, 0, 3*192*sizeof(int));
  hipMemset(d_C180_sign, 0, 180*sizeof(int));

  hipMemset(d_XP, 0, 192*MaxNoofC180s*sizeof(float));
  hipMemset(d_YP, 0, 192*MaxNoofC180s*sizeof(float));
  hipMemset(d_ZP, 0, 192*MaxNoofC180s*sizeof(float));
  hipMemset(d_X, 0, 192*MaxNoofC180s*sizeof(float));
  hipMemset(d_Y, 0, 192*MaxNoofC180s*sizeof(float));
  hipMemset(d_Z, 0, 192*MaxNoofC180s*sizeof(float));
  hipMemset(d_XM, 0, 192*MaxNoofC180s*sizeof(float));
  hipMemset(d_YM, 0, 192*MaxNoofC180s*sizeof(float));
  hipMemset(d_ZM, 0, 192*MaxNoofC180s*sizeof(float));
  CudaErrorCheck();

  hipMemset(d_CMx, 0, MaxNoofC180s*sizeof(float));
  hipMemset(d_CMy, 0, MaxNoofC180s*sizeof(float));
  hipMemset(d_CMz, 0, MaxNoofC180s*sizeof(float));
  hipMemset(d_VCMx, 0, MaxNoofC180s*sizeof(float));
  hipMemset(d_VCMy, 0, MaxNoofC180s*sizeof(float));
  hipMemset(d_VCMz, 0, MaxNoofC180s*sizeof(float));
  CudaErrorCheck();

  hipMemset(d_R0, 0, 3*192*sizeof(float));
  hipMemset(d_pressList, 0, MaxNoofC180s*sizeof(float));
  hipMemset(d_Youngs_mod, 0, MaxNoofC180s*sizeof(float));
  hipMemset(d_Growth_rate, 0, MaxNoofC180s*sizeof(float));
  hipMemset(d_CellINdex, 0, MaxNoofC180s*sizeof(int));
  hipMemset(d_ScaleFactor, 0, MaxNoofC180s*sizeof(int));
  hipMemset(d_DivisionVolume, 0, MaxNoofC180s*sizeof(float));
  hipMemset(d_gamma_env, 0, MaxNoofC180s*sizeof(float));
  hipMemset(d_viscotic_damp, 0, MaxNoofC180s*sizeof(float));
  //hipMemset(d_bounding_xyz, 0, MaxNoofC180s*6*sizeof(float));
  //hipMemset(d_Minx, 0, 1024*sizeof(float));
  //hipMemset(d_Maxx, 0, 1024*sizeof(float));
  //hipMemset(d_Miny, 0, 1024*sizeof(float));
  //hipMemset(d_Maxy, 0, 1024*sizeof(float));
  //hipMemset(d_Minz, 0, 1024*sizeof(float));
  //hipMemset(d_Maxz, 0, 1024*sizeof(float)); 
  CudaErrorCheck();

  hipMemset(d_velListX, 0, 192*MaxNoofC180s*sizeof(float));
  hipMemset(d_velListY, 0, 192*MaxNoofC180s*sizeof(float));
  hipMemset(d_velListZ, 0, 192*MaxNoofC180s*sizeof(float));


  hipMemset(d_fConList.x, 0, 192*MaxNoofC180s*sizeof(float));
  hipMemset(d_fConList.y, 0, 192*MaxNoofC180s*sizeof(float));
  hipMemset(d_fConList.z, 0, 192*MaxNoofC180s*sizeof(float));

  hipMemset(d_fDisList.x, 0, 192*MaxNoofC180s*sizeof(float));
  hipMemset(d_fDisList.y, 0, 192*MaxNoofC180s*sizeof(float));
  hipMemset(d_fDisList.z, 0, 192*MaxNoofC180s*sizeof(float));

  hipMemset(d_fRanList.x, 0, 192*MaxNoofC180s*sizeof(float));
  hipMemset(d_fRanList.y, 0, 192*MaxNoofC180s*sizeof(float));
  hipMemset(d_fRanList.z, 0, 192*MaxNoofC180s*sizeof(float));
  
  hipMemset(d_ExtForces.x, 0, 192*MaxNoofC180s*sizeof(float));
  hipMemset(d_ExtForces.y, 0, 192*MaxNoofC180s*sizeof(float));
  hipMemset(d_ExtForces.z, 0, 192*MaxNoofC180s*sizeof(float));

  hipMemset(d_area, 0, MaxNoofC180s*sizeof(float));
  CudaErrorCheck();

  hipMemset(d_boxMin, 0, 3*sizeof(float));
  hipMemset(d_contactForces.x, 0, 192*MaxNoofC180s*sizeof(float));
  hipMemset(d_contactForces.y, 0, 192*MaxNoofC180s*sizeof(float));
  hipMemset(d_contactForces.z, 0, 192*MaxNoofC180s*sizeof(float));
  
  thrust::host_vector<angles3> theta0(192);
  thrust::device_vector<angles3> d_theta0V(192);
  angles3* d_theta0 = thrust::raw_pointer_cast(&d_theta0V[0]);

  h_contactForces.x = (float *)calloc(192*MaxNoofC180s, sizeof(float));
  h_contactForces.y = (float *)calloc(192*MaxNoofC180s, sizeof(float));
  h_contactForces.z = (float *)calloc(192*MaxNoofC180s, sizeof(float));


  h_ExtForces.x = (float *)calloc(192*MaxNoofC180s, sizeof(float));
  h_ExtForces.y = (float *)calloc(192*MaxNoofC180s, sizeof(float));
  h_ExtForces.z = (float *)calloc(192*MaxNoofC180s, sizeof(float));


  thrust::device_vector<float> d_XMMV(MaxNoofC180s*192);
  thrust::fill(d_XMMV.begin(), d_XMMV.end(), 0.f);
  float *d_XMM = thrust::raw_pointer_cast(&d_XMMV[0]);
  
  thrust::device_vector<float> d_YMMV(MaxNoofC180s*192);
  thrust::fill(d_YMMV.begin(), d_YMMV.end(), 0.f);
  float *d_YMM = thrust::raw_pointer_cast(&d_YMMV[0]); 
  
  thrust::device_vector<float> d_ZMMV(MaxNoofC180s*192);
  thrust::fill(d_ZMMV.begin(), d_ZMMV.end(), 0.f);
  float *d_ZMM = thrust::raw_pointer_cast(&d_ZMMV[0]);

  thrust::device_vector<int> d_NoofNNlistV(1024*1024);
  thrust::fill(d_NoofNNlistV.begin(), d_NoofNNlistV.end(), 0);
  int *d_NoofNNlist = thrust::raw_pointer_cast(&d_NoofNNlistV[0]);

  thrust::device_vector<float> d_timeV(192*MaxNoofC180s);
  thrust::fill(d_timeV.begin(), d_timeV.end(), delta_t); 
  float *d_time = thrust::raw_pointer_cast(&d_timeV[0]);
  thrust::host_vector<float> h_timeV(192*MaxNoofC180s);

  thrust::host_vector<float> dt_listV(Time_steps + equiStepCount);
  thrust::fill(dt_listV.begin(), dt_listV.end(), delta_t);  
  thrust::device_vector<float> d_XtV(192*MaxNoofC180s);
  thrust::fill(d_XtV.begin(), d_XtV.end(), 0.f);
  float *d_Xt = thrust::raw_pointer_cast(&d_XtV[0]);

  thrust::device_vector<float> d_YtV(192*MaxNoofC180s);
  thrust::fill(d_YtV.begin(), d_YtV.end(), 0.f);
  float *d_Yt = thrust::raw_pointer_cast(&d_YtV[0]);

  thrust::device_vector<float> d_ZtV(192*MaxNoofC180s);
  thrust::fill(d_ZtV.begin(), d_ZtV.end(), 0.f);
  float *d_Zt = thrust::raw_pointer_cast(&d_ZtV[0]);

  thrust::device_vector<float> d_AdpErrorsV(192*MaxNoofC180s); 
  thrust::fill(d_AdpErrorsV.begin(), d_AdpErrorsV.end(), 0); 
  float *d_AdpErrors = thrust::raw_pointer_cast(&d_AdpErrorsV[0]); 

  thrust::host_vector<float> h_AdpErrors(192*MaxNoofC180s);

  thrust::device_vector<float> d_volumeV(MaxNoofC180s);
  thrust::host_vector<float> h_volume(MaxNoofC180s);
  thrust::fill(d_volumeV.begin(), d_volumeV.end(), 0.f);
  d_volume = thrust::raw_pointer_cast(&d_volumeV[0]);
  volume = thrust::raw_pointer_cast(&h_volume[0]);


  // thrust::device_vector<float> d_R0V(MaxNoofC180s);
  // thrust::host_vector<float> h_R0V(MaxNoofC180s);
  // d_R0 = thrust::raw_pointer_cast(&d_R0V[0]);
  // thrust::fill(h_R0V.begin(), h_R0V.end(), R0);
  // d_R0V = h_R0V;

  if (hipSuccess != hipMemcpy(d_R0, h_R0, 3*192*sizeof(float), hipMemcpyHostToDevice)) return -1; 

  // velocities

  

  
  CMx   = (float *)calloc(MaxNoofC180s, sizeof(float));
  CMy   = (float *)calloc(MaxNoofC180s, sizeof(float));
  CMz   = (float *)calloc(MaxNoofC180s, sizeof(float));
  VCMx   = (float *)calloc(MaxNoofC180s, sizeof(float));
  VCMy   = (float *)calloc(MaxNoofC180s, sizeof(float));
  VCMz   = (float *)calloc(MaxNoofC180s, sizeof(float));
  //volume= (float *)calloc(MaxNoofC180s, sizeof(float));
  area= (float *)calloc(MaxNoofC180s, sizeof(float));
  cell_div = (char *)calloc(MaxNoofC180s, sizeof(char));
  cell_div_inds = (int *)calloc(MaxNoofC180s, sizeof(int));
  cell_Apo = (char *)calloc(MaxNoofC180s, sizeof(char));
  cell_Apo_inds = (int *)calloc(MaxNoofC180s, sizeof(int));
  Minx  = (float *)calloc(1024, sizeof(float));
  Maxx  = (float *)calloc(1024, sizeof(float));
  Miny  = (float *)calloc(1024, sizeof(float));
  Maxy  = (float *)calloc(1024, sizeof(float));
  Minz  = (float *)calloc(1024, sizeof(float));
  Maxz  = (float *)calloc(1024, sizeof(float));
  NoofNNlist = (int *)calloc( 1024*1024,sizeof(int));
  NNlist =  (int *)calloc(32*1024*1024, sizeof(int));
  resetIndices = (int *)calloc(MaxNoofC180s, sizeof(int));
  
  CPUMemory += MaxNoofC180s*7L*sizeof(float);
  CPUMemory += MaxNoofC180s*sizeof(float);
  CPUMemory += 3L*MaxNoofC180s*sizeof(float);
  CPUMemory += 6L*1024L*sizeof(float);
  CPUMemory += MaxNoofC180s*sizeof(char);
  CPUMemory += MaxNoofC180s*sizeof(int);
  CPUMemory += MaxNoofC180s*sizeof(int); 
  CPUMemory += 3*180*sizeof(float); 


  //hipMemcpy(d_pressList, pressList, MaxNoofC180s*sizeof(float), hipMemcpyHostToDevice);
  


  if (colloidal_dynamics && rand_vel & !Restart ){
  	 
  	if ( initialize_Vel(Orig_No_of_C180s) != 0 ) return(-1);
  	hipMemcpy(d_velListX, velListX, 192*No_of_C180s*sizeof(float), hipMemcpyHostToDevice);
      	hipMemcpy(d_velListY, velListY, 192*No_of_C180s*sizeof(float), hipMemcpyHostToDevice);
  	hipMemcpy(d_velListZ, velListZ, 192*No_of_C180s*sizeof(float), hipMemcpyHostToDevice);	 
  	 
  }


  

  // Better way to see how much GPU memory is being used.
  size_t totalGPUMem;
  size_t freeGPUMem;

  if ( hipSuccess != hipMemGetInfo ( &freeGPUMem, &totalGPUMem ) ) {
      printf("Couldn't read GPU Memory status\nExiting...\n");
      CudaErrorCheck();
      exit(1);
  }

  GPUMemory = totalGPUMem - freeGPUMem;


  hipMemcpy(d_C180_nn,   C180_nn,   3*192*sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(d_C180_sign, C180_sign, 180*sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(d_C180_56,   C180_56,   7*92*sizeof(int),hipMemcpyHostToDevice);


  float tempX[192*No_of_C180s];


  hipMemcpy(d_cell_div, cell_div, MaxNoofC180s*sizeof(char), hipMemcpyHostToDevice);
  CudaErrorCheck();
  hipMemcpy(d_cell_Apo, cell_Apo, MaxNoofC180s*sizeof(char), hipMemcpyHostToDevice);
  CudaErrorCheck();



if (Restart == 0) {	


  	for (int cell = 0; cell < MaxNoofC180s; cell++){
		if (colloidal_dynamics){
			pressList[cell] = 0;
		}else{
		 	pressList[cell] = minPressure; 
  		}
  	}
	
  	for (int i =  0; i < MaxNoofC180s; ++i){
       	if (colloidal_dynamics){
			Growth_rate[i] = 0;
		}else{
			Growth_rate[i] = rMax; 
  		}
  	}
	
	
  	for (int i =  0; i < MaxNoofC180s; ++i){
      		youngsModArray[i] = stiffness1; 
  	}
  
    	for (int i =  0; i < MaxNoofC180s; ++i){
      		DivisionVolume[i] = divVol; 
  	}
  	
  	for (int i =  0; i < MaxNoofC180s; ++i){
      		gamma_env[i] = gamma_visc; 
  	}
  	
  	for (int i =  0; i < MaxNoofC180s; ++i){
      		viscotic_damp[i] = viscotic_damping; 
  	}
  	
  
	if(!colloidal_dynamics && useDifferentCell) SecondCell(Orig_No_of_C180s);
  
 } // end of restart if else

 
 // copy to device after reading restart file or initial values
  hipMemcpy(d_XP, X, 192*No_of_C180s*sizeof(float),hipMemcpyHostToDevice);
  hipMemcpy(d_YP, Y, 192*No_of_C180s*sizeof(float),hipMemcpyHostToDevice);
  hipMemcpy(d_ZP, Z, 192*No_of_C180s*sizeof(float),hipMemcpyHostToDevice);
  CudaErrorCheck();
  hipMemcpy(d_X,  X, 192*No_of_C180s*sizeof(float),hipMemcpyHostToDevice);
  hipMemcpy(d_Y,  Y, 192*No_of_C180s*sizeof(float),hipMemcpyHostToDevice);
  hipMemcpy(d_Z,  Z, 192*No_of_C180s*sizeof(float),hipMemcpyHostToDevice);
  CudaErrorCheck();
  hipMemcpy(d_XM, X, 192*No_of_C180s*sizeof(float),hipMemcpyHostToDevice);
  hipMemcpy(d_YM, Y, 192*No_of_C180s*sizeof(float),hipMemcpyHostToDevice);
  hipMemcpy(d_ZM, Z, 192*No_of_C180s*sizeof(float),hipMemcpyHostToDevice);
  CudaErrorCheck();
  hipMemcpy(d_velListX, velListX, 192*No_of_C180s*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_velListY, velListY, 192*No_of_C180s*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_velListZ, velListZ, 192*No_of_C180s*sizeof(float), hipMemcpyHostToDevice);
  CudaErrorCheck();
  hipMemcpy(d_Youngs_mod, youngsModArray, MaxNoofC180s*sizeof(float), hipMemcpyHostToDevice);
  CudaErrorCheck();
  hipMemcpy(d_Growth_rate, Growth_rate, MaxNoofC180s*sizeof(float), hipMemcpyHostToDevice);
  CudaErrorCheck();
  hipMemcpy(d_CellINdex, CellINdex, MaxNoofC180s*sizeof(int), hipMemcpyHostToDevice);
  CudaErrorCheck(); 
  hipMemcpy(d_pressList, pressList, No_of_C180s*sizeof(float), hipMemcpyHostToDevice);
  CudaErrorCheck();
  hipMemcpy(d_ScaleFactor, ScaleFactor, No_of_C180s*sizeof(float), hipMemcpyHostToDevice);
  CudaErrorCheck();
  hipMemcpy(d_DivisionVolume, DivisionVolume, No_of_C180s*sizeof(float), hipMemcpyHostToDevice);
  CudaErrorCheck();
  hipMemcpy(d_gamma_env, gamma_env, No_of_C180s*sizeof(float), hipMemcpyHostToDevice);
  CudaErrorCheck();
  hipMemcpy(d_viscotic_damp, viscotic_damp, No_of_C180s*sizeof(float), hipMemcpyHostToDevice);
  CudaErrorCheck();


/**************************************************************************************************************/


  // initialize device rng


    if (add_rands){
      hiprandGenerator_t gen;
      
      if (hipMalloc((void **)&d_rngStates, sizeof(hiprandState)*192*MaxNoofC180s) != hipSuccess){
          fprintf(stderr, "ERROR: Failed to allocate rng state memory in %s, at %d\n", __FILE__, __LINE__);
          return 1;
      }
          

      if (hipMalloc((void **)&d_seeds, sizeof(unsigned int)*192*MaxNoofC180s) != hipSuccess){
          fprintf(stderr, "ERROR: Failed to allocate rng seeds in %s, at %d\n", __FILE__, __LINE__);
          return 1;
      }
      
      time_t secs_since_1970;
      
      
      hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MT19937);
      CudaErrorCheck();

      //hiprandSetPseudoRandomGeneratorSeed(gen, time(NULL));
      hiprandSetPseudoRandomGeneratorSeed(gen, time(&secs_since_1970));
      CudaErrorCheck();

      if ( rand_seed <= 0 ) printf("   rand_seed = %lu\n",(unsigned long)secs_since_1970);

      if (rand_seed > 0){
          hiprandSetPseudoRandomGeneratorSeed(gen, rand_seed);
          CudaErrorCheck();
      }

      hiprandGenerate(gen, d_seeds, MaxNoofC180s*192);
      CudaErrorCheck();
  
      DeviceRandInit<<<(192*MaxNoofC180s)/256 + 1, 256>>>(d_rngStates, d_seeds, 192*MaxNoofC180s);
      CudaErrorCheck();
  }


  prevnoofblocks  = No_of_C180s;
  noofblocks      = No_of_C180s;
  threadsperblock = 192;
  printf("   no of blocks = %d, threadsperblock = %d, no of threads = %ld\n",
         noofblocks, threadsperblock, ((long) noofblocks)*((long) threadsperblock));

//  CenterOfMass<<<No_of_C180s,256>>>(No_of_C180s,
//                                    d_X, d_Y, d_Z,
//                                    d_CMx, d_CMy, d_CMz);
  
  //bounding_boxes<<<No_of_C180s,32>>>(No_of_C180s,d_X,d_Y,d_Z,
  //                                   d_bounding_xyz, d_CMx, d_CMy, d_CMz);

  //CudaErrorCheck(); 

  reductionblocks = (No_of_C180s-1)/1024+1;
  //minmaxpre<<<reductionblocks,1024>>>( No_of_C180s, d_bounding_xyz,
  //                                     d_Minx, d_Maxx, d_Miny, d_Maxy, d_Minz, d_Maxz);
 // CudaErrorCheck(); 
  //minmaxpost<<<1,1024>>>(reductionblocks, d_Minx, d_Maxx, d_Miny, d_Maxy, d_Minz, d_Maxz);
 // CudaErrorCheck(); 
 // hipMemcpy(Minx, d_Minx, 6*sizeof(float),hipMemcpyDeviceToHost);
  //  DL = 3.8f;
  
  globalrank = 0;


  // open trajectory file
  
   if (Restart == 0){
  	trajfile = fopen (trajFileName, "w");
  }else{
  	trajfile = fopen (trajFileName, "a+");
  }
  
  if ( trajfile == NULL)
  {
      printf("Failed to open %s \n", trajFileName);
      return -1;
  }

  FILE* forceFile;
  if (Restart == 0){
    	 forceFile = fopen(forces_file, "w");
  }else{
   	 forceFile = fopen(forces_file, "a");
  }
  

  FILE* velFile;
  if (Restart == 0){
    	 velFile = fopen("velocity.xyz", "w");
  }else{
   	 velFile = fopen("velocity.xyz", "a+");
  }	


#ifdef OUTPUT_ADP_ERROR
  FILE* timeFile = fopen("times", "w");
  FILE* errFile = fopen("errors", "w"); 
#endif 


  if (correct_com == true){
      CenterOfMass<<<No_of_C180s,256>>>(No_of_C180s,
                                        d_X, d_Y, d_Z,
                                        d_CMx, d_CMy, d_CMz);
      hipMemcpy(CMx, d_CMx, No_of_C180s*sizeof(float),hipMemcpyDeviceToHost);
      hipMemcpy(CMy, d_CMy, No_of_C180s*sizeof(float),hipMemcpyDeviceToHost);
      hipMemcpy(CMz, d_CMz, No_of_C180s*sizeof(float),hipMemcpyDeviceToHost);
      calc_sys_CM();
      //printf("sysCMx = %f, sysCMy = %f, sysCmz = %f\n", sysCMx, sysCMy, sysCMz);
      CorrectCoMMotion<<<(No_of_C180s*192)/1024 + 1, 1024>>>(d_X, d_Y, d_Z,
                                                             sysCMx, sysCMy, sysCMz,
                                                             No_of_C180s*192);
          
      CorrectCoMMotion<<<(No_of_C180s*192)/1024 + 1, 1024>>>(d_XP, d_YP, d_ZP,
                                                             sysCMx, sysCMy, sysCMz,
                                                             No_of_C180s*192);

      CorrectCoMMotion<<<(No_of_C180s*192)/1024 + 1, 1024>>>(d_XM, d_YM, d_ZM,
                                                             sysCMx, sysCMy, sysCMz,
                                                             No_of_C180s*192);
      CudaErrorCheck(); 
  }
  

  if ( correct_Vcom == true){
     
      VelocityCenterOfMass<<<No_of_C180s,256>>>(No_of_C180s,
                                        d_velListX, d_velListY, d_velListZ,
                                        d_VCMx, d_VCMy, d_VCMz);
      hipMemcpy(VCMx, d_VCMx, No_of_C180s*sizeof(float),hipMemcpyDeviceToHost);
      hipMemcpy(VCMy, d_VCMy, No_of_C180s*sizeof(float),hipMemcpyDeviceToHost);
      hipMemcpy(VCMz, d_VCMz, No_of_C180s*sizeof(float),hipMemcpyDeviceToHost);
      calc_sys_VCM();
      CorrectCoMVelocity<<<(No_of_C180s*192)/1024 + 1, 1024>>>(d_velListX, d_velListY, d_velListZ,
                                                             sysVCMx, sysVCMy, sysVCMz,
                                                             No_of_C180s*192);
          
      CudaErrorCheck(); 
      //printf("sysVCMx = %f, sysVCMy = %f, sysVCmz = %f\n", sysVCMx, sysVCMy, sysVCMz);
  }  
  
  

  // Set up walls if needed
  if (useWalls == 1){
      // First we must make sure that the walls surround the
      // starting system.
      CenterOfMass<<<No_of_C180s,256>>>(No_of_C180s,
                                        d_X, d_Y, d_Z,
                                        d_CMx, d_CMy, d_CMz);
      hipMemcpy(CMx, d_CMx, No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);
      hipMemcpy(CMy, d_CMy, No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);
      hipMemcpy(CMz, d_CMz, No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);
      float COMx = 0, COMy = 0, COMz = 0;

      for(int cell = 0; cell < No_of_C180s; cell++){
          COMx += CMx[cell];
          COMy += CMy[cell];
          COMz += CMz[cell];
      }

      COMx = COMx/No_of_C180s;
      COMy = COMy/No_of_C180s;
      COMz = COMz/No_of_C180s;


      if (perpAxis[0] == 'Z' || perpAxis[0] == 'z' ){
          // Check that the walls are far enough from the beginning cells
          float tempZ[192*No_of_C180s];
          hipMemcpy(tempZ, d_Z, 192*No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);
          std::sort(tempZ, tempZ+No_of_C180s);
          float radius = 3.0 * divVol / 4.0;
          radius = radius/3.14159;
          radius = std::pow(radius, 0.33333333333);
          dAxis = dAxis * 2 * radius;

          if (dAxis < (tempZ[No_of_C180s] - tempZ[0])){
                  printf("Distance between walls is too small\nExiting...");
                  printf("Starting system size= %f\nWall gap = %f",
                         tempZ[No_of_C180s] - tempZ[0], dAxis);
                  return(-1);
              }

          wall1 = COMz - (dAxis/2.0);
          wall2 = COMz + (dAxis/2.0);
          wallLStart = COMx - (wallLen/2.0);
          wallLEnd = COMx + (wallLen/2.0);
          wallWStart = COMy - (wallWidth/2.0);
          wallWEnd = COMy + (wallWidth/2.0);
      }
      else {
          printf(" Invalid wall axis selection %s\nExiting...", perpAxis);
          return(-1);
      }

  }


  float rGrowth = 0;
  bool growthDone = false;
  
  boxMin[0] = BoxMin.x;
  boxMin[1] = BoxMin.y;
  boxMin[2] = BoxMin.z;
  
  // Setup simulation box, if needed (non-pbc)
  if (useRigidSimulationBox){
      printf("   Setup rigid (non-PBC) box...\n"); 
     // boxLength = boxLength*ceil(max( (Minx[5]-Minx[4]), max( (Minx[1]-Minx[0]), (Minx[3]-Minx[2]) ) ));
      //if (boxLength < minBoxLength) boxLength = minBoxLength
      //if (Side_length < 5) boxLength = boxLength * 5; 
 
      
      if ((boxMax.z - BoxMin.z) < divVol){
      	//DL = divVol; 
      	DL = divisionV;
      } else {
      	DL = 1.5;
      }
      
      
      Xdiv = ceil((boxMax.x - BoxMin.x)/DL);
      printf (" %d \n",Xdiv);
      Ydiv = ceil((boxMax.y - BoxMin.y)/DL);
      printf (" %d \n",Ydiv);
      Zdiv = ceil((boxMax.z - BoxMin.z)/DL);
      printf (" %d \n",Zdiv); 

      printf("   Done!\n");
      printf("   Simulation box minima:\n   X: %f, Y: %f, Z: %f\n", BoxMin.x, BoxMin.y, BoxMin.z);
      printf("   Simulation box maximum:\n   X: %f, Y: %f, Z: %f\n", boxMax.x, boxMax.y, boxMax.z);
     // printf("   Simulation box length = %f\n", boxLength);
  }

  /*
  hipMemcpy(CMx, d_CMx, No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);
  CudaErrorCheck();
  for(int i=0;i<No_of_C180s;i++){
  printf (" %f \n",CMx[i]);
  }
  */

 

  // Code to set up pbc things
  if (usePBCs || useLEbc){
    printf("   Setup PBC box...\n"); 
    
    boxMin[0] = 0;
    boxMin[1] = 0;
    boxMin[2] = 0;
    BoxMin.x = 0.0;
    BoxMin.y = 0.0;
    BoxMin.z = 0.0;
  

    if ((boxMax.z - BoxMin.z) < divVol){
      	//DL = divVol;
      	DL = divisionV; 
    } else {
      	DL = 1.5;
    }
    
    Xdiv = ceil((boxMax.x - boxMin[0])/DL);
    DLp.x = (boxMax.x - boxMin[0])/Xdiv;
    printf (" %d \n",Xdiv);
    Ydiv = ceil((boxMax.y - boxMin[1])/DL);
    DLp.y = (boxMax.y - boxMin[1])/Ydiv;
    printf (" %d \n",Ydiv);
    Zdiv = ceil((boxMax.z - boxMin[2])/DL);
    DLp.z = (boxMax.z - boxMin[2])/Zdiv;
    printf (" %d \n",Zdiv);  
    
    printf (" %f \n",DLp.x);
    printf (" %f \n",DLp.y);
    printf (" %f \n",DLp.z);

    printf("   Done!\n");
    printf("   Simulation box minima:\n   X: %f, Y: %f, Z: %f\n", BoxMin.x, BoxMin.y, BoxMin.z);
    printf("   Simulation box maximum:\n   X: %f, Y: %f, Z: %f\n", boxMax.x, boxMax.y, boxMax.z);
   // printf("   Simulation box length = %f\n", boxLength);
  }



  hipMemcpy(d_boxMin, boxMin, 3*sizeof(float), hipMemcpyHostToDevice);
  CudaErrorCheck(); 

  CenterOfMass<<<No_of_C180s,256>>>(No_of_C180s, d_X, d_Y, d_Z, d_CMx, d_CMy, d_CMz);
  //DL = divVol; 
  CudaErrorCheck(); 


   if(usePBCs ){
        
       CoorUpdatePBC <<<No_of_C180s, threadsperblock>>> (d_X, d_Y, d_Z,
                                                          d_XM, d_YM, d_ZM,
                                                          d_CMx, d_CMy, d_CMz,
                                                          boxMax, divVol, No_of_C180s,
                                                          useRigidBoxZ, useRigidBoxY);
  


       CudaErrorCheck();
   }
   if(useLEbc){
        
       UpdateLEbc <<<No_of_C180s, threadsperblock>>> (d_X, d_Y, d_Z, d_XM, d_YM, d_ZM,
                        d_velListX, d_velListY, d_velListZ, d_CMx, d_CMy, d_CMz,
                        boxMax, divVol, No_of_C180s, Pshift, Vshift);
                        
        CudaErrorCheck();	
	
    }





      if (useRigidSimulationBox){	
      		makeNNlist<<<No_of_C180s/512+1,512>>>( No_of_C180s, d_CMx, d_CMy, d_CMz,
        	Xdiv, Ydiv, Zdiv, BoxMin, d_NoofNNlist, d_NNlist, DL);
        
        	CudaErrorCheck(); 
       }
	if(usePBCs){
       	makeNNlistPBC<<<No_of_C180s/512+1,512>>>( No_of_C180s, d_CMx, d_CMy, d_CMz,
        	attraction_range, Xdiv, Ydiv, Zdiv, boxMax, d_NoofNNlist, d_NNlist, DLp, useRigidBoxZ,useRigidBoxY);
        
        	CudaErrorCheck(); 
       }
       if(useLEbc){
       	makeNNlistLEbc<<<No_of_C180s/512+1,512>>>( No_of_C180s, d_CMx, d_CMy, d_CMz,
        	attraction_range, Xdiv, Ydiv, Zdiv, boxMax, d_NoofNNlist, d_NNlist, DLp, Pshift, useRigidBoxZ);
        	
        	CudaErrorCheck();
       
       } 
  



  if (constrainAngles){
      // Code to initialize equillibrium angles
      float3 p, ni, nj, nk;
      for (int n = 0; n<180; n++){
          p = make_float3(X[n], Y[n], Z[n]); 

          ni = make_float3(X[C180_nn[0*192 + n]], Y[C180_nn[0*192 + n]], 
                           Z[C180_nn[0*192 + n]]); 
          
          nj = make_float3(X[C180_nn[1*192 + n]], Y[C180_nn[1*192 + n]], 
                           Z[C180_nn[1*192 + n]]);
          
          nk = make_float3(X[C180_nn[2*192 + n]], Y[C180_nn[2*192 + n]],
                           Z[C180_nn[2*192 + n]]);

          ni = ni-p;
          nj = nj-p;
          nk = nk-p; 

          theta0[n].aij = acos(dot(ni, nj)/(mag(ni)*mag(nj)));
          
          theta0[n].ajk = acos(dot(nj, nk)/(mag(nj)*mag(nk)));
          
          theta0[n].aik = acos(dot(ni, nk)/(mag(ni)*mag(nk)));

        }

      d_theta0V = theta0; 
      CudaErrorCheck(); 
  }

  // if (useDifferentStiffnesses && recalc_r0){

//       CalculateR0<<<No_of_C180s/1024 + 1, 1024>>>(d_R0,
//                                                   d_X, d_Y, d_Z,
//                                                   d_C180_nn,
//                                                   d_Youngs_mod,
//                                                   stiffness2,
//                                                   No_of_C180s);
// #ifdef RO_DEBUG
//       h_R0V = d_R0V;

//       hipMemcpy(youngsModArray, d_Youngs_mod, sizeof(float)*MaxNoofC180s, hipMemcpyDeviceToHost);
      
//       for (int i =0; i < No_of_C180s; ++i){
//           std::cout << "Cell " << i << " R0 = "
//                     << h_R0V[i] << " E = " << youngsModArray[i] << std::endl;
//       }
// #endif
  
//   }

  // Different kind of pressure stuff

  float r_CM_o = pow((3.0/4.0) * (1/3.14159) * divVol*2.0, 1.0/3);



  printf("   Total amount of GPU memory used =    %8.2lf MB\n",GPUMemory/(1024*1024.0));
  printf("   Total amount of CPU memory used =    %8.2lf MB\n",CPUMemory/(1024*1024.0));

  bool phase = false;

  // initial conditions

  // X[10] *= 1.1;

  // hipMemcpy(d_X, X, 192*No_of_C180s*sizeof(float), hipMemcpyHostToDevice);

  if (useRigidSimulationBox){	
  	CalculateConForce<<<No_of_C180s,threadsperblock>>>(   No_of_C180s, d_C180_nn, d_C180_sign,
                                                     	d_X,  d_Y,  d_Z,
                                                     	d_CMx, d_CMy, d_CMz,
                                                     	d_R0, d_ScaleFactor, d_pressList, d_Youngs_mod , stiffness1, 
                                                     	internal_damping, d_time,
                                                     	attraction_strength, attraction_range,
                                                     	repulsion_strength, repulsion_range,
                                                     	d_viscotic_damp,
                                                     	Xdiv, Ydiv, Zdiv, usePBCs, boxMax, d_NoofNNlist, d_NNlist, DL, d_gamma_env,
                                                     	wall1, wall2,
                                                     	threshDist, useWalls,
                                                     	d_velListX, d_velListY, d_velListZ,
                                                     	useRigidSimulationBox, boxLength, BoxMin, Youngs_mod,
                                                     	constrainAngles, d_theta0, d_fConList, r_CM_o, d_contactForces, d_ExtForces, d_volume, divVol,
                                                     	impurityNum); 
                                                     	
       CudaErrorCheck();
                                                     	
      CalculateDisForce<<<No_of_C180s, threadsperblock>>>(No_of_C180s, d_C180_nn, d_C180_sign, 
                                                        d_X, d_Y, d_Z,
                                                        d_CMx, d_CMy, d_CMz,r_CM_o,
                                                        internal_damping,
                                                        attraction_range,
                                                        d_viscotic_damp,
                                                        Xdiv, Ydiv, Zdiv, usePBCs, boxMax,BoxMin,
                                                        d_NoofNNlist, d_NNlist, DL, d_gamma_env,
                                                        d_velListX, d_velListY, d_velListZ,
                                                        d_fDisList,impurityNum);
                                                        
                                                        
       CudaErrorCheck();                                                  
  }
  
  if(usePBCs){
  
    	CalculateConForcePBC<<<No_of_C180s,threadsperblock>>>( No_of_C180s, d_C180_nn, d_C180_sign,
                                                     	d_X,  d_Y,  d_Z,
                                                     	d_CMx, d_CMy, d_CMz,
                                                     	d_R0,d_ScaleFactor, d_pressList, d_Youngs_mod , stiffness1, 
                                                     	internal_damping, d_time,
                                                     	attraction_strength, attraction_range,
                                                     	repulsion_strength, repulsion_range,
                                                     	d_viscotic_damp,
                                                     	Xdiv, Ydiv, Zdiv, usePBCs, boxMax, d_NoofNNlist, d_NNlist, DLp, d_gamma_env,
                                                     	wall1, wall2,
                                                     	threshDist, useWalls,
                                                     	d_velListX, d_velListY, d_velListZ,
                                                     	useRigidSimulationBox, boxLength, BoxMin, Youngs_mod,
                                                     	constrainAngles, d_theta0, d_fConList, r_CM_o, d_contactForces, d_volume, divVol,
                                                     	useRigidBoxZ,useRigidBoxY);
                                                     	
       CudaErrorCheck();                                             	
  	
  	                                                     	
      CalculateDisForcePBC<<<No_of_C180s, threadsperblock>>>(No_of_C180s, d_C180_nn, d_C180_sign, 
                                                        d_X, d_Y, d_Z,
                                                        d_CMx, d_CMy, d_CMz,r_CM_o,
                                                        internal_damping,
                                                        attraction_range,
                                                        d_viscotic_damp,
                                                        Xdiv, Ydiv, Zdiv, usePBCs, boxMax,
                                                        d_NoofNNlist, d_NNlist, DLp, d_gamma_env,
                                                        d_velListX, d_velListY, d_velListZ,
                                                        d_fDisList, useRigidBoxZ,useRigidBoxY);
    CudaErrorCheck();	
  
  }
  if(useLEbc){
  
  CalculateConForceLEbc<<<No_of_C180s,threadsperblock>>>( No_of_C180s, d_C180_nn, d_C180_sign,
                                                     	d_X,  d_Y,  d_Z,
                                                     	d_CMx, d_CMy, d_CMz,
                                                     	d_R0,d_ScaleFactor, d_pressList, d_Youngs_mod , stiffness1, 
                                                     	internal_damping, d_time,
                                                     	attraction_strength, attraction_range,
                                                     	repulsion_strength, repulsion_range,
                                                     	d_viscotic_damp,
                                                     	Xdiv, Ydiv, Zdiv, usePBCs, boxMax, d_NoofNNlist, d_NNlist, DLp, d_gamma_env,
                                                     	wall1, wall2,
                                                     	threshDist, useWalls,
                                                     	d_velListX, d_velListY, d_velListZ,
                                                     	useRigidSimulationBox, boxLength, BoxMin, Youngs_mod,
                                                     	constrainAngles, d_theta0, d_fConList, r_CM_o, d_contactForces, d_volume, divVol,
                                                     	Pshift,useRigidBoxZ);
                                                     	
       CudaErrorCheck();                                             	
  	
  	                                                     	
      CalculateDisForceLEbc<<<No_of_C180s, threadsperblock>>>(No_of_C180s, d_C180_nn, d_C180_sign, 
                                                        d_X, d_Y, d_Z,
                                                        d_CMx, d_CMy, d_CMz,r_CM_o,
                                                        internal_damping,
                                                        attraction_range,
                                                        d_viscotic_damp,
                                                        Xdiv, Ydiv, Zdiv, usePBCs, boxMax,
                                                        d_NoofNNlist, d_NNlist, DLp, d_gamma_env,
                                                        d_velListX, d_velListY, d_velListZ,
                                                        d_fDisList, Pshift, Vshift, useRigidBoxZ);
    CudaErrorCheck();	
  
  
  
  }
  
  	  volumes<<<No_of_C180s,192>>>(No_of_C180s, d_C180_56,
                                     d_X, d_Y, d_Z,
                                     d_CMx , d_CMy, d_CMz,
                                     d_volume, d_cell_div, d_DivisionVolume,
                                     checkSphericity, d_area, phase_count, step,
                                     stiffness1, useDifferentCell, d_Youngs_mod,
                                     recalc_r0, ApoVol ,d_cell_Apo, d_ScaleFactor);
  

	
  int t = MaxNoofC180s;	
  if (Restart ==0){
  
  	if (binaryOutput){
  
      		
      		fwrite(&t, sizeof(int), 1, trajfile);
      
      		t = (int)useDifferentCell;
      		fwrite(&t, sizeof(int), 1, trajfile);
      
      		t = (Time_steps+equiStepCount+1) / trajWriteInt;
      		fwrite(&t, sizeof(int), 1, trajfile);
      
    
     		WriteBinaryTraj(0, trajfile, 1); 
  	} else {
      		fprintf(trajfile, "Header Start:\n");
      		fprintf(trajfile, "Maximum number of cells:\n%d\n", MaxNoofC180s);

      		fprintf(trajfile, "Using variable stiffness:\n");
      		if (useDifferentCell) 
          		fprintf(trajfile, "True\n");
      		else
          		fprintf(trajfile, "False\n");

      		fprintf(trajfile, "Maximum number of frames:\n%d\n", (Time_steps+equiStepCount+1) / trajWriteInt);
     	 	fprintf(trajfile, "Header End\n");
      		write_traj(0, trajfile);
  	}
  	if (write_cont_force){
  
      		fprintf(forceFile, "step,num_cells,cell_ind,node_ind,glob_node_ind,FX,FY,FZ,F,FX_ext,FY_ext,FZ_ext,F_ext,VX,VY,VZ,V,X,Y,Z,P,Vol,Area\n");
      
      		hipMemcpy(h_contactForces.x, d_contactForces.x, 192*No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);
      		hipMemcpy(h_contactForces.y, d_contactForces.y, 192*No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);
      		hipMemcpy(h_contactForces.z, d_contactForces.z, 192*No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);
      		hipMemcpy(h_ExtForces.x, d_ExtForces.x, 192*No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);
      		hipMemcpy(h_ExtForces.y, d_ExtForces.y, 192*No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);
		hipMemcpy(h_ExtForces.z, d_ExtForces.z, 192*No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);
      		
      		hipMemcpy(velListX, d_velListX, 192*No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);
      		hipMemcpy(velListY, d_velListY, 192*No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);
      		hipMemcpy(velListZ, d_velListZ, 192*No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);

		hipMemcpy(pressList, d_pressList, No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);      
      		hipMemcpy(volume, d_volume, No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);
      		hipMemcpy(area, d_area, No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);	
      		writeForces(forceFile, 0, No_of_C180s);
  	}
  	if(write_vel_file){
  	          
               hipMemcpy(velListX, d_velListX, 192*No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);
               hipMemcpy(velListY, d_velListY, 192*No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);
               hipMemcpy(velListZ, d_velListZ, 192*No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);
               
               t = MaxNoofC180s;
               fwrite(&t, sizeof(int), 1, velFile);
      
      		t = (int)useDifferentCell;
      		fwrite(&t, sizeof(int), 1, velFile);
      
      		t = (Time_steps+equiStepCount+1) / trajWriteInt;
      		fwrite(&t, sizeof(int), 1, velFile);
               
               write_vel(0, velFile,1);
       }
  	
		  	
  	

  }

  //return 0;


  int numNodes = No_of_C180s*192;
  int NewCellInd = No_of_C180s;
  asym[0] = 0.5;
  WithoutApo = true;
  // Simulation loop
  for ( step = 1; step < Time_steps+1 + equiStepCount; step++)
  {
	Pshift = shear_rate*step*boxMax.x*delta_t;     



    
    //printf("step %d\n", step);
      numNodes = No_of_C180s*192;
      Integrate<<<No_of_C180s, threadsperblock>>>(d_XP, d_YP, d_ZP,
                                                 d_X, d_Y, d_Z,
                                                 d_XM, d_YM, d_ZM, 
                                                 d_velListX, d_velListY, d_velListZ, 
                                                 d_time,  mass,
                                                 d_fConList, d_fDisList, d_fRanList,
                                                 No_of_C180s, add_rands, d_rngStates, rand_scale_factor, impurityNum);
      CudaErrorCheck();


      ForwardTime<<<No_of_C180s, threadsperblock>>>(d_XP, d_YP, d_ZP, 
                                                   d_X , d_Y , d_Z ,
                                                   d_XM, d_YM, d_ZM,
                                                   No_of_C180s);
      CudaErrorCheck();



// ------------------------------------------------------------------------------------------------

	if ( apoptosis && WithoutApo && No_of_C180s > popToStartApo && !colloidal_dynamics)
	{
	        printf("Cell apoptosis starts.\n");
		WithoutApo = false;
	}

	
	// ----------------------------------------- Begin Cell Death ------------	
	if (apoptosis && !WithoutApo) {	
     
		int ApoNum1;
		int ApoNum2;
      		float rans[1];
        	int DInd;
        	int Aporank;
        	int count;
        	int Rcell = 0;
        	
        	
		if(!useDifferentCell) No_of_Cell1 = No_of_C180s;
      		
      		
      		if ((step)%1000 == 0){
	 		      		
      			ApoNum1 = ceil (Apo_rate * (No_of_Cell1- impurityNum));
      		        
	 		count = 0;
	 		Rcell = floor((No_of_Cell1 - impurityNum) * 0.9);


      		 	while (count < ApoNum1){
      		 		  	 	
      		 		if ( CellInApo1 >= Rcell) break;  	
      		 		 		
      		 		ranmar(rans, 1);
        	        	DInd = floor(rans[0]*(No_of_C180s - impurityNum)) + impurityNum;
        	        	
        	        	if (Growth_rate[DInd] == - squeeze_rate || CellINdex[DInd] < 0) continue;
        	        	
        	        	
        	        		Growth_rate[DInd] = - squeeze_rate;
    					count++;
    					CellInApo1 ++;
        	        	
        	        }
        	        
        	        if(useDifferentCell){
        	        
        	        	ApoNum2 = ceil (Apo_rate2 * (No_of_Cell2- impurityNum));
        	        		
        	        	count = 0;
        	        	Rcell = floor((No_of_Cell2 - impurityNum) * 0.9);
        	        	
      		 		while (count < ApoNum2){
      		 		  	 	
      		 			if ( CellInApo2 >= Rcell ) break;  	
      		 		 		
      		 			ranmar(rans, 1);
        	        		DInd = floor(rans[0]*(No_of_C180s - impurityNum)) + impurityNum;
        	        	
        	        		if (Growth_rate[DInd] == - squeeze_rate2 || CellINdex[DInd] >= 0) continue;
        	        	
        	        	
        	        		Growth_rate[DInd] = - squeeze_rate2;
    					count++;
    					CellInApo2 ++;
        	        	
        	        	}
        	        }        	        
        	        
			count_and_die();
			
			if (num_cell_Apo> 0){
			
			        hipMemcpy(X , d_X, No_of_C180s*192*sizeof(float), hipMemcpyDeviceToHost);
  				CudaErrorCheck();
  				hipMemcpy(Y , d_Y, No_of_C180s*192*sizeof(float), hipMemcpyDeviceToHost);
  				CudaErrorCheck();
  				hipMemcpy(Z , d_Z, No_of_C180s*192*sizeof(float), hipMemcpyDeviceToHost);
  				CudaErrorCheck();
  				hipMemcpy(velListX , d_velListX , No_of_C180s*192*sizeof(float), hipMemcpyDeviceToHost);
  				CudaErrorCheck();
  				hipMemcpy(velListY , d_velListY , No_of_C180s*192*sizeof(float), hipMemcpyDeviceToHost);
  				CudaErrorCheck();
  				hipMemcpy(velListZ , d_velListZ , No_of_C180s*192*sizeof(float), hipMemcpyDeviceToHost);
  				CudaErrorCheck();
        	               hipMemcpy(pressList, d_pressList, No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);
        	               CudaErrorCheck();
        	               hipMemcpy(youngsModArray, d_Youngs_mod ,No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);
  				CudaErrorCheck();
  				hipMemcpy(Growth_rate, d_Growth_rate, No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);
        	               CudaErrorCheck();			
			        hipMemcpy(ScaleFactor, d_ScaleFactor, sizeof(float)*No_of_C180s, hipMemcpyDeviceToHost);
            			CudaErrorCheck();
            			hipMemcpy(DivisionVolume, d_DivisionVolume, sizeof(float)*No_of_C180s, hipMemcpyDeviceToHost);
            			CudaErrorCheck();
            			hipMemcpy(gamma_env, d_gamma_env, sizeof(float)*No_of_C180s, hipMemcpyDeviceToHost);
            			CudaErrorCheck();
            			hipMemcpy(viscotic_damp, d_viscotic_damp, sizeof(float)*No_of_C180s, hipMemcpyDeviceToHost);
           			CudaErrorCheck();
			        hipMemcpy(CellINdex, d_CellINdex, No_of_C180s*sizeof(int), hipMemcpyDeviceToHost);
        	               CudaErrorCheck();

				size_t EndShift;
				for (int ApoCell = 0; ApoCell < num_cell_Apo; ApoCell++) {
          			
          			
          				Aporank = cell_Apo_inds[ApoCell] - ApoCell; 
          				EndShift =  (No_of_C180s - Aporank-1)*192*sizeof(float); 
          				
          				if (CellINdex[Aporank] >= 0) {
          					-- CellInApo1;
          					-- No_of_Cell1;
					} else {
						-- CellInApo2;
						-- No_of_Cell2;
					}

  				
  					hipMemcpy(X + 192*Aporank,  d_X + 192*(Aporank + 1), EndShift ,hipMemcpyDeviceToHost);
  					CudaErrorCheck();
  					hipMemcpy(Y + 192*Aporank,  d_Y + 192*(Aporank + 1), EndShift ,hipMemcpyDeviceToHost);
  					CudaErrorCheck();
  					hipMemcpy(Z + 192*Aporank,  d_Z + 192*(Aporank + 1), EndShift ,hipMemcpyDeviceToHost);
  					CudaErrorCheck();	                  	                  
   	     	               	hipMemcpy(velListX + 192*Aporank, d_velListX + 192*(Aporank + 1), EndShift , hipMemcpyDeviceToHost);
               			CudaErrorCheck();
               			hipMemcpy(velListY + 192*Aporank, d_velListY + 192*(Aporank + 1), EndShift , hipMemcpyDeviceToHost);
               			CudaErrorCheck();
               			hipMemcpy(velListZ + 192*Aporank, d_velListZ + 192*(Aporank + 1), EndShift , hipMemcpyDeviceToHost);
        	               	CudaErrorCheck();	                  
        	               	hipMemcpy(pressList + Aporank, d_pressList + Aporank+1, (MaxNoofC180s - Aporank - 1)*sizeof(float), hipMemcpyDeviceToHost);
        	               	CudaErrorCheck();
        	               	hipMemcpy(Growth_rate + Aporank, d_Growth_rate + Aporank+1, (MaxNoofC180s - Aporank - 1)*sizeof(float), hipMemcpyDeviceToHost);
        	               	CudaErrorCheck();
        	               	hipMemcpy(youngsModArray + Aporank, d_Youngs_mod + Aporank+1, (MaxNoofC180s - Aporank - 1)*sizeof(float), hipMemcpyDeviceToHost);
  					CudaErrorCheck();
        	               	hipMemcpy(ScaleFactor + Aporank, d_ScaleFactor + Aporank+1, (MaxNoofC180s - Aporank - 1)*sizeof(float), hipMemcpyDeviceToHost);
  					CudaErrorCheck();
        	               	hipMemcpy(DivisionVolume + Aporank, d_DivisionVolume + Aporank+1, (MaxNoofC180s - Aporank - 1)*sizeof(float), hipMemcpyDeviceToHost);
  					CudaErrorCheck();  					  					
        	               	hipMemcpy(gamma_env + Aporank, d_gamma_env + Aporank+1, (MaxNoofC180s - Aporank - 1)*sizeof(float), hipMemcpyDeviceToHost);
  					CudaErrorCheck();  					
        	               	hipMemcpy(viscotic_damp + Aporank, d_viscotic_damp + Aporank+1, (MaxNoofC180s - Aporank - 1)*sizeof(float), hipMemcpyDeviceToHost);
  					CudaErrorCheck();			
        	               	hipMemcpy(CellINdex + Aporank, d_CellINdex + Aporank+1, (MaxNoofC180s - Aporank - 1)*sizeof(int), hipMemcpyDeviceToHost);
        	               	CudaErrorCheck();

      				
      					-- No_of_C180s;
      				
      							
      					hipMemcpy(d_X , X, No_of_C180s*192*sizeof(float), hipMemcpyHostToDevice);
  					CudaErrorCheck();
  					hipMemcpy(d_Y , Y, No_of_C180s*192*sizeof(float), hipMemcpyHostToDevice);
  					CudaErrorCheck();
  					hipMemcpy(d_Z , Z, No_of_C180s*192*sizeof(float), hipMemcpyHostToDevice);
  					CudaErrorCheck();
  					hipMemcpy(d_velListX , velListX , No_of_C180s*192*sizeof(float), hipMemcpyHostToDevice);
  					CudaErrorCheck();
  					hipMemcpy(d_velListY , velListY , No_of_C180s*192*sizeof(float), hipMemcpyHostToDevice);
  					CudaErrorCheck();
  					hipMemcpy(d_velListZ , velListZ , No_of_C180s*192*sizeof(float), hipMemcpyHostToDevice);
  					CudaErrorCheck();
        	               	hipMemcpy(d_pressList, pressList, No_of_C180s*sizeof(float), hipMemcpyHostToDevice);
        	               	CudaErrorCheck();
  					hipMemcpy(d_Growth_rate, Growth_rate, No_of_C180s*sizeof(float), hipMemcpyHostToDevice);
        	               	CudaErrorCheck();
        	               	hipMemcpy(d_Youngs_mod, youngsModArray ,No_of_C180s*sizeof(float), hipMemcpyHostToDevice );
  					CudaErrorCheck();
        	               	hipMemcpy(d_ScaleFactor, ScaleFactor ,No_of_C180s*sizeof(float), hipMemcpyHostToDevice );
  					CudaErrorCheck();  					
        	               	hipMemcpy(d_DivisionVolume, DivisionVolume ,No_of_C180s*sizeof(float), hipMemcpyHostToDevice );
  					CudaErrorCheck();  					
        	               	hipMemcpy(d_gamma_env, gamma_env ,No_of_C180s*sizeof(float), hipMemcpyHostToDevice );
  					CudaErrorCheck();  					
        	               	hipMemcpy(d_viscotic_damp, viscotic_damp ,No_of_C180s*sizeof(float), hipMemcpyHostToDevice );
  					CudaErrorCheck();
        	               	hipMemcpy(d_CellINdex, CellINdex, No_of_C180s*sizeof(int), hipMemcpyHostToDevice);
        	               	CudaErrorCheck();  				

      					
      					NumApoCell ++;

      				}
      			}       
      		}	

	}
// ----------------------------------------- End Cell Death --------------



      CenterOfMass<<<No_of_C180s,256>>>(No_of_C180s, d_X, d_Y, d_Z, d_CMx, d_CMy, d_CMz);
      //DL = divVol; 
      CudaErrorCheck(); 

      hipMemset(d_NoofNNlist, 0, 1024*1024);

      if (useRigidSimulationBox){	
      		makeNNlist<<<No_of_C180s/512+1,512>>>( No_of_C180s, d_CMx, d_CMy, d_CMz,
        	Xdiv, Ydiv, Zdiv, BoxMin, d_NoofNNlist, d_NNlist, DL);
        
        	CudaErrorCheck(); 
       }
	if(usePBCs){
       	makeNNlistPBC<<<No_of_C180s/512+1,512>>>( No_of_C180s, d_CMx, d_CMy, d_CMz,
        	attraction_range, Xdiv, Ydiv, Zdiv, boxMax, d_NoofNNlist, d_NNlist, DLp, useRigidBoxZ,useRigidBoxY);
        
        	CudaErrorCheck(); 
       }
       if(useLEbc){
       	makeNNlistLEbc<<<No_of_C180s/512+1,512>>>( No_of_C180s, d_CMx, d_CMy, d_CMz,
        	attraction_range, Xdiv, Ydiv, Zdiv, boxMax, d_NoofNNlist, d_NNlist, DLp, Pshift, useRigidBoxZ);
        	
        	CudaErrorCheck();
       
       }


// ---------------------------------------------------------------------------------------------------




      // save previous step forces in g
      
      if (doPopModel == 1){
            rGrowth = rMax * (1 - (No_of_C180s*1.0/maxPop));
            // dr = -rGrowth(a + b*rGrowth)
            // rGrowth += dr * delta_t ;
            // dN/dT = N*R
            // dR/dT = -R(a+bR)
            // 
            if (rGrowth < 0) rGrowth =0; 
      }
      else {
      		rGrowth = rMax;
      }

      if (!colloidal_dynamics){      
      	PressureUpdate <<<No_of_C180s/1024 + 1, 1024>>> (d_pressList, minPressure, maxPressure, d_Growth_rate, No_of_C180s,
        	                                               useDifferentCell, stiffness1, d_Youngs_mod, step,
        	                                               phase_count, impurityNum);
      		CudaErrorCheck(); 
      }
      
      if ( (step)%1000 == 0)
      {
          printf("   time %-8d %d cells, CellInApoptosis %d, NumCellDeath %d\n",step,No_of_C180s, CellInApo1 + CellInApo2, NumApoCell);
      }


#ifdef FORCE_DEBUG
      printf("time %d  pressure = %f\n", step, Pressure);
#endif

    if (useRigidSimulationBox){	
  	CalculateConForce<<<No_of_C180s,threadsperblock>>>( No_of_C180s, d_C180_nn, d_C180_sign,
                                                     	d_X,  d_Y,  d_Z,
                                                     	d_CMx, d_CMy, d_CMz,
                                                     	d_R0, d_ScaleFactor, d_pressList, d_Youngs_mod , stiffness1, 
                                                     	internal_damping, d_time,
                                                     	attraction_strength, attraction_range,
                                                     	repulsion_strength, repulsion_range,
                                                     	d_viscotic_damp,
                                                     	Xdiv, Ydiv, Zdiv, usePBCs, boxMax, d_NoofNNlist, d_NNlist, DL, d_gamma_env,
                                                     	wall1, wall2,
                                                     	threshDist, useWalls,
                                                     	d_velListX, d_velListY, d_velListZ,
                                                     	useRigidSimulationBox, boxLength, BoxMin, Youngs_mod,
                                                     	constrainAngles, d_theta0, d_fConList, r_CM_o, d_contactForces, d_ExtForces, d_volume, divVol,
                                                     	impurityNum); 
                                                     	
                                                     	
       CudaErrorCheck();
                                                     	
      CalculateDisForce<<<No_of_C180s, threadsperblock>>>(No_of_C180s, d_C180_nn, d_C180_sign, 
                                                        d_X, d_Y, d_Z,
                                                        d_CMx, d_CMy, d_CMz,r_CM_o,
                                                        internal_damping,
                                                        attraction_range,
                                                        d_viscotic_damp,
                                                        Xdiv, Ydiv, Zdiv, usePBCs, boxMax,BoxMin,
                                                        d_NoofNNlist, d_NNlist, DL, d_gamma_env,
                                                        d_velListX, d_velListY, d_velListZ,
                                                        d_fDisList, impurityNum);
                                                        
       CudaErrorCheck();                                                  
  }
  
  if(usePBCs){
  
    	CalculateConForcePBC<<<No_of_C180s,threadsperblock>>>( No_of_C180s, d_C180_nn, d_C180_sign,
                                                     	d_X,  d_Y,  d_Z,
                                                     	d_CMx, d_CMy, d_CMz,
                                                     	d_R0,d_ScaleFactor, d_pressList, d_Youngs_mod , stiffness1, 
                                                     	internal_damping, d_time,
                                                     	attraction_strength, attraction_range,
                                                     	repulsion_strength, repulsion_range,
                                                     	d_viscotic_damp,
                                                     	Xdiv, Ydiv, Zdiv, usePBCs, boxMax, d_NoofNNlist, d_NNlist, DLp, d_gamma_env,
                                                     	wall1, wall2,
                                                     	threshDist, useWalls,
                                                     	d_velListX, d_velListY, d_velListZ,
                                                     	useRigidSimulationBox, boxLength, BoxMin, Youngs_mod,
                                                     	constrainAngles, d_theta0, d_fConList, r_CM_o, d_contactForces, d_volume, divVol,
                                                     	useRigidBoxZ,useRigidBoxY);
                                                     	
       CudaErrorCheck();                                             	
  	
  	                                                     	
      CalculateDisForcePBC<<<No_of_C180s, threadsperblock>>>(No_of_C180s, d_C180_nn, d_C180_sign, 
                                                        d_X, d_Y, d_Z,
                                                        d_CMx, d_CMy, d_CMz,r_CM_o,
                                                        internal_damping,
                                                        attraction_range,
                                                        d_viscotic_damp,
                                                        Xdiv, Ydiv, Zdiv, usePBCs, boxMax,
                                                        d_NoofNNlist, d_NNlist, DLp, d_gamma_env,
                                                        d_velListX, d_velListY, d_velListZ,
                                                        d_fDisList, useRigidBoxZ,useRigidBoxY);
    CudaErrorCheck();	
  
  }
  if(useLEbc){
  
  CalculateConForceLEbc<<<No_of_C180s,threadsperblock>>>( No_of_C180s, d_C180_nn, d_C180_sign,
                                                     	d_X,  d_Y,  d_Z,
                                                     	d_CMx, d_CMy, d_CMz,
                                                     	d_R0,d_ScaleFactor, d_pressList, d_Youngs_mod , stiffness1, 
                                                     	internal_damping, d_time,
                                                     	attraction_strength, attraction_range,
                                                     	repulsion_strength, repulsion_range,
                                                     	d_viscotic_damp,
                                                     	Xdiv, Ydiv, Zdiv, usePBCs, boxMax, d_NoofNNlist, d_NNlist, DLp, d_gamma_env,
                                                     	wall1, wall2,
                                                     	threshDist, useWalls,
                                                     	d_velListX, d_velListY, d_velListZ,
                                                     	useRigidSimulationBox, boxLength, BoxMin, Youngs_mod,
                                                     	constrainAngles, d_theta0, d_fConList, r_CM_o, d_contactForces, d_volume, divVol,
                                                     	Pshift,useRigidBoxZ);
                                                     	
       CudaErrorCheck();                                             	
  	
  	                                                     	
      CalculateDisForceLEbc<<<No_of_C180s, threadsperblock>>>(No_of_C180s, d_C180_nn, d_C180_sign, 
                                                        d_X, d_Y, d_Z,
                                                        d_CMx, d_CMy, d_CMz,r_CM_o,
                                                        internal_damping,
                                                        attraction_range,
                                                        d_viscotic_damp,
                                                        Xdiv, Ydiv, Zdiv, usePBCs, boxMax,
                                                        d_NoofNNlist, d_NNlist, DLp, d_gamma_env,
                                                        d_velListX, d_velListY, d_velListZ,
                                                        d_fDisList, Pshift, Vshift, useRigidBoxZ);
    CudaErrorCheck();	
  
  
  
  }
      

      // Calculate random Force here...
      if (add_rands){
          CalculateRanForce<<<No_of_C180s, threadsperblock>>>(No_of_C180s, d_rngStates, rand_scale_factor,
                                                              d_fRanList, impurityNum);
          CudaErrorCheck();
      }
      
      VelocityUpdateA<<<No_of_C180s, threadsperblock>>>(d_velListX, d_velListY, d_velListZ,
                                                        d_fConList, d_fRanList, delta_t, numNodes, mass, impurityNum);
      CudaErrorCheck();


      // Dissipative velocity update part...
      for (int s = 0; s < 1; ++s){
      
          VelocityUpdateB<<<No_of_C180s, threadsperblock>>>(d_velListX, d_velListY, d_velListZ,
                                                           d_fDisList, delta_t, numNodes, mass, impurityNum);
          CudaErrorCheck();
          
          if (useRigidSimulationBox){	
                                                     	
     		CalculateDisForce<<<No_of_C180s, threadsperblock>>>(No_of_C180s, d_C180_nn, d_C180_sign, 
                                                        d_X, d_Y, d_Z,
                                                        d_CMx, d_CMy, d_CMz,r_CM_o,
                                                        internal_damping,
                                                        attraction_range,
                                                        d_viscotic_damp,
                                                        Xdiv, Ydiv, Zdiv, usePBCs, boxMax,BoxMin,
                                                        d_NoofNNlist, d_NNlist, DL, d_gamma_env,
                                                        d_velListX, d_velListY, d_velListZ,
                                                        d_fDisList, impurityNum);
                                                        
       	CudaErrorCheck();                                                  
  	}
  	if(usePBCs){              	
      		CalculateDisForcePBC<<<No_of_C180s, threadsperblock>>>(No_of_C180s, d_C180_nn, d_C180_sign, 
                	                                        d_X, d_Y, d_Z,
                	                                        d_CMx, d_CMy, d_CMz,r_CM_o,
                	                                        internal_damping,
                	                                        attraction_range,
                	                                        d_viscotic_damp,
                	                                        Xdiv, Ydiv, Zdiv, usePBCs, boxMax,
                	                                        d_NoofNNlist, d_NNlist, DLp, d_gamma_env,
                	                                        d_velListX, d_velListY, d_velListZ,
                	                                        d_fDisList,useRigidBoxZ,useRigidBoxY);
    		CudaErrorCheck();	
  
  	}
  	if(useLEbc){
  	
  	  	                                                     	
      		CalculateDisForceLEbc<<<No_of_C180s, threadsperblock>>>(No_of_C180s, d_C180_nn, d_C180_sign, 
               	                                         d_X, d_Y, d_Z,
               	                                         d_CMx, d_CMy, d_CMz,r_CM_o,
               	                                         internal_damping,
               	                                         attraction_range,
               	                                         d_viscotic_damp,
               	                                         Xdiv, Ydiv, Zdiv, usePBCs, boxMax,
               	                                         d_NoofNNlist, d_NNlist, DLp, d_gamma_env,
               	                                         d_velListX, d_velListY, d_velListZ,
               	                                         d_fDisList, Pshift, Vshift, useRigidBoxZ);
    CudaErrorCheck();
  	
  	}
  	
     }


      
      CenterOfMass<<<No_of_C180s,256>>>(No_of_C180s,d_X, d_Y, d_Z,
                                        d_CMx, d_CMy, d_CMz);

        CudaErrorCheck();



      if (step <= Time_steps && rGrowth > 0 && !colloidal_dynamics){
        // ------------------------------ Begin Cell Division ------------------------------------------------


        volumes<<<No_of_C180s,192>>>(No_of_C180s, d_C180_56,
                                     d_X, d_Y, d_Z,
                                     d_CMx , d_CMy, d_CMz,
                                     d_volume, d_cell_div, d_DivisionVolume,
                                     checkSphericity, d_area, phase_count, step,
                                     stiffness1, useDifferentCell, d_Youngs_mod,
                                     recalc_r0,ApoVol,d_cell_Apo, d_ScaleFactor);
        CudaErrorCheck();

#if defined(FORCE_DEBUG) || defined(PRINT_VOLUMES)
      if (checkSphericity){
          //hipMemcpy(volume, d_volume, No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);
          h_volume = d_volumeV; 
          hipMemcpy(area, d_area, No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);
          hipMemcpy(pressList, d_pressList, No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);
          printf("time: %d\n", step); 
          for (int i = 0; i < No_of_C180s; i++){
              printf ("Cell: %d, volume= %f, area=%f, psi=%f, p = %f" , i, h_volume[i], area[i],
                      4.835975862049408*pow(h_volume[i], 2.0/3.0)/area[i], pressList[i]);
          
              if (h_volume[i] > divVol)
                  printf(", I'm too big :(");
          
              printf("\n"); 
          }
      } else{
          h_volume = d_volumeV; 
          for (int i = 0; i < No_of_C180s; i++){
              printf ("Cell: %d, volume= %f", i, h_volume[i]); 
          
              if (h_volume[i] > divVol)
                  printf(", I'm too big :(");
          
              printf("\n"); 
          }
      }
#endif

        count_and_get_div();

#ifdef PRINT_VOLUMES
        if (num_cell_div > 0){
            printf("Dividing cells: ");
            for (int i = 0; i<num_cell_div; i++){
                printf("%d ", cell_div_inds[i]);
            }
          
            printf("\n");
        }
#endif
        for (int divCell = 0; divCell < num_cell_div; divCell++) {
          globalrank = cell_div_inds[divCell];
          float norm[3];

          norm[0] = 0; 
          norm[1] = 1; 
          norm[2] = 0;
          
          if (useDivPlaneBasis)
              GetRandomVectorBasis(norm, divPlaneBasis);
          else
              GetRandomVector(norm);

#ifdef TURNOFF_RAN
          norm[0] = 0; 
          norm[1] = 1; 
          norm[2] = 0;
#endif
          
          hipMemcpy( d_ran2, norm, 3*sizeof(float), hipMemcpyHostToDevice);
          CudaErrorCheck();
          
          NDIV[globalrank] += 1;
          

	  if (asymDivision){ 
	      ranmar(asym, 1);	
	  } 

          cell_division<<<1,256>>>(globalrank,
                                   d_XP, d_YP, d_ZP, 
                                   d_X, d_Y, d_Z,
                                   d_XM, d_YM, d_ZM, 
                                   d_CMx, d_CMy, d_CMz,
                                   d_velListX, d_velListY, d_velListZ, d_Growth_rate, rMax,
                                   d_CellINdex, NewCellInd, No_of_C180s, d_ran2, repulsion_range,asym[0]);
                                   
          CudaErrorCheck()
          resetIndices[divCell] = globalrank;
          resetIndices[divCell + num_cell_div] = No_of_C180s;
	 
	 


		
	if (useDifferentCell && daughtSame){
	
		ScaleFactor[No_of_C180s] = ScaleFactor[globalrank];
        	youngsModArray[No_of_C180s] = youngsModArray[globalrank];
        	Growth_rate[No_of_C180s] = Growth_rate[globalrank];
        	DivisionVolume[No_of_C180s] = DivisionVolume[globalrank];      
       	gamma_env[No_of_C180s] = gamma_env[globalrank];
       	viscotic_damp[No_of_C180s] = viscotic_damp[globalrank];
       	
       	if(CellINdex[globalrank] < 0 ){
       		CellINdex[No_of_C180s] = - NewCellInd;
       	} else {
       		CellINdex[No_of_C180s] = NewCellInd;
       	}
        
        } else {
        
              	youngsModArray[No_of_C180s] = stiffness1; 
          	ScaleFactor[No_of_C180s] = 1;
          	Growth_rate[No_of_C180s] = rMax;
          	DivisionVolume[No_of_C180s] = divVol;
          	gamma_env[No_of_C180s] = gamma_visc;
          	viscotic_damp[No_of_C180s] = viscotic_damping;
          	CellINdex[No_of_C180s] = NewCellInd;
          	  	
        }

          
          if (CellINdex[No_of_C180s] > 0) ++No_of_Cell1;
          if (CellINdex[No_of_C180s] < 0) ++No_of_Cell2;
          
          
          ++No_of_C180s;
          ++NewCellInd;
          
          
          if (No_of_C180s > MaxNoofC180s){
              printf("ERROR: Population is %d, only allocated enough memory for %d\n",
                     No_of_C180s, MaxNoofC180s);
              printf("ERROR: Fatal error, crashing...\n");
              return -69;
          }
        
         
        }
        
        if (num_cell_div>0){
            
            hipMemcpy(d_resetIndices, resetIndices, 2*num_cell_div*sizeof(int),
                       hipMemcpyHostToDevice);

            CudaErrorCheck(); 

            PressureReset <<<(2*num_cell_div)/512 + 1, 512>>> (d_resetIndices, d_pressList, minPressure, 2*num_cell_div); 
            CudaErrorCheck();

            hipMemcpy(d_Youngs_mod, youngsModArray, sizeof(float)*No_of_C180s, hipMemcpyHostToDevice);
            CudaErrorCheck();
            
            hipMemcpy(d_ScaleFactor, ScaleFactor, sizeof(float)*No_of_C180s, hipMemcpyHostToDevice);
            CudaErrorCheck();
            
            hipMemcpy(d_Growth_rate, Growth_rate, No_of_C180s*sizeof(float), hipMemcpyHostToDevice);
            CudaErrorCheck();
            
            hipMemcpy(d_DivisionVolume, DivisionVolume, sizeof(float)*No_of_C180s, hipMemcpyHostToDevice);
            CudaErrorCheck();
            
            hipMemcpy(d_gamma_env, gamma_env, sizeof(float)*No_of_C180s, hipMemcpyHostToDevice);
            CudaErrorCheck();
            
            hipMemcpy(d_viscotic_damp, viscotic_damp, sizeof(float)*No_of_C180s, hipMemcpyHostToDevice);
            CudaErrorCheck();
            
            hipMemcpy(d_CellINdex, CellINdex, No_of_C180s*sizeof(int), hipMemcpyHostToDevice);
            CudaErrorCheck();            
            

        }

        if (countOnlyInternal == 1){
          num_cell_div -= num_cells_far();
        }

        num_new_cells_per_step[step-1] = num_cell_div;
        if (step%newCellCountInt == 0){
          newcells = 0;
          for (int i = 0; i < newCellCountInt; i++) {
            newcells += num_new_cells_per_step[countOffset + i];
          }
          dividingCells[(step-1)/newCellCountInt] = newcells;
          totalCells[(step-1)/newCellCountInt] = No_of_C180s - newcells;
          // Need to make sure this is how MIs are even calculated
          countOffset += newCellCountInt;
        }
        // --------------------------------------- End Cell Division -----------
      }


      CenterOfMass<<<No_of_C180s,256>>>(No_of_C180s, d_X, d_Y, d_Z, d_CMx, d_CMy, d_CMz);
      //DL = divVol; 
      CudaErrorCheck(); 


     // bounding_boxes<<<No_of_C180s,32>>>(No_of_C180s,
     //                                    d_X,d_Y,d_Z,
     //                                    d_bounding_xyz, d_CMx, d_CMy, d_CMz);
     // CudaErrorCheck();
      
      

          
      //reductionblocks = (No_of_C180s-1)/1024+1;
     // minmaxpre<<<reductionblocks,1024>>>( No_of_C180s, d_bounding_xyz,
     //                                      d_Minx, d_Maxx, d_Miny, d_Maxy, d_Minz, d_Maxz);
    //  CudaErrorCheck(); 

    //  minmaxpost<<<1,1024>>>( reductionblocks, d_Minx, d_Maxx, d_Miny, d_Maxy, d_Minz, d_Maxz);
      
    //  CudaErrorCheck(); 




   if(usePBCs && (step)%1000 == 0){
        
            CoorUpdatePBC <<<No_of_C180s, threadsperblock>>> (d_X, d_Y, d_Z,
                                                              d_XM, d_YM, d_ZM,
                                                              d_CMx, d_CMy, d_CMz,
                                                              boxMax, divVol, No_of_C180s,
                                                              useRigidBoxZ, useRigidBoxY);
  

		
            CudaErrorCheck();
        }
        
        if(useLEbc){
       
            UpdateLEbc <<<No_of_C180s, threadsperblock>>> (d_X, d_Y, d_Z, d_XM, d_YM, d_ZM,
                        d_velListX, d_velListY, d_velListZ, d_CMx, d_CMy, d_CMz,
                        boxMax, divVol, No_of_C180s, Pshift, Vshift);
	
	
	}


    //  hipMemcpy(Minx, d_Minx, 6*sizeof(float), hipMemcpyDeviceToHost);
    //  Xdiv = (int)((Minx[1]-Minx[0])/DL+1);
   //   Ydiv = (int)((Minx[3]-Minx[2])/DL+1);
   //   Zdiv = (int)((Minx[5]-Minx[4])/DL+1);


    //  makeNNlist<<<No_of_C180s/512+1,512>>>( No_of_C180s, d_bounding_xyz, Minx[0], Minx[2], Minx[4],
    //                                         attraction_range, Xdiv, Ydiv, Zdiv, d_NoofNNlist, d_NNlist, DL);
    //  CudaErrorCheck();


      if (!growthDone && step > Time_steps+1){
          printf("Cell growth halted.\nProceeding with MD simulation without growth...\n");
          growthDone = true;
          
          if (useDifferentCell && !duringGrowth && !colloidal_dynamics){
              printf("Now making some cells softer...\n");
              int softCellCounter = 0;
              if (fractionOfCells > 0){
                  numberOfCells = roundf(fractionOfCells*No_of_C180s); 
              }

              printf("Will make %d cells softer\n", numberOfCells); 
              
              if (chooseRandomCellIndices){
                  float rnd[1];
                  //int* chosenIndices = (int*)malloc(numberOfCells, sizeof(int));
                  int chosenIndices[numberOfCells]; 
                  
                  for (int i = 0; i < numberOfCells; i++){
                      chosenIndices[i] = -1; 
                  }
                  
                  bool indexChosen = false;
                  int cellInd = -1;

                  printf("Make cells with indices "); 
                  
                  while (softCellCounter < numberOfCells){
                      ranmar(rnd, 1);
                      cellInd = roundf(rnd[0] * No_of_C180s);

                      for (int i = 0; i < softCellCounter; i++){
                          if (chosenIndices[i] == cellInd){
                              indexChosen = true;
                              break;
                          }
                      }

                      if (!indexChosen){
                          chosenIndices[softCellCounter] = cellInd;
                          softCellCounter++;
                          printf("%d ", cellInd); 
                      } else
                          indexChosen = false;
                      
                  }

                  //free(chosenIndices);

                  for (int i = 0; i < numberOfCells; i++){
                      youngsModArray[chosenIndices[i]] = Stiffness2; 
                  }
              }
              else {
                  // search for the oldest cells near the center of the system, and make them soft
                  hipMemcpy(CMx, d_CMx, No_of_C180s*sizeof(float),hipMemcpyDeviceToHost);
                  hipMemcpy(CMy, d_CMy, No_of_C180s*sizeof(float),hipMemcpyDeviceToHost);
                  hipMemcpy(CMz, d_CMz, No_of_C180s*sizeof(float),hipMemcpyDeviceToHost);

                  float Rmax2 = getRmax2();
                  float R2, dx, dy, dz;
                  int cellInd = 0; 
                  calc_sys_CM();

                  float f = 1 - closenessToCenter;
              
                  printf("Made cells with indices "); 

                  while (softCellCounter < numberOfCells && cellInd < No_of_C180s){
                      dx = CMx[cellInd] - sysCMx; 
                      dy = CMy[cellInd] - sysCMy; 
                      dz = CMz[cellInd] - sysCMz;

                      R2 = dx*dx + dy*dy + dz*dz;

                      if (R2 <= f*f*Rmax2){
                          printf("%d, ", cellInd); 
                          softCellCounter++; 
                          youngsModArray[cellInd] = Stiffness2; 

                      }
                      cellInd++; 
                  }
              }
              
              hipMemcpy(d_Youngs_mod, youngsModArray, No_of_C180s*sizeof(float), hipMemcpyHostToDevice);
              printf("\b\b softer\n"); 
          }

      }

      if (correct_com == true){
          hipMemcpy(CMx, d_CMx, No_of_C180s*sizeof(float),hipMemcpyDeviceToHost);
          hipMemcpy(CMy, d_CMy, No_of_C180s*sizeof(float),hipMemcpyDeviceToHost);
          hipMemcpy(CMz, d_CMz, No_of_C180s*sizeof(float),hipMemcpyDeviceToHost);
          calc_sys_CM();
          //printf("sysCMx = %f, sysCMy = %f, sysCmz = %f\n", sysCMx, sysCMy, sysCMz);
          CorrectCoMMotion<<<(No_of_C180s*192)/1024 + 1, 1024>>>(d_X, d_Y, d_Z,
                                                                 sysCMx, sysCMy, sysCMz,
                                                                 No_of_C180s*192);
          CudaErrorCheck(); 
      }

    if ( correct_Vcom == true){
     
     
        VelocityCenterOfMass<<<No_of_C180s,256>>>(No_of_C180s,
                                          d_velListX, d_velListY, d_velListZ,
                                          d_VCMx, d_VCMy, d_VCMz);
        hipMemcpy(VCMx, d_VCMx, No_of_C180s*sizeof(float),hipMemcpyDeviceToHost);
        hipMemcpy(VCMy, d_VCMy, No_of_C180s*sizeof(float),hipMemcpyDeviceToHost);
        hipMemcpy(VCMz, d_VCMz, No_of_C180s*sizeof(float),hipMemcpyDeviceToHost);
        calc_sys_VCM();
        //calc_Multiplier();
        CorrectCoMVelocity<<<(No_of_C180s*192)/1024 + 1, 1024>>>(d_velListX, d_velListY, d_velListZ,
                                                             sysVCMx, sysVCMy, sysVCMz,
                                                             No_of_C180s*192);
          
        CudaErrorCheck(); 
        
        
      //printf("sysVCMx = %f, sysVCMy = %f, sysVCmz = %f\n", sysVCMx, sysVCMy, sysVCMz);
    }


      if ( step%trajWriteInt == 0 )
      {
          //printf("   Writing trajectory to traj.xyz...\n");
          frameCount++; 
          hipMemcpy(X, d_X, 192*No_of_C180s*sizeof(float),hipMemcpyDeviceToHost);
          hipMemcpy(Y, d_Y, 192*No_of_C180s*sizeof(float),hipMemcpyDeviceToHost);
          hipMemcpy(Z, d_Z, 192*No_of_C180s*sizeof(float),hipMemcpyDeviceToHost);
          hipMemcpy(CellINdex, d_CellINdex, No_of_C180s*sizeof(int), hipMemcpyDeviceToHost);
          
          if (binaryOutput)
              WriteBinaryTraj(step + Laststep, trajfile, frameCount + Lastframe);
          else
              write_traj(step + Laststep, trajfile);

          if (write_cont_force == true){

              hipMemcpy(h_contactForces.x, d_contactForces.x, 192*No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);
              hipMemcpy(h_contactForces.y, d_contactForces.y, 192*No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);
              hipMemcpy(h_contactForces.z, d_contactForces.z, 192*No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);
              hipMemcpy(h_ExtForces.x, d_ExtForces.x, 192*No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);
              hipMemcpy(h_ExtForces.y, d_ExtForces.y, 192*No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);
              hipMemcpy(h_ExtForces.z, d_ExtForces.z, 192*No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);
              
              hipMemcpy(velListX, d_velListX, 192*No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);
              hipMemcpy(velListY, d_velListY, 192*No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);
              hipMemcpy(velListZ, d_velListZ, 192*No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);
              
              hipMemcpy(X, d_X, 192*No_of_C180s, hipMemcpyDeviceToHost);
              hipMemcpy(Y, d_Y, 192*No_of_C180s, hipMemcpyDeviceToHost);
              hipMemcpy(Z, d_Z, 192*No_of_C180s, hipMemcpyDeviceToHost);
              
              hipMemcpy(pressList, d_pressList, No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);
              hipMemcpy(volume, d_volume, No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);
              hipMemcpy(area, d_area, No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);	
                    
              writeForces(forceFile, step + Laststep, No_of_C180s);
          }
          if(write_vel_file){
                         
               hipMemcpy(velListX, d_velListX, 192*No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);
               hipMemcpy(velListY, d_velListY, 192*No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);
               hipMemcpy(velListZ, d_velListZ, 192*No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);
          	write_vel(step + Laststep, velFile,frameCount + Lastframe);
          }  
      }

      myError = hipGetLastError();
      if ( hipSuccess != myError )
      {
          printf( "Error %d: %s!\n",myError,hipGetErrorString(myError) );return(-1);
      }

      if (step > phase_count && phase){
          printf("In phase 2\n");
          phase = false;
          if (useDifferentCell && recalc_r0){
              CalculateR0<<<No_of_C180s/1024 + 1, 1024>>>(d_R0,
                                                          d_X, d_Y, d_Z,
                                                          d_C180_nn,
                                                          d_Youngs_mod,
                                                          Stiffness2,
                                                          No_of_C180s);
#ifdef RO_DEBUG
              thrust::fill(h_R0V.begin(), h_R0V.end(), 0.f);
              h_R0V = d_R0V;
          
              hipMemcpy(youngsModArray, d_Youngs_mod, sizeof(float)*MaxNoofC180s, hipMemcpyDeviceToHost);
      
              for (int i =0; i < No_of_C180s; ++i){
                  std::cout << "Cell " << i << " R0 = "
                            << h_R0V[i] << " E = " << youngsModArray[i] << std::endl;
              }
#endif
          }
      }
  }


  t = (Time_steps+equiStepCount+Laststep+1) / trajWriteInt; 
  
  if(write_vel_file){
       
       fclose(velFile);   
        
       velFile = fopen("velocity.xyz", "r+");
       fseek(velFile, 0, SEEK_SET);
       fwrite(&No_of_C180s, sizeof(int), 1, velFile);  
       fseek(velFile, 8, SEEK_SET);
       fwrite(&t, sizeof(int), 1, velFile);
       
  }
  
    if (binaryOutput){
  
      fclose(trajfile);
      
      trajfile = fopen (trajFileName, "r+");
      fseek(trajfile, 0, SEEK_SET);
      fwrite(&No_of_C180s, sizeof(int), 1, trajfile);    
      fseek(trajfile, 8, SEEK_SET);
      fwrite(&t, sizeof(int), 1, trajfile);
  }
  
  printf("Xdiv = %d, Ydiv = %d, Zdiv = %d\n", Xdiv, Ydiv, Zdiv );

  FILE* MitIndFile;
  std::fstream MitIndFile2;
  std::string datFileName = inpFile; 
  
  if (overWriteMitInd == 0){
      
      MitIndFile = fopen(mitIndFileName, "a");
      //MitIndFile2.open(datFileName, "a"); 
  }
  else{
      MitIndFile = fopen(mitIndFileName, "w");
      //MitIndFile2.open(datFileName, "w"); 
  }
  if (MitIndFile == NULL)
  {
      printf("Failed to open mit-index.dat\n");
      exit(1);
  }


  for (int i = 0; i < (Time_steps/newCellCountInt) + 1; i++)
  {
      if ( dividingCells[i]!=0 && totalCells[i]!=0 ){
          fprintf(MitIndFile, "%f\n", (float)dividingCells[i]/totalCells[i]);
          // totalCells is number of non-dividing cells
          
      }
      else {
          fprintf(MitIndFile, "%f\n", 0.0);

      }

  }
  
  if ( writeRestartFile(step + Laststep, frameCount + Lastframe) != 0 ){
 	printf("Unable to call Restart Kernel. \n");
	return(-1);
   }
 
 
   

  //hipFree( (void *)d_bounding_xyz );
  hipFree( (void *)d_XP );
  hipFree( (void *)d_YP );
  hipFree( (void *)d_ZP );
  hipFree( (void *)d_X  );
  hipFree( (void *)d_Y  );
  hipFree( (void *)d_Z  );
  hipFree( (void *)d_XM );
  hipFree( (void *)d_YM );
  hipFree( (void *)d_ZM );
  hipFree( (void *)d_CMx );
  hipFree( (void *)d_CMy );
  hipFree( (void *)d_CMz );
  hipFree( (void *)d_ran2 );

  hipFree( (void *)d_C180_nn);
  hipFree( (void *)d_C180_sign);
  hipFree( (void *)d_cell_div);
  hipFree( (void *)d_cell_Apo);
  free(X); free(Y); free(Z);
  //free(bounding_xyz);
  free(CMx); free(CMy); free(CMz);
  free(dividingCells); free(totalCells);
  free(NDIV);
  //free(volume);
  free(Minx); free(Miny); free(Minz);
  free(Maxx); free(Maxy); free(Maxz);
  free(NoofNNlist);
  free(NNlist);
  free(ran2);
  free(num_new_cells_per_step);
  free(cell_div_inds);
  free(cell_Apo_inds);
  free(pressList);

  free(velListX); 
  free(velListY); 
  free(velListZ); 

  fclose(trajfile);
  fclose(forceFile);
  fclose(velFile);
  fclose(MitIndFile);
#ifdef OUTPUT_ADP_ERROR
  fclose(timeFile);
  fclose(errFile);
#endif
  return(0);
}



int initialize_C180s(int Orig_No_of_C180s)
{
  int rank;
  int atom;
  float initx[181], inity[181], initz[181];
  FILE *infil;

  printf("      Initializing positions for %d fullerenes...\n", Orig_No_of_C180s);


  CPUMemory += 3L*192L*MaxNoofC180s*sizeof(float);
  CPUMemory += MaxNoofC180s*6L*sizeof(float);

  infil = fopen("C180","r");
  if ( infil == NULL ) {printf("Unable to open file C180\n");return(-1);}
  for ( atom = 0 ; atom < 180 ; ++atom)
  {
      if ( fscanf(infil,"%f %f %f",&initx[atom], &inity[atom], &initz[atom]) != 3 )
      {
          printf("   Unable to read file C180 on line %d\n",atom+1);
          fclose(infil);
          return(-1);
      }
  }
  fclose(infil);

  // first correct for the cells com

  float sumx = 0; 
  float sumy = 0; 
  float sumz = 0;
      
  for (int i =0; i < 180; ++i){
      sumx += initx[i]; 
      sumy += inity[i]; 
      sumz += initz[i]; 
  }

  sumx /= 180.0; 
  sumy /= 180.0; 
  sumz /= 180.0;

  // calculate initial cell volume

  
      
  for (int i =0; i < 180; ++i){
      initx[i] -= sumx; 
      inity[i] -= sumy; 
      initz[i] -= sumz; 
  }

  float rCheck = powf(0.75*(1.f/3.14159)*0.786, 1.f/3.f); // this code is magical
  printf("Check radius = %f\n", rCheck);
  float3 allCMs[Orig_No_of_C180s];

  float vol = 0;
  int k = 0;
      
  vol = (boxMax.x - BoxMin.x)*(boxMax.y - BoxMin.y)*(boxMax.z - BoxMin.z);
  k = floor(vol/0.786);
      
  if (k < Orig_No_of_C180s){
      fprintf(stderr, "ERROR: Simulation box is too small\n");
      fprintf(stderr, "       Big enough for %d\n", k);
      return 27;
  }

  printf("Can fit upto %d cells\n", k);

  int c = 0;
  float rands[3];
  float3 center = 0.5*boxMax;	
  float3 CM;
  float yoffset;
  yoffset = BoxMin.y + 1;
  if (LineCenter == 1) {
	yoffset = center.y; 
  }

  if (colloidal_dynamics){
  
  	while (true){
  	
              ranmar(rands, 3);
              CM = make_float3(rands[0]*((boxMax.x - BoxMin.x) - 1.f)  + BoxMin.x + 1.f,
                                      rands[1]*((boxMax.y - BoxMin.y) - 1.f)  + BoxMin.y + 1.f,
                                      0.f);
              if (flatbox == 1){
                  CM.z = (boxMax.z - BoxMin.z)/2;
              } else {
                  CM.z = rands[2]*((boxMax.z - BoxMin.z) - 1.f)  + BoxMin.z + 1.f;
              }

	      	
              bool farEnough = true;
              
              
              farEnough = !(CM.x + ScaleFactor[c]*rCheck > boxMax.x || CM.x-ScaleFactor[c]*rCheck < BoxMin.x ||
                            CM.y+ScaleFactor[c]*rCheck > boxMax.y || CM.y-ScaleFactor[c]*rCheck < BoxMin.y ||
                            CM.z+ScaleFactor[c]*rCheck > boxMax.z || CM.z-ScaleFactor[c]*rCheck < BoxMin.z);
              
              
              for (int nInd = 0; nInd < c; ++nInd){
                  if (mag(allCMs[nInd] - CM) < 1.2*rCheck*(ScaleFactor[nInd]+ScaleFactor[c])){
                      farEnough = false;
                      break;
                  }
              }
          
 
              if (farEnough){

                  allCMs[c] = CM; 
                  c++;
              }
          
              if (c == Orig_No_of_C180s){
		break;
              }
          }
  
  
  
  	   for (int cellInd = 0; cellInd < Orig_No_of_C180s; cellInd++){
  	     for(int nodeInd = 0; nodeInd < 180; ++nodeInd){
                  X[cellInd*192 + nodeInd] = ScaleFactor[cellInd]*initx[nodeInd] + allCMs[cellInd].x;
                  Y[cellInd*192 + nodeInd] = ScaleFactor[cellInd]*inity[nodeInd] + allCMs[cellInd].y;
                  Z[cellInd*192 + nodeInd] = ScaleFactor[cellInd]*initz[nodeInd] + allCMs[cellInd].z;
  	     }
  	   }
  

  } else {

  	if (rand_pos){
          
		while (true){
        	      ranmar(rands, 3);
        	      CM = make_float3(rands[0]*((boxMax.x - BoxMin.x) - 1.f)  + BoxMin.x + 1.f,
                                      rands[1]*((boxMax.y - BoxMin.y) - 1.f)  + BoxMin.y + 1.f,
                                      0.f);
        	      if (flatbox == 1){
        	          CM.z = (boxMax.z - BoxMin.z)/2;
        	      } else {
        	          CM.z = rands[2]*((boxMax.z - BoxMin.z) - 1.f)  + BoxMin.z + 1.f;
        	      }

	      	
        	      bool farEnough = true;
              
              
        	      farEnough = !(CM.x+rCheck > boxMax.x || CM.x-rCheck < BoxMin.x ||
        	                    CM.y+rCheck > boxMax.y || CM.y-rCheck < BoxMin.y ||
        	                    CM.z+rCheck > boxMax.z || CM.z-rCheck < BoxMin.z);
              
              
        	      for (int nInd = 0; nInd < c; ++nInd){
        	          if (mag(allCMs[nInd] - CM) < 2*rCheck){
        	              farEnough = false;
        	              break;
        	          }
        	      }
          
 
        	      if (farEnough){
	
        	          allCMs[c] = CM; 
        	          c++;
        	      }
          
        	      if (c == Orig_No_of_C180s){
			break;
        	      }
        	  }
	
  	} else if ( line ){
	
			c = Orig_No_of_C180s-1;
				
			for ( rank = 0; rank < Orig_No_of_C180s - impurityNum ; rank++ )
        	        {
        	                         
        	   	        CM.x = L*rank + 0.5*L + BoxMin.x;
        	    	      	CM.y = yoffset;
        	    	      	CM.z = center.z;
				allCMs[c] = CM; 
        	    	  	c--;

        	   	}
           	
        	   	if (impurity){
        	   	
        	   		while (true){
        		      
				      ranmar(rands, 3);
        	      		      CM = make_float3(rands[0]*((boxMax.x - BoxMin.x) - 1.f)  + BoxMin.x + 1.f,
        	       	                       rands[1]*((boxMax.y - BoxMin.y) - 1.f)  + BoxMin.y + 1.f,
        	       	                       0.f);
        	      			if (flatbox == 1){
        	       		   CM.z = (boxMax.z - BoxMin.z)/2;
        	      			}else {
        	       	   		CM.z = rands[2]*((boxMax.z - BoxMin.z) - 1.f)  + BoxMin.z + 1.f;
        	      			}

	      		
        			      bool farEnough = true;
        	      
        			      farEnough = !(CM.x+rCheck > boxMax.x || CM.x-rCheck < BoxMin.x ||
        			                    CM.y+rCheck > boxMax.y || CM.y-rCheck < BoxMin.y ||
        			                    CM.z+rCheck > boxMax.z || CM.z-rCheck < BoxMin.z );
              	
        			      for (int nInd = Orig_No_of_C180s-1; nInd > c; --nInd){
        			          if (mag(allCMs[nInd] - CM) < 2*rCheck){
        			              farEnough = false;
        			              break;
        			          }
        			      }
          

				      if (farEnough){
	
        			          allCMs[c] = CM; 
        			          c--;
        			      }
        	  
        			      if (c == -1){
					break;
        			      }
        	       	}
				
			}
  	} else if (plane) {
  
  			rCheck *= 1.2;
  			c = Orig_No_of_C180s-1;
  			float l = 2.5;
			int Side = int (((boxMax.x - BoxMin.x) / l) + 0.1 );
			printf(" Max number of initial cells:  %d\n", Side*Side);
			
			for ( rank = 0; rank < Orig_No_of_C180s - impurityNum ; rank++ )
        	        {
                        
        	               ey=rank%Side;
        			ex=rank/Side;         
        	          	CM.x = l*ex + 0.5*l + BoxMin.x;
        	          	CM.y = l*ey + 0.5*l + BoxMin.x;
        	    	      	CM.z = BoxMin.z + 1 ;
				allCMs[c] = CM; 
        	    	  	c--;
        	   	}
  
  
  
  	} else {			
	
			rCheck *= 1.2;
			c = Orig_No_of_C180s-1;
		
        	 	for ( rank = 0; rank < Orig_No_of_C180s - impurityNum; ++rank )
        	 	{
        	 		 ey=rank%Side_length;
        			 ex=rank/Side_length;
        	          	 CM.x = L1*ex + 0.5*L1 + center.x;
        	          	 CM.y = L1*ey + 0.5*L1 + center.y;
        	          	 CM.z = center.z;
        	          	 allCMs[c] = CM;
        	          	 c--;

        	  	}  
	
			if (impurity){	
			
				while (true){
        		      
			      		ranmar(rands, 3);
        	      	      		CM = make_float3(rands[0]*((boxMax.x - BoxMin.x) - 1.f)  + BoxMin.x + 1.f,
        	                       	       rands[1]*((boxMax.y - BoxMin.y) - 1.f)  + BoxMin.y + 1.f,
        	                       	       0.f);
        	      			if (flatbox == 1){
        	       	   		CM.z = (boxMax.z - BoxMin.z)/2;
        	      			}else {
        	          			CM.z = rands[2]*((boxMax.z - BoxMin.z) - 1.f)  + BoxMin.z + 1.f;
        	      			}

	      		
        		      		bool farEnough = true;
              
        		      		farEnough = !(CM.x+rCheck > boxMax.x || CM.x-rCheck < BoxMin.x ||
        		               	     CM.y+rCheck > boxMax.y || CM.y-rCheck < BoxMin.y ||
        		               	     CM.z+rCheck > boxMax.z || CM.z-rCheck < BoxMin.z );
              
        		      		for (int nInd = Orig_No_of_C180s-1; nInd > c; --nInd){
        		          		if (mag(allCMs[nInd] - CM) < 2*rCheck){
        		              			farEnough = false;
        		              			break;
        		          		}
        		      		}
          

			      		if (farEnough){
	
        		          		allCMs[c] = CM; 
        		          		c--;
        		      		}
        	  
        		 	     if (c == -1){
					break;
        		 	     }
        	       	}
	  	 }
   	}

			 


   	for (int cellInd = 0; cellInd < Orig_No_of_C180s; cellInd++){
       	for(int nodeInd = 0; nodeInd < 180; ++nodeInd){
               	   X[cellInd*192 + nodeInd] = initx[nodeInd] + allCMs[cellInd].x;
               	   Y[cellInd*192 + nodeInd] = inity[nodeInd] + allCMs[cellInd].y;
               	   Z[cellInd*192 + nodeInd] = initz[nodeInd] + allCMs[cellInd].z;
       	}
   	}
   }	
       
      // check all the fucking positions...
      for (int i = 0; i < Orig_No_of_C180s*192; ++i){
          if (X[i] > boxMax.x || X[i] < BoxMin.x ||
              Y[i] > boxMax.y || Y[i] < BoxMin.y ||
              Z[i] > boxMax.z || Z[i] < BoxMin.z ){

              printf("shit is in the fan\n");
              printf("%f %f %f\n", X[i], Y[i], Z[i]);
              //exit(4); 
          }
                               
      }


	for (int cellInd = 0; cellInd < Orig_No_of_C180s; cellInd++)
	{
		CellINdex[cellInd] = cellInd;
	}


  return(0);
}


int initialize_Vel(int Orig_No_of_C180s)
{

int rank;
int atom;
float rands[3];
float3 Vcom;

printf("      Initializing velocities for %d fullerenes...\n", Orig_No_of_C180s);
  
	      
	for ( rank = 0; rank < Orig_No_of_C180s; ++rank )
	{	
		ranmar(rands, 3);
		Vcom = make_float3(rands[0]*6 - 3, rands[1]*6 - 3, rands[2]*6 - 3);
		if (flatbox == 1) Vcom.z = 0;
			
               for ( atom = 0 ; atom < 180 ; ++atom)
		{    
                    velListX[rank*192+atom] = Vcom.x;
                    velListY[rank*192+atom] = Vcom.y;
                    velListZ[rank*192+atom] = Vcom.z;
          	}

        }

  return(0);
  
}

int SecondCell (int Orig_No_of_C180s){


  float sumx = 0; 
  float sumy = 0; 
  float sumz = 0;
	  
	  
	 if (closenessToCenter > 0.f && closenessToCenter < 1.f){
          printf("Only making cells within %f of max radius different\n", closenessToCenter);
          	CenterOfMass<<<No_of_C180s,256>>>(No_of_C180s,
                              	              d_XP, d_YP, d_ZP,
                                      	      d_CMx, d_CMy, d_CMz);
          
         	 hipMemcpy(CMx, d_CMx, No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);
         	 hipMemcpy(CMy, d_CMy, No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);
         	 hipMemcpy(CMz, d_CMz, No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);

         	 float3 sysCM = make_float3(0.f, 0.f, 0.f);

         	 for(int i =0; i < No_of_C180s; ++i){
         	     sysCM = sysCM + make_float3(CMx[i], CMy[i], CMz[i]);
         	 }

         	 sysCM = sysCM/No_of_C180s; 
          
         	 printf("COM = (%f, %f, %f)\n", sysCM.x, sysCM.y, sysCM.z);

         	 float rMax = 0;
         	 float mags[No_of_C180s];
          
         	 for (int i =0; i < No_of_C180s; ++i){
         	     float3 pos = make_float3(CMx[i], CMy[i], CMz[i]) - sysCM;
         	     mags[i] = mag(pos);
         	     rMax = max(rMax, mags[i]);
         	 }
        	 int c = 0; 
          	 for (int i = 0; i < No_of_C180s; ++i){
              		if (mags[i]/rMax <= closenessToCenter){
              		
              			ScaleFactor[i] = SizeFactor;
                  		youngsModArray[i] = Stiffness2;
                  		Growth_rate[i] = gRate;
                  		DivisionVolume[i] = divisionV;
                  		gamma_env[i] = gEnv;
                  		viscotic_damp[i] = gVis;
                  		CellINdex[i] = - CellINdex[i];
                  		
                  		
                  		for (int j =0; j < 180; ++j){
      					sumx += X[i*192 + j]; 
      					sumy += Y[i*192 + j]; 
      					sumz += Z[i*192 + j]; 
  				}

  				sumx /= 180.0; 
  				sumy /= 180.0; 
  				sumz /= 180.0;


  				for (int j =0; j < 180; ++j){
      					X[i*192 + j] -= sumx; 
      					Y[i*192 + j] -= sumy; 
      					Z[i*192 + j] -= sumz; 
  				}
                  		
		
				for(int j = 0; j < 180; ++j){
       	           		X[i*192 + j] = SizeFactor*X[i*192 + j] + sumx;
       	           		Y[i*192 + j] = SizeFactor*Y[i*192 + j] + sumy;
       	           		Z[i*192 + j] = SizeFactor*Z[i*192 + j] + sumz;
       			}

				sumx = 0;
				sumy = 0;
				sumz = 0;
				
                  		++c;
                  		No_of_Cell2++; 
              		}
          	}
          	
		No_of_Cell1 = No_of_C180s - No_of_Cell2;
          	
          	printf("Made %d cells softer\n", c);

	  }else{

		printf("Choosing second cell randomly\n");
        	int c = numberOfCells;
                if (fractionOfCells > 0.f && fractionOfCells < 1.f){
                	c = round(fractionOfCells*(float)(Orig_No_of_C180s - impurityNum));
                	
                }

                if (c > Orig_No_of_C180s - impurityNum){
              		printf("ERROR: Too many different cells requested\n");
              		return 12517;
          	}
	 
          	float rands[1];
	  	int coun;
	  	coun = c;	
	  	while(true){
			
	  		ranmar(rands, 1);
                	int i = round(rands[0]*(No_of_C180s - impurityNum) + impurityNum );

			if ( ScaleFactor[i] == SizeFactor ||  youngsModArray[i] == Stiffness2) continue;                 
		
			ScaleFactor[i] = SizeFactor;
			youngsModArray[i] = Stiffness2;
			Growth_rate[i] = gRate;
			DivisionVolume[i] = divisionV;
			gamma_env[i] = gEnv;
			viscotic_damp[i] = gVis;
			CellINdex[i] = - CellINdex[i];
			
                  	for (int j =0; j < 180; ++j){
      				sumx += X[i*192 + j]; 
      				sumy += Y[i*192 + j]; 
      				sumz += Z[i*192 + j]; 
  			}

  			sumx /= 180.0; 
  			sumy /= 180.0; 
  			sumz /= 180.0;


  			for (int j =0; j < 180; ++j){
      				X[i*192 + j] -= sumx; 
      				Y[i*192 + j] -= sumy; 
      				Z[i*192 + j] -= sumz; 
  			}
                  		
		
			for(int j = 0; j < 180; ++j){
       	           	X[i*192 + j] = SizeFactor*X[i*192 + j] + sumx;
       	           	Y[i*192 + j] = SizeFactor*Y[i*192 + j] + sumy;
       	           	Z[i*192 + j] = SizeFactor*Z[i*192 + j] + sumz;
       		}

			sumx = 0;
			sumy = 0;
			sumz = 0;
			
			coun--;
			No_of_Cell2 ++;
			
			if (coun == 0 ) break;

	      }
	      
	      No_of_Cell1 = No_of_C180s - No_of_Cell2;

	}	      

	return 0;

}


int DispersityFunc(int Orig_No_of_C180s){

	
	if(dispersity && colloidal_dynamics){
	
		float rands[1];
		
		for (int rank = 0; rank < Orig_No_of_C180s; ++rank )
		{
			ranmar(rands,1);
			ScaleFactor[rank] = rands[0]*0.35 + 0.65 ;
		}

	}

	return(0);

}


int generate_random(int no_of_ran1_vectors)
{
  // This function uses marsaglia random number generator
  // Defined in marsaglia.h
  int seed_ij, seed_kl ,ij,kl;

  

  ran2 = (float *)calloc(MaxNoofC180s+1,sizeof(float));
  CPUMemory += (MaxNoofC180s+1L)*sizeof(float);
  if (rand_seed < 0){
      time_t current_time;
      time(&current_time);
      seed_ij = (int)current_time;
      localtime(&current_time);
      seed_kl = (int)current_time;
  
      ij = seed_ij%31328;
      kl = seed_kl%30081;
  } else {
      ij = rand_seed;
      kl = rand_seed*100;
  }

#ifdef TURNOFF_RAN
  ij = 80;
  kl = 97;
#endif
  
  rmarin(ij,kl);

  printf("RNG seeds: %d, %d\n", ij, kl); 
  return(0);
}



int read_fullerene_nn(void)
{
  int i,end;
  int N1, N2, N3, N4, N5, N6, Sign;
  FILE *infil;

  printf("   Reading C180NN ..\n");

  infil = fopen("C180NN","r");
  if ( infil == NULL ) {printf("Unable to open file C180NN\n");return(-1);}
  
  end = 180;
  for ( i = 0; i < 180 ; ++i )
  {
      if ( fscanf(infil,"%d,%d,%d,%d", &N1, &N2, &N3, &Sign) != 4 ) {end = i; break;}
      C180_nn[0 + i] = N1-1;
      C180_nn[192+i] = N2-1;
      C180_nn[384+i] = N3-1;
      C180_sign[i] = Sign;
  }
  fclose(infil);

  if ( end < 180 ) {printf("Error: Unable to read line %d in file C180NN\n",end);return(-1);}

  printf("   Reading C180C ..\n");

  infil = fopen("C180C","r");
  if ( infil == NULL ) {printf("Unable to open file C180C\n");return(-1);}

  end = 270;
  for ( i = 0; i < 270 ; ++i )
  {
      if ( fscanf(infil,"%d,%d", &N1, &N2) != 2 ) {end = i; break;}
      CCI[0][i] = N1-1;
      CCI[1][i] = N2-1;
  }
  fclose(infil);

  if ( end < 270 ) {printf("Error: Unable to read line %d in file C180C\n",end);return(-1);}

  printf("      read nearest neighbour ids for atoms in C180\n");

  printf("   Reading C180 pentagons, hexagons ..\n");

  infil = fopen("C180_pentahexa","r");
  if ( infil == NULL ) {printf("Unable to open file C180_pentahexa\n");return(-1);}

  end = 12;
  for ( i = 0; i < 12 ; ++i )
  {
      if ( fscanf(infil,"%d %d %d %d %d", &N1, &N2, &N3, &N4, &N5) != 5 ) {end = i; break;}
      C180_56[i*7+0] = N1;
      C180_56[i*7+1] = N2;
      C180_56[i*7+2] = N3;
      C180_56[i*7+3] = N4;
      C180_56[i*7+4] = N5;
      C180_56[i*7+5] = N1;
      C180_56[i*7+6] = N1;
  }
  if ( end != 12 ) {printf("Error: Unable to read line %d in file C180_pentahexa\n",end);return(-1);}
  end = 80;
  for ( i = 0; i < 80 ; ++i )
  {
      if ( fscanf(infil,"%d %d %d %d %d %d", &N1, &N2, &N3, &N4, &N5, &N6) != 6 ) {end = i; break;}
      C180_56[84+i*7+0] = N1;
      C180_56[84+i*7+1] = N2;
      C180_56[84+i*7+2] = N3;
      C180_56[84+i*7+3] = N4;
      C180_56[84+i*7+4] = N5;
      C180_56[84+i*7+5] = N6;
      C180_56[84+i*7+6] = N1;
  }
  if ( end != 80 ) {printf("Error: Unable to read line %d in file C180_pentahexa\n",end);return(-1);}

  fclose(infil);

  printf("Calculating equilibrium bond lengths\n");

  float initX[181], initY[181], initZ[181];

  infil = fopen("C180","r");
  if ( infil == NULL ) {printf("Unable to open file C180\n");return(-1);}
  for ( int atom = 0 ; atom < 180 ; ++atom)
  {
  	if ( fscanf(infil,"%f %f %f",&initX[atom], &initY[atom], &initZ[atom]) != 3 )
  	{
              printf("   Unable to read file C180 on line %d\n",atom+1);
              fclose(infil);
              return(-1);
          }
  }
  fclose(infil);
		

  for (int i = 0; i < 180; ++i){
      int N1 = C180_nn[0 + i];
      int N2 = C180_nn[192 + i];
      int N3 = C180_nn[384 + i];

      float3 a, b; 
      a = make_float3(initX[i], initY[i], initZ[i]);

      b = make_float3(initX[N1], initY[N1], initZ[N1]);
      h_R0[0 + i] = mag(a-b);

      b = make_float3(initX[N2], initY[N2], initZ[N2]);
      h_R0[192 + i] = mag(a-b);

      b = make_float3(initX[N3], initY[N3], initZ[N3]);
      h_R0[384 + i] = mag(a-b);
  }
  return(0);
}


int read_json_params(const char* inpFile){
    // Function to parse a json input file using the jsoncpp library

    // variable to hold the root of the json input
    Json::Value inpRoot;
    Json::Reader inpReader;

    std::ifstream inpStream(inpFile);
    std::string inpString((std::istreambuf_iterator<char>(inpStream)),
                          std::istreambuf_iterator<char>());

    bool parsingSuccess = inpReader.parse(inpString, inpRoot);
    if (!parsingSuccess){
        printf("Failed to parse %s\n", inpFile);
        // There must be a way to keep from converting from string to char*
        // Maybe by making inpString a char*
        printf("%s", inpReader.getFormattedErrorMessages().c_str());
        return -1;
    }
    else
        printf("%s parsed successfully\n", inpFile);

    // begin detailed parameter extraction

    Json::Value coreParams = inpRoot.get("core", Json::nullValue);

    // load core simulation parameters
    if (coreParams == Json::nullValue){
        printf("ERROR: Cannot load core simulation parameters\nExiting");
        return -1;
    }
    else {
        MaxNoofC180s = coreParams["MaxNoofC180s"].asInt(); 
        mass = coreParams["particle_mass"].asFloat();
        repulsion_range = coreParams["repulsion_range"].asFloat();
        attraction_range = coreParams["attraction_range"].asFloat();
        repulsion_strength = coreParams["repulsion_strength"].asFloat();
        attraction_strength = coreParams["attraction_strength"].asFloat();
        Youngs_mod = coreParams["Youngs_mod"].asFloat(); 
        stiffness1 = coreParams["stiffFactor1"].asFloat()*Youngs_mod;
        viscotic_damping = coreParams["viscotic_damping"].asFloat();
        internal_damping = coreParams["internal_damping"].asFloat();
        divVol = coreParams["division_Vol"].asFloat();
        ranZOffset = coreParams["random_z_offset?"].asInt();
        zOffset = coreParams["z_offset"].asFloat();
        Time_steps = coreParams["div_time_steps"].asFloat();
        delta_t = coreParams["time_interval"].asFloat();
        Restart = coreParams["Restart"].asInt();
        trajWriteInt = coreParams["trajWriteInt"].asInt();
        equiStepCount = coreParams["non_div_time_steps"].asInt();

        std::strcpy (trajFileName, coreParams["trajFileName"].asString().c_str());
        binaryOutput = coreParams["binaryOutput"].asBool(); 

        maxPressure = coreParams["maxPressure"].asFloat();
        minPressure = coreParams["minPressure"].asFloat();
        gamma_visc = coreParams["gamma_visc"].asFloat();
        shear_rate = coreParams["shear_rate"].asFloat();
        rMax = coreParams["growth_rate"].asFloat();
        checkSphericity = coreParams["checkSphericity"].asBool();
        constrainAngles = coreParams["constrainAngles"].asBool();
        dt_max = coreParams["dt_max"].asFloat();
        dt_tol = coreParams["dt_tol"].asFloat();
        doAdaptive_dt = coreParams["doAdaptive_dt"].asBool();
        phase_count = coreParams["phase_count"].asInt();
        write_cont_force = coreParams["write_cont_force"].asBool();
        write_vel_file = coreParams["write_vel_file"].asBool();
        std::strcpy(forces_file, coreParams["forces_file"].asString().c_str());
        dispersity = coreParams["dispersity"].asBool();
        colloidal_dynamics = coreParams["colloidal_dynamics"].asBool();
        correct_com = coreParams["correct_com"].asBool();
        correct_Vcom = coreParams["correct_Vcom"].asBool();
                                 
    }

    Json::Value countParams = inpRoot.get("counting", Json::nullValue);
    if (countParams == Json::nullValue){
        // countCells = FALSE;
        printf("ERROR: Cannot load counting parameters\nExiting");
        return -1;
    }
    else {
        // countCells = countParams["countcells"].asBool();
        std::strcpy(mitIndFileName, countParams["mit-index_file_name"].asString().c_str()); 
        countOnlyInternal = countParams["count_only_internal_cells?"].asBool();
        radFrac = countParams["radius_cutoff"].asFloat();
        overWriteMitInd = countParams["overwrite_mit_ind_file?"].asBool();
        newCellCountInt = countParams["cell_count_int"].asInt();
    }
    

    Json::Value popParams = inpRoot.get("population", Json::nullValue);
    if (popParams == Json::nullValue){
        printf("ERROR: Cannot load population parameters\nExiting");
        return -1;
    }
    else{
        doPopModel = popParams["doPopModel"].asInt();
        totalFood = popParams["totalFood"].asFloat();
        cellFoodCons = popParams["regular_consumption"].asFloat();
        cellFoodConsDiv = popParams["division_consumption"].asFloat();
        cellFoodRel = popParams["death_release_food"].asFloat();
        cellLifeTime = popParams["cellLifeTime"].asInt();
        maxPop = popParams["max_pop"].asFloat(); 
    }

    
    Json::Value apoParams = inpRoot.get("apoptosis", Json::nullValue);
    if (apoParams == Json::nullValue){
        printf("ERROR: Cannot load apoptosis parameters\nExiting");
        return -1;
    }
    else{
	apoptosis = apoParams["apoptosis"].asBool();
    	popToStartApo = apoParams["popToStartApo"].asFloat();
    	Apo_rate = apoParams["Apo_ratio"].asFloat();
    	squeeze_rate = apoParams["squeeze_rate"].asFloat();
    	ApoVol = apoParams["apoptosis_Vol"].asFloat();
    	
    }	


    Json::Value wallParams = inpRoot.get("walls", Json::nullValue);

    if (wallParams == Json::nullValue){
        printf("ERROR: Cannot load wall parameters\nExiting");
        return -1;
    }
    else{
        useWalls = wallParams["useWalls"].asInt();
        std::strcpy(perpAxis, wallParams["perpAxis"].asString().c_str());
        dAxis = wallParams["dAxis"].asFloat();
        wallLen = wallParams["wallLen"].asFloat();
        wallWidth = wallParams["wallWidth"].asFloat();
        threshDist = wallParams["threshDist"].asFloat();
    }

    Json::Value divParams = inpRoot.get("divParams", Json::nullValue);
    
    if (divParams == Json::nullValue){
        printf("ERROR: Cannot load division parameters\n");
        return -1;
    } else{
        useDivPlaneBasis = divParams["useDivPlaneBasis"].asInt();
        divPlaneBasis[0] = divParams["divPlaneBasisX"].asFloat();
        divPlaneBasis[1] = divParams["divPlaneBasisY"].asFloat();
        divPlaneBasis[2] = divParams["divPlaneBasisZ"].asFloat();
	asymDivision = divParams["asymDivision"].asBool();
    }

    Json::Value NewCell = inpRoot.get("NewCell", Json::nullValue);

    if (NewCell == Json::nullValue){
        printf("ERROR: Cannot load New Cell parameters\n");
        return -1;
    } else {
        useDifferentCell = NewCell["useDifferentCell"].asBool();
        SizeFactor = NewCell["SizeFactor"].asFloat();
        Stiffness2 = NewCell["StiffFactor"].asFloat() * Youngs_mod;
        gRate = NewCell["GrowthRate"].asFloat();
        divisionV = NewCell["divisionV"].asFloat();
        gEnv = NewCell["gamma"].asFloat();
        gVis = NewCell["VisDamping"].asFloat();
        Apo_rate2 = NewCell["Apo_rate2"].asFloat();
        squeeze_rate2 = NewCell["squeeze_rate2"].asFloat();
        numberOfCells = NewCell["numberOfCells"].asInt();
        fractionOfCells = NewCell["fractionOfCells"].asFloat();
        closenessToCenter = NewCell["closenessToCenter"].asFloat();
        chooseRandomCellIndices = NewCell["chooseRandomCellIndices"].asBool(); 
        daughtSame = NewCell["daughtSame"].asBool(); 
        duringGrowth = NewCell["duringGrowth"].asBool();
        recalc_r0 = NewCell["recalc_r0"].asBool(); 
    }
    

    Json::Value boxParams = inpRoot.get("boxParams", Json::nullValue);

    if (boxParams == Json::nullValue){
        printf("ERROR: Cannot load box parameters\n");
        return -1;
    } else{
        useRigidSimulationBox = boxParams["useRigidSimulationBox"].asBool();
        usePBCs = boxParams["usePBCs"].asBool();
        useLEbc = boxParams["useLEbc"].asBool();
        useRigidBoxZ = boxParams["useRigidBoxZ"].asBool();
        useRigidBoxY = boxParams["useRigidBoxY"].asBool();
        boxLength = boxParams["boxLength"].asFloat();
        boxMax.x = boxParams["box_len_x"].asFloat();
        boxMax.y = boxParams["box_len_y"].asFloat(); 
        boxMax.z = boxParams["box_len_z"].asFloat();
        BoxMin.x = boxParams["BoxMin_x"].asFloat();
        BoxMin.y = boxParams["BoxMin_y"].asFloat(); 
        BoxMin.z = boxParams["BoxMin_z"].asFloat();
        flatbox = boxParams["flatbox"].asBool();
        LineCenter = boxParams["LineCenter"].asBool();
        rand_vel = boxParams["rand_vel"].asBool();
        rand_pos = boxParams["rand_pos"].asBool();
	impurity = boxParams["impurity"].asBool();
	impurityNum = boxParams["impurityNum"].asInt();
	line = boxParams["line"].asBool();
	plane = boxParams["plane"].asBool();
	
    }
    
    
    if (ranZOffset == 0)
        zOffset = 0.0;

    if (dt_tol > dt_max || dt_max <= 0 || dt_tol < 0){
        printf("ERROR: Invalid time step parameters\n");
        printf("FATAL ERROR\n");
        exit(-60); 
    }

    Json::Value randParams = inpRoot.get("rand_params", Json::nullValue);
	impurity = boxParams["impurity"].asBool();
	line = boxParams["line"].asBool();
    if (randParams == Json::nullValue){
        printf("ERROR: Cannot load randomness parameters\n");
        return -1;
    }
    else {
        add_rands = randParams["add_rands"].asBool();
        rand_seed = randParams["rand_seed"].asInt();
        rand_dist = randParams["rand_dist"].asInt();
        rand_scale_factor = randParams["rand_scale_factor"].asFloat();
    }


    printf("      mass                = %f\n",mass);
    printf("      spring equilibrium  = %f\n",R0);
    printf("      repulsion range     = %f\n",repulsion_range);
    printf("      attraction range    = %f\n",attraction_range);
    printf("      repulsion strength  = %f\n",repulsion_strength);
    printf("      attraction strength = %f\n",attraction_strength);
    printf("      Youngs modulus      = %f\n",stiffness1);
    printf("      viscotic damping    = %f\n",viscotic_damping);
    printf("      internal damping    = %f\n",internal_damping);
    printf("      division volume     = %f\n",divVol);
    printf("      ran_z_offset?       = %d\n", ranZOffset);
    printf("      z_offset            = %f\n", zOffset);
    printf("      Time steps          = %d\n",Time_steps);
    printf("      delta t             = %f\n",delta_t);
    printf("      Restart             = %d\n",Restart);
    printf("      trajWriteInterval   = %d\n",trajWriteInt);
    printf("      countOnlyInternal   = %d\n", countOnlyInternal);
    printf("      radFrac             = %f\n", radFrac);
    printf("      newCellCountInt     = %d\n", newCellCountInt);
    printf("      equiStepCount       = %d\n", equiStepCount);
    printf("      trajFileName        = %s\n", trajFileName);
    printf("      doPopModel          = %d\n", doPopModel);
    printf("      totalFood           = %f\n", totalFood);
    printf("      cellFoodCons        = %f\n", cellFoodCons);
    printf("      cellFoodConsDiv     = %f\n", cellFoodConsDiv);
    printf("      cellFoodRel         = %f\n", cellFoodRel);
    printf("      useWalls            = %d\n", useWalls);
    printf("      perpAxis            = %s\n", perpAxis);
    printf("      dAxis               = %f\n", dAxis);
    printf("      wallLen             = %f\n", wallLen);
    printf("      wallWidth           = %f\n", wallWidth);
    printf("      thresDist           = %f\n", threshDist);
    printf("      maxPressure         = %f\n", maxPressure);
    printf("      minPressure         = %f\n", minPressure);
    printf("      growth_rate         = %f\n", rMax);
    printf("      squeeze_rate         = %f\n", squeeze_rate);
    printf("      checkSphericity     = %d\n", checkSphericity);
    printf("      gamma_visc          = %f\n", gamma_visc);
    printf("      useDivPlanebasis    = %d\n", useDivPlaneBasis);
    printf("      divPlaneBasisX      = %f\n", divPlaneBasis[0]);
    printf("      divPlaneBasisY      = %f\n", divPlaneBasis[1]);
    printf("      divPlaneBasisZ      = %f\n", divPlaneBasis[2]);
    printf("      useDifferentCell = %d\n", useDifferentCell);
    printf("      SizeFactor  	=%f\n", SizeFactor);
    printf("      Stiffness2  	=%f\n", Stiffness2);
    printf("      GrowthRate2  	=%f\n", gRate);
    printf("      division volume2  	=%f\n", divisionV);
    printf("      gamma_visc2  	=%f\n", gEnv); 
    printf("      viscotic damping2  	=%f\n", gVis);            
    printf("      softYoungsMod       = %f\n", softYoungsMod);
    printf("      numberOfCells       = %d\n", numberOfCells);
    printf("      duringGrowth        = %d\n", duringGrowth);
    printf("      closenesstoCenter   = %f\n", closenessToCenter);
    printf("      fractionOfCells     = %f\n", fractionOfCells);
    printf("      chooseRandomCellIndices = %d\n", chooseRandomCellIndices);
    printf("      daughtSame = 	%d\n", daughtSame);
    printf("      recalc_r0           = %d\n", recalc_r0);
    printf("      useRigidSimulationBox = %d\n", useRigidSimulationBox);
    printf("      usePBCs             = %d\n", usePBCs);
    printf("      boxLength           = %f\n", boxLength);
    printf("      box_len_x           = %f\n", boxMax.x);
    printf("      box_len_y           = %f\n", boxMax.y);
    printf("      box_len_z           = %f\n", boxMax.z);
    printf("      BoxMin_x            = %f\n", BoxMin.x);
    printf("      BoxMin_y            = %f\n", BoxMin.y);
    printf("      BoxMin_z            = %f\n", BoxMin.z);
    printf("      flatbox             = %d\n", flatbox); 
    printf("      doAdaptive_dt       = %d\n", doAdaptive_dt); 
    printf("      dt_max              = %f\n", dt_max); 
    printf("      dt_tol              = %f\n", dt_tol);
    printf("      add_rands           = %d\n", add_rands);
    printf("      rand_seed           = %d\n", rand_seed);
    printf("      rand_scale_factor   = %f\n", rand_scale_factor);
    printf("      phase_count         = %d\n", phase_count);
    printf("      correct_com         = %d\n", correct_com);
    printf("      correct_Vcom         = %d\n", correct_Vcom);    
    printf("      impurityNum         = %d\n", impurityNum);
    printf("      apoptosis           = %d\n",apoptosis);
    printf("      Apoptosis ratio     = %f\n",Apo_rate);
    printf("      apoptosis volume    = %f\n",ApoVol);
    printf("      squeeze rate        = %f\n",squeeze_rate);
    
    
    

    if (asymDivision){
        printf("asymmetric cell division \n");
    }    

    if ( radFrac < 0.4 || radFrac > 0.8 || radFrac < 0 ){
        printf("radFrac not in [0.4, 0.8] setting to 1.\n");
        countOnlyInternal = 0;
    }

    if (trajWriteInt == 0){
        trajWriteInt = 1;
    }

    if (newCellCountInt == 0){
        newCellCountInt = 1;
    }

    if ( trajWriteInt > Time_steps + equiStepCount){
        printf ("Trajectory write interval is too large\n");
        return -1;
    }

    if ((Time_steps + equiStepCount)%trajWriteInt != 0){
        printf ("Invalid trajectory write interval. Time steps must be divisible by it. \n");
        return -1;
    }

    if (newCellCountInt > Time_steps + 1){
        printf("New cell counting interval is too large. \n");
        return -1;
    }

    if (equiStepCount <= 0){
        equiStepCount = 0;
    }

    if (doPopModel != 1){ // This ensures that Pop modelling is only done if this
        // var is only 1
        doPopModel = 0;
    }

    if (maxPressure < 0){
        printf("Invalid maximum pressure value of %f\n", maxPressure);
        printf("Disabling population modelling...");
        doPopModel = 0;
    }


    if ( (useRigidSimulationBox && usePBCs) || (useRigidSimulationBox && useLEbc) || (useLEbc && usePBCs)){
  
      printf ("More than one boundary condition.... \n");
      return -1;
    }	
    
    if ( line && rand_pos && plane){
  
      printf ("More than one initial condition.... \n");
      return -1;
    }

	


    /*

    // The if statement below is not a very good one
    // think about rewriting.
    if (totalFood < 0.0
    || No_of_threads*100 < totalFood
    || cellFoodCons < 0.0
    || cellFoodCons*No_of_threads*10 < totalFood
    || cellFoodConsDiv < 0.0
    || cellFoodConsDiv*No_of_threads*10 < totalFood
    ){
    doPopModel = 0;
    printf("Food parameters invalid. Skipping population modelling.\n");
    }
    */

    if ( !(closenessToCenter >=0 && closenessToCenter <= 1) ){
        printf("ERROR: closenessToCenter is not in [0, 1]\n");
        printf("ERROR: invalid input parameter\n");
        return -1;
    }

    if (useWalls && useRigidSimulationBox){
        printf("ERROR: Cannot use infinite XY walls and rigid simulation box simultaneously.\n");
        printf("ERROR: Only use on or the other.\n");
        return -1;
    }

    if (fractionOfCells > 1.0){
        printf("ERROR: Softer cell fraction is > 1\n");
        return -1;
    }
        

    return 0;
}


int read_global_params(void)
{
  int error;
  FILE *infil;

  printf("   Reading inp.dat ..\n");

  infil = fopen("inp.dat","r");
  if ( infil == NULL ) {printf("Error: Unable to open file inp.dat\n");return(-1);}

  error = 0;


  if ( fscanf(infil,"%f",&mass)                != 1 ) {error =  1 ;}
  if ( fscanf(infil,"%f",&repulsion_range)     != 1 ) {error =  2 ;}
  if ( fscanf(infil,"%f",&attraction_range)    != 1 ) {error =  3 ;}
  if ( fscanf(infil,"%f",&repulsion_strength)  != 1 ) {error =  4 ;}
  if ( fscanf(infil,"%f",&attraction_strength) != 1 ) {error =  5 ;}
//  if ( fscanf(infil,"%f",&Youngs_mod)          != 1 ) {error =  6 ;}
  if ( fscanf(infil,"%f",&viscotic_damping)    != 1 ) {error =  7 ;}
  if ( fscanf(infil,"%f",&internal_damping)    != 1 ) {error =  8 ;}
  if ( fscanf(infil,"%f",&divVol)              != 1 ) {error =  9 ;}
  if ( fscanf(infil,"%d",&Time_steps)          != 1 ) {error = 10 ;}
  if ( fscanf(infil,"%f",&delta_t)             != 1 ) {error = 11 ;}
  if ( fscanf(infil,"%d",&Restart)             != 1 ) {error = 12 ;}
  if ( fscanf(infil,"%d",&trajWriteInt)        != 1 ) {error = 13 ;}
  if ( fscanf(infil,"%d",&countOnlyInternal)   != 1 ) {error = 14 ;}
  if ( fscanf(infil,"%f",&radFrac)             != 1 ) {error = 15 ;}
  if ( fscanf(infil,"%d",&overWriteMitInd)     != 1 ) {error = 16 ;}
  if ( fscanf(infil,"%d",&newCellCountInt)     != 1 ) {error = 17 ;}
  if ( fscanf(infil,"%d",&equiStepCount)       != 1 ) {error = 18 ;}
  if ( fscanf(infil,"%s",trajFileName)         != 1 ) {error = 19 ;}
  if ( fscanf(infil,"%d",&doPopModel)          != 1 ) {error = 20 ;}
  if ( fscanf(infil,"%f",&totalFood)           != 1 ) {error = 21 ;}
  if ( fscanf(infil,"%f",&cellFoodCons)        != 1 ) {error = 22 ;}
  if ( fscanf(infil,"%f",&cellFoodConsDiv)     != 1 ) {error = 23 ;}
  if ( fscanf(infil,"%f",&cellFoodRel)         != 1 ) {error = 24 ;}
  if ( fscanf(infil,"%d",&haylimit)            != 1 ) {error = 25 ;}
  if ( fscanf(infil,"%d",&cellLifeTime)        != 1 ) {error = 26 ;}
  if ( fscanf(infil,"%f",&maxPressure)         != 1 ) {error = 27 ;}
  if ( fscanf(infil,"%d",&useWalls)            != 1 ) {error = 28 ;}
  if ( fscanf(infil,"%s",perpAxis)             != 1 ) {error = 29 ;}
  if ( fscanf(infil,"%f",&dAxis)               != 1 ) {error = 30 ;}
  if ( fscanf(infil,"%f",&wallLen)             != 1 ) {error = 31 ;}
  if ( fscanf(infil,"%f",&wallWidth)           != 1 ) {error = 32 ;}
  if ( fscanf(infil,"%f",&threshDist)          != 1 ) {error = 33 ;}





  fclose(infil);

  if ( error != 0 ){
      printf("   Error reading line %d from file inp.dat\n",error);
      return(-1);
  }

  if ( radFrac < 0.4 || radFrac > 0.8 || radFrac < 0 ){
      printf("radFrac not in [0.4, 0.8] setting to 1.\n");
      countOnlyInternal = 0;
  }

  if (trajWriteInt == 0){
      trajWriteInt = 1;
  }

  if (newCellCountInt == 0){
      newCellCountInt = 1;
  }

  if ( trajWriteInt > Time_steps){
      printf ("Trajectory write interval is too large\n");
      return -1;
  }
  

  if (Time_steps%trajWriteInt != 0){
      printf ("Invalid trajectory write interval. Time steps must be divisible by it. \n");
      return -1;
  }

  if (newCellCountInt > Time_steps){
      printf("New cell counting interval is too large. \n");
      return -1;
  }

  if (equiStepCount <= 0){
    equiStepCount = 0;
  }

  if (doPopModel != 1){ // This ensures that Pop modelling is only done if this
                        // var is only 1
      doPopModel = 0;
  }

  if (maxPressure < 0){
      printf("Invalid maximum pressure value of %f\n", maxPressure);
      printf("Disabling population modelling...");
      doPopModel = 0;
  }


  /*

  // The if statement below is not a very good one
  // think about rewriting.
  if (totalFood < 0.0
      || No_of_threads*100 < totalFood
      || cellFoodCons < 0.0
      || cellFoodCons*No_of_threads*10 < totalFood
      || cellFoodConsDiv < 0.0
      || cellFoodConsDiv*No_of_threads*10 < totalFood
       ){
      doPopModel = 0;
      printf("Food parameters invalid. Skipping population modelling.\n");
  }
  */


  printf("      mass                = %f\n",mass);
  printf("      spring equilibrium  = %f\n",R0);
  printf("      repulsion range     = %f\n",repulsion_range);
  printf("      attraction range    = %f\n",attraction_range);
  printf("      repulsion strength  = %f\n",repulsion_strength);
  printf("      attraction strength = %f\n",attraction_strength);
//  printf("      Youngs modulus      = %f\n",Youngs_mod);
  printf("      viscotic damping    = %f\n",viscotic_damping);
  printf("      internal damping    = %f\n",internal_damping);
  printf("      division volume     = %f\n",divVol);
  printf("      Time steps          = %d\n",Time_steps);
  printf("      delta t             = %f\n",delta_t);
  printf("      Restart             = %d\n",Restart);
  printf("      trajWriteInterval   = %d\n",trajWriteInt);
  printf("      countOnlyInternal   = %d\n", countOnlyInternal);
  printf("      radFrac             = %f\n", radFrac);
  printf("      newCellCountInt     = %d\n", newCellCountInt);
  printf("      equiStepCount       = %d\n", equiStepCount);
  printf("      trajFileName        = %s\n", trajFileName);
  printf("      doPopModel          = %d\n", doPopModel);
  printf("      totalFood           = %f\n", totalFood);
  printf("      cellFoodCons        = %f\n", cellFoodCons);
  printf("      cellFoodConsDiv     = %f\n", cellFoodConsDiv);
  printf("      cellFoodRel         = %f\n", cellFoodRel);
  printf("      useWalls            = %d\n", useWalls);
  printf("      perpAxis            = %s\n", perpAxis);
  printf("      dAxis               = %f\n", dAxis);
  printf("      wallLen             = %f\n", wallLen);
  printf("      wallWidth           = %f\n", wallWidth);
  printf("      thresDist           = %f\n", threshDist);


  return(0);
}




//C *****************************************************************



void write_traj(int t_step, FILE* trajfile)
{

  fprintf(trajfile, "%d\n", No_of_C180s * 192);
  fprintf(trajfile, "Step: %d frame: %d\n", t_step, t_step/trajWriteInt);
  
  if (useDifferentCell){
      for (int c = 0; c < No_of_C180s; c++){
          if (youngsModArray[c] == stiffness1)
              fprintf(trajfile, "cell: %d H\n", CellINdex[c]);
          else if(youngsModArray[c] == Stiffness2)
              fprintf(trajfile, "cell: %d C\n", CellINdex[c]);
          else
              fprintf(trajfile, "cell: %d UnknownStiffness\n", CellINdex[c]);

          for (int p = 0; p < 192; p++)
          {
              fprintf(trajfile, "%.7f,  %.7f,  %.7f\n", X[(c*192)+p], Y[(c*192)+p], Z[(c*192)+p]);
          }
      }
        
  } else {
      for (int c = 0; c < No_of_C180s; c++){
              fprintf(trajfile, "cell: %d\n", CellINdex[c]);
              
              for (int p = 0; p < 192; p++)
              {
                  fprintf(trajfile, "%.7f,  %.7f,  %.7f\n", X[(c*192)+p], Y[(c*192)+p], Z[(c*192)+p]);
              }
      }
      
  }
}

void WriteBinaryTraj(int t_step, FILE* trajFile, int frameCount){
    
    fwrite(&t_step, sizeof(int), 1, trajFile);
    fwrite(&frameCount, sizeof(int), 1, trajFile); 
    fwrite(&No_of_C180s, sizeof(int), 1, trajFile);
    if (useDifferentCell){
        int cellType = 0; 
        for (int c = 0; c < No_of_C180s; c++){ 
            fwrite(&CellINdex[c], sizeof(int), 1, trajFile);
            fwrite(X + (c*192), sizeof(float), 192, trajFile); 
            fwrite(Y + (c*192), sizeof(float), 192, trajFile); 
            fwrite(Z + (c*192), sizeof(float), 192, trajFile);
            
            if (youngsModArray[c] == stiffness1)
                cellType = 0;
            else
                cellType = 1; 
            
            fwrite(&cellType, sizeof(int), 1, trajFile);
        }
    } else {
        for (int c = 0; c < No_of_C180s; c++){
        
            fwrite(&CellINdex[c], sizeof(int), 1, trajFile);            
            fwrite(X + (c*192), sizeof(float), 192, trajFile); 
            fwrite(Y + (c*192), sizeof(float), 192, trajFile); 
            fwrite(Z + (c*192), sizeof(float), 192, trajFile); 
        }
    }
        
    
}

void write_vel(int t_step, FILE* velFile,int frameCount){
    
    fwrite(&t_step, sizeof(int), 1, velFile);
    fwrite(&frameCount, sizeof(int), 1, velFile); 
    fwrite(&No_of_C180s, sizeof(int), 1, velFile);
    
    if (useDifferentCell){
        int cellType = 0; 
        for (int c = 0; c < No_of_C180s; c++){
            fwrite(&c, sizeof(int), 1, velFile);
            fwrite(velListX + (c*192), sizeof(float), 192, velFile); 
            fwrite(velListY + (c*192), sizeof(float), 192, velFile); 
            fwrite(velListZ + (c*192), sizeof(float), 192, velFile);
            
            if (youngsModArray[c] == stiffness1)
                cellType = 0;
            else
                cellType = 1; 
            
            fwrite(&cellType, sizeof(int), 1, velFile);
        }
    } else {
        for (int c = 0; c < No_of_C180s; c++){
        
            fwrite(&c, sizeof(int), 1, velFile);
            fwrite(velListX + (c*192), sizeof(float), 192, velFile); 
            fwrite(velListY + (c*192), sizeof(float), 192, velFile); 
            fwrite(velListZ + (c*192), sizeof(float), 192, velFile);
        }
    }
}


inline void count_and_get_div(){
  num_cell_div = 0;
  hipMemcpy(cell_div, d_cell_div, No_of_C180s*sizeof(char), hipMemcpyDeviceToHost);
  
  for (int cellInd = 0; cellInd < No_of_C180s; cellInd++) {
    if (cell_div[cellInd] == 1){
      cell_div[cellInd] = 0;
      cell_div_inds[num_cell_div] = cellInd;
      num_cell_div++;
    }
  }
  hipMemcpy(d_cell_div, cell_div, No_of_C180s*sizeof(char), hipMemcpyHostToDevice);
}


inline void count_and_die(){
  
  num_cell_Apo = 0;
  hipMemcpy(cell_Apo, d_cell_Apo, No_of_C180s*sizeof(char), hipMemcpyDeviceToHost);
  hipMemcpy(d_Growth_rate, Growth_rate, No_of_C180s*sizeof(float), hipMemcpyHostToDevice);
  
  for (int cellInd = 0; cellInd < No_of_C180s; cellInd++) {
    if (cell_Apo[cellInd] == 1){ 
      if(Growth_rate[cellInd] == -squeeze_rate || Growth_rate[cellInd] == -squeeze_rate2){
      	cell_Apo[cellInd] = 0;
      	cell_Apo_inds[num_cell_Apo] = cellInd;
      	num_cell_Apo++;
      }
    }
  }
  hipMemcpy(d_cell_Apo, cell_Apo, No_of_C180s*sizeof(char), hipMemcpyHostToDevice);
}



inline void calc_sys_CM(){ // Put this into a kernel at some point

  sysCMx = 0;
  sysCMy = 0;
  sysCMz = 0;

  for (int cellInd = 0; cellInd < No_of_C180s; cellInd++) {
    sysCMx += CMx[cellInd];
    sysCMy += CMy[cellInd];
    sysCMz += CMz[cellInd];
  }

  sysCMx = sysCMx/No_of_C180s;
  sysCMy = sysCMy/No_of_C180s;
  sysCMz = sysCMz/No_of_C180s;

}


inline void calc_sys_VCM(){ // Put this into a kernel at some point

  sysVCMx = 0;
  sysVCMy = 0;
  sysVCMz = 0;

  for (int cellInd = 0; cellInd < No_of_C180s; cellInd++) {
    sysVCMx += VCMx[cellInd];
    sysVCMy += VCMy[cellInd];
    sysVCMz += VCMz[cellInd];
  }

  sysVCMx = sysVCMx/No_of_C180s;
  sysVCMy = sysVCMy/No_of_C180s;
  sysVCMz = sysVCMz/No_of_C180s;

}



inline float getRmax2(){
  float dx, dy, dz, Rmax2 = 0;
  for (int cell = 0; cell < No_of_C180s; cell++) {
    dx = CMx[cell] - sysCMx;
    dy = CMy[cell] - sysCMy;
    dz = CMz[cell] - sysCMz;

    Rmax2 = max(Rmax2, dx*dx + dy*dy + dz*dz);

  }

  return Rmax2;

}

inline int num_cells_far(){

  if (num_cell_div == 0 || No_of_C180s < 50) return 0;

  hipMemcpy(CMx, d_CMx, No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(CMy, d_CMy, No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(CMz, d_CMz, No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);

  calc_sys_CM();

  float dx, dy, dz, dr2;
  float Rmax2 = getRmax2();
  int farCellCount = 0;

  for (int cell = No_of_C180s - num_cell_div; cell < No_of_C180s; cell++) { // Only check the newest cells
    dx = CMx[cell] - sysCMx;
    dy = CMy[cell] - sysCMy;
    dz = CMz[cell] - sysCMz;

    dr2 = dx*dx + dy*dy + dz*dz;

    if (dr2 > radFrac*radFrac*Rmax2)
      farCellCount++;
  }

  return farCellCount;

}

void writeForces(FILE* forceFile, int t_step, int num_cells){
    if(forceFile == NULL){
        printf("ERROR: forces file not available\n");
        exit(1);
    }

    for (int c =0; c < num_cells; ++c){
        for (int n = 0; n < 180; ++n){
            fprintf(forceFile, "%d,%d,%d,%d,%d,%.8f,%.8f,%.8f,%.8f,%.8f,%.8f,%.8f,%.8f,%.8f,%.8f,%.8f,%.8f,%.8f,%.8f,%.8f,%.8f,%.8f,%.8f\n",
                    t_step, num_cells, c, n, c*192+n,
                    h_contactForces.x[c*192 + n],
                    h_contactForces.y[c*192 + n],
                    h_contactForces.z[c*192 + n],
                    mag(make_float3(h_contactForces.x[c*192 + n],
                                    h_contactForces.y[c*192 + n],
                                    h_contactForces.z[c*192 + n])),
                                        h_ExtForces.x[c*192 + n],
                    h_ExtForces.y[c*192 + n],
                    h_ExtForces.z[c*192 + n],
                    mag(make_float3(h_ExtForces.x[c*192 + n],
                                    h_ExtForces.y[c*192 + n],
                                    h_ExtForces.z[c*192 + n])),
                    velListX[c*192+n],
                    velListY[c*192+n],
                    velListZ[c*192+n],
                    mag(make_float3(velListX[c*192+n],
                                    velListY[c*192+n],
                                    velListZ[c*192+n])),
                    X[c*192+n],
                    Y[c*192+n],
                    Z[c*192+n],
                    pressList[c],
                    volume[c],
                    area[c]
                );
                        
        }


    }
}





int writeRestartFile(int t_step, int frameCount){
	
	FILE *Restartfile;
	Restartfile = fopen ("Restart.xyz", "w");
        if ( Restartfile == NULL)
  	{
      	    printf("Failed to open Restart file \n" );
      	    return -1;
  	}

	int cellType = 0;
	float p = 0;
	float y = 0;
	float g = 0;
	float s = 0;
	float d = 0;
	float ge = 0;
	float vd = 0;
	int I = 0;	

	fwrite(&t_step, sizeof(int), 1, Restartfile);
	fwrite(&frameCount, sizeof(int), 1, Restartfile); 
	fwrite(&No_of_C180s, sizeof(int), 1, Restartfile); 
	fwrite(&impurityNum, sizeof(int), 1, Restartfile);
	fwrite(&No_of_Cell1, sizeof(int), 1, Restartfile);
	fwrite(&No_of_Cell2, sizeof(int), 1, Restartfile);
	fwrite(&CellInApo1, sizeof(int), 1, Restartfile);
	fwrite(&CellInApo2, sizeof(int), 1, Restartfile);
	fwrite(&NumApoCell, sizeof(int), 1, Restartfile);  
 
        	
        for (int c = 0; c < No_of_C180s; c++){
		p = pressList[c];
		y = youngsModArray[c];
        	g = Growth_rate[c];
        	s = ScaleFactor[c];
        	d = DivisionVolume[c];
        	ge = gamma_env[c];
        	vd =  viscotic_damp[c];
        	I = CellINdex[c];
        		
		fwrite(&c, sizeof(int), 1, Restartfile);
        	fwrite(X + (c*192), sizeof(float), 192, Restartfile); 
        	fwrite(Y + (c*192), sizeof(float), 192, Restartfile); 
        	fwrite(Z + (c*192), sizeof(float), 192, Restartfile);
		fwrite(velListX + (c*192), sizeof(float), 192, Restartfile);
            	fwrite(velListY + (c*192), sizeof(float), 192, Restartfile);
		fwrite(velListZ + (c*192), sizeof(float), 192, Restartfile);
		fwrite(&p, sizeof(float), 1, Restartfile);
            	fwrite(&y, sizeof(float), 1, Restartfile);
            	fwrite(&g, sizeof(float), 1, Restartfile);
            	fwrite(&s, sizeof(float), 1, Restartfile);
            	fwrite(&d, sizeof(float), 1, Restartfile);
            	fwrite(&ge, sizeof(float), 1, Restartfile);
            	fwrite(&vd, sizeof(float), 1, Restartfile);
            	fwrite(&I, sizeof(int), 1, Restartfile);
            	
        }

   
   fclose(Restartfile);
   return 0;

}

int ReadRestartFile( ){

  FILE *infil;
  int s;
  int f;
  int nCell;
  int nImp;
  int CellType;  
  int CellInd;
  int shift;
  int CA1;
  int CA2;
  int NC1;
  int NC2;
  int NCA;
  

  printf("Reading Restart.xyz ...\n");
  infil = fopen("Restart.xyz","rb");
  
  if ( infil == NULL ) {
    printf("Unable to open file Restart.xyz \n");
    return(-1);
  }



  if ( fread(&s, sizeof(int),1,infil) != 1 ){ 
	printf("Data missing from trajectory. \n");
	return(-1);
  } else printf("\nstep %d \n",s -1);

  if ( fread(&f, sizeof(int),1,infil) != 1 ){ 
	printf("Data missing from trajectory. \n");
	return(-1);
  } else printf("frame number is: %d \n",f - 1);

  if ( fread(&nCell, sizeof(int),1,infil) != 1 ) { 
	printf("Data missing from trajectory. \n");
	return(-1);
  }
  
  if ( fread(&nImp, sizeof(int),1,infil) != 1 ) { 
	printf("Data missing from trajectory. \n");
	return(-1);
  }

  if ( fread(&NC1, sizeof(int),1,infil) != 1 ) { 
	printf("Data missing from trajectory. \n");
	return(-1);
  }

  if ( fread(&NC2, sizeof(int),1,infil) != 1 ) { 
	printf("Data missing from trajectory. \n");
	return(-1);
  }

  if ( fread(&CA1, sizeof(int),1,infil) != 1 ) { 
	printf("Data missing from trajectory. \n");
	return(-1);
  }

  if ( fread(&CA2, sizeof(int),1,infil) != 1 ) { 
	printf("Data missing from trajectory. \n");
	return(-1);
  }

  if ( fread(&NCA, sizeof(int),1,infil) != 1 ) { 
	printf("Data missing from trajectory. \n");
	return(-1);
  }


  Laststep = s-1;
  Lastframe = f-1;
  No_of_threads = nCell;	
  No_of_C180s = nCell;
  Orig_No_of_C180s = nCell;
  impurityNum = nImp;
  CellInApo1 = CA1;
  CellInApo2 = CA2;
  No_of_Cell1 = NC1;
  No_of_Cell2 = NC2;
  NumApoCell = NCA;  

  printf("Number of the initial Cells is: %d \n",Orig_No_of_C180s);
  printf("Number of the  impurity is: %d \n",impurityNum);
  

  for (int c = 0; c < Orig_No_of_C180s; c++){

    shift = c*192;
    
    if ( fread(&CellInd, sizeof(int),1,infil) != 1 ) printf("Data missing from trajectory. \n");
    if ( fread(&X[shift], sizeof(float),192,infil) != 192 ) printf("Data missing from trajectory. \n");
    if ( fread(&Y[shift], sizeof(float),192,infil) != 192 ) printf("Data missing from trajectory. \n");
    if ( fread(&Z[shift], sizeof(float),192,infil) != 192 ) printf("Data missing from trajectory. \n");
    if ( fread(&velListX[shift], sizeof(float),192,infil) != 192 ) printf("Data missing from trajectory. \n");
    if ( fread(&velListY[shift], sizeof(float),192,infil) != 192 ) printf("Data missing from trajectory. \n");
    if ( fread(&velListZ[shift], sizeof(float),192,infil) != 192 ) printf("Data missing from trajectory. \n");
    if ( fread(&pressList[c], sizeof(float),1,infil) != 1 ) printf("Data missing from trajectory. \n");
    if ( fread(&youngsModArray[c], sizeof(float),1,infil) != 1 ) printf("Data missing from trajectory. \n");
    if ( fread(&Growth_rate[c], sizeof(float),1,infil) != 1 ) printf("Data missing from trajectory. \n");
    if ( fread(&ScaleFactor[c], sizeof(float),1,infil) != 1 ) printf("Data missing from trajectory. \n");
    if ( fread(&DivisionVolume[c], sizeof(float),1,infil) != 1 ) printf("Data missing from trajectory. \n");
    if ( fread(&gamma_env[c], sizeof(float),1,infil) != 1 ) printf("Data missing from trajectory. \n");
    if ( fread(&viscotic_damp[c], sizeof(float),1,infil) != 1 ) printf("Data missing from trajectory. \n");
    if ( fread(&CellINdex[c], sizeof(int),1,infil) != 1 ) printf("Data missing from trajectory. \n");
	
   }

   fclose(infil);
   return 0;

}
	

