#include "hip/hip_runtime.h"
//#define FORCE_DEBUG
//#define PRINT_VOLUMES
//#define TURNOFF_RAN
//#define DEBUG_RAND
//#define OUTPUT_ADP_ERROR
//#degine RO_DEBUG
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <locale.h>
#include <algorithm>
#include <iostream>
#include <fstream>
#include <streambuf>
#include <cstring>
#include <string>

#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <vector_functions.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/fill.h>
#include <thrust/sort.h>
#include <thrust/extrema.h>
//#include "hip/hip_runtime_api.h"
#include "postscript.h"
#include "marsaglia.h"
//#include "IntegrationKernels.h"
#include "RandomVector.h"
#include "VectorFunctions.hpp"
#include "AdaptiveTimeKernels.cuh"

#include "json/json.h"


void CudaFailure();

#define CudaErrorCheck() { \
        hipError_t e = hipPeekAtLastError();        \
        if (e!=hipSuccess){                                            \
            printf("Cuda failure in %s, line %d, code %d %s\n", __FILE__, __LINE__, e, hipGetErrorString(e)); \
            exit(0); \
        }\
    }



// the three nearest neighbours of C180 atoms
int   C180_nn[3*192];
int   C180_sign[180];
// device: the three nearest neighbours of C180 atoms
int   *d_C180_nn;
int   *d_C180_sign;
int   CCI[2][271];       // list of nearest neighbor carbon pairs in the fullerne
// number of pairs = 270
int   C180_56[92*7];     // 12 lists of atoms forming pentagons 1 2 3 4 5 1 1 and
// 80 lists of atoms forming hexagons  1 2 3 4 5 6 1
int   *d_C180_56;


float mass;                                           //  M
float repulsion_range,    attraction_range;        //  LL1, LL2
float repulsion_strength, attraction_strength;     //  ST1, ST2
float viscotic_damping, internal_damping;          //  C, DMP    
float divVol;
float ApoVol;
float gamma_visc;
float shear_rate;
float Pshift;
float Vshift;
float maxPressure;
float minPressure;
float rMax;
float squeeze_rate;
int Restart;
int Laststep = 0;
int Lastframe = 0;
int   Time_steps;
int   trajWriteInt; // trajectory write interval

float delta_t;

//__constant__ float d_dt;

float dt_max;
float dt_tol;
bool doAdaptive_dt;
float c1 = 0; float c2 = 0; 

bool write_cont_force=false;
bool write_vel_file = false;
char forces_file[256];
int   overWriteMitInd; // 0 No, 1 yes
const char* ptrajFileName;
char trajFileName[256];
bool binaryOutput; 
char mitIndFileName[256]; 


bool asymDivision;
float* asym;
float* d_asym;
bool checkSphericity; 
bool useDivPlaneBasis;
float divPlaneBasis[3]; 



int   countOnlyInternal; // 0 - Count all new cells
                         // 1 - Count only the cells born within 0.6Rmax from
                         //     the center of mass of the system
float radFrac; 	// The factor to count cells within a raduys (<Rmax)
int newCellCountInt; // Interval at which to count the divided cells
int equiStepCount;
bool countCells;

// equilibrium length of springs between fullerene atoms
float* d_R0;
float* h_R0;

bool constrainAngles;
angles3* theta0;


float L1  = 3.0f;       // the initial fullerenes are placed in
			// an X x Y grid of size L1 x L1


float *d_volume;
float *volume;
float *d_area; 
float *area; 


char* cell_div;
char* d_cell_div;
int num_cell_div;
int* cell_div_inds;

char* cell_Apo;
char* d_cell_Apo;
int num_cell_Apo;
int* cell_Apo_inds;



// Params related to population modelling
int doPopModel;
char* didCellDie;
float totalFood;
float* d_totalFood;
int haylimit;
int cellLifeTime;
float cellFoodCons; // baseline food consumption
float cellFoodConsDiv; // Extra good consumption when cell divides
float cellFoodRel; // Food released when cell dies (should < total consumed food)
float maxPop;
 

float3 boxMax;
float3 BoxMin;
float3 BoxCen;
bool flatbox; 
bool LineCenter; 
bool useRigidSimulationBox;
float threshDist;
bool usePBCs; 
bool useLEbc;
bool useRigidBoxZ; 
bool useRigidBoxY; 
bool rand_pos;
bool impurity;
int impurityNum;
bool line;
bool plane;
float L  = 2.5f;  

int No_of_threads; // ie number of staring cells
int Side_length;
int ex, ey;



// randomness parameters

bool add_rands;
int rand_seed;
int rand_dist;
float rand_scale_factor;
hiprandState *d_rngStates;
unsigned int *d_seeds; 



float  *X,  *Y,  *Z;     // host: atom positions
//float *d_XP, *d_YP, *d_ZP;     // device: time propagated atom positions
float  *d_X,  *d_Y,  *d_Z;     // device: present atom positions
float* d_velListX, *d_velListY, *d_velListZ; 
float* velListX, *velListY, *velListZ; 


R3Nptrs d_fConList;
R3Nptrs d_fDisList;
R3Nptrs d_fRanList; 
R3Nptrs d_ExtForces;
R3Nptrs h_contactForces;
R3Nptrs h_ExtForces;


float DL;
float3 DLp;
int Xdiv, Ydiv, Zdiv;

//int *d_NoofNNlist;
int *d_NNlist;
int *NoofNNlist;
int *NNlist;


bool correct_com = false;
bool correct_Vcom = false;
int reductionblocks;

float *d_CMx, *d_CMy, *d_CMz;
float *CMx, *CMy, *CMz;
float *d_VCMx, *d_VCMy, *d_VCMz;
float *VCMx, *VCMy, *VCMz;
float *d_SysCx, *d_SysCy, *d_SysCz; 

R3Nptrs h_sysCM;
R3Nptrs d_sysCM;
R3Nptrs d_sysVCM;
R3Nptrs h_sysVCM;


//float Pressure;          // pressure
//float Temperature;       // equation of state relates Pressure and Temperature

int  No_of_C180s;        // the global number of C180 fullerenes
int  No_of_C180s_in;     // the number of C180s near the center of mass of the system
int MaxNoofC180s;
int NewCellInd; 

float *ran2;             // host: ran2[]
float *d_ran2;           // device: ran2[], used in celldivision

int *NDIV;               // # of divisions

// Parameters related to division

long int GPUMemory;
long int CPUMemory;


int frameCount = 1;


int Orig_No_of_C180s;
 

bool apoptosis;
float Apo_rate;
int popToStartApo;
bool WithoutApo;
int NumApoCell;
int NumRemoveCell;



bool colloidal_dynamics;
bool dispersity;
bool rand_vel;
float Xratio;
float Yratio;
float Zratio;
float shapeLim;
bool RandInitDir;



bool useDifferentCell;
float* ScaleFactor;
float* d_ScaleFactor;
float* DivisionVolume;
float* d_DivisionVolume;
float* gamma_env;
float* d_gamma_env;
float* viscotic_damp;
float* d_viscotic_damp;
float* d_Growth_rate;
float* Growth_rate;
float* d_Youngs_mod;
float* youngsModArray;
float *d_pressList;
float *pressList;
int* d_resetIndices;
int* resetIndices; 
int* CellINdex;
int* d_CellINdex;


float SizeFactor;
float Stiffness2;
float stiffness1;
float Youngs_mod; 
float gRate;
float divisionV;
float gEnv;
float gVis;
float Apo_rate2;
float squeeze_rate2;
int   numberOfCells;
float fractionOfCells;
float closenessToCenter;
bool  chooseRandomCellIndices;
bool daughtSame;
bool duringGrowth;
bool recalc_r0; 


R3Nptrs DivPlane;
R3Nptrs d_DivPlane;
float f_range;

int main(int argc, char *argv[])
{


  int globalrank;
  int step = 0;
  int noofblocks, threadsperblock;
  int newcells;
  
  NumApoCell = 0;
  NumRemoveCell = 0;
  
  
  FILE *outfile;
  FILE *trajfile; // pointer to xyz file
  hipError_t myError;

  int* dividingCells; //Cells that are about to divide
  int* totalCells; // No. of cells at every Dividing_steps


  int* num_new_cells_per_step;
  int countOffset = 0;

  //int min_no_of_cells = 10;

  printf("CellDiv version 0.9\n");

  if ( argc !=4 )
  {
      printf("Usage: CellDiv no_of_threads inpFile.json gpuid\n");
      return(0);
  }
  

  No_of_threads = atoi(argv[1]);

  char inpFile[256];
  strcpy(inpFile, argv[2]);

  if ( read_json_params(inpFile)          != 0 ) return(-1);

  printf("%d\n", MaxNoofC180s); 
  
  Vshift = shear_rate*boxMax.x;
  Pshift = 0;
  
  float Vscale;
  float scale;
	
  Vscale = 1.0/(Xratio*Yratio*Zratio);
  scale = pow(Vscale, 1.0/3.0);
	
  Xratio *= scale;
  Yratio *= scale;
  Zratio *= scale;

  
  if((Xratio >= Yratio) && (Xratio >= Zratio)){
        shapeLim = Xratio;
  }else if ((Yratio >= Xratio) && (Yratio >= Zratio)){
        shapeLim = Yratio;
  }else{
        shapeLim = Zratio;
  }

  if(!colloidal_dynamics) shapeLim = 1.0f;
  
  f_range = (attraction_range + 0.9*shapeLim) * (attraction_range + 0.9*shapeLim);
  	
  if ( line ) {
  
  	int LineCell = 0;
  	LineCell = (int) ((boxMax.x - BoxMin.x)/L);
  	printf(" Max Number of initial cells in line is:   %d\n", LineCell); 
  	if ( No_of_threads > LineCell ) {
  	printf(" Number of initial cells in line is greater than Max \n ");
  	return(-1);
  	}	
  }	

  Side_length   = (int)( sqrt( (double)No_of_threads )+0.5);
  if ( No_of_threads > MaxNoofC180s // Side_length*Side_length != No_of_threads
      )
  {
      printf("Usage: Celldiv no_of_threads\n");
      printf("       no_of_threads should be a square, n^2, < %d\n", MaxNoofC180s);
      return(0);
  }

  if (!impurity) impurityNum = 0;	
  No_of_C180s      = No_of_threads + impurityNum;
  Orig_No_of_C180s = No_of_C180s;
  GPUMemory = 0L;
  CPUMemory = 0L;



  X = (float *)calloc(192*MaxNoofC180s,sizeof(float));
  Y = (float *)calloc(192*MaxNoofC180s,sizeof(float));
  Z = (float *)calloc(192*MaxNoofC180s,sizeof(float));
  velListX = (float *)calloc(192*MaxNoofC180s, sizeof(float)); 
  velListY = (float *)calloc(192*MaxNoofC180s, sizeof(float)); 
  velListZ = (float *)calloc(192*MaxNoofC180s, sizeof(float));
  youngsModArray = (float *)calloc(MaxNoofC180s, sizeof(float));
  Growth_rate = (float *)calloc(MaxNoofC180s, sizeof(float));
  pressList = (float *)calloc(MaxNoofC180s, sizeof(float));  
  CellINdex = (int *)calloc(MaxNoofC180s, sizeof(int));
  ScaleFactor = (float *)calloc(MaxNoofC180s, sizeof(float)); 
  DivisionVolume = (float *)calloc(MaxNoofC180s, sizeof(float));
  gamma_env = (float *)calloc(MaxNoofC180s, sizeof(float));
  viscotic_damp = (float *)calloc(MaxNoofC180s, sizeof(float));
  h_contactForces.x = (float *)calloc(192*MaxNoofC180s, sizeof(float));
  h_contactForces.y = (float *)calloc(192*MaxNoofC180s, sizeof(float));
  h_contactForces.z = (float *)calloc(192*MaxNoofC180s, sizeof(float));
  DivPlane.x = (float *)calloc(MaxNoofC180s, sizeof(float));
  DivPlane.y = (float *)calloc(MaxNoofC180s, sizeof(float));
  DivPlane.z = (float *)calloc(MaxNoofC180s, sizeof(float));
  h_ExtForces.x = (float *)calloc(192*MaxNoofC180s, sizeof(float));
  h_ExtForces.y = (float *)calloc(192*MaxNoofC180s, sizeof(float));
  h_ExtForces.z = (float *)calloc(192*MaxNoofC180s, sizeof(float));
  CMx = (float *)calloc(MaxNoofC180s, sizeof(float));
  CMy = (float *)calloc(MaxNoofC180s, sizeof(float));
  CMz = (float *)calloc(MaxNoofC180s, sizeof(float));
  VCMx = (float *)calloc(MaxNoofC180s, sizeof(float));
  VCMy = (float *)calloc(MaxNoofC180s, sizeof(float));
  VCMz = (float *)calloc(MaxNoofC180s, sizeof(float));
  area= (float *)calloc(MaxNoofC180s, sizeof(float));
  cell_div = (char *)calloc(MaxNoofC180s, sizeof(char));
  cell_div_inds = (int *)calloc(MaxNoofC180s, sizeof(int));
  cell_Apo = (char *)calloc(MaxNoofC180s, sizeof(char));
  cell_Apo_inds = (int *)calloc(MaxNoofC180s, sizeof(int));
  NoofNNlist = (int *)calloc( 1024*1024,sizeof(int));
  NNlist =  (int *)calloc(32*1024*1024, sizeof(int));
  asym = (float *)calloc(MaxNoofC180s, sizeof(float));
  resetIndices = (int *)calloc(MaxNoofC180s, sizeof(int));
  h_sysCM.x = (float *)calloc(1, sizeof(float));
  h_sysCM.y = (float *)calloc(1, sizeof(float));
  h_sysCM.z = (float *)calloc(1, sizeof(float));
  h_sysVCM.x = (float *)calloc(1, sizeof(float));
  h_sysVCM.y = (float *)calloc(1, sizeof(float));
  h_sysVCM.z = (float *)calloc(1, sizeof(float));
//volume= (float *)calloc(MaxNoofC180s, sizeof(float));  
//  SysCx = (float *) calloc(1024 , sizeof(float));
//  SysCy = (float *) calloc(1024 , sizeof(float));
//  SysCz = (float *) calloc(1024 , sizeof(float));



  CPUMemory += 6L*192L*MaxNoofC180s*sizeof(float);
  CPUMemory += MaxNoofC180s*10L*sizeof(float);
  CPUMemory += MaxNoofC180s*7L*sizeof(float);
  CPUMemory += MaxNoofC180s*sizeof(float);
  CPUMemory += 3L*MaxNoofC180s*sizeof(float);
  CPUMemory += 6L*1024L*sizeof(float);
  CPUMemory += MaxNoofC180s*sizeof(char);
  CPUMemory += MaxNoofC180s*sizeof(int);
  CPUMemory += MaxNoofC180s*sizeof(int); 
  CPUMemory += 3*180*sizeof(float);
  CPUMemory += 2*MaxNoofC180s*sizeof(int); 



  hipDeviceProp_t deviceProp = getDevice();
  if (hipSuccess != hipSetDevice(atoi(argv[3]))){
      CudaErrorCheck();
      printf("Could not set to divice %d\n", atoi(argv[3]));
      return -1;
  }


  thrust::device_vector<float> d_volumeV(MaxNoofC180s);
  thrust::host_vector<float> h_volume(MaxNoofC180s);
  thrust::fill(d_volumeV.begin(), d_volumeV.end(), 0.f);
  d_volume = thrust::raw_pointer_cast(&d_volumeV[0]);
  volume = thrust::raw_pointer_cast(&h_volume[0]);
  
  thrust::host_vector<angles3> h_theta0(192);
  thrust::device_vector<angles3> d_theta0V(192);
  angles3* d_theta0 = thrust::raw_pointer_cast(&d_theta0V[0]);
  theta0 = thrust::raw_pointer_cast(&h_theta0[0]);


  h_R0 = (float *)calloc(192*3, sizeof(float));
  for (int i =  0; i < MaxNoofC180s; ++i) ScaleFactor[i] = 1.0;

  //if ( read_global_params()               != 0 ) return(-1);
  if (Restart == 1 ) if( ReadRestartFile() != 0 ) return(-1); 
  if (generate_random(Orig_No_of_C180s)  != 0 ) return(-1);
  if (DispersityFunc(Orig_No_of_C180s) != 0 ) return(-1);
  if (Restart == 0 ) if ( initialize_C180s(Orig_No_of_C180s) != 0 ) return(-1);
  if ( read_fullerene_nn()                != 0 ) return(-1);
  
 
  // empty the psfil from previous results
  outfile = fopen("psfil","w");
  if ( outfile == NULL ) {printf("Unable to open file psfil\n");return(-1);}
  fclose(outfile);

  /* PM
     Allocate memory for the dividingCells array that will be used to
     calculate the mitotic index.
  */

  dividingCells = (int *)calloc((Time_steps/newCellCountInt), sizeof(int));
  totalCells = (int *)calloc((Time_steps/newCellCountInt), sizeof(int));
  num_new_cells_per_step = (int *)calloc(Time_steps, sizeof(int));
  


  CPUMemory += (2L*(long)(Time_steps/newCellCountInt) + 1L + (long)Time_steps) * sizeof(int);
  CPUMemory += (long)MaxNoofC180s * sizeof(char);





  if ( hipSuccess != hipMalloc((void **)&d_sysVCM.x, sizeof(float))) return -1;
  if ( hipSuccess != hipMalloc((void **)&d_sysVCM.y, sizeof(float))) return -1;
  if ( hipSuccess != hipMalloc((void **)&d_sysVCM.z, sizeof(float))) return -1;  
  if ( hipSuccess != hipMalloc((void **)&d_sysCM.x, sizeof(float))) return -1;
  if ( hipSuccess != hipMalloc((void **)&d_sysCM.y, sizeof(float))) return -1;
  if ( hipSuccess != hipMalloc((void **)&d_sysCM.z, sizeof(float))) return -1;    
  if ( hipSuccess != hipMalloc( (void **)&d_C180_nn, 3*192*sizeof(int))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_C180_sign, 180*sizeof(int))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_X  , 192*MaxNoofC180s*sizeof(float))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_Y  , 192*MaxNoofC180s*sizeof(float))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_Z  , 192*MaxNoofC180s*sizeof(float))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_CMx , MaxNoofC180s*sizeof(float))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_CMy , MaxNoofC180s*sizeof(float))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_CMz , MaxNoofC180s*sizeof(float))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_VCMx ,          MaxNoofC180s*sizeof(float))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_VCMy ,          MaxNoofC180s*sizeof(float))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_VCMz ,          MaxNoofC180s*sizeof(float))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_area ,       MaxNoofC180s*sizeof(float))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_cell_div ,     MaxNoofC180s*sizeof(char))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_cell_Apo ,     MaxNoofC180s*sizeof(char))) return(-1); 
  if ( hipSuccess != hipMalloc( (void **)&d_C180_56,       92*7*sizeof(int))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_ran2 , 10000*sizeof(float))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_pressList, MaxNoofC180s*sizeof(float))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_resetIndices, MaxNoofC180s*sizeof(int))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_Youngs_mod, MaxNoofC180s*sizeof(float))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_Growth_rate, MaxNoofC180s*sizeof(float))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_CellINdex, MaxNoofC180s*sizeof(int))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_ScaleFactor, MaxNoofC180s*sizeof(float))) return(-1); 
  if ( hipSuccess != hipMalloc( (void **)&d_DivisionVolume, MaxNoofC180s*sizeof(float))) return(-1); 
  if ( hipSuccess != hipMalloc( (void **)&d_gamma_env, MaxNoofC180s*sizeof(float))) return(-1); 
  if ( hipSuccess != hipMalloc( (void **)&d_viscotic_damp, MaxNoofC180s*sizeof(float))) return(-1); 
  if ( hipSuccess != hipMalloc( (void **)&d_R0, 192*3*sizeof(float))) return(-1);
  if ( hipSuccess != hipMalloc((void **)&d_velListX, 192*MaxNoofC180s*sizeof(float))) return -1; 
  if ( hipSuccess != hipMalloc((void **)&d_velListY, 192*MaxNoofC180s*sizeof(float))) return -1; 
  if ( hipSuccess != hipMalloc((void **)&d_velListZ, 192*MaxNoofC180s*sizeof(float))) return -1;
  if ( hipSuccess != hipMalloc((void **)&d_ExtForces.x, 192*MaxNoofC180s*sizeof(float))) return -1;
  if ( hipSuccess != hipMalloc((void **)&d_ExtForces.y, 192*MaxNoofC180s*sizeof(float))) return -1;
  if ( hipSuccess != hipMalloc((void **)&d_ExtForces.z, 192*MaxNoofC180s*sizeof(float))) return -1;
  if ( hipSuccess != hipMalloc((void **)&d_fConList.x, 192*MaxNoofC180s*sizeof(float))) return -1;
  if ( hipSuccess != hipMalloc((void **)&d_fConList.y, 192*MaxNoofC180s*sizeof(float))) return -1;
  if ( hipSuccess != hipMalloc((void **)&d_fConList.z, 192*MaxNoofC180s*sizeof(float))) return -1;
  if ( hipSuccess != hipMalloc((void **)&d_fDisList.x, 192*MaxNoofC180s*sizeof(float))) return -1;
  if ( hipSuccess != hipMalloc((void **)&d_fDisList.y, 192*MaxNoofC180s*sizeof(float))) return -1;
  if ( hipSuccess != hipMalloc((void **)&d_fDisList.z, 192*MaxNoofC180s*sizeof(float))) return -1;
  if ( hipSuccess != hipMalloc((void **)&d_fRanList.x, 192*MaxNoofC180s*sizeof(float))) return -1;
  if ( hipSuccess != hipMalloc((void **)&d_fRanList.y, 192*MaxNoofC180s*sizeof(float))) return -1;
  if ( hipSuccess != hipMalloc((void **)&d_fRanList.z, 192*MaxNoofC180s*sizeof(float))) return -1;
  if ( hipSuccess != hipMalloc((void **)&d_DivPlane.x, MaxNoofC180s*sizeof(float))) return -1;
  if ( hipSuccess != hipMalloc((void **)&d_DivPlane.y, MaxNoofC180s*sizeof(float))) return -1;
  if ( hipSuccess != hipMalloc((void **)&d_DivPlane.z, MaxNoofC180s*sizeof(float))) return -1;
  if ( hipSuccess != hipMalloc((void **)&d_asym, MaxNoofC180s*sizeof(float))) return -1;
  if ( hipSuccess != hipMalloc((void **)&d_SysCx, 1024*sizeof(float))) return -1;
  if ( hipSuccess != hipMalloc((void **)&d_SysCy, 1024*sizeof(float))) return -1;
  if ( hipSuccess != hipMalloc((void **)&d_SysCz, 1024*sizeof(float))) return -1;



  hipMemset(d_C180_nn, 0, 3*192*sizeof(int));
  hipMemset(d_C180_sign, 0, 180*sizeof(int));
  CudaErrorCheck();

  hipMemset(d_X, 0, 192*MaxNoofC180s*sizeof(float));
  hipMemset(d_Y, 0, 192*MaxNoofC180s*sizeof(float));
  hipMemset(d_Z, 0, 192*MaxNoofC180s*sizeof(float));
  CudaErrorCheck();

  hipMemset(d_CMx, 0, MaxNoofC180s*sizeof(float));
  hipMemset(d_CMy, 0, MaxNoofC180s*sizeof(float));
  hipMemset(d_CMz, 0, MaxNoofC180s*sizeof(float));
  hipMemset(d_VCMx, 0, MaxNoofC180s*sizeof(float));
  hipMemset(d_VCMy, 0, MaxNoofC180s*sizeof(float));
  hipMemset(d_VCMz, 0, MaxNoofC180s*sizeof(float));
  CudaErrorCheck();

  hipMemset(d_R0, 0, 3*192*sizeof(float));
  hipMemset(d_pressList, 0, MaxNoofC180s*sizeof(float));
  hipMemset(d_Youngs_mod, 0, MaxNoofC180s*sizeof(float));
  hipMemset(d_Growth_rate, 0, MaxNoofC180s*sizeof(float));
  hipMemset(d_CellINdex, 0, MaxNoofC180s*sizeof(int));
  hipMemset(d_ScaleFactor, 0, MaxNoofC180s*sizeof(int));
  hipMemset(d_DivisionVolume, 0, MaxNoofC180s*sizeof(float));
  hipMemset(d_gamma_env, 0, MaxNoofC180s*sizeof(float));
  hipMemset(d_viscotic_damp, 0, MaxNoofC180s*sizeof(float));
  hipMemset(d_area, 0, MaxNoofC180s*sizeof(float));
  CudaErrorCheck();

  hipMemset(d_velListX, 0, 192*MaxNoofC180s*sizeof(float));
  hipMemset(d_velListY, 0, 192*MaxNoofC180s*sizeof(float));
  hipMemset(d_velListZ, 0, 192*MaxNoofC180s*sizeof(float));
  CudaErrorCheck();

  hipMemset(d_fConList.x, 0, 192*MaxNoofC180s*sizeof(float));
  hipMemset(d_fConList.y, 0, 192*MaxNoofC180s*sizeof(float));
  hipMemset(d_fConList.z, 0, 192*MaxNoofC180s*sizeof(float));
  CudaErrorCheck();
  
  hipMemset(d_fDisList.x, 0, 192*MaxNoofC180s*sizeof(float));
  hipMemset(d_fDisList.y, 0, 192*MaxNoofC180s*sizeof(float));
  hipMemset(d_fDisList.z, 0, 192*MaxNoofC180s*sizeof(float));
  CudaErrorCheck();
  hipMemset(d_fRanList.x, 0, 192*MaxNoofC180s*sizeof(float));
  hipMemset(d_fRanList.y, 0, 192*MaxNoofC180s*sizeof(float));
  hipMemset(d_fRanList.z, 0, 192*MaxNoofC180s*sizeof(float));
  CudaErrorCheck();
  hipMemset(d_ExtForces.x, 0, 192*MaxNoofC180s*sizeof(float));
  hipMemset(d_ExtForces.y, 0, 192*MaxNoofC180s*sizeof(float));
  hipMemset(d_ExtForces.z, 0, 192*MaxNoofC180s*sizeof(float));
  CudaErrorCheck();

  
  hipMemset(d_DivPlane.x, 0, MaxNoofC180s*sizeof(float));
  hipMemset(d_DivPlane.y, 0, MaxNoofC180s*sizeof(float));
  hipMemset(d_DivPlane.z, 0, MaxNoofC180s*sizeof(float));
  CudaErrorCheck();
  
  
  hipMemset(d_SysCx, 0, 1024*sizeof(float));
  hipMemset(d_SysCy, 0, 1024*sizeof(float));
  hipMemset(d_SysCz, 0, 1024*sizeof(float));
  CudaErrorCheck();
  
  
  //hipMemcpyToSymbol(HIP_SYMBOL(d_dt), &delta_t, sizeof(float),0, hipMemcpyHostToDevice);


  if (hipSuccess != hipMemcpy(d_R0, h_R0, 3*192*sizeof(float), hipMemcpyHostToDevice)) return -1; 

  //hipMemcpy(d_pressList, pressList, MaxNoofC180s*sizeof(float), hipMemcpyHostToDevice);
  


  if (colloidal_dynamics && rand_vel && !Restart ){
  	 
  	if ( initialize_Vel(Orig_No_of_C180s) != 0 ) return(-1);
  	hipMemcpy(d_velListX, velListX, 192*No_of_C180s*sizeof(float), hipMemcpyHostToDevice);
      	hipMemcpy(d_velListY, velListY, 192*No_of_C180s*sizeof(float), hipMemcpyHostToDevice);
  	hipMemcpy(d_velListZ, velListZ, 192*No_of_C180s*sizeof(float), hipMemcpyHostToDevice);	 
  	 
  }



  hipMemcpy(d_C180_nn,   C180_nn,   3*192*sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(d_C180_sign, C180_sign, 180*sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(d_C180_56,   C180_56,   7*92*sizeof(int),hipMemcpyHostToDevice);
  CudaErrorCheck();

  hipMemcpy(d_cell_div, cell_div, MaxNoofC180s*sizeof(char), hipMemcpyHostToDevice);
  CudaErrorCheck();
  hipMemcpy(d_cell_Apo, cell_Apo, MaxNoofC180s*sizeof(char), hipMemcpyHostToDevice);
  CudaErrorCheck();



if (Restart == 0) {	


  	for (int cell = 0; cell < MaxNoofC180s; cell++){
		if (colloidal_dynamics){
			pressList[cell] = 0;
		}else{
		 	pressList[cell] = minPressure; 
  		}
  	}
	
  	for (int i =  0; i < MaxNoofC180s; ++i){
       	if (colloidal_dynamics){
			Growth_rate[i] = 0;
		}else{
			Growth_rate[i] = rMax; 
  		}
  	}
	
	
  	for (int i =  0; i < MaxNoofC180s; ++i){
      		youngsModArray[i] = stiffness1; 
  	}
  
    	for (int i =  0; i < MaxNoofC180s; ++i){
      		DivisionVolume[i] = divVol; 
  	}
  	
  	for (int i =  0; i < MaxNoofC180s; ++i){
      		gamma_env[i] = gamma_visc; 
  	}
  	
  	for (int i =  0; i < MaxNoofC180s; ++i){
      		viscotic_damp[i] = viscotic_damping; 
  	}
  	
  
	if(!colloidal_dynamics && useDifferentCell) SecondCell(Orig_No_of_C180s);
  
 } // end of restart if else


  hipMemcpy(d_X,  X, 192*No_of_C180s*sizeof(float),hipMemcpyHostToDevice);
  hipMemcpy(d_Y,  Y, 192*No_of_C180s*sizeof(float),hipMemcpyHostToDevice);
  hipMemcpy(d_Z,  Z, 192*No_of_C180s*sizeof(float),hipMemcpyHostToDevice);
  CudaErrorCheck();
  hipMemcpy(d_velListX, velListX, 192*No_of_C180s*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_velListY, velListY, 192*No_of_C180s*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_velListZ, velListZ, 192*No_of_C180s*sizeof(float), hipMemcpyHostToDevice);
  CudaErrorCheck();
  hipMemcpy(d_Youngs_mod, youngsModArray, MaxNoofC180s*sizeof(float), hipMemcpyHostToDevice);
  CudaErrorCheck();
  hipMemcpy(d_Growth_rate, Growth_rate, MaxNoofC180s*sizeof(float), hipMemcpyHostToDevice);
  CudaErrorCheck();
  hipMemcpy(d_CellINdex, CellINdex, MaxNoofC180s*sizeof(int), hipMemcpyHostToDevice);
  CudaErrorCheck(); 
  hipMemcpy(d_pressList, pressList, No_of_C180s*sizeof(float), hipMemcpyHostToDevice);
  CudaErrorCheck();
  hipMemcpy(d_ScaleFactor, ScaleFactor, No_of_C180s*sizeof(float), hipMemcpyHostToDevice);
  CudaErrorCheck();
  hipMemcpy(d_DivisionVolume, DivisionVolume, No_of_C180s*sizeof(float), hipMemcpyHostToDevice);
  CudaErrorCheck();
  hipMemcpy(d_gamma_env, gamma_env, No_of_C180s*sizeof(float), hipMemcpyHostToDevice);
  CudaErrorCheck();
  hipMemcpy(d_viscotic_damp, viscotic_damp, No_of_C180s*sizeof(float), hipMemcpyHostToDevice);
  CudaErrorCheck();


/**************************************************************************************************************/


  // initialize device rng


    if (add_rands){
      hiprandGenerator_t gen;
      
      if (hipMalloc((void **)&d_rngStates, sizeof(hiprandState)*192*MaxNoofC180s) != hipSuccess){
          fprintf(stderr, "ERROR: Failed to allocate rng state memory in %s, at %d\n", __FILE__, __LINE__);
          return 1;
      }
          

      if (hipMalloc((void **)&d_seeds, sizeof(unsigned int)*192*MaxNoofC180s) != hipSuccess){
          fprintf(stderr, "ERROR: Failed to allocate rng seeds in %s, at %d\n", __FILE__, __LINE__);
          return 1;
      }
      
      time_t secs_since_1970;
      
      
      hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MT19937);
      CudaErrorCheck();

      hiprandSetPseudoRandomGeneratorSeed(gen, time(&secs_since_1970));
      CudaErrorCheck();

      if ( rand_seed <= 0 ) printf("   rand_seed = %lu\n",(unsigned long)secs_since_1970);

      if (rand_seed > 0){
          hiprandSetPseudoRandomGeneratorSeed(gen, rand_seed);
          CudaErrorCheck();
      }

      hiprandGenerate(gen, d_seeds, MaxNoofC180s*192);
      CudaErrorCheck();
  
      DeviceRandInit<<<(192*MaxNoofC180s)/256 + 1, 256>>>(d_rngStates, d_seeds, 192*MaxNoofC180s);
      CudaErrorCheck();
  }


  noofblocks      = No_of_C180s;
  threadsperblock = 192;
  printf("   no of blocks = %d, threadsperblock = %d, no of threads = %ld\n",
         noofblocks, threadsperblock, ((long) noofblocks)*((long) threadsperblock));


  
  globalrank = 0;


  // open trajectory file
  
   if (Restart == 0){
  	trajfile = fopen (trajFileName, "w");
  }else{
  	trajfile = fopen (trajFileName, "a+");
  }
  
  if ( trajfile == NULL)
  {
      printf("Failed to open %s \n", trajFileName);
      return -1;
  }

  FILE* forceFile;
  if (Restart == 0){
    	 forceFile = fopen(forces_file, "w");
  }else{
   	 forceFile = fopen(forces_file, "a");
  }
  

  FILE* velFile;
  if (Restart == 0){
    	 velFile = fopen("velocity.xyz", "w");
  }else{
   	 velFile = fopen("velocity.xyz", "a+");
  }	


#ifdef OUTPUT_ADP_ERROR
  FILE* timeFile = fopen("times", "w");
  FILE* errFile = fopen("errors", "w"); 
#endif 


  


  float rGrowth = 0;
  bool growthDone = false;
  
  
  // Setup simulation box, if needed (non-pbc)
  if (useRigidSimulationBox){
      
      printf("   Setup rigid (non-PBC) box...\n"); 
 
      BoxCen.x = (boxMax.x - BoxMin.x)/2;
      BoxCen.y = (boxMax.y - BoxMin.y)/2;
      BoxCen.z = (boxMax.z - BoxMin.z)/2;
 
      
      if ((boxMax.z - BoxMin.z) < divVol){
      	//DL = divVol; 
      	DL = divisionV;
      } else {
      	DL = 1.4;
      }
      
      
      Xdiv = ceil((boxMax.x - BoxMin.x)/DL);
      printf (" %d \n",Xdiv);
      Ydiv = ceil((boxMax.y - BoxMin.y)/DL);
      printf (" %d \n",Ydiv);
      Zdiv = ceil((boxMax.z - BoxMin.z)/DL);
      printf (" %d \n",Zdiv); 

      printf("   Done!\n");
      printf("   Simulation box minima:\n   X: %f, Y: %f, Z: %f\n", BoxMin.x, BoxMin.y, BoxMin.z);
      printf("   Simulation box maximum:\n   X: %f, Y: %f, Z: %f\n", boxMax.x, boxMax.y, boxMax.z);
  }


  // Code to set up pbc things
  if (usePBCs || useLEbc){
    printf("   Setup PBC box...\n"); 
    
    BoxMin.x = 0.0;
    BoxMin.y = 0.0;
    BoxMin.z = 0.0;
    
    BoxCen.x = (boxMax.x - BoxMin.x)/2;
    BoxCen.y = (boxMax.y - BoxMin.y)/2;
    BoxCen.z = (boxMax.z - BoxMin.z)/2;
  

    if ((boxMax.z - BoxMin.z) < divVol){
      	//DL = divVol;
      	DL = divisionV; 
    } else {
      	DL = 1.4;
    }
    
    Xdiv = ceil((boxMax.x - BoxMin.x)/DL);
    DLp.x = (boxMax.x - BoxMin.x)/Xdiv;
    printf (" %d \n",Xdiv);
    Ydiv = ceil((boxMax.y - BoxMin.y)/DL);
    DLp.y = (boxMax.y - BoxMin.y)/Ydiv;
    printf (" %d \n",Ydiv);
    Zdiv = ceil((boxMax.z - BoxMin.y)/DL);
    DLp.z = (boxMax.z - BoxMin.y)/Zdiv;
    printf (" %d \n",Zdiv);  
    
    printf (" %f \n",DLp.x);
    printf (" %f \n",DLp.y);
    printf (" %f \n",DLp.z);

    printf("   Done!\n");
    printf("   Simulation box minima:\n   X: %f, Y: %f, Z: %f\n", BoxMin.x, BoxMin.y, BoxMin.z);
    printf("   Simulation box maximum:\n   X: %f, Y: %f, Z: %f\n", boxMax.x, boxMax.y, boxMax.z);
  }

  if (correct_com == true && !Restart){
  
      CenterOfMass<<<No_of_C180s,256>>>(No_of_C180s,
                                        d_X, d_Y, d_Z,
                                        d_CMx, d_CMy, d_CMz);

      CudaErrorCheck();
     
     
      reductionblocks = (No_of_C180s-1)/1024+1;
      SysCMpost<<<reductionblocks,1024>>> ( No_of_C180s, d_CMx, d_CMy, d_CMz, 
			   		      d_SysCx, d_SysCy, d_SysCz);
      CudaErrorCheck(); 


      SysCM<<<1,1024>>> (No_of_C180s, reductionblocks,
        		    d_SysCx, d_SysCy, d_SysCz,
			    d_sysCM);
      
         
      CudaErrorCheck();
      
      CorrectCoMMotion<<<(No_of_C180s*192)/1024 + 1, 1024>>>(d_X, d_Y, d_Z,
                                                             d_sysCM,BoxCen,
                                                             No_of_C180s*192);
      CudaErrorCheck();
      

      
  }
  

  if ( correct_Vcom == true && !Restart){
  
     
      VelocityCenterOfMass<<<No_of_C180s,256>>>(No_of_C180s,
                                        	  d_velListX, d_velListY, d_velListZ,
                                        	  d_VCMx, d_VCMy, d_VCMz);
      CudaErrorCheck();
      
      reductionblocks = (No_of_C180s-1)/1024+1;
      SysCMpost<<<reductionblocks,1024>>> ( No_of_C180s, d_VCMx, d_VCMy, d_VCMz, 
			   			d_SysCx, d_SysCy, d_SysCz);
      CudaErrorCheck(); 


      SysCM<<<1,1024>>> (No_of_C180s, reductionblocks,
        		    d_SysCx, d_SysCy, d_SysCz,
			    d_sysVCM);
      
     CudaErrorCheck();
      
      
      CorrectCoMVelocity<<<(No_of_C180s*192)/1024 + 1, 1024>>>(d_velListX, d_velListY, d_velListZ,
                                                               d_sysVCM,
                                                               No_of_C180s*192);
          
      CudaErrorCheck(); 
  }  
  


  CenterOfMass<<<No_of_C180s,256>>>(No_of_C180s, d_X, d_Y, d_Z, d_CMx, d_CMy, d_CMz);
  //DL = divVol; 
  CudaErrorCheck(); 

  if ( hipSuccess != hipMalloc( (void **)&d_NNlist ,    Xdiv*Ydiv*Zdiv*64*sizeof(int))) return(-1); 

  thrust::device_vector<int> d_NoofNNlistV(Xdiv*Ydiv*Zdiv);
  thrust::fill(d_NoofNNlistV.begin(), d_NoofNNlistV.end(), 0);
  int *d_NoofNNlist = thrust::raw_pointer_cast(&d_NoofNNlistV[0]);


  // Better way to see how much GPU memory is being used.
  size_t totalGPUMem;
  size_t freeGPUMem;

  if ( hipSuccess != hipMemGetInfo ( &freeGPUMem, &totalGPUMem ) ) {
      printf("Couldn't read GPU Memory status\nExiting...\n");
      CudaErrorCheck();
      exit(1);
  }

  GPUMemory = totalGPUMem - freeGPUMem;

 // Precalculate random plane
  initialize_Plane(MaxNoofC180s);


   if(usePBCs ){
        
       CoorUpdatePBC <<<No_of_C180s, threadsperblock>>> (d_X, d_Y, d_Z,
                                                          d_CMx, d_CMy, d_CMz,
                                                          boxMax, divVol, No_of_C180s,
                                                          useRigidBoxZ, useRigidBoxY, impurityNum);
  


       CudaErrorCheck();
   }
   if(useLEbc){
        
       UpdateLEbc <<<No_of_C180s, threadsperblock>>> (d_X, d_Y, d_Z,
                       				 d_velListX, d_velListY, d_velListZ, d_CMx, d_CMy, d_CMz,
                       				 boxMax, divVol, No_of_C180s, Pshift, Vshift, useRigidBoxZ, impurityNum);
                        
        CudaErrorCheck();	
	
    }





      if (useRigidSimulationBox){	
      		makeNNlist<<<No_of_C180s/512+1,512>>>( No_of_C180s, d_CMx, d_CMy, d_CMz,
        	Xdiv, Ydiv, Zdiv, BoxMin, d_NoofNNlist, d_NNlist, DL);
        
        	CudaErrorCheck(); 
       }
	if(usePBCs){
       	makeNNlistPBC<<<No_of_C180s/512+1,512>>>( No_of_C180s, d_CMx, d_CMy, d_CMz,
        	attraction_range, Xdiv, Ydiv, Zdiv, boxMax, d_NoofNNlist, d_NNlist, DLp, useRigidBoxZ,useRigidBoxY);
        
        	CudaErrorCheck(); 
       }
       if(useLEbc){
       	makeNNlistLEbc<<<No_of_C180s/512+1,512>>>( No_of_C180s, d_CMx, d_CMy, d_CMz,
        	attraction_range, Xdiv, Ydiv, Zdiv, boxMax, d_NoofNNlist, d_NNlist, DLp, Pshift, useRigidBoxZ);
        	
        	CudaErrorCheck();
       
       } 
  



  if (constrainAngles){


      d_theta0V = h_theta0; 
      CudaErrorCheck(); 
  }


  float r_CM_o = pow((3.0/4.0) * (1/3.14159) * divVol*2.0, 1.0/3);



  printf("   Total amount of GPU memory used =    %8.2lf MB\n",GPUMemory/(1024*1024.0));
  printf("   Total amount of CPU memory used =    %8.2lf MB\n",CPUMemory/(1024*1024.0));

  // initial conditions

  // X[10] *= 1.1;

  // hipMemcpy(d_X, X, 192*No_of_C180s*sizeof(float), hipMemcpyHostToDevice);

  if (useRigidSimulationBox){	
  	CalculateConForce<<<No_of_C180s,threadsperblock>>>(   No_of_C180s, d_C180_nn, d_C180_sign,
                                                     	d_X,  d_Y,  d_Z,
                                                     	d_CMx, d_CMy, d_CMz,
                                                     	d_R0, d_ScaleFactor, d_pressList, d_Youngs_mod, 
                                                     	attraction_strength, attraction_range,
                                                     	repulsion_strength, repulsion_range,
                                                     	d_viscotic_damp,
                                                     	Xdiv, Ydiv, Zdiv, boxMax,
                                                     	d_NoofNNlist, d_NNlist, DL, d_gamma_env,
                                                     	threshDist,
								BoxMin, Youngs_mod,
                                                     	constrainAngles, d_theta0, d_fConList, d_ExtForces,
                                                     	impurityNum,f_range); 
                                                     	
       CudaErrorCheck();
                                                     	
      CalculateDisForce<<<No_of_C180s, threadsperblock>>>(No_of_C180s, d_C180_nn, d_C180_sign, 
                                                        	d_X, d_Y, d_Z,
                                                        	d_CMx, d_CMy, d_CMz,
                                                        	internal_damping,
                                                        	attraction_range,
                                                        	d_viscotic_damp,
                                                        	Xdiv, Ydiv, Zdiv,BoxMin,
                                                        	d_NoofNNlist, d_NNlist, DL, d_gamma_env,
                                                        	d_velListX, d_velListY, d_velListZ,
                                                        	d_fDisList,impurityNum,f_range);
                                                        
                                                        
       CudaErrorCheck();                                                  
  }
  
  if(usePBCs){
  
    	CalculateConForcePBC<<<No_of_C180s,threadsperblock>>>( No_of_C180s, d_C180_nn, d_C180_sign,
                                                     	d_X,  d_Y,  d_Z,
                                                     	d_CMx, d_CMy, d_CMz,
                                                     	d_R0,d_ScaleFactor, d_pressList, d_Youngs_mod , 
                                                     	attraction_strength, attraction_range,
                                                     	repulsion_strength, repulsion_range,
                                                     	d_viscotic_damp,
                                                     	Xdiv, Ydiv, Zdiv, boxMax,
                                                     	d_NoofNNlist, d_NNlist, DLp, d_gamma_env,
                                                     	threshDist,
                                                     	BoxMin, Youngs_mod,
                                                     	constrainAngles, d_theta0, d_fConList,
                                                     	useRigidBoxZ,useRigidBoxY,impurityNum,f_range);
                                                     	
       CudaErrorCheck();                                             	
  	
  	                                                     	
      CalculateDisForcePBC<<<No_of_C180s, threadsperblock>>>(No_of_C180s, d_C180_nn, d_C180_sign, 
                                                        d_X, d_Y, d_Z,
                                                        d_CMx, d_CMy, d_CMz,
                                                        internal_damping,
                                                        attraction_range,
                                                        d_viscotic_damp,
                                                        Xdiv, Ydiv, Zdiv, boxMax,
                                                        d_NoofNNlist, d_NNlist, DLp, d_gamma_env,
                                                        d_velListX, d_velListY, d_velListZ,
                                                        d_fDisList, useRigidBoxZ,useRigidBoxY,impurityNum,f_range);
    CudaErrorCheck();	
  
  }
  if(useLEbc){
  
  CalculateConForceLEbc<<<No_of_C180s,threadsperblock>>>(No_of_C180s, d_C180_nn, d_C180_sign,
                                                     	d_X,  d_Y,  d_Z,
                                                     	d_CMx, d_CMy, d_CMz,
                                                     	d_R0,d_ScaleFactor, d_pressList, d_Youngs_mod , 
                                                     	attraction_strength, attraction_range,
                                                     	repulsion_strength, repulsion_range,
                                                     	d_viscotic_damp,
                                                     	Xdiv, Ydiv, Zdiv, boxMax,
                                                     	d_NoofNNlist, d_NNlist, DLp, d_gamma_env,
                                                     	threshDist,
                                                     	BoxMin, Youngs_mod,
                                                     	constrainAngles, d_theta0, d_fConList,
                                                     	Pshift,useRigidBoxZ,impurityNum,f_range);
                                                     	
       CudaErrorCheck();                                             	
  	
  	                                                     	
      CalculateDisForceLEbc<<<No_of_C180s, threadsperblock>>>(No_of_C180s, d_C180_nn, d_C180_sign, 
                                                        	d_X, d_Y, d_Z,
                                                        	d_CMx, d_CMy, d_CMz,
                                                        	internal_damping,
                                                        	attraction_range,
                                                       	d_viscotic_damp,
                                                        	Xdiv, Ydiv, Zdiv, boxMax,
                                                        	d_NoofNNlist, d_NNlist, DLp, d_gamma_env,
                                                        	d_velListX, d_velListY, d_velListZ,
                                                        	d_fDisList, Pshift, Vshift, useRigidBoxZ,impurityNum,f_range);
    CudaErrorCheck();	
  
  
  
  }
  
  	  volumes<<<No_of_C180s,192>>>(No_of_C180s, d_C180_56,
                                     d_X, d_Y, d_Z,
                                     d_CMx , d_CMy, d_CMz,
                                     d_volume, d_cell_div, d_DivisionVolume,
                                     checkSphericity, d_area, 
                                     stiffness1, useDifferentCell, d_Youngs_mod,
                                     recalc_r0, ApoVol ,d_cell_Apo, d_ScaleFactor);
  

	
  int t = MaxNoofC180s;	
  if (Restart ==0){
  
  	if (binaryOutput){
  
      		
      		fwrite(&t, sizeof(int), 1, trajfile);
      
      		t = (int)useDifferentCell;
      		fwrite(&t, sizeof(int), 1, trajfile);
      
      		t = (Time_steps+equiStepCount+1) / trajWriteInt;
      		fwrite(&t, sizeof(int), 1, trajfile);
      
    
     		WriteBinaryTraj(0, trajfile, 1); 
  	} else {
      		fprintf(trajfile, "Header Start:\n");
      		fprintf(trajfile, "Maximum number of cells:\n%d\n", MaxNoofC180s);

      		fprintf(trajfile, "Using variable stiffness:\n");
      		if (useDifferentCell) 
          		fprintf(trajfile, "True\n");
      		else
          		fprintf(trajfile, "False\n");

      		fprintf(trajfile, "Maximum number of frames:\n%d\n", (Time_steps+equiStepCount+1) / trajWriteInt);
     	 	fprintf(trajfile, "Header End\n");
      		write_traj(0, trajfile);
  	}
  	if (write_cont_force){
  
      		fprintf(forceFile, "step,num_cells,cell_ind,node_ind,glob_node_ind,FX,FY,FZ,F,FX_ext,FY_ext,FZ_ext,F_ext,VX,VY,VZ,V,X,Y,Z,P,Vol,Area\n");
      
      		hipMemcpy(h_contactForces.x, d_fConList.x, 192*No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);
      		hipMemcpy(h_contactForces.y, d_fConList.y, 192*No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);
      		hipMemcpy(h_contactForces.z, d_fConList.z, 192*No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);
      		hipMemcpy(h_ExtForces.x, d_ExtForces.x, 192*No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);
      		hipMemcpy(h_ExtForces.y, d_ExtForces.y, 192*No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);
		hipMemcpy(h_ExtForces.z, d_ExtForces.z, 192*No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);
      		
      		hipMemcpy(velListX, d_velListX, 192*No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);
      		hipMemcpy(velListY, d_velListY, 192*No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);
      		hipMemcpy(velListZ, d_velListZ, 192*No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);

		hipMemcpy(pressList, d_pressList, No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);      
      		hipMemcpy(volume, d_volume, No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);
      		hipMemcpy(area, d_area, No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);	
      		writeForces(forceFile, 0, No_of_C180s);
  	}
  	if(write_vel_file){
  	          
               hipMemcpy(velListX, d_velListX, 192*No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);
               hipMemcpy(velListY, d_velListY, 192*No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);
               hipMemcpy(velListZ, d_velListZ, 192*No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);
               
               t = MaxNoofC180s;
               fwrite(&t, sizeof(int), 1, velFile);
      
      		t = (int)useDifferentCell;
      		fwrite(&t, sizeof(int), 1, velFile);
      
      		t = (Time_steps+equiStepCount+1) / trajWriteInt;
      		fwrite(&t, sizeof(int), 1, velFile);
               
               write_vel(0, velFile,1);
       }
  	
		  	
  	

  }

  //return 0;


  int numNodes = No_of_C180s*192;
  NewCellInd = No_of_C180s;
  WithoutApo = true;
  // Simulation loop
  for ( step = 1; step < Time_steps+1 + equiStepCount; step++)
  {
	Pshift = shear_rate*step*boxMax.x*delta_t;     



    
    //printf("step %d\n", step);
      numNodes = No_of_C180s*192;
      Integrate<<<No_of_C180s, threadsperblock>>>(d_X, d_Y, d_Z, 
                                                 d_velListX, d_velListY, d_velListZ, 
                                                 delta_t,  mass,
                                                 d_fConList, d_fDisList, d_fRanList,
                                                 No_of_C180s, impurityNum);
      CudaErrorCheck();


      //ForwardTime<<<No_of_C180s, threadsperblock>>>(d_XP, d_YP, d_ZP, 
      //                                             d_X , d_Y , d_Z ,
      //                                             No_of_C180s, impurityNum);
      //CudaErrorCheck();



// ------------------------------------------------------------------------------------------------

	if ( apoptosis && WithoutApo && No_of_C180s > popToStartApo && !colloidal_dynamics)
	{
	        printf("Cell apoptosis starts.\n");
		WithoutApo = false;
	}

	
	// ----------------------------------------- Begin Cell Death ------------	
	if (apoptosis && !WithoutApo) {	
     
		
      		float rans[1];
        	int DInd;
        	int Aporank;
      		
      		
      		if ((step)%1000 == 0){
      		
            		
            		hipMemcpy(Growth_rate, d_Growth_rate, No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);
            		CudaErrorCheck(); 
            		
                       hipMemcpy(CellINdex, d_CellINdex, No_of_C180s*sizeof(int), hipMemcpyDeviceToHost);
        	        CudaErrorCheck();
	 		      		
     		 	for (int s= impurityNum; s < No_of_C180s - impurityNum; s++){
      		 		  	 	
 	
      		 		if (CellINdex[s] >= 0){
      		 		
      		 			if (Growth_rate[s] == - squeeze_rate) continue; 		
      		 		
        	        		ranmar(rans, 1);
        	        		if (rans[0] < Apo_rate*0.1){
        	        			Growth_rate[s] = - squeeze_rate;
        	        			NumApoCell ++;
        	        		}
        	        	
        	        	} else {
        	        		
        	        	       	
        	        	
        	        	       if (Growth_rate[DInd] == - squeeze_rate2 || !useDifferentCell) continue;	
      		 			
        	        		ranmar(rans, 1);
        	        		if(rans[0] < Apo_rate2*0.1){
        	        			Growth_rate[DInd] = - squeeze_rate2;
        	        			NumApoCell ++;	
        	        		}
        	        	
        	        	
        	        	
        	        	
        	        	}			
        	        	
        	        }

        	        
        	        hipMemcpy(d_Growth_rate, Growth_rate, No_of_C180s*sizeof(float), hipMemcpyHostToDevice);
            		CudaErrorCheck();      	        
        	        
			count_and_die();
			
			if (num_cell_Apo> 0){
			
			        hipMemcpy(X , d_X, No_of_C180s*192*sizeof(float), hipMemcpyDeviceToHost);
  				CudaErrorCheck();
  				hipMemcpy(Y , d_Y, No_of_C180s*192*sizeof(float), hipMemcpyDeviceToHost);
  				CudaErrorCheck();
  				hipMemcpy(Z , d_Z, No_of_C180s*192*sizeof(float), hipMemcpyDeviceToHost);
  				CudaErrorCheck();
  				hipMemcpy(velListX , d_velListX , No_of_C180s*192*sizeof(float), hipMemcpyDeviceToHost);
  				CudaErrorCheck();
  				hipMemcpy(velListY , d_velListY , No_of_C180s*192*sizeof(float), hipMemcpyDeviceToHost);
  				CudaErrorCheck();
  				hipMemcpy(velListZ , d_velListZ , No_of_C180s*192*sizeof(float), hipMemcpyDeviceToHost);
  				CudaErrorCheck();
        	               hipMemcpy(pressList, d_pressList, No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);
        	               CudaErrorCheck();
        	               hipMemcpy(youngsModArray, d_Youngs_mod ,No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);
  				CudaErrorCheck();
  				hipMemcpy(Growth_rate, d_Growth_rate, No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);
        	               CudaErrorCheck();			
			        hipMemcpy(ScaleFactor, d_ScaleFactor, sizeof(float)*No_of_C180s, hipMemcpyDeviceToHost);
            			CudaErrorCheck();
            			hipMemcpy(DivisionVolume, d_DivisionVolume, sizeof(float)*No_of_C180s, hipMemcpyDeviceToHost);
            			CudaErrorCheck();
            			hipMemcpy(gamma_env, d_gamma_env, sizeof(float)*No_of_C180s, hipMemcpyDeviceToHost);
            			CudaErrorCheck();
            			hipMemcpy(viscotic_damp, d_viscotic_damp, sizeof(float)*No_of_C180s, hipMemcpyDeviceToHost);
           			CudaErrorCheck();
			        hipMemcpy(CellINdex, d_CellINdex, No_of_C180s*sizeof(int), hipMemcpyDeviceToHost);
        	               CudaErrorCheck();

				size_t EndShift;
				for (int ApoCell = 0; ApoCell < num_cell_Apo; ApoCell++) {
          			
          			
          				Aporank = cell_Apo_inds[ApoCell] - ApoCell; 
          				EndShift =  (No_of_C180s - Aporank-1)*192*sizeof(float); 


  				
  					hipMemcpy(X + 192*Aporank,  d_X + 192*(Aporank + 1), EndShift ,hipMemcpyDeviceToHost);
  					CudaErrorCheck();
  					hipMemcpy(Y + 192*Aporank,  d_Y + 192*(Aporank + 1), EndShift ,hipMemcpyDeviceToHost);
  					CudaErrorCheck();
  					hipMemcpy(Z + 192*Aporank,  d_Z + 192*(Aporank + 1), EndShift ,hipMemcpyDeviceToHost);
  					CudaErrorCheck();	                  	                  
   	     	               	hipMemcpy(velListX + 192*Aporank, d_velListX + 192*(Aporank + 1), EndShift , hipMemcpyDeviceToHost);
               			CudaErrorCheck();
               			hipMemcpy(velListY + 192*Aporank, d_velListY + 192*(Aporank + 1), EndShift , hipMemcpyDeviceToHost);
               			CudaErrorCheck();
               			hipMemcpy(velListZ + 192*Aporank, d_velListZ + 192*(Aporank + 1), EndShift , hipMemcpyDeviceToHost);
        	               	CudaErrorCheck();	                  
        	               	hipMemcpy(pressList + Aporank, d_pressList + Aporank+1, (MaxNoofC180s - Aporank - 1)*sizeof(float), hipMemcpyDeviceToHost);
        	               	CudaErrorCheck();
        	               	hipMemcpy(Growth_rate + Aporank, d_Growth_rate + Aporank+1, (MaxNoofC180s - Aporank - 1)*sizeof(float), hipMemcpyDeviceToHost);
        	               	CudaErrorCheck();
        	               	hipMemcpy(youngsModArray + Aporank, d_Youngs_mod + Aporank+1, (MaxNoofC180s - Aporank - 1)*sizeof(float), hipMemcpyDeviceToHost);
  					CudaErrorCheck();
        	               	hipMemcpy(ScaleFactor + Aporank, d_ScaleFactor + Aporank+1, (MaxNoofC180s - Aporank - 1)*sizeof(float), hipMemcpyDeviceToHost);
  					CudaErrorCheck();
        	               	hipMemcpy(DivisionVolume + Aporank, d_DivisionVolume + Aporank+1, (MaxNoofC180s - Aporank - 1)*sizeof(float), hipMemcpyDeviceToHost);
  					CudaErrorCheck();  					  					
        	               	hipMemcpy(gamma_env + Aporank, d_gamma_env + Aporank+1, (MaxNoofC180s - Aporank - 1)*sizeof(float), hipMemcpyDeviceToHost);
  					CudaErrorCheck();  					
        	               	hipMemcpy(viscotic_damp + Aporank, d_viscotic_damp + Aporank+1, (MaxNoofC180s - Aporank - 1)*sizeof(float), hipMemcpyDeviceToHost);
  					CudaErrorCheck();			
        	               	hipMemcpy(CellINdex + Aporank, d_CellINdex + Aporank+1, (MaxNoofC180s - Aporank - 1)*sizeof(int), hipMemcpyDeviceToHost);
        	               	CudaErrorCheck();

      				
      					-- No_of_C180s;
      				
      							
      					hipMemcpy(d_X , X, No_of_C180s*192*sizeof(float), hipMemcpyHostToDevice);
  					CudaErrorCheck();
  					hipMemcpy(d_Y , Y, No_of_C180s*192*sizeof(float), hipMemcpyHostToDevice);
  					CudaErrorCheck();
  					hipMemcpy(d_Z , Z, No_of_C180s*192*sizeof(float), hipMemcpyHostToDevice);
  					CudaErrorCheck();
  					hipMemcpy(d_velListX , velListX , No_of_C180s*192*sizeof(float), hipMemcpyHostToDevice);
  					CudaErrorCheck();
  					hipMemcpy(d_velListY , velListY , No_of_C180s*192*sizeof(float), hipMemcpyHostToDevice);
  					CudaErrorCheck();
  					hipMemcpy(d_velListZ , velListZ , No_of_C180s*192*sizeof(float), hipMemcpyHostToDevice);
  					CudaErrorCheck();
        	               	hipMemcpy(d_pressList, pressList, No_of_C180s*sizeof(float), hipMemcpyHostToDevice);
        	               	CudaErrorCheck();
  					hipMemcpy(d_Growth_rate, Growth_rate, No_of_C180s*sizeof(float), hipMemcpyHostToDevice);
        	               	CudaErrorCheck();
        	               	hipMemcpy(d_Youngs_mod, youngsModArray ,No_of_C180s*sizeof(float), hipMemcpyHostToDevice );
  					CudaErrorCheck();
        	               	hipMemcpy(d_ScaleFactor, ScaleFactor ,No_of_C180s*sizeof(float), hipMemcpyHostToDevice );
  					CudaErrorCheck();  					
        	               	hipMemcpy(d_DivisionVolume, DivisionVolume ,No_of_C180s*sizeof(float), hipMemcpyHostToDevice );
  					CudaErrorCheck();  					
        	               	hipMemcpy(d_gamma_env, gamma_env ,No_of_C180s*sizeof(float), hipMemcpyHostToDevice );
  					CudaErrorCheck();  					
        	               	hipMemcpy(d_viscotic_damp, viscotic_damp ,No_of_C180s*sizeof(float), hipMemcpyHostToDevice );
  					CudaErrorCheck();
        	               	hipMemcpy(d_CellINdex, CellINdex, No_of_C180s*sizeof(int), hipMemcpyHostToDevice);
        	               	CudaErrorCheck();  				

      					
      					NumRemoveCell ++;

      				}
      			}       
      		}	

	}
// ----------------------------------------- End Cell Death --------------



      CenterOfMass<<<No_of_C180s,256>>>(No_of_C180s, d_X, d_Y, d_Z, d_CMx, d_CMy, d_CMz);
      //DL = divVol; 
      CudaErrorCheck(); 

      hipMemset(d_NoofNNlist, 0, Xdiv*Ydiv*Zdiv*sizeof(int));

      if (useRigidSimulationBox){	
      		makeNNlist<<<No_of_C180s/512+1,512>>>( No_of_C180s, d_CMx, d_CMy, d_CMz,
        	Xdiv, Ydiv, Zdiv, BoxMin, d_NoofNNlist, d_NNlist, DL);
        
        	CudaErrorCheck(); 
       }
	if(usePBCs){
       	makeNNlistPBC<<<No_of_C180s/512+1,512>>>( No_of_C180s, d_CMx, d_CMy, d_CMz,
        	attraction_range, Xdiv, Ydiv, Zdiv, boxMax, d_NoofNNlist, d_NNlist, DLp, useRigidBoxZ,useRigidBoxY);
        
        	CudaErrorCheck(); 
       }
       if(useLEbc){
       	makeNNlistLEbc<<<No_of_C180s/512+1,512>>>( No_of_C180s, d_CMx, d_CMy, d_CMz,
        	attraction_range, Xdiv, Ydiv, Zdiv, boxMax, d_NoofNNlist, d_NNlist, DLp, Pshift, useRigidBoxZ);
        	
        	CudaErrorCheck();
       
       }


// ---------------------------------------------------------------------------------------------------




      // save previous step forces in g
      
      if (doPopModel == 1){
            rGrowth = rMax * (1 - (No_of_C180s*1.0/maxPop));
            // dr = -rGrowth(a + b*rGrowth)
            // rGrowth += dr * delta_t ;
            // dN/dT = N*R
            // dR/dT = -R(a+bR)
            // 
            if (rGrowth < 0) rGrowth =0; 
      }
      else {
      		rGrowth = rMax;
      }

      if (!colloidal_dynamics){      
      	PressureUpdate <<<No_of_C180s/1024 + 1, 1024>>> (d_pressList, maxPressure, d_Growth_rate, No_of_C180s,
        	                                           d_Youngs_mod, impurityNum);
      		CudaErrorCheck(); 
      }
      
      if ( (step)%1000 == 0)
      {
          printf("   time %-8d %d cells, CellInApoptosis %d, NumCellDeath %d\n",step,No_of_C180s, NumApoCell, NumRemoveCell);
      }


#ifdef FORCE_DEBUG
      printf("time %d  pressure = %f\n", step, Pressure);
#endif

 if (useRigidSimulationBox){	
  	CalculateConForce<<<No_of_C180s,threadsperblock>>>(   No_of_C180s, d_C180_nn, d_C180_sign,
                                                     	d_X,  d_Y,  d_Z,
                                                     	d_CMx, d_CMy, d_CMz,
                                                     	d_R0, d_ScaleFactor, d_pressList, d_Youngs_mod, 
                                                     	attraction_strength, attraction_range,
                                                     	repulsion_strength, repulsion_range,
                                                     	d_viscotic_damp,
                                                     	Xdiv, Ydiv, Zdiv, boxMax,
                                                     	d_NoofNNlist, d_NNlist, DL, d_gamma_env,
                                                     	threshDist,
								BoxMin, Youngs_mod,
                                                     	constrainAngles, d_theta0, d_fConList, d_ExtForces,
                                                     	impurityNum,f_range); 
                                                     	
       CudaErrorCheck();
                                                     	
      CalculateDisForce<<<No_of_C180s, threadsperblock>>>(No_of_C180s, d_C180_nn, d_C180_sign, 
                                                        	d_X, d_Y, d_Z,
                                                        	d_CMx, d_CMy, d_CMz,
                                                        	internal_damping,
                                                        	attraction_range,
                                                        	d_viscotic_damp,
                                                        	Xdiv, Ydiv, Zdiv,BoxMin,
                                                        	d_NoofNNlist, d_NNlist, DL, d_gamma_env,
                                                        	d_velListX, d_velListY, d_velListZ,
                                                        	d_fDisList,impurityNum,f_range);
                                                        
                                                        
       CudaErrorCheck();                                                  
  }
  
  if(usePBCs){
  
    	CalculateConForcePBC<<<No_of_C180s,threadsperblock>>>( No_of_C180s, d_C180_nn, d_C180_sign,
                                                     	d_X,  d_Y,  d_Z,
                                                     	d_CMx, d_CMy, d_CMz,
                                                     	d_R0,d_ScaleFactor, d_pressList, d_Youngs_mod , 
                                                     	attraction_strength, attraction_range,
                                                     	repulsion_strength, repulsion_range,
                                                     	d_viscotic_damp,
                                                     	Xdiv, Ydiv, Zdiv, boxMax,
                                                     	d_NoofNNlist, d_NNlist, DLp, d_gamma_env,
                                                     	threshDist,
                                                     	BoxMin, Youngs_mod,
                                                     	constrainAngles, d_theta0, d_fConList,
                                                     	useRigidBoxZ,useRigidBoxY,impurityNum,f_range);
                                                     	
       CudaErrorCheck();                                             	
  	
  	                                                     	
      CalculateDisForcePBC<<<No_of_C180s, threadsperblock>>>(No_of_C180s, d_C180_nn, d_C180_sign, 
                                                        d_X, d_Y, d_Z,
                                                        d_CMx, d_CMy, d_CMz,
                                                        internal_damping,
                                                        attraction_range,
                                                        d_viscotic_damp,
                                                        Xdiv, Ydiv, Zdiv,boxMax,
                                                        d_NoofNNlist, d_NNlist, DLp, d_gamma_env,
                                                        d_velListX, d_velListY, d_velListZ,
                                                        d_fDisList, useRigidBoxZ,useRigidBoxY,impurityNum,f_range);
    CudaErrorCheck();	
  
  }
  if(useLEbc){
  
  CalculateConForceLEbc<<<No_of_C180s,threadsperblock>>>( No_of_C180s, d_C180_nn, d_C180_sign,
                                                     	d_X,  d_Y,  d_Z,
                                                     	d_CMx, d_CMy, d_CMz,
                                                     	d_R0,d_ScaleFactor, d_pressList, d_Youngs_mod , 
                                                     	attraction_strength, attraction_range,
                                                     	repulsion_strength, repulsion_range,
                                                     	d_viscotic_damp,
                                                     	Xdiv, Ydiv, Zdiv, boxMax,
                                                     	d_NoofNNlist, d_NNlist, DLp, d_gamma_env,
                                                     	threshDist,
                                                     	BoxMin, Youngs_mod,
                                                     	constrainAngles, d_theta0, d_fConList,
                                                     	Pshift,useRigidBoxZ,impurityNum,f_range);
                                                     	
       CudaErrorCheck();                                             	
  	
  	                                                     	
      CalculateDisForceLEbc<<<No_of_C180s, threadsperblock>>>(No_of_C180s, d_C180_nn, d_C180_sign, 
                                                        d_X, d_Y, d_Z,
                                                        d_CMx, d_CMy, d_CMz,
                                                        internal_damping,
                                                        attraction_range,
                                                        d_viscotic_damp,
                                                        Xdiv, Ydiv, Zdiv,boxMax,
                                                        d_NoofNNlist, d_NNlist, DLp, d_gamma_env,
                                                        d_velListX, d_velListY, d_velListZ,
                                                        d_fDisList, Pshift, Vshift, useRigidBoxZ,impurityNum,f_range);
    CudaErrorCheck();	
  
  
  
  }
       

      // Calculate random Force here...
      if (add_rands){
          CalculateRanForce<<<No_of_C180s, threadsperblock>>>(No_of_C180s, d_rngStates, rand_scale_factor,
                                                              d_fRanList, impurityNum);
          CudaErrorCheck();
      }
      
      VelocityUpdateA<<<No_of_C180s, threadsperblock>>>(d_velListX, d_velListY, d_velListZ,
                                                        d_fConList, d_fRanList, delta_t, numNodes, mass, impurityNum);
      CudaErrorCheck();


      // Dissipative velocity update part...
      for (int s = 0; s < 1; ++s){
      
          VelocityUpdateB<<<No_of_C180s, threadsperblock>>>(d_velListX, d_velListY, d_velListZ,
                                                           d_fDisList, delta_t, numNodes, mass, impurityNum);
          CudaErrorCheck();
          
          if (useRigidSimulationBox){	
                                                     	
     		CalculateDisForce<<<No_of_C180s, threadsperblock>>>(No_of_C180s, d_C180_nn, d_C180_sign, 
                                                        d_X, d_Y, d_Z,
                                                        d_CMx, d_CMy, d_CMz,
                                                        internal_damping,
                                                        attraction_range,
                                                        d_viscotic_damp,
                                                        Xdiv, Ydiv, Zdiv, BoxMin,
                                                        d_NoofNNlist, d_NNlist, DL, d_gamma_env,
                                                        d_velListX, d_velListY, d_velListZ,
                                                        d_fDisList, impurityNum,f_range);
                                                        
       	CudaErrorCheck();                                                  
  	}
  	if(usePBCs){              	
      		CalculateDisForcePBC<<<No_of_C180s, threadsperblock>>>(No_of_C180s, d_C180_nn, d_C180_sign, 
                	                                        d_X, d_Y, d_Z,
                	                                        d_CMx, d_CMy, d_CMz,
                	                                        internal_damping,
                	                                        attraction_range,
                	                                        d_viscotic_damp,
                	                                        Xdiv, Ydiv, Zdiv,boxMax,
                	                                        d_NoofNNlist, d_NNlist, DLp, d_gamma_env,
                	                                        d_velListX, d_velListY, d_velListZ,
                	                                        d_fDisList,useRigidBoxZ,useRigidBoxY, impurityNum,f_range);
    		CudaErrorCheck();	
  
  	}
  	if(useLEbc){
  	
  	  	                                                     	
      		CalculateDisForceLEbc<<<No_of_C180s, threadsperblock>>>(No_of_C180s, d_C180_nn, d_C180_sign, 
               	                                         d_X, d_Y, d_Z,
               	                                         d_CMx, d_CMy, d_CMz,
               	                                         internal_damping,
               	                                         attraction_range,
               	                                         d_viscotic_damp,
               	                                         Xdiv, Ydiv, Zdiv,boxMax,
               	                                         d_NoofNNlist, d_NNlist, DLp, d_gamma_env,
               	                                         d_velListX, d_velListY, d_velListZ,
               	                                         d_fDisList, Pshift, Vshift, useRigidBoxZ, impurityNum,f_range);
    CudaErrorCheck();
  	
  	}
  	
     }


      
      CenterOfMass<<<No_of_C180s,256>>>(No_of_C180s,d_X, d_Y, d_Z,
                                        d_CMx, d_CMy, d_CMz);

        CudaErrorCheck();



      if (step <= Time_steps && rGrowth > 0 && !colloidal_dynamics){
        // ------------------------------ Begin Cell Division ------------------------------------------------


        volumes<<<No_of_C180s,192>>>(No_of_C180s, d_C180_56,
                                     d_X, d_Y, d_Z,
                                     d_CMx , d_CMy, d_CMz,
                                     d_volume, d_cell_div, d_DivisionVolume,
                                     checkSphericity, d_area,
                                     stiffness1, useDifferentCell, d_Youngs_mod,
                                     recalc_r0,ApoVol,d_cell_Apo, d_ScaleFactor);
        CudaErrorCheck();



        count_and_get_div();

	//hipStream_t *streams = (hipStream_t *)malloc(num_cell_div*sizeof(hipStream_t));
	
	//for (int i = 0 ; i < num_cell_div; i++) hipStreamCreate(&streams[i]);
	//CudaErrorCheck();
	
        
        for (int divCell = 0; divCell < num_cell_div; divCell++) {
          
          globalrank = cell_div_inds[divCell];
   

          cell_division<<<1,256>>>(globalrank, 
                                   d_X, d_Y, d_Z, 
                                   d_CMx, d_CMy, d_CMz,
                                   d_velListX, d_velListY, d_velListZ,
                                   No_of_C180s, repulsion_range, d_asym,
                                   useDifferentCell, daughtSame,
                                   NewCellInd, stiffness1, rMax, divVol, gamma_visc, viscotic_damping,
                                   d_ScaleFactor, d_Youngs_mod, d_Growth_rate, d_DivisionVolume,
                                   d_gamma_env, d_viscotic_damp, d_CellINdex,
                                   d_DivPlane);
                                   
          CudaErrorCheck()
          
          resetIndices[divCell] = globalrank;
          resetIndices[divCell + num_cell_div] = No_of_C180s;
          
          
          ++No_of_C180s;
          ++NewCellInd;
          
          
          if (No_of_C180s > MaxNoofC180s){
          
              printf("ERROR: Population is %d, only allocated enough memory for %d\n",
                     No_of_C180s, MaxNoofC180s);
              printf("ERROR: Fatal error, crashing...\n");
              return -69;
          }
        
         
        }
        
        
        //for (int i = 0 ; i < num_cell_div; i++) hipStreamDestroy(streams[i]);
	//CudaErrorCheck();
        
        if (num_cell_div>0){
            
            hipMemcpy(d_resetIndices, resetIndices, 2*num_cell_div*sizeof(int), hipMemcpyHostToDevice);
            CudaErrorCheck(); 

            PressureReset <<<(2*num_cell_div)/512 + 1, 512>>> (d_resetIndices, d_pressList, minPressure, 2*num_cell_div); 
            CudaErrorCheck();	 		      	            

        }


	if (countCells) {

        	if (countOnlyInternal == 1){
        
         		CenterOfMass<<<No_of_C180s,256>>>(No_of_C180s,
               		                         d_X, d_Y, d_Z,
               	        	                 d_CMx, d_CMy, d_CMz);

      			CudaErrorCheck();
     
     
      			reductionblocks = (No_of_C180s-1)/1024+1;
      			SysCMpost<<<reductionblocks,1024>>> ( No_of_C180s, d_CMx, d_CMy, d_CMz, 
		    		   		          d_SysCx, d_SysCy, d_SysCz);
      			CudaErrorCheck(); 


      			SysCM<<<1,1024>>> (No_of_C180s, reductionblocks,
        		  		      d_SysCx, d_SysCy, d_SysCz,
					      d_sysCM);
      
         
      			CudaErrorCheck();
      			
      			hipMemcpy(h_sysCM.x, d_sysCM.x, sizeof(float), hipMemcpyHostToDevice);
      			hipMemcpy(h_sysCM.y, d_sysCM.y, sizeof(float), hipMemcpyHostToDevice);
      			hipMemcpy(h_sysCM.z, d_sysCM.z, sizeof(float), hipMemcpyHostToDevice);
                       CudaErrorCheck();

//			h_SCM = d_SCM;
        
        
          		num_cell_div -= num_cells_far();
          
        	}

        	num_new_cells_per_step[step-1] = num_cell_div;
        	
        	if (step%newCellCountInt == 0){
          		newcells = 0;
          		for (int i = 0; i < newCellCountInt; i++) {
            			newcells += num_new_cells_per_step[countOffset + i];
          		}
          		dividingCells[(step-1)/newCellCountInt] = newcells;
          		totalCells[(step-1)/newCellCountInt] = No_of_C180s - newcells;
          		// Need to make sure this is how MIs are even calculated
          		countOffset += newCellCountInt;
        	}
      
       }
      
      
#if defined(FORCE_DEBUG) || defined(PRINT_VOLUMES)
      
      if (checkSphericity){
          //hipMemcpy(volume, d_volume, No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);
          h_volume = d_volumeV; 
          hipMemcpy(area, d_area, No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);
          hipMemcpy(pressList, d_pressList, No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);
          printf("time: %d\n", step); 
          for (int i = 0; i < No_of_C180s; i++){
              printf ("Cell: %d, volume= %f, area=%f, psi=%f, p = %f" , i, h_volume[i], area[i],
                      4.835975862049408*pow(h_volume[i], 2.0/3.0)/area[i], pressList[i]);
          
              if (h_volume[i] > divVol)
                  printf(", I'm too big :(");
          
              printf("\n"); 
          }
      } else{
          h_volume = d_volumeV; 
          for (int i = 0; i < No_of_C180s; i++){
              printf ("Cell: %d, volume= %f", i, h_volume[i]); 
          
              if (h_volume[i] > divVol)
                  printf(", I'm too big :(");
          
              printf("\n"); 
          }
      }
#endif
      
      
        // --------------------------------------- End Cell Division -----------
      }


      CenterOfMass<<<No_of_C180s,256>>>(No_of_C180s, d_X, d_Y, d_Z, d_CMx, d_CMy, d_CMz);
      //DL = divVol; 
      CudaErrorCheck(); 



   if(usePBCs && (step)%2000 == 0){
        
            CoorUpdatePBC <<<No_of_C180s, threadsperblock>>> (d_X, d_Y, d_Z,
                                                              d_CMx, d_CMy, d_CMz,
                                                              boxMax, divVol, No_of_C180s,
                                                              useRigidBoxZ, useRigidBoxY, impurityNum);
  

		
            CudaErrorCheck();
        }
        
        if(useLEbc && step%1000 == 0){
       
            UpdateLEbc <<<No_of_C180s, threadsperblock>>> (d_X, d_Y, d_Z,
                        					d_velListX, d_velListY, d_velListZ, d_CMx, d_CMy, d_CMz,
                        					boxMax, divVol, No_of_C180s, Pshift, Vshift, useRigidBoxZ, impurityNum);
	
	
	}



      if (!growthDone && step > Time_steps+1){
          printf("Cell growth halted.\nProceeding with MD simulation without growth...\n");
          growthDone = true;
          
          if (useDifferentCell && !duringGrowth && !colloidal_dynamics){
              printf("Now making some cells softer...\n");
              int softCellCounter = 0;
              if (fractionOfCells > 0){
                  numberOfCells = roundf(fractionOfCells*No_of_C180s); 
              }

              printf("Will make %d cells softer\n", numberOfCells); 
              
              if (chooseRandomCellIndices){
                  float rnd[1];
                  //int* chosenIndices = (int*)malloc(numberOfCells, sizeof(int));
                  int chosenIndices[numberOfCells]; 
                  
                  for (int i = 0; i < numberOfCells; i++){
                      chosenIndices[i] = -1; 
                  }
                  
                  bool indexChosen = false;
                  int cellInd = -1;

                  printf("Make cells with indices "); 
                  
                  while (softCellCounter < numberOfCells){
                      ranmar(rnd, 1);
                      cellInd = roundf(rnd[0] * No_of_C180s);

                      for (int i = 0; i < softCellCounter; i++){
                          if (chosenIndices[i] == cellInd){
                              indexChosen = true;
                              break;
                          }
                      }

                      if (!indexChosen){
                          chosenIndices[softCellCounter] = cellInd;
                          softCellCounter++;
                          printf("%d ", cellInd); 
                      } else
                          indexChosen = false;
                      
                  }

                  //free(chosenIndices);

                  for (int i = 0; i < numberOfCells; i++){
                      youngsModArray[chosenIndices[i]] = Stiffness2; 
                  }
              }
              else {
                  // search for the oldest cells near the center of the system, and make them soft

      		   CenterOfMass<<<No_of_C180s,256>>>(No_of_C180s,
               	                         d_X, d_Y, d_Z,
                       	                 d_CMx, d_CMy, d_CMz);

      		   CudaErrorCheck();
     
     
      		   reductionblocks = (No_of_C180s-1)/1024+1;
      		   SysCMpost<<<reductionblocks,1024>>> ( No_of_C180s, d_CMx, d_CMy, d_CMz, 
		    		   		          d_SysCx, d_SysCy, d_SysCz);
      		   CudaErrorCheck(); 


      		   SysCM<<<1,1024>>> (No_of_C180s, reductionblocks,
        	  		      d_SysCx, d_SysCy, d_SysCz,
				      d_sysCM);
      
         
      		   CudaErrorCheck();


      		   hipMemcpy(h_sysCM.x, d_sysCM.x, sizeof(float), hipMemcpyHostToDevice);
      		   hipMemcpy(h_sysCM.y, d_sysCM.y, sizeof(float), hipMemcpyHostToDevice);
      		   hipMemcpy(h_sysCM.z, d_sysCM.z, sizeof(float), hipMemcpyHostToDevice);
                   CudaErrorCheck();


//		   h_SCM = d_SCM;	
                  
                  float Rmax2 = getRmax2();
                  float R2, dx, dy, dz;
                  int cellInd = 0; 


                  float f = 1 - closenessToCenter;
              
                  printf("Made cells with indices "); 

                  while (softCellCounter < numberOfCells && cellInd < No_of_C180s){
                      dx = CMx[cellInd] - *h_sysCM.x; 
                      dy = CMy[cellInd] - *h_sysCM.y; 
                      dz = CMz[cellInd] - *h_sysCM.z;

                      R2 = dx*dx + dy*dy + dz*dz;

                      if (R2 <= f*f*Rmax2){
                          printf("%d, ", cellInd); 
                          softCellCounter++; 
                          youngsModArray[cellInd] = Stiffness2; 

                      }
                      cellInd++; 
                  }
              }
              
              hipMemcpy(d_Youngs_mod, youngsModArray, No_of_C180s*sizeof(float), hipMemcpyHostToDevice);
              printf("\b\b softer\n"); 
          }

      }

      if (correct_com == true){


      		CenterOfMass<<<No_of_C180s,256>>>(No_of_C180s,
               	                         d_X, d_Y, d_Z,
                       	                 d_CMx, d_CMy, d_CMz);

      		CudaErrorCheck();
     
     
      		reductionblocks = (No_of_C180s-1)/1024+1;
      		SysCMpost<<<reductionblocks,1024>>> ( No_of_C180s, d_CMx, d_CMy, d_CMz, 
				   		      d_SysCx, d_SysCy, d_SysCz);
      		CudaErrorCheck(); 


      		SysCM<<<1,1024>>> (No_of_C180s, reductionblocks,
        			    d_SysCx, d_SysCy, d_SysCz,
				    d_sysCM);
      
         
      		CudaErrorCheck();
      
      		CorrectCoMMotion<<<(No_of_C180s*192)/1024 + 1, 1024>>>(d_X, d_Y, d_Z,
               	                                              d_sysCM, BoxCen,
               	                                              No_of_C180s*192);
      		CudaErrorCheck();
      
      		//hipMemcpy(h_sysCM.x, d_sysCM.x, sizeof(float), hipMemcpyHostToDevice);
      		//hipMemcpy(h_sysCM.y, d_sysCM.y, sizeof(float), hipMemcpyHostToDevice);
      		//hipMemcpy(h_sysCM.z, d_sysCM.z, sizeof(float), hipMemcpyHostToDevice);
                //CudaErrorCheck();
      		//printf("sysCMx = 	%f, sysCMy = 		%f, sysCmz = 		%f\n", h_sysCM.x, h_sysCM.y, h_sysCM.z);
      
      }

    if ( correct_Vcom == true ){
     
     
       VelocityCenterOfMass<<<No_of_C180s,256>>>(No_of_C180s,
                                          	    d_velListX, d_velListY, d_velListZ,
                                          	    d_VCMx, d_VCMy, d_VCMz);
                                          
        CudaErrorCheck();                        
                  
        reductionblocks = (No_of_C180s-1)/1024+1;
        
        SysCMpost<<<reductionblocks,1024>>> ( No_of_C180s, d_VCMx, d_VCMy, d_VCMz, 
			   			d_SysCx, d_SysCy, d_SysCz);
        CudaErrorCheck(); 


        SysCM<<<1,1024>>> (No_of_C180s, reductionblocks,
        		    d_SysCx, d_SysCy, d_SysCz,
			    d_sysVCM);
      
        CudaErrorCheck(); 

        CorrectCoMVelocity<<<(No_of_C180s*192)/1024 + 1, 1024>>>(d_velListX, d_velListY, d_velListZ,
                                                                 d_sysVCM,
                                                                 No_of_C180s*192);
          
        CudaErrorCheck(); 
        
        //h_SVCM = d_SVCM;
        //printf("sysVCMx = 	%f, sysVCMy = 		%f, sysVCmz = 		%f\n", h_SVCM[0].x, h_SVCM[0].y, h_SVCM[0].z);
        
 
    }


      if ( step%trajWriteInt == 0 )
      {
          //printf("   Writing trajectory to traj.xyz...\n");
          frameCount++; 
          hipMemcpy(X, d_X, 192*No_of_C180s*sizeof(float),hipMemcpyDeviceToHost);
          hipMemcpy(Y, d_Y, 192*No_of_C180s*sizeof(float),hipMemcpyDeviceToHost);
          hipMemcpy(Z, d_Z, 192*No_of_C180s*sizeof(float),hipMemcpyDeviceToHost);
          hipMemcpy(CellINdex, d_CellINdex, No_of_C180s*sizeof(int), hipMemcpyDeviceToHost);
          
          if (binaryOutput)
              WriteBinaryTraj(step + Laststep, trajfile, frameCount + Lastframe);
          else
              write_traj(step + Laststep, trajfile);

          if (write_cont_force == true){

              hipMemcpy(h_contactForces.x, d_fConList.x, 192*No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);
              hipMemcpy(h_contactForces.y, d_fConList.y, 192*No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);
              hipMemcpy(h_contactForces.z, d_fConList.z, 192*No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);
              hipMemcpy(h_ExtForces.x, d_ExtForces.x, 192*No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);
              hipMemcpy(h_ExtForces.y, d_ExtForces.y, 192*No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);
              hipMemcpy(h_ExtForces.z, d_ExtForces.z, 192*No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);
              
              hipMemcpy(velListX, d_velListX, 192*No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);
              hipMemcpy(velListY, d_velListY, 192*No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);
              hipMemcpy(velListZ, d_velListZ, 192*No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);
              
              hipMemcpy(X, d_X, 192*No_of_C180s, hipMemcpyDeviceToHost);
              hipMemcpy(Y, d_Y, 192*No_of_C180s, hipMemcpyDeviceToHost);
              hipMemcpy(Z, d_Z, 192*No_of_C180s, hipMemcpyDeviceToHost);
              
              hipMemcpy(pressList, d_pressList, No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);
              hipMemcpy(volume, d_volume, No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);
              hipMemcpy(area, d_area, No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);	
                    
              writeForces(forceFile, step + Laststep, No_of_C180s);
          }
          if(write_vel_file){
                         
               hipMemcpy(velListX, d_velListX, 192*No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);
               hipMemcpy(velListY, d_velListY, 192*No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);
               hipMemcpy(velListZ, d_velListZ, 192*No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);
          	write_vel(step + Laststep, velFile,frameCount + Lastframe);
          }  
      }

      myError = hipGetLastError();
      if ( hipSuccess != myError )
      {
          printf( "Error %d: %s!\n",myError,hipGetErrorString(myError) );return(-1);
      }
  }


  t = (Time_steps+equiStepCount+Laststep+1) / trajWriteInt; 
  
  if(write_vel_file){
       
       fclose(velFile);   
        
       velFile = fopen("velocity.xyz", "r+");
       fseek(velFile, 0, SEEK_SET);
       fwrite(&No_of_C180s, sizeof(int), 1, velFile);  
       fseek(velFile, 8, SEEK_SET);
       fwrite(&t, sizeof(int), 1, velFile);
       
  }
  
    if (binaryOutput){
  
      fclose(trajfile);
      
      trajfile = fopen (trajFileName, "r+");
      fseek(trajfile, 0, SEEK_SET);
      fwrite(&No_of_C180s, sizeof(int), 1, trajfile);    
      fseek(trajfile, 8, SEEK_SET);
      fwrite(&t, sizeof(int), 1, trajfile);
  }
  
  printf("Xdiv = %d, Ydiv = %d, Zdiv = %d\n", Xdiv, Ydiv, Zdiv );

  FILE* MitIndFile;
  std::fstream MitIndFile2;
  std::string datFileName = inpFile; 
  
  if (overWriteMitInd == 0){
      
      MitIndFile = fopen(mitIndFileName, "a");
      //MitIndFile2.open(datFileName, "a"); 
  }
  else{
      MitIndFile = fopen(mitIndFileName, "w");
      //MitIndFile2.open(datFileName, "w"); 
  }
  if (MitIndFile == NULL)
  {
      printf("Failed to open mit-index.dat\n");
      exit(1);
  }


  for (int i = 0; i < (Time_steps/newCellCountInt) + 1; i++)
  {
      if ( dividingCells[i]!=0 && totalCells[i]!=0 ){
          fprintf(MitIndFile, "%f\n", (float)dividingCells[i]/totalCells[i]);
          // totalCells is number of non-dividing cells
          
      }
      else {
          fprintf(MitIndFile, "%f\n", 0.0);

      }

  }
  
  if ( writeRestartFile(step + Laststep, frameCount + Lastframe) != 0 ){
 	printf("Unable to call Restart Kernel. \n");
	return(-1);
   }
 
 
 
  hipFree( (void *)d_X  );
  hipFree( (void *)d_Y  );
  hipFree( (void *)d_Z  );
  hipFree( (void *)d_CMx );
  hipFree( (void *)d_CMy );
  hipFree( (void *)d_CMz );
  hipFree( (void *)d_ran2 );

  hipFree( (void *)d_C180_nn);
  hipFree( (void *)d_C180_sign);
  hipFree( (void *)d_cell_div);
  hipFree( (void *)d_cell_Apo);
  
  hipFree( (void *)d_DivPlane.x);
  hipFree( (void *)d_DivPlane.y);
  hipFree( (void *)d_DivPlane.z);
  
  
  free(X); free(Y); free(Z);
  free(CMx); free(CMy); free(CMz);
  free(dividingCells); free(totalCells);
  free(NoofNNlist);
  free(NNlist);
  free(ran2);
  free(num_new_cells_per_step);
  free(cell_div_inds);
  free(cell_Apo_inds);
  free(pressList);

  free(velListX); 
  free(velListY); 
  free(velListZ); 
  
  free(DivPlane.x);
  free(DivPlane.y);
  free(DivPlane.z);

  fclose(trajfile);
  fclose(forceFile);
  fclose(velFile);
  fclose(MitIndFile);
#ifdef OUTPUT_ADP_ERROR
  fclose(timeFile);
  fclose(errFile);
#endif
  return(0);
}



int initialize_C180s(int Orig_No_of_C180s)
{
  int rank;
  int atom;
  float initx[181], inity[181], initz[181];
  FILE *infil;

  printf("      Initializing positions for %d fullerenes...\n", Orig_No_of_C180s);


  CPUMemory += 3L*192L*MaxNoofC180s*sizeof(float);
  CPUMemory += MaxNoofC180s*6L*sizeof(float);

  infil = fopen("C180","r");
  if ( infil == NULL ) {printf("Unable to open file C180\n");return(-1);}
  for ( atom = 0 ; atom < 180 ; ++atom)
  {
      if ( fscanf(infil,"%f %f %f",&initx[atom], &inity[atom], &initz[atom]) != 3 )
      {
          printf("   Unable to read file C180 on line %d\n",atom+1);
          fclose(infil);
          return(-1);
      }
  }
  fclose(infil);

  // first correct for the cells com

  float sumx = 0; 
  float sumy = 0; 
  float sumz = 0;
      
  for (int i =0; i < 180; ++i){
      sumx += initx[i]; 
      sumy += inity[i]; 
      sumz += initz[i]; 
  }

  sumx /= 180.0; 
  sumy /= 180.0; 
  sumz /= 180.0;

  // calculate initial cell volume

  
      
  for (int i =0; i < 180; ++i){
      initx[i] -= sumx; 
      inity[i] -= sumy; 
      initz[i] -= sumz; 
  }

  float rCheck = powf(0.75*(1.f/3.14159)*0.786, 1.f/3.f); // this code is magical
  printf("Check radius = %f\n", rCheck);
  float3 allCMs[Orig_No_of_C180s];

  float vol = 0;
  int k = 0;
      
  vol = (boxMax.x - BoxMin.x)*(boxMax.y - BoxMin.y)*(boxMax.z - BoxMin.z);
  k = floor(vol/0.786);
      
  if (k < Orig_No_of_C180s){
      fprintf(stderr, "ERROR: Simulation box is too small\n");
      fprintf(stderr, "       Big enough for %d\n", k);
      return 27;
  }

  printf("Can fit upto %d cells\n", k);

  int c = 0;
  float rands[3];
  float3 center = 0.5*boxMax;	
  float3 CM;
  float yoffset;
  yoffset = BoxMin.y + 1;
  if (LineCenter == 1) {
	yoffset = center.y; 
  }

  if (colloidal_dynamics){
  
  	
  	ShapeScaler (initx,inity,initz);
  
  	while (true){
  	
              ranmar(rands, 3);
              CM = make_float3(rands[0]*((boxMax.x - BoxMin.x) - 1.f)  + BoxMin.x + 1.f,
                                      rands[1]*((boxMax.y - BoxMin.y) - 1.f)  + BoxMin.y + 1.f,
                                      0.f);
              if (flatbox == 1){
                  CM.z = (boxMax.z - BoxMin.z)/2;
              } else {
                  CM.z = rands[2]*((boxMax.z - BoxMin.z) - 1.f)  + BoxMin.z + 1.f;
              }

	      	
              bool farEnough = true;
              
              
              farEnough = !(CM.x + ScaleFactor[c]*rCheck*shapeLim > boxMax.x || CM.x-ScaleFactor[c]*rCheck*shapeLim < BoxMin.x ||
                            CM.y+ScaleFactor[c]*rCheck*shapeLim > boxMax.y || CM.y-ScaleFactor[c]*rCheck*shapeLim < BoxMin.y ||
                            CM.z+ScaleFactor[c]*rCheck*shapeLim > boxMax.z || CM.z-ScaleFactor[c]*rCheck*shapeLim < BoxMin.z);
              
              
              for (int nInd = 0; nInd < c; ++nInd){
                  if (mag(allCMs[nInd] - CM) < 1.8*rCheck*shapeLim){
                      //(ScaleFactor[nInd]+ScaleFactor[c])
                      farEnough = false;
                      break;
                  }
              }
          
 
              if (farEnough){

                  allCMs[c] = CM; 
                  c++;
              }
          
              if (c == Orig_No_of_C180s){
		break;
              }
          }
  
  	  if(RandInitDir){	
  	  
  	  	float axis[3];
	  	float RMat[9];
	  	float theta[1];
	  	float tempS[3];
	  	float tempR[3];
	  	
  	  	for (int cellInd = 0; cellInd < Orig_No_of_C180s; cellInd++){
  	  
  	  		axis[0] = 0; 
          		axis[1] = 1; 
          		axis[2] = 0;	 	  
  	  		GetRandomVector(axis);	  
  	  
  	  		ranmar(theta,1);
  	  		theta[0] = theta[0]*2*3.14159265;
  	  
  	  		RotationMatrix(RMat,axis,theta);
  	     
  	        	for(int nodeInd = 0; nodeInd < 180; ++nodeInd){
                  		
                  		tempS[0] = ScaleFactor[cellInd]*initx[nodeInd];
                  		tempS[1] = ScaleFactor[cellInd]*inity[nodeInd];
                  		tempS[2] = ScaleFactor[cellInd]*initz[nodeInd];
                  		
                  		tempR[0] = RMat[0]*tempS[0] + RMat[1]*tempS[1] + RMat[2]*tempS[2];
                  		tempR[1] = RMat[3]*tempS[0] + RMat[4]*tempS[1] + RMat[5]*tempS[2];
                  		tempR[2] = RMat[6]*tempS[0] + RMat[7]*tempS[1] + RMat[8]*tempS[2];
                  		                  		
                		X[cellInd*192 + nodeInd] = tempR[0] + allCMs[cellInd].x;
                  		Y[cellInd*192 + nodeInd] = tempR[1] + allCMs[cellInd].y;
                  		Z[cellInd*192 + nodeInd] = tempR[2] + allCMs[cellInd].z;
  	     		}
  	   
  	   
  	   	}
  	  
  
  	 } else{
  	 
  	 	for (int cellInd = 0; cellInd < Orig_No_of_C180s; cellInd++){
  	     
  	     		for(int nodeInd = 0; nodeInd < 180; ++nodeInd){
                  
                  		X[cellInd*192 + nodeInd] = ScaleFactor[cellInd]*initx[nodeInd] + allCMs[cellInd].x;
                  		Y[cellInd*192 + nodeInd] = ScaleFactor[cellInd]*inity[nodeInd] + allCMs[cellInd].y;
                  		Z[cellInd*192 + nodeInd] = ScaleFactor[cellInd]*initz[nodeInd] + allCMs[cellInd].z;
  	     
  	     		}
  	   
  	   	}
  
  	}


  } else {

  	if (rand_pos){
          
		while (true){
        	      ranmar(rands, 3);
        	      CM = make_float3(rands[0]*((boxMax.x - BoxMin.x) - 1.f)  + BoxMin.x + 1.f,
                                      rands[1]*((boxMax.y - BoxMin.y) - 1.f)  + BoxMin.y + 1.f,
                                      0.f);
        	      if (flatbox == 1){
        	          CM.z = (boxMax.z - BoxMin.z)/2;
        	      } else {
        	          CM.z = rands[2]*((boxMax.z - BoxMin.z) - 1.f)  + BoxMin.z + 1.f;
        	      }

	      	
        	      bool farEnough = true;
              
              
        	      farEnough = !(CM.x+rCheck > boxMax.x || CM.x-rCheck < BoxMin.x ||
        	                    CM.y+rCheck > boxMax.y || CM.y-rCheck < BoxMin.y ||
        	                    CM.z+rCheck > boxMax.z || CM.z-rCheck < BoxMin.z);
              
              
        	      for (int nInd = 0; nInd < c; ++nInd){
        	          if (mag(allCMs[nInd] - CM) < 2*rCheck){
        	              farEnough = false;
        	              break;
        	          }
        	      }
          
 
        	      if (farEnough){
	
        	          allCMs[c] = CM; 
        	          c++;
        	      }
          
        	      if (c == Orig_No_of_C180s){
			break;
        	      }
        	  }
	
  	} else if ( line ){
	
			c = Orig_No_of_C180s-1;
				
			for ( rank = 0; rank < Orig_No_of_C180s - impurityNum ; rank++ )
        	        {
        	                         
        	   	        CM.x = L*rank + 0.5*L + BoxMin.x;
        	    	      	CM.y = yoffset;
        	    	      	CM.z = center.z;
				allCMs[c] = CM; 
        	    	  	c--;

        	   	}
           	
        	   	if (impurity){
        	   	
        	   		while (true){
        		      
				      ranmar(rands, 3);
        	      		      CM = make_float3(rands[0]*((boxMax.x - BoxMin.x) - 1.f)  + BoxMin.x + 1.f,
        	       	                       rands[1]*((boxMax.y - BoxMin.y) - 1.f)  + BoxMin.y + 1.f,
        	       	                       0.f);
        	      			if (flatbox == 1){
        	       		   CM.z = (boxMax.z - BoxMin.z)/2;
        	      			}else {
        	       	   		CM.z = rands[2]*((boxMax.z - BoxMin.z) - 1.f)  + BoxMin.z + 1.f;
        	      			}

	      		
        			      bool farEnough = true;
        	      
        			      farEnough = !(CM.x+rCheck > boxMax.x || CM.x-rCheck < BoxMin.x ||
        			                    CM.y+rCheck > boxMax.y || CM.y-rCheck < BoxMin.y ||
        			                    CM.z+rCheck > boxMax.z || CM.z-rCheck < BoxMin.z );
              	
        			      for (int nInd = Orig_No_of_C180s-1; nInd > c; --nInd){
        			          if (mag(allCMs[nInd] - CM) < 2*rCheck){
        			              farEnough = false;
        			              break;
        			          }
        			      }
          

				      if (farEnough){
	
        			          allCMs[c] = CM; 
        			          c--;
        			      }
        	  
        			      if (c == -1){
					break;
        			      }
        	       	}
				
			}
  	} else if (plane) {
  
  			rCheck *= 1.2;
  			c = Orig_No_of_C180s-1;
  			float l = 2.5;
			int Side = int (((boxMax.x - BoxMin.x) / l) + 0.1 );
			printf(" Max number of initial cells:  %d\n", Side*Side);
			
			for ( rank = 0; rank < Orig_No_of_C180s - impurityNum ; rank++ )
        	        {
                        
        	               ey=rank%Side;
        			ex=rank/Side;         
        	          	CM.x = l*ex + 0.5*l + BoxMin.x;
        	          	CM.y = l*ey + 0.5*l + BoxMin.x;
        	    	      	CM.z = BoxMin.z + 1 ;
				allCMs[c] = CM; 
        	    	  	c--;
        	   	}
  
  
  
  	} else {			
	
			rCheck *= 1.2;
			c = Orig_No_of_C180s-1;
		
        	 	for ( rank = 0; rank < Orig_No_of_C180s - impurityNum; ++rank )
        	 	{
        	 		 ey=rank%Side_length;
        			 ex=rank/Side_length;
        	          	 CM.x = L1*ex + 0.5*L1 + center.x;
        	          	 CM.y = L1*ey + 0.5*L1 + center.y;
        	          	 CM.z = center.z;
        	          	 allCMs[c] = CM;
        	          	 c--;

        	  	}  
	
			if (impurity){	
			
				while (true){
        		      
			      		ranmar(rands, 3);
        	      	      		CM = make_float3(rands[0]*((boxMax.x - BoxMin.x) - 1.f)  + BoxMin.x + 1.f,
        	                       	       rands[1]*((boxMax.y - BoxMin.y) - 1.f)  + BoxMin.y + 1.f,
        	                       	       0.f);
        	      			if (flatbox == 1){
        	       	   		CM.z = (boxMax.z - BoxMin.z)/2;
        	      			}else {
        	          			CM.z = rands[2]*((boxMax.z - BoxMin.z) - 1.f)  + BoxMin.z + 1.f;
        	      			}

	      		
        		      		bool farEnough = true;
              
        		      		farEnough = !(CM.x+rCheck > boxMax.x || CM.x-rCheck < BoxMin.x ||
        		               	     CM.y+rCheck > boxMax.y || CM.y-rCheck < BoxMin.y ||
        		               	     CM.z+rCheck > boxMax.z || CM.z-rCheck < BoxMin.z );
              
        		      		for (int nInd = Orig_No_of_C180s-1; nInd > c; --nInd){
        		          		if (mag(allCMs[nInd] - CM) < 2*rCheck){
        		              			farEnough = false;
        		              			break;
        		          		}
        		      		}
          

			      		if (farEnough){
	
        		          		allCMs[c] = CM; 
        		          		c--;
        		      		}
        	  
        		 	     if (c == -1){
					break;
        		 	     }
        	       	}
	  	 }
   	}

			 


   	for (int cellInd = 0; cellInd < Orig_No_of_C180s; cellInd++){
       	for(int nodeInd = 0; nodeInd < 180; ++nodeInd){
               	   X[cellInd*192 + nodeInd] = initx[nodeInd] + allCMs[cellInd].x;
               	   Y[cellInd*192 + nodeInd] = inity[nodeInd] + allCMs[cellInd].y;
               	   Z[cellInd*192 + nodeInd] = initz[nodeInd] + allCMs[cellInd].z;
       	}
   	}
   }	
       
      // check all the fucking positions...
      for (int i = 0; i < Orig_No_of_C180s*192; ++i){
          if (X[i] > boxMax.x || X[i] < BoxMin.x ||
              Y[i] > boxMax.y || Y[i] < BoxMin.y ||
              Z[i] > boxMax.z || Z[i] < BoxMin.z ){

              printf("shit is in the fan\n");
              printf("%f %f %f\n", X[i], Y[i], Z[i]);
              //exit(4); 
          }
                               
      }


	for (int cellInd = 0; cellInd < Orig_No_of_C180s; cellInd++)
	{
		CellINdex[cellInd] = cellInd;
	}


  return(0);
}


void RotationMatrix(float* RMat,float* axis,float* theta){

	float C,S;
  	C = cos(theta[0]);
  	S = sin(theta[0]);
  	  
  	RMat[0] = 1.0f + (1.0f - C)*(-axis[2]*axis[2] - axis[1]*axis[1]);
  	RMat[1] = -S*axis[2] + (1.0f - C)*axis[0]*axis[1];
  	RMat[2] =  S*axis[1] + (1.0f - C)*axis[0]*axis[2];
  	  
  	RMat[3] =  S*axis[2] + (1.0f - C)*axis[0]*axis[1];
  	RMat[4] =  1.0f + (1.0f - C)*(-axis[2]*axis[2] - axis[0]*axis[0]);
  	RMat[5] = -S*axis[0] + (1.0f - C)*axis[1]*axis[2];
  	  
  	RMat[6] = -S*axis[1] + (1.0f - C)*axis[0]*axis[2];
  	RMat[7] =  S*axis[0] + (1.0f - C)*axis[1]*axis[2];
  	RMat[8] =  1.0f + (1.0f - C)*(-axis[0]*axis[0] - axis[1]*axis[1]);

}


inline void initialize_Plane(int MaxNoofC180s){



   float v[3], w[3];
     
   if (useDivPlaneBasis){
          
              
              if (divPlaneBasis[1] != 0){
        		
        		v[0] = 0;
        		v[1] = divPlaneBasis[2];
        		v[2] = -1*divPlaneBasis[1];

        		w[0] = divPlaneBasis[1];
        		w[1] = -1*divPlaneBasis[0];
        		w[2] = 0;
    		}else{ // this branch is very unlikely, placed for correctness
        		v[0] = 0;
        		v[1] = 1;
        		v[2] = 0;

        		w[0] = divPlaneBasis[2];
        		w[1] = 0;
       		w[2] = -1*divPlaneBasis[0];
    		}

    		// Orthogonalize
   		float f = (w[0]*v[0] + w[1]*v[1] + w[2]*w[2])/(v[0]*v[0] + v[1]*v[1] + v[2]*v[2]);

    		w[0] = w[0] - f*v[0];
    		w[1] = w[1] - f*v[1];
      		w[2] = w[2] - f*v[2];

    		// normalize
    		f = sqrt(v[0]*v[0] + v[1]*v[1] + v[2]*v[2]);

    		v[0] = v[0]/f;
    		v[1] = v[1]/f;
    		v[2] = v[2]/f;

    		f = sqrt(w[0]*w[0] + w[1]*w[1] + w[2]*w[2]);

    		w[0] = w[0]/f;
    		w[1] = w[1]/f;
    		w[2] = w[2]/f;
    
    }
    
    
    for (int i = 0; i < MaxNoofC180s; i++) {
     
          
          float norm[3];
          
          if (useDivPlaneBasis)
              	
              	GetRandomVectorBasis(norm,v,w);
          
          else
          
              GetRandomVector(norm);

#ifdef TURNOFF_RAN

          norm[0] = 0; 
          norm[1] = 1; 
          norm[2] = 0;
          
#endif
          
          DivPlane.x[i] = norm[0];
          DivPlane.y[i] = norm[1];
          DivPlane.z[i] = norm[2]; 

   }

          hipMemcpy( d_DivPlane.x, DivPlane.x, MaxNoofC180s*sizeof(float), hipMemcpyHostToDevice);
          hipMemcpy( d_DivPlane.y, DivPlane.y, MaxNoofC180s*sizeof(float), hipMemcpyHostToDevice);
          hipMemcpy( d_DivPlane.z, DivPlane.z, MaxNoofC180s*sizeof(float), hipMemcpyHostToDevice);
          CudaErrorCheck();

   if (asymDivision){	
   
	ranmar(asym, MaxNoofC180s); 
	 
   } else {
   	for (int i = 0; i < MaxNoofC180s; i++) asym[i] = 0.5;
   }
   
   hipMemcpy( d_asym, asym, MaxNoofC180s*sizeof(float), hipMemcpyHostToDevice);
   CudaErrorCheck();	

}




int initialize_Vel(int Orig_No_of_C180s)
{

int rank;
int atom;
float rands[3];
float3 Vcom;

printf("      Initializing velocities for %d fullerenes...\n", Orig_No_of_C180s);
  
	      
	for ( rank = 0; rank < Orig_No_of_C180s; ++rank )
	{	
		ranmar(rands, 3);
		Vcom = make_float3(rands[0]*6 - 3, rands[1]*6 - 3, rands[2]*6 - 3);
		if (flatbox == 1) Vcom.z = 0;
			
               for ( atom = 0 ; atom < 180 ; ++atom)
		{    
                    velListX[rank*192+atom] = Vcom.x;
                    velListY[rank*192+atom] = Vcom.y;
                    velListZ[rank*192+atom] = Vcom.z;
          	}

        }

  return(0);
  
}

int SecondCell (int Orig_No_of_C180s){


  float sumx = 0; 
  float sumy = 0; 
  float sumz = 0;
	  
	  
	 if (closenessToCenter > 0.f && closenessToCenter < 1.f){
          printf("Only making cells within %f of max radius different\n", closenessToCenter);
          	CenterOfMass<<<No_of_C180s,256>>>(No_of_C180s,
                              	              d_X, d_Y, d_Z,
                                      	      d_CMx, d_CMy, d_CMz);
          
         	 hipMemcpy(CMx, d_CMx, No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);
         	 hipMemcpy(CMy, d_CMy, No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);
         	 hipMemcpy(CMz, d_CMz, No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);

         	 float3 sysCM = make_float3(0.f, 0.f, 0.f);

         	 for(int i =0; i < No_of_C180s; ++i){
         	     sysCM = sysCM + make_float3(CMx[i], CMy[i], CMz[i]);
         	 }

         	 sysCM = sysCM/No_of_C180s; 
          
         	 printf("COM = (%f, %f, %f)\n", sysCM.x, sysCM.y, sysCM.z);

         	 float rMax = 0;
         	 float mags[No_of_C180s];
          
         	 for (int i =0; i < No_of_C180s; ++i){
         	     float3 pos = make_float3(CMx[i], CMy[i], CMz[i]) - sysCM;
         	     mags[i] = mag(pos);
         	     rMax = max(rMax, mags[i]);
         	 }
        	 int c = 0; 
          	 for (int i = 0; i < No_of_C180s; ++i){
              		if (mags[i]/rMax <= closenessToCenter){
              		
              			ScaleFactor[i] = SizeFactor;
                  		youngsModArray[i] = Stiffness2;
                  		Growth_rate[i] = gRate;
                  		DivisionVolume[i] = divisionV;
                  		gamma_env[i] = gEnv;
                  		viscotic_damp[i] = gVis;
                  		CellINdex[i] = - CellINdex[i];
                  		
                  		
                  		for (int j =0; j < 180; ++j){
      					sumx += X[i*192 + j]; 
      					sumy += Y[i*192 + j]; 
      					sumz += Z[i*192 + j]; 
  				}

  				sumx /= 180.0; 
  				sumy /= 180.0; 
  				sumz /= 180.0;


  				for (int j =0; j < 180; ++j){
      					X[i*192 + j] -= sumx; 
      					Y[i*192 + j] -= sumy; 
      					Z[i*192 + j] -= sumz; 
  				}
                  		
		
				for(int j = 0; j < 180; ++j){
       	           		X[i*192 + j] = SizeFactor*X[i*192 + j] + sumx;
       	           		Y[i*192 + j] = SizeFactor*Y[i*192 + j] + sumy;
       	           		Z[i*192 + j] = SizeFactor*Z[i*192 + j] + sumz;
       			}

				sumx = 0;
				sumy = 0;
				sumz = 0;
				
                  		++c;
                  		
              		}
          	}
          	
          	printf("Made %d cells softer\n", c);

	  }else{

		printf("Choosing second cell randomly\n");
        	int c = numberOfCells;
                if (fractionOfCells > 0.f && fractionOfCells < 1.f){
                	c = round(fractionOfCells*(float)(Orig_No_of_C180s - impurityNum));
                	
                }

                if (c > Orig_No_of_C180s - impurityNum){
              		printf("ERROR: Too many different cells requested\n");
              		return 12517;
          	}
	 
          	float rands[1];
	  	int coun;
	  	coun = c;	
	  	while(true){
			
	  		ranmar(rands, 1);
                	int i = round(rands[0]*(No_of_C180s - impurityNum) + impurityNum );

			if ( ScaleFactor[i] == SizeFactor ||  youngsModArray[i] == Stiffness2) continue;                 
		
			ScaleFactor[i] = SizeFactor;
			youngsModArray[i] = Stiffness2;
			Growth_rate[i] = gRate;
			DivisionVolume[i] = divisionV;
			gamma_env[i] = gEnv;
			viscotic_damp[i] = gVis;
			CellINdex[i] = - CellINdex[i];
			
                  	for (int j =0; j < 180; ++j){
      				sumx += X[i*192 + j]; 
      				sumy += Y[i*192 + j]; 
      				sumz += Z[i*192 + j]; 
  			}

  			sumx /= 180.0; 
  			sumy /= 180.0; 
  			sumz /= 180.0;


  			for (int j =0; j < 180; ++j){
      				X[i*192 + j] -= sumx; 
      				Y[i*192 + j] -= sumy; 
      				Z[i*192 + j] -= sumz; 
  			}
                  		
		
			for(int j = 0; j < 180; ++j){
       	           	X[i*192 + j] = SizeFactor*X[i*192 + j] + sumx;
       	           	Y[i*192 + j] = SizeFactor*Y[i*192 + j] + sumy;
       	           	Z[i*192 + j] = SizeFactor*Z[i*192 + j] + sumz;
       		}

			sumx = 0;
			sumy = 0;
			sumz = 0;
			
			coun--;
			
			
			if (coun == 0 ) break;

	      }

	}	      

	return 0;

}


int DispersityFunc(int Orig_No_of_C180s){

	
	if(dispersity && colloidal_dynamics && !Restart){
	
		float rands[1];
		
		for (int rank = 0; rank < Orig_No_of_C180s; ++rank )
		{
			ranmar(rands,1);
			ScaleFactor[rank] = rands[0]*0.35 + 0.65 ;
		}

	}

	return(0);

}


void ShapeScaler (float* initX,float* initY,float* initZ){


	
	float sumx = 0; 
  	float sumy = 0; 
  	float sumz = 0;

  	
  	for (int i =0; i < 180; ++i){
      		sumx += initX[i]; 
      		sumy += initY[i]; 
      		sumz += initZ[i]; 
  	}

  	sumx /= 180.0; 
  	sumy /= 180.0; 
  	sumz /= 180.0;  
      
  	for (int i =0; i < 180; ++i){
      		initX[i] -= sumx; 
      		initY[i] -= sumy; 
      		initZ[i] -= sumz; 
  	}
  	
  	
  	for (int i =0; i < 180; ++i){
  	
  		initX[i] *= Xratio;
  		initY[i] *= Yratio;
  		initZ[i] *= Zratio;
  			
  	}
	

}


int generate_random(int no_of_ran1_vectors)
{
  // This function uses marsaglia random number generator
  // Defined in marsaglia.h
  int seed_ij, seed_kl ,ij,kl;

  

  ran2 = (float *)calloc(MaxNoofC180s+1,sizeof(float));
  CPUMemory += (MaxNoofC180s+1L)*sizeof(float);
  if (rand_seed < 0){
      time_t current_time;
      time(&current_time);
      seed_ij = (int)current_time;
      localtime(&current_time);
      seed_kl = (int)current_time;
  
      ij = seed_ij%31328;
      kl = seed_kl%30081;
  } else {
      ij = rand_seed;
      kl = rand_seed*100;
  }

#ifdef TURNOFF_RAN
  ij = 80;
  kl = 97;
#endif
  
  rmarin(ij,kl);

  printf("RNG seeds: %d, %d\n", ij, kl); 
  return(0);
}



int read_fullerene_nn(void)
{
  int i,end;
  int N1, N2, N3, N4, N5, N6, Sign;
  FILE *infil;

  printf("   Reading C180NN ..\n");

  infil = fopen("C180NN","r");
  if ( infil == NULL ) {printf("Unable to open file C180NN\n");return(-1);}
  
  end = 180;
  for ( i = 0; i < 180 ; ++i )
  {
      if ( fscanf(infil,"%d,%d,%d,%d", &N1, &N2, &N3, &Sign) != 4 ) {end = i; break;}
      C180_nn[0 + i] = N1-1;
      C180_nn[192+i] = N2-1;
      C180_nn[384+i] = N3-1;
      C180_sign[i] = Sign;
  }
  fclose(infil);

  if ( end < 180 ) {printf("Error: Unable to read line %d in file C180NN\n",end);return(-1);}

  printf("   Reading C180C ..\n");

  infil = fopen("C180C","r");
  if ( infil == NULL ) {printf("Unable to open file C180C\n");return(-1);}

  end = 270;
  for ( i = 0; i < 270 ; ++i )
  {
      if ( fscanf(infil,"%d,%d", &N1, &N2) != 2 ) {end = i; break;}
      CCI[0][i] = N1-1;
      CCI[1][i] = N2-1;
  }
  fclose(infil);

  if ( end < 270 ) {printf("Error: Unable to read line %d in file C180C\n",end);return(-1);}

  printf("      read nearest neighbour ids for atoms in C180\n");

  printf("   Reading C180 pentagons, hexagons ..\n");

  infil = fopen("C180_pentahexa","r");
  if ( infil == NULL ) {printf("Unable to open file C180_pentahexa\n");return(-1);}

  end = 12;
  for ( i = 0; i < 12 ; ++i )
  {
      if ( fscanf(infil,"%d %d %d %d %d", &N1, &N2, &N3, &N4, &N5) != 5 ) {end = i; break;}
      C180_56[i*7+0] = N1;
      C180_56[i*7+1] = N2;
      C180_56[i*7+2] = N3;
      C180_56[i*7+3] = N4;
      C180_56[i*7+4] = N5;
      C180_56[i*7+5] = N1;
      C180_56[i*7+6] = N1;
  }
  if ( end != 12 ) {printf("Error: Unable to read line %d in file C180_pentahexa\n",end);return(-1);}
  end = 80;
  for ( i = 0; i < 80 ; ++i )
  {
      if ( fscanf(infil,"%d %d %d %d %d %d", &N1, &N2, &N3, &N4, &N5, &N6) != 6 ) {end = i; break;}
      C180_56[84+i*7+0] = N1;
      C180_56[84+i*7+1] = N2;
      C180_56[84+i*7+2] = N3;
      C180_56[84+i*7+3] = N4;
      C180_56[84+i*7+4] = N5;
      C180_56[84+i*7+5] = N6;
      C180_56[84+i*7+6] = N1;
  }
  if ( end != 80 ) {printf("Error: Unable to read line %d in file C180_pentahexa\n",end);return(-1);}

  fclose(infil);

  printf("Calculating equilibrium bond lengths\n");

  float initX[181], initY[181], initZ[181];

  infil = fopen("C180","r");
  if ( infil == NULL ) {printf("Unable to open file C180\n");return(-1);}
  for ( int atom = 0 ; atom < 180 ; ++atom)
  {
  	if ( fscanf(infil,"%f %f %f",&initX[atom], &initY[atom], &initZ[atom]) != 3 )
  	{
              printf("   Unable to read file C180 on line %d\n",atom+1);
              fclose(infil);
              return(-1);
          }
  }
  fclose(infil);

  if (colloidal_dynamics) ShapeScaler (initX,initY,initZ);			

  for (int i = 0; i < 180; ++i){
      int N1 = C180_nn[0 + i];
      int N2 = C180_nn[192 + i];
      int N3 = C180_nn[384 + i];

      float3 a, b; 
      a = make_float3(initX[i], initY[i], initZ[i]);

      b = make_float3(initX[N1], initY[N1], initZ[N1]);
      h_R0[0 + i] = mag(a-b);

      b = make_float3(initX[N2], initY[N2], initZ[N2]);
      h_R0[192 + i] = mag(a-b);

      b = make_float3(initX[N3], initY[N3], initZ[N3]);
      h_R0[384 + i] = mag(a-b);
  }
  
  float3 p, ni, nj, nk;
  for (int n = 0; n<180; n++){
  
          p = make_float3(initX[n], initY[n], initZ[n]); 

          ni = make_float3(initX[C180_nn[0*192 + n]], initY[C180_nn[0*192 + n]], 
                           initZ[C180_nn[0*192 + n]]); 
          
          nj = make_float3(initX[C180_nn[1*192 + n]], initY[C180_nn[1*192 + n]], 
                           initZ[C180_nn[1*192 + n]]);
          
          nk = make_float3(initX[C180_nn[2*192 + n]], initY[C180_nn[2*192 + n]],
                           initZ[C180_nn[2*192 + n]]);

          ni = ni-p;
          nj = nj-p;
          nk = nk-p; 

          theta0[n].aij = acos(dot(ni, nj)/(mag(ni)*mag(nj)));
          
          theta0[n].ajk = acos(dot(nj, nk)/(mag(nj)*mag(nk)));
          
          theta0[n].aik = acos(dot(ni, nk)/(mag(ni)*mag(nk)));

  } 
  
  return(0);
}


int read_json_params(const char* inpFile){
    // Function to parse a json input file using the jsoncpp library

    // variable to hold the root of the json input
    Json::Value inpRoot;
    Json::Reader inpReader;

    std::ifstream inpStream(inpFile);
    std::string inpString((std::istreambuf_iterator<char>(inpStream)),
                          std::istreambuf_iterator<char>());

    bool parsingSuccess = inpReader.parse(inpString, inpRoot);
    if (!parsingSuccess){
        printf("Failed to parse %s\n", inpFile);
        // There must be a way to keep from converting from string to char*
        // Maybe by making inpString a char*
        printf("%s", inpReader.getFormattedErrorMessages().c_str());
        return -1;
    }
    else
        printf("%s parsed successfully\n", inpFile);

    // begin detailed parameter extraction

    Json::Value coreParams = inpRoot.get("core", Json::nullValue);

    // load core simulation parameters
    if (coreParams == Json::nullValue){
        printf("ERROR: Cannot load core simulation parameters\nExiting");
        return -1;
    }
    else {
        MaxNoofC180s = coreParams["MaxNoofC180s"].asInt(); 
        mass = coreParams["particle_mass"].asFloat();
        repulsion_range = coreParams["repulsion_range"].asFloat();
        attraction_range = coreParams["attraction_range"].asFloat();
        repulsion_strength = coreParams["repulsion_strength"].asFloat();
        attraction_strength = coreParams["attraction_strength"].asFloat();
        Youngs_mod = coreParams["Youngs_mod"].asFloat(); 
        stiffness1 = coreParams["stiffFactor1"].asFloat()*Youngs_mod;
        viscotic_damping = coreParams["viscotic_damping"].asFloat();
        internal_damping = coreParams["internal_damping"].asFloat();
        divVol = coreParams["division_Vol"].asFloat();
        Time_steps = coreParams["div_time_steps"].asFloat();
        delta_t = coreParams["time_interval"].asFloat();
        Restart = coreParams["Restart"].asInt();
        trajWriteInt = coreParams["trajWriteInt"].asInt();
        equiStepCount = coreParams["non_div_time_steps"].asInt();

        std::strcpy (trajFileName, coreParams["trajFileName"].asString().c_str());
        binaryOutput = coreParams["binaryOutput"].asBool(); 

        maxPressure = coreParams["maxPressure"].asFloat();
        minPressure = coreParams["minPressure"].asFloat();
        gamma_visc = coreParams["gamma_visc"].asFloat();
        shear_rate = coreParams["shear_rate"].asFloat();
        rMax = coreParams["growth_rate"].asFloat();
        checkSphericity = coreParams["checkSphericity"].asBool();
        constrainAngles = coreParams["constrainAngles"].asBool();
        dt_max = coreParams["dt_max"].asFloat();
        dt_tol = coreParams["dt_tol"].asFloat();
        doAdaptive_dt = coreParams["doAdaptive_dt"].asBool();
        write_cont_force = coreParams["write_cont_force"].asBool();
        write_vel_file = coreParams["write_vel_file"].asBool();
        std::strcpy(forces_file, coreParams["forces_file"].asString().c_str());
        correct_com = coreParams["correct_com"].asBool();
        correct_Vcom = coreParams["correct_Vcom"].asBool();
                                 
    }

    Json::Value countParams = inpRoot.get("counting", Json::nullValue);
    if (countParams == Json::nullValue){
        // countCells = FALSE;
        printf("ERROR: Cannot load counting parameters\nExiting");
        return -1;
    }
    else {
        countCells = countParams["countcells"].asBool();
        std::strcpy(mitIndFileName, countParams["mit-index_file_name"].asString().c_str()); 
        countOnlyInternal = countParams["count_only_internal_cells?"].asBool();
        radFrac = countParams["radius_cutoff"].asFloat();
        overWriteMitInd = countParams["overwrite_mit_ind_file?"].asBool();
        newCellCountInt = countParams["cell_count_int"].asInt();
    }
    

    Json::Value popParams = inpRoot.get("population", Json::nullValue);
    if (popParams == Json::nullValue){
        printf("ERROR: Cannot load population parameters\nExiting");
        return -1;
    }
    else{
        doPopModel = popParams["doPopModel"].asInt();
        totalFood = popParams["totalFood"].asFloat();
        cellFoodCons = popParams["regular_consumption"].asFloat();
        cellFoodConsDiv = popParams["division_consumption"].asFloat();
        cellFoodRel = popParams["death_release_food"].asFloat();
        cellLifeTime = popParams["cellLifeTime"].asInt();
        maxPop = popParams["max_pop"].asFloat(); 
    }

    
    Json::Value apoParams = inpRoot.get("apoptosis", Json::nullValue);
    if (apoParams == Json::nullValue){
        printf("ERROR: Cannot load apoptosis parameters\nExiting");
        return -1;
    }
    else{
	apoptosis = apoParams["apoptosis"].asBool();
    	popToStartApo = apoParams["popToStartApo"].asFloat();
    	Apo_rate = apoParams["Apo_ratio"].asFloat();
    	squeeze_rate = apoParams["squeeze_rate"].asFloat();
    	ApoVol = apoParams["apoptosis_Vol"].asFloat();
    	
    }	

    Json::Value divParams = inpRoot.get("divParams", Json::nullValue);
    
    if (divParams == Json::nullValue){
        printf("ERROR: Cannot load division parameters\n");
        return -1;
    } else{
        useDivPlaneBasis = divParams["useDivPlaneBasis"].asInt();
        divPlaneBasis[0] = divParams["divPlaneBasisX"].asFloat();
        divPlaneBasis[1] = divParams["divPlaneBasisY"].asFloat();
        divPlaneBasis[2] = divParams["divPlaneBasisZ"].asFloat();
	asymDivision = divParams["asymDivision"].asBool();
    }

    Json::Value NewCell = inpRoot.get("NewCell", Json::nullValue);

    if (NewCell == Json::nullValue){
        printf("ERROR: Cannot load New Cell parameters\n");
        return -1;
    } else {
        useDifferentCell = NewCell["useDifferentCell"].asBool();
        SizeFactor = NewCell["SizeFactor"].asFloat();
        Stiffness2 = NewCell["StiffFactor"].asFloat() * Youngs_mod;
        gRate = NewCell["GrowthRate"].asFloat();
        divisionV = NewCell["divisionV"].asFloat();
        gEnv = NewCell["gamma"].asFloat();
        gVis = NewCell["VisDamping"].asFloat();
        Apo_rate2 = NewCell["Apo_rate2"].asFloat();
        squeeze_rate2 = NewCell["squeeze_rate2"].asFloat();
        numberOfCells = NewCell["numberOfCells"].asInt();
        fractionOfCells = NewCell["fractionOfCells"].asFloat();
        closenessToCenter = NewCell["closenessToCenter"].asFloat();
        chooseRandomCellIndices = NewCell["chooseRandomCellIndices"].asBool(); 
        daughtSame = NewCell["daughtSame"].asBool(); 
        duringGrowth = NewCell["duringGrowth"].asBool();
        recalc_r0 = NewCell["recalc_r0"].asBool(); 
    }
    
    Json::Value ColloidParams = inpRoot.get("ColloidParams", Json::nullValue);

    if (ColloidParams == Json::nullValue){
        printf("ERROR: Cannot load Colloidal parameters\n");
        return -1;
    } else {
    
        colloidal_dynamics = ColloidParams["colloidal_dynamics"].asBool();
    	dispersity = ColloidParams["dispersity"].asBool();
    	rand_vel = ColloidParams["rand_vel"].asBool();
        Xratio = ColloidParams["Xratio"].asFloat();
        Yratio = ColloidParams["Yratio"].asFloat();
        Zratio = ColloidParams["Zratio"].asFloat();
        RandInitDir = ColloidParams["RandInitDir"].asBool();
    
    }
  

    Json::Value boxParams = inpRoot.get("boxParams", Json::nullValue);

    if (boxParams == Json::nullValue){
        printf("ERROR: Cannot load box parameters\n");
        return -1;
    } else{
        useRigidSimulationBox = boxParams["useRigidSimulationBox"].asBool();
        usePBCs = boxParams["usePBCs"].asBool();
        useLEbc = boxParams["useLEbc"].asBool();
        useRigidBoxZ = boxParams["useRigidBoxZ"].asBool();
        useRigidBoxY = boxParams["useRigidBoxY"].asBool();
        threshDist = boxParams["threshDist"].asFloat();
        boxMax.x = boxParams["box_len_x"].asFloat();
        boxMax.y = boxParams["box_len_y"].asFloat(); 
        boxMax.z = boxParams["box_len_z"].asFloat();
        BoxMin.x = boxParams["BoxMin_x"].asFloat();
        BoxMin.y = boxParams["BoxMin_y"].asFloat(); 
        BoxMin.z = boxParams["BoxMin_z"].asFloat();
        flatbox = boxParams["flatbox"].asBool();
        LineCenter = boxParams["LineCenter"].asBool();
        rand_pos = boxParams["rand_pos"].asBool();
	impurity = boxParams["impurity"].asBool();
	impurityNum = boxParams["impurityNum"].asInt();
	line = boxParams["line"].asBool();
	plane = boxParams["plane"].asBool();
	
    }

    if (dt_tol > dt_max || dt_max <= 0 || dt_tol < 0){
        printf("ERROR: Invalid time step parameters\n");
        printf("FATAL ERROR\n");
        exit(-60); 
    }

    Json::Value randParams = inpRoot.get("rand_params", Json::nullValue);
	impurity = boxParams["impurity"].asBool();
	line = boxParams["line"].asBool();
    if (randParams == Json::nullValue){
        printf("ERROR: Cannot load randomness parameters\n");
        return -1;
    }
    else {
        add_rands = randParams["add_rands"].asBool();
        rand_seed = randParams["rand_seed"].asInt();
        rand_dist = randParams["rand_dist"].asInt();
        rand_scale_factor = randParams["rand_scale_factor"].asFloat();
    }


    printf("      mass                = %f\n",mass);
    printf("      repulsion range     = %f\n",repulsion_range);
    printf("      attraction range    = %f\n",attraction_range);
    printf("      repulsion strength  = %f\n",repulsion_strength);
    printf("      attraction strength = %f\n",attraction_strength);
    printf("      Youngs modulus      = %f\n",stiffness1);
    printf("      viscotic damping    = %f\n",viscotic_damping);
    printf("      internal damping    = %f\n",internal_damping);
    printf("      division volume     = %f\n",divVol);
    printf("      Time steps          = %d\n",Time_steps);
    printf("      delta t             = %f\n",delta_t);
    printf("      Restart             = %d\n",Restart);
    printf("      trajWriteInterval   = %d\n",trajWriteInt);
    printf("      countOnlyInternal   = %d\n", countOnlyInternal);
    printf("      radFrac             = %f\n", radFrac);
    printf("      newCellCountInt     = %d\n", newCellCountInt);
    printf("      equiStepCount       = %d\n", equiStepCount);
    printf("      trajFileName        = %s\n", trajFileName);
//    printf("      doPopModel          = %d\n", doPopModel);
//    printf("      totalFood           = %f\n", totalFood);
//    printf("      cellFoodCons        = %f\n", cellFoodCons);
//    printf("      cellFoodConsDiv     = %f\n", cellFoodConsDiv);
//    printf("      cellFoodRel         = %f\n", cellFoodRel);
    printf("      thresDist           = %f\n", threshDist);
    printf("      maxPressure         = %f\n", maxPressure);
    printf("      minPressure         = %f\n", minPressure);
    printf("      growth_rate         = %f\n", rMax);
    printf("      squeeze_rate         = %f\n", squeeze_rate);
    printf("      checkSphericity     = %d\n", checkSphericity);
    printf("      gamma_visc          = %f\n", gamma_visc);
    printf("      useDivPlanebasis    = %d\n", useDivPlaneBasis);
    printf("      divPlaneBasisX      = %f\n", divPlaneBasis[0]);
    printf("      divPlaneBasisY      = %f\n", divPlaneBasis[1]);
    printf("      divPlaneBasisZ      = %f\n", divPlaneBasis[2]);
    printf("      useDifferentCell = %d\n", useDifferentCell);
    printf("      SizeFactor  	=%f\n", SizeFactor);
    printf("      Stiffness2  	=%f\n", Stiffness2);
    printf("      GrowthRate2  	=%f\n", gRate);
    printf("      division volume2  	=%f\n", divisionV);
    printf("      gamma_visc2  	=%f\n", gEnv); 
    printf("      viscotic damping2  	=%f\n", gVis);            
    printf("      numberOfCells       = %d\n", numberOfCells);
    printf("      duringGrowth        = %d\n", duringGrowth);
    printf("      closenesstoCenter   = %f\n", closenessToCenter);
    printf("      fractionOfCells     = %f\n", fractionOfCells);
    printf("      chooseRandomCellIndices = %d\n", chooseRandomCellIndices);
    printf("      daughtSame = 	%d\n", daughtSame);
    printf("      recalc_r0           = %d\n", recalc_r0);
    printf("      useRigidSimulationBox = %d\n", useRigidSimulationBox);
    printf("      usePBCs             = %d\n", usePBCs);
    printf("      box_len_x           = %f\n", boxMax.x);
    printf("      box_len_y           = %f\n", boxMax.y);
    printf("      box_len_z           = %f\n", boxMax.z);
    printf("      BoxMin_x            = %f\n", BoxMin.x);
    printf("      BoxMin_y            = %f\n", BoxMin.y);
    printf("      BoxMin_z            = %f\n", BoxMin.z);
    printf("      flatbox             = %d\n", flatbox); 
    printf("      doAdaptive_dt       = %d\n", doAdaptive_dt); 
    printf("      dt_max              = %f\n", dt_max); 
    printf("      dt_tol              = %f\n", dt_tol);
    printf("      add_rands           = %d\n", add_rands);
    printf("      rand_seed           = %d\n", rand_seed);
    printf("      rand_scale_factor   = %f\n", rand_scale_factor);
    printf("      correct_com         = %d\n", correct_com);
    printf("      correct_Vcom         = %d\n", correct_Vcom);    
    printf("      impurityNum         = %d\n", impurityNum);
    printf("      apoptosis           = %d\n",apoptosis);
    printf("      Apoptosis ratio     = %f\n",Apo_rate);
    printf("      apoptosis volume    = %f\n",ApoVol);
    printf("      squeeze rate        = %f\n",squeeze_rate);
    
    
    

    if (asymDivision){
        printf("asymmetric cell division \n");
    }    

    if ( radFrac < 0.4 || radFrac > 0.8 || radFrac < 0 ){
        printf("radFrac not in [0.4, 0.8] setting to 1.\n");
        countOnlyInternal = 0;
    }

    if (trajWriteInt == 0){
        trajWriteInt = 1;
    }

    if (newCellCountInt == 0){
        newCellCountInt = 1;
    }

    if ( trajWriteInt > Time_steps + equiStepCount){
        printf ("Trajectory write interval is too large\n");
        return -1;
    }

    if ((Time_steps + equiStepCount)%trajWriteInt != 0){
        printf ("Invalid trajectory write interval. Time steps must be divisible by it. \n");
        return -1;
    }

    if (newCellCountInt > Time_steps + 1){
        printf("New cell counting interval is too large. \n");
        return -1;
    }

    if (equiStepCount <= 0){
        equiStepCount = 0;
    }

    if (doPopModel != 1){ // This ensures that Pop modelling is only done if this
        // var is only 1
        doPopModel = 0;
    }

    if (maxPressure < 0){
        printf("Invalid maximum pressure value of %f\n", maxPressure);
        printf("Disabling population modelling...");
        doPopModel = 0;
    }


    if ( (useRigidSimulationBox && usePBCs) || (useRigidSimulationBox && useLEbc) || (useLEbc && usePBCs)){
  
      printf ("More than one boundary condition.... \n");
      return -1;
    }	
    
    if ( line && rand_pos && plane){
  
      printf ("More than one initial condition.... \n");
      return -1;
    }

	


    /*

    // The if statement below is not a very good one
    // think about rewriting.
    if (totalFood < 0.0
    || No_of_threads*100 < totalFood
    || cellFoodCons < 0.0
    || cellFoodCons*No_of_threads*10 < totalFood
    || cellFoodConsDiv < 0.0
    || cellFoodConsDiv*No_of_threads*10 < totalFood
    ){
    doPopModel = 0;
    printf("Food parameters invalid. Skipping population modelling.\n");
    }
    */

    if ( !(closenessToCenter >=0 && closenessToCenter <= 1) ){
        printf("ERROR: closenessToCenter is not in [0, 1]\n");
        printf("ERROR: invalid input parameter\n");
        return -1;
    }


    if (fractionOfCells > 1.0){
        printf("ERROR: Softer cell fraction is > 1\n");
        return -1;
    }
        

    return 0;
}


int read_global_params(void)
{
  int error;
  FILE *infil;

  printf("   Reading inp.dat ..\n");

  infil = fopen("inp.dat","r");
  if ( infil == NULL ) {printf("Error: Unable to open file inp.dat\n");return(-1);}

  error = 0;


  if ( fscanf(infil,"%f",&mass)                != 1 ) {error =  1 ;}
  if ( fscanf(infil,"%f",&repulsion_range)     != 1 ) {error =  2 ;}
  if ( fscanf(infil,"%f",&attraction_range)    != 1 ) {error =  3 ;}
  if ( fscanf(infil,"%f",&repulsion_strength)  != 1 ) {error =  4 ;}
  if ( fscanf(infil,"%f",&attraction_strength) != 1 ) {error =  5 ;}
//  if ( fscanf(infil,"%f",&Youngs_mod)          != 1 ) {error =  6 ;}
  if ( fscanf(infil,"%f",&viscotic_damping)    != 1 ) {error =  7 ;}
  if ( fscanf(infil,"%f",&internal_damping)    != 1 ) {error =  8 ;}
  if ( fscanf(infil,"%f",&divVol)              != 1 ) {error =  9 ;}
  if ( fscanf(infil,"%d",&Time_steps)          != 1 ) {error = 10 ;}
  if ( fscanf(infil,"%f",&delta_t)             != 1 ) {error = 11 ;}
  if ( fscanf(infil,"%d",&Restart)             != 1 ) {error = 12 ;}
  if ( fscanf(infil,"%d",&trajWriteInt)        != 1 ) {error = 13 ;}
  if ( fscanf(infil,"%d",&countOnlyInternal)   != 1 ) {error = 14 ;}
  if ( fscanf(infil,"%f",&radFrac)             != 1 ) {error = 15 ;}
  if ( fscanf(infil,"%d",&overWriteMitInd)     != 1 ) {error = 16 ;}
  if ( fscanf(infil,"%d",&newCellCountInt)     != 1 ) {error = 17 ;}
  if ( fscanf(infil,"%d",&equiStepCount)       != 1 ) {error = 18 ;}
  if ( fscanf(infil,"%s",trajFileName)         != 1 ) {error = 19 ;}
  if ( fscanf(infil,"%d",&doPopModel)          != 1 ) {error = 20 ;}
  if ( fscanf(infil,"%f",&totalFood)           != 1 ) {error = 21 ;}
  if ( fscanf(infil,"%f",&cellFoodCons)        != 1 ) {error = 22 ;}
  if ( fscanf(infil,"%f",&cellFoodConsDiv)     != 1 ) {error = 23 ;}
  if ( fscanf(infil,"%f",&cellFoodRel)         != 1 ) {error = 24 ;}
  if ( fscanf(infil,"%d",&haylimit)            != 1 ) {error = 25 ;}
  if ( fscanf(infil,"%d",&cellLifeTime)        != 1 ) {error = 26 ;}
  if ( fscanf(infil,"%f",&maxPressure)         != 1 ) {error = 27 ;}
  if ( fscanf(infil,"%f",&threshDist)          != 1 ) {error = 33 ;}





  fclose(infil);

  if ( error != 0 ){
      printf("   Error reading line %d from file inp.dat\n",error);
      return(-1);
  }

  if ( radFrac < 0.4 || radFrac > 0.8 || radFrac < 0 ){
      printf("radFrac not in [0.4, 0.8] setting to 1.\n");
      countOnlyInternal = 0;
  }

  if (trajWriteInt == 0){
      trajWriteInt = 1;
  }

  if (newCellCountInt == 0){
      newCellCountInt = 1;
  }

  if ( trajWriteInt > Time_steps){
      printf ("Trajectory write interval is too large\n");
      return -1;
  }
  

  if (Time_steps%trajWriteInt != 0){
      printf ("Invalid trajectory write interval. Time steps must be divisible by it. \n");
      return -1;
  }

  if (newCellCountInt > Time_steps){
      printf("New cell counting interval is too large. \n");
      return -1;
  }

  if (equiStepCount <= 0){
    equiStepCount = 0;
  }

  if (doPopModel != 1){ // This ensures that Pop modelling is only done if this
                        // var is only 1
      doPopModel = 0;
  }

  if (maxPressure < 0){
      printf("Invalid maximum pressure value of %f\n", maxPressure);
      printf("Disabling population modelling...");
      doPopModel = 0;
  }


  /*

  // The if statement below is not a very good one
  // think about rewriting.
  if (totalFood < 0.0
      || No_of_threads*100 < totalFood
      || cellFoodCons < 0.0
      || cellFoodCons*No_of_threads*10 < totalFood
      || cellFoodConsDiv < 0.0
      || cellFoodConsDiv*No_of_threads*10 < totalFood
       ){
      doPopModel = 0;
      printf("Food parameters invalid. Skipping population modelling.\n");
  }
  */


  printf("      mass                = %f\n",mass);
  printf("      repulsion range     = %f\n",repulsion_range);
  printf("      attraction range    = %f\n",attraction_range);
  printf("      repulsion strength  = %f\n",repulsion_strength);
  printf("      attraction strength = %f\n",attraction_strength);
//  printf("      Youngs modulus      = %f\n",Youngs_mod);
  printf("      viscotic damping    = %f\n",viscotic_damping);
  printf("      internal damping    = %f\n",internal_damping);
  printf("      division volume     = %f\n",divVol);
  printf("      Time steps          = %d\n",Time_steps);
  printf("      delta t             = %f\n",delta_t);
  printf("      Restart             = %d\n",Restart);
  printf("      trajWriteInterval   = %d\n",trajWriteInt);
  printf("      countOnlyInternal   = %d\n", countOnlyInternal);
  printf("      radFrac             = %f\n", radFrac);
  printf("      newCellCountInt     = %d\n", newCellCountInt);
  printf("      equiStepCount       = %d\n", equiStepCount);
  printf("      trajFileName        = %s\n", trajFileName);
//  printf("      doPopModel          = %d\n", doPopModel);
//  printf("      totalFood           = %f\n", totalFood);
//  printf("      cellFoodCons        = %f\n", cellFoodCons);
//  printf("      cellFoodConsDiv     = %f\n", cellFoodConsDiv);
//  printf("      cellFoodRel         = %f\n", cellFoodRel);
  printf("      thresDist           = %f\n", threshDist);


  return(0);
}




//C *****************************************************************



void write_traj(int t_step, FILE* trajfile)
{

  fprintf(trajfile, "%d\n", No_of_C180s * 192);
  fprintf(trajfile, "Step: %d frame: %d\n", t_step, t_step/trajWriteInt);
  
  if (useDifferentCell){
      for (int c = 0; c < No_of_C180s; c++){
          if (youngsModArray[c] == stiffness1)
              fprintf(trajfile, "cell: %d H\n", CellINdex[c]);
          else if(youngsModArray[c] == Stiffness2)
              fprintf(trajfile, "cell: %d C\n", CellINdex[c]);
          else
              fprintf(trajfile, "cell: %d UnknownStiffness\n", CellINdex[c]);

          for (int p = 0; p < 192; p++)
          {
              fprintf(trajfile, "%.7f,  %.7f,  %.7f\n", X[(c*192)+p], Y[(c*192)+p], Z[(c*192)+p]);
          }
      }
        
  } else {
      for (int c = 0; c < No_of_C180s; c++){
              fprintf(trajfile, "cell: %d\n", CellINdex[c]);
              
              for (int p = 0; p < 192; p++)
              {
                  fprintf(trajfile, "%.7f,  %.7f,  %.7f\n", X[(c*192)+p], Y[(c*192)+p], Z[(c*192)+p]);
              }
      }
      
  }
}

void WriteBinaryTraj(int t_step, FILE* trajFile, int frameCount){
    
    fwrite(&t_step, sizeof(int), 1, trajFile);
    fwrite(&frameCount, sizeof(int), 1, trajFile); 
    fwrite(&No_of_C180s, sizeof(int), 1, trajFile);
    if (useDifferentCell){
        int cellType = 0; 
        for (int c = 0; c < No_of_C180s; c++){ 
            fwrite(&CellINdex[c], sizeof(int), 1, trajFile);
            fwrite(X + (c*192), sizeof(float), 192, trajFile); 
            fwrite(Y + (c*192), sizeof(float), 192, trajFile); 
            fwrite(Z + (c*192), sizeof(float), 192, trajFile);
            
            if (youngsModArray[c] == stiffness1)
                cellType = 0;
            else
                cellType = 1; 
            
            fwrite(&cellType, sizeof(int), 1, trajFile);
        }
    } else {
        for (int c = 0; c < No_of_C180s; c++){
        
            fwrite(&CellINdex[c], sizeof(int), 1, trajFile);            
            fwrite(X + (c*192), sizeof(float), 192, trajFile); 
            fwrite(Y + (c*192), sizeof(float), 192, trajFile); 
            fwrite(Z + (c*192), sizeof(float), 192, trajFile); 
        }
    }
        
    
}

void write_vel(int t_step, FILE* velFile,int frameCount){
    
    fwrite(&t_step, sizeof(int), 1, velFile);
    fwrite(&frameCount, sizeof(int), 1, velFile); 
    fwrite(&No_of_C180s, sizeof(int), 1, velFile);
    
    if (useDifferentCell){
        int cellType = 0; 
        for (int c = 0; c < No_of_C180s; c++){
            fwrite(&c, sizeof(int), 1, velFile);
            fwrite(velListX + (c*192), sizeof(float), 192, velFile); 
            fwrite(velListY + (c*192), sizeof(float), 192, velFile); 
            fwrite(velListZ + (c*192), sizeof(float), 192, velFile);
            
            if (youngsModArray[c] == stiffness1)
                cellType = 0;
            else
                cellType = 1; 
            
            fwrite(&cellType, sizeof(int), 1, velFile);
        }
    } else {
        for (int c = 0; c < No_of_C180s; c++){
        
            fwrite(&c, sizeof(int), 1, velFile);
            fwrite(velListX + (c*192), sizeof(float), 192, velFile); 
            fwrite(velListY + (c*192), sizeof(float), 192, velFile); 
            fwrite(velListZ + (c*192), sizeof(float), 192, velFile);
        }
    }
}


inline void count_and_get_div(){
  num_cell_div = 0;
  hipMemcpy(cell_div, d_cell_div, No_of_C180s*sizeof(char), hipMemcpyDeviceToHost);
  
  for (int cellInd = 0; cellInd < No_of_C180s; cellInd++) {
    if (cell_div[cellInd] == 1){
      cell_div[cellInd] = 0;
      cell_div_inds[num_cell_div] = cellInd;
      num_cell_div++;
    }
  }
  hipMemcpy(d_cell_div, cell_div, No_of_C180s*sizeof(char), hipMemcpyHostToDevice);

#ifdef PRINT_VOLUMES
        if (num_cell_div > 0){
            printf("Dividing cells: ");
            for (int i = 0; i<num_cell_div; i++){
                printf("%d ", cell_div_inds[i]);
            }
          
            printf("\n");
        }
#endif

}


inline void count_and_die(){
  
  num_cell_Apo = 0;
  hipMemcpy(cell_Apo, d_cell_Apo, No_of_C180s*sizeof(char), hipMemcpyDeviceToHost);
  hipMemcpy(d_Growth_rate, Growth_rate, No_of_C180s*sizeof(float), hipMemcpyHostToDevice);
  
  for (int cellInd = 0; cellInd < No_of_C180s; cellInd++) {
    if (cell_Apo[cellInd] == 1){ 
      if(Growth_rate[cellInd] == -squeeze_rate || Growth_rate[cellInd] == -squeeze_rate2){
      	cell_Apo[cellInd] = 0;
      	cell_Apo_inds[num_cell_Apo] = cellInd;
      	num_cell_Apo++;
      }
    }
  }
  hipMemcpy(d_cell_Apo, cell_Apo, No_of_C180s*sizeof(char), hipMemcpyHostToDevice);
}


inline float getRmax2(){
  float dx, dy, dz, Rmax2 = 0;
  for (int cell = 0; cell < No_of_C180s; cell++) {
    dx = CMx[cell] - *h_sysCM.x;
    dy = CMy[cell] - *h_sysCM.y;
    dz = CMz[cell] - *h_sysCM.z;

    Rmax2 = max(Rmax2, dx*dx + dy*dy + dz*dz);

  }

  return Rmax2;

}

inline int num_cells_far(){

  if (num_cell_div == 0 || No_of_C180s < 50) return 0;



  float dx, dy, dz, dr2;
  float Rmax2 = getRmax2();
  int farCellCount = 0;


  for (int cell = No_of_C180s - num_cell_div; cell < No_of_C180s; cell++) { // Only check the newest cells
    dx = CMx[cell] - *h_sysCM.x;
    dy = CMy[cell] - *h_sysCM.y;
    dz = CMz[cell] - *h_sysCM.z;

    dr2 = dx*dx + dy*dy + dz*dz;

    if (dr2 > radFrac*radFrac*Rmax2)
      farCellCount++;
  }

  return farCellCount;

}

void writeForces(FILE* forceFile, int t_step, int num_cells){
    if(forceFile == NULL){
        printf("ERROR: forces file not available\n");
        exit(1);
    }

    for (int c =0; c < num_cells; ++c){
        for (int n = 0; n < 180; ++n){
            fprintf(forceFile, "%d,%d,%d,%d,%d,%.8f,%.8f,%.8f,%.8f,%.8f,%.8f,%.8f,%.8f,%.8f,%.8f,%.8f,%.8f,%.8f,%.8f,%.8f,%.8f,%.8f,%.8f\n",
                    t_step, num_cells, c, n, c*192+n,
                    h_contactForces.x[c*192 + n],
                    h_contactForces.y[c*192 + n],
                    h_contactForces.z[c*192 + n],
                    mag(make_float3(h_contactForces.x[c*192 + n],
                                    h_contactForces.y[c*192 + n],
                                    h_contactForces.z[c*192 + n])),
                                        h_ExtForces.x[c*192 + n],
                    h_ExtForces.y[c*192 + n],
                    h_ExtForces.z[c*192 + n],
                    mag(make_float3(h_ExtForces.x[c*192 + n],
                                    h_ExtForces.y[c*192 + n],
                                    h_ExtForces.z[c*192 + n])),
                    velListX[c*192+n],
                    velListY[c*192+n],
                    velListZ[c*192+n],
                    mag(make_float3(velListX[c*192+n],
                                    velListY[c*192+n],
                                    velListZ[c*192+n])),
                    X[c*192+n],
                    Y[c*192+n],
                    Z[c*192+n],
                    pressList[c],
                    volume[c],
                    area[c]
                );
                        
        }


    }
}





int writeRestartFile(int t_step, int frameCount){
	
	FILE *Restartfile;
	Restartfile = fopen ("Restart.xyz", "w");
        if ( Restartfile == NULL)
  	{
      	    printf("Failed to open Restart file \n" );
      	    return -1;
  	}

	float p = 0;
	float y = 0;
	float g = 0;
	float s = 0;
	float d = 0;
	float ge = 0;
	float vd = 0;
	int I = 0;	

	fwrite(&t_step, sizeof(int), 1, Restartfile);
	fwrite(&frameCount, sizeof(int), 1, Restartfile); 
	fwrite(&No_of_C180s, sizeof(int), 1, Restartfile); 
	fwrite(&impurityNum, sizeof(int), 1, Restartfile);
	fwrite(&NumApoCell, sizeof(int), 1, Restartfile);
	fwrite(&NumRemoveCell, sizeof(int), 1, Restartfile);  
 
        	
        for (int c = 0; c < No_of_C180s; c++){
		p = pressList[c];
		y = youngsModArray[c];
        	g = Growth_rate[c];
        	s = ScaleFactor[c];
        	d = DivisionVolume[c];
        	ge = gamma_env[c];
        	vd =  viscotic_damp[c];
        	I = CellINdex[c];
        		
		fwrite(&c, sizeof(int), 1, Restartfile);
        	fwrite(X + (c*192), sizeof(float), 192, Restartfile); 
        	fwrite(Y + (c*192), sizeof(float), 192, Restartfile); 
        	fwrite(Z + (c*192), sizeof(float), 192, Restartfile);
		fwrite(velListX + (c*192), sizeof(float), 192, Restartfile);
            	fwrite(velListY + (c*192), sizeof(float), 192, Restartfile);
		fwrite(velListZ + (c*192), sizeof(float), 192, Restartfile);
		fwrite(&p, sizeof(float), 1, Restartfile);
            	fwrite(&y, sizeof(float), 1, Restartfile);
            	fwrite(&g, sizeof(float), 1, Restartfile);
            	fwrite(&s, sizeof(float), 1, Restartfile);
            	fwrite(&d, sizeof(float), 1, Restartfile);
            	fwrite(&ge, sizeof(float), 1, Restartfile);
            	fwrite(&vd, sizeof(float), 1, Restartfile);
            	fwrite(&I, sizeof(int), 1, Restartfile);
            	
        }

   
   fclose(Restartfile);
   return 0;

}

int ReadRestartFile( ){

  FILE *infil;
  int s;
  int f;
  int nCell;
  int nImp;  
  int CellInd;
  int shift;
  int NCA;
  int NCR;
  

  printf("Reading Restart.xyz ...\n");
  infil = fopen("Restart.xyz","rb");
  
  if ( infil == NULL ) {
    printf("Unable to open file Restart.xyz \n");
    return(-1);
  }



  if ( fread(&s, sizeof(int),1,infil) != 1 ){ 
	printf("Data missing from trajectory. \n");
	return(-1);
  } else printf("\nstep %d \n",s -1);

  if ( fread(&f, sizeof(int),1,infil) != 1 ){ 
	printf("Data missing from trajectory. \n");
	return(-1);
  } else printf("frame number is: %d \n",f - 1);

  if ( fread(&nCell, sizeof(int),1,infil) != 1 ) { 
	printf("Data missing from trajectory. \n");
	return(-1);
  }
  
  if ( fread(&nImp, sizeof(int),1,infil) != 1 ) { 
	printf("Data missing from trajectory. \n");
	return(-1);
  }

  if ( fread(&NCA, sizeof(int),1,infil) != 1 ) { 
	printf("Data missing from trajectory. \n");
	return(-1);
  }

  if ( fread(&NCR, sizeof(int),1,infil) != 1 ) { 
	printf("Data missing from trajectory. \n");
	return(-1);
  }

  Laststep = s-1;
  Lastframe = f-1;
  No_of_threads = nCell;	
  No_of_C180s = nCell;
  Orig_No_of_C180s = nCell;
  impurityNum = nImp;
  NumApoCell = NCA;
  NumRemoveCell = NCR;  

  printf("Number of the initial Cells is: %d \n",Orig_No_of_C180s);
  printf("Number of the  impurity is: %d \n",impurityNum);
  

  for (int c = 0; c < Orig_No_of_C180s; c++){

    shift = c*192;
    
    if ( fread(&CellInd, sizeof(int),1,infil) != 1 ) printf("Data missing from trajectory. \n");
    if ( fread(&X[shift], sizeof(float),192,infil) != 192 ) printf("Data missing from trajectory. \n");
    if ( fread(&Y[shift], sizeof(float),192,infil) != 192 ) printf("Data missing from trajectory. \n");
    if ( fread(&Z[shift], sizeof(float),192,infil) != 192 ) printf("Data missing from trajectory. \n");
    if ( fread(&velListX[shift], sizeof(float),192,infil) != 192 ) printf("Data missing from trajectory. \n");
    if ( fread(&velListY[shift], sizeof(float),192,infil) != 192 ) printf("Data missing from trajectory. \n");
    if ( fread(&velListZ[shift], sizeof(float),192,infil) != 192 ) printf("Data missing from trajectory. \n");
    if ( fread(&pressList[c], sizeof(float),1,infil) != 1 ) printf("Data missing from trajectory. \n");
    if ( fread(&youngsModArray[c], sizeof(float),1,infil) != 1 ) printf("Data missing from trajectory. \n");
    if ( fread(&Growth_rate[c], sizeof(float),1,infil) != 1 ) printf("Data missing from trajectory. \n");
    if ( fread(&ScaleFactor[c], sizeof(float),1,infil) != 1 ) printf("Data missing from trajectory. \n");
    if ( fread(&DivisionVolume[c], sizeof(float),1,infil) != 1 ) printf("Data missing from trajectory. \n");
    if ( fread(&gamma_env[c], sizeof(float),1,infil) != 1 ) printf("Data missing from trajectory. \n");
    if ( fread(&viscotic_damp[c], sizeof(float),1,infil) != 1 ) printf("Data missing from trajectory. \n");
    if ( fread(&CellINdex[c], sizeof(int),1,infil) != 1 ) printf("Data missing from trajectory. \n");
	
   }

   fclose(infil);
   return 0;

}

